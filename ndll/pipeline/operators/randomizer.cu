#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.

#include "ndll/pipeline/operators/randomizer.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

namespace ndll {


__global__
void initializeStates(const int N, unsigned int seed, hiprandState *states) {
  for (int idx = threadIdx.x + blockIdx.x * blockDim.x;
       idx < N;
       idx += blockDim.x * gridDim.x) {
    hiprand_init(seed, idx, 0, &states[idx]);
  }
}

template <>
Randomizer<GPUBackend>::Randomizer(int seed, size_t len) {
  len_ = len;
  states_ = GPUBackend::New(sizeof(hiprandState) * len, true);
  initializeStates<<<128, 256>>>(len_, seed, reinterpret_cast<hiprandState*>(states_));
}

template <>
__device__
int Randomizer<GPUBackend>::rand(int idx) {
  return hiprand(reinterpret_cast<hiprandState*>(states_) + idx);
}

template <>
void Randomizer<GPUBackend>::Cleanup() {
  GPUBackend::Delete(states_, sizeof(hiprandState) * len_, true);
}

}  // namespace ndll

