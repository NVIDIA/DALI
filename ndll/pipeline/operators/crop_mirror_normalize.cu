#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
#include "ndll/pipeline/operators/crop_mirror_normalize.h"

namespace ndll {

namespace {

// Crop, mirror, mean sub, stddev div, NHWC->NCHW, Npp8u->fp32
template <NDLLTensorLayout Layout, typename Out, bool pad>
__global__ void BatchedCropMirrorNormalizePermuteKernel(
    const int N,
    const int C,
    const int H,
    const int W,
    const bool *mirror,
    const float* mean,
    const float* inv_std,
    const uint8* const * img_ptrs,
    const int *input_steps,
    Out* out) {
  const int n = blockIdx.x;

  const int pad_C = pad ? 4 : C;
  const int nStride = pad_C*H*W;

  // pointers to data for this image
  const uint8* input_ptr = img_ptrs[n];
  int in_step = input_steps[n];
  Out* output_ptr = &out[n*nStride];
  bool mirror_image = mirror[n];

  if (Layout == NDLL_NCHW) {
    if (mirror_image) {
      // Mirror the image - coalesced writes
      for (int c=0; c < C; ++c) {
        for (int h=threadIdx.y; h < H; h += blockDim.y) {
          for (int w=threadIdx.x; w < W; w += blockDim.x) {
            int mirrored_width = (W - 1) - w;
            int in_idx = c + C*mirrored_width + in_step*h;  // HWC, mirrored
            int out_idx = c*H*W + h*W + w;  // CHW

            output_ptr[out_idx] = static_cast<Out>(
                (static_cast<float>(input_ptr[in_idx])-mean[c]) * inv_std[c]);
          }
        }
      }
    } else {
      // Copy normally - coalesced writes
      for (int c=0; c < C; ++c) {
        for (int h=threadIdx.y; h < H; h += blockDim.y) {
          for (int w=threadIdx.x; w < W; w += blockDim.x) {
            int in_idx = c + C*w + in_step*h;  // HWC
            int out_idx = c*H*W + h*W + w;  // CHW

            output_ptr[out_idx] = static_cast<Out>(
                (static_cast<float>(input_ptr[in_idx])-mean[c]) * inv_std[c]);
          }
        }
      }
    }
    // Pad to 4 channels with 0s
    if (pad) {
      for (int c=C; c < 4; ++c) {
        for (int h=threadIdx.y; h < H; h += blockDim.y) {
          for (int w=threadIdx.x; w < W; w += blockDim.x) {
            int out_idx = c*H*W + h*W + w;  // CHW

            output_ptr[out_idx] = 0;
          }
        }
      }
    }
  } else {
    for (int tid = threadIdx.x + threadIdx.y * blockDim.x;
         tid < pad_C * H * W;
         tid += blockDim.x * blockDim.y) {
      const int c = tid % pad_C;
      const int w = (tid / pad_C) % W;
      const int h = tid / (pad_C * W);

      const int in_c = c;
      const int in_w = mirror_image ? (W - 1) - w : w;
      const int in_h = h;

      const int in_idx = in_c + in_w * C + in_step * in_h;
      const int out_idx = c + w*pad_C + h*W*pad_C;

      float input;
      if (pad && c == 3) {
        input = 0;
      } else {
        input = (static_cast<float>(input_ptr[in_idx])-mean[c]) * inv_std[c];
      }

      output_ptr[out_idx] = static_cast<Out>(input);
    }
  }
}

/**
 * @brief Takes in a jagged batch of images and crops, (optional) mirrors,
 * performs mean subtraction & stddev division per channel, cast to output
 * type, and NHWC->NCHW permutation
 *
 * The crop is performed by offsetting the ptrs in 'in_batch' to the beginning
 * of the crop region, and then passing in the stride of each image so that
 * the kernel can correctly process the crop region.
 *
 * @param in_batch device pointer to pointer to the beginning of the crop
 * region for each image
 * @param in_strides device pointer to `N` ints whose value is the stride
 * of each input image
 * @param mirror device pointer to `N` bools whose values indicate whether
 * the image should be mirrored or not
 * @param N number of elements in the batch
 * @param H output height for all images in the batch
 * @param W output width for all images in the batch
 * @param C number of channels of images in the batch
 * @param mean device pointer of length `C` to the mean to subtract for
 * each image channel
 * @param std device pointer of length `C` to the inverse std dev. to multiply by
 * for each image channel
 * @param out_batch pointer of size `N*C*H*W` to store the dense, cropped,
 * NCHW output batch
 * @param stream cuda stream to operate in
 */
template <NDLLTensorLayout L, typename OUT>
NDLLError_t BatchedCropMirrorNormalizePermute(const uint8 * const *in_batch,
    const int *in_strides, int N, int H, int W, int C, bool pad, const bool *mirror,
    const float *mean, const float *inv_std, OUT *out_batch, hipStream_t stream) {
  NDLL_ASSERT(in_batch != nullptr);
  NDLL_ASSERT(in_strides != nullptr);
  NDLL_ASSERT(mirror != nullptr);
  NDLL_ASSERT(mean != nullptr);
  NDLL_ASSERT(inv_std != nullptr);
  NDLL_ASSERT(out_batch != nullptr);
  if (pad) {
    BatchedCropMirrorNormalizePermuteKernel<L, OUT, true><<<N, dim3(32, 32), 0, stream>>>(
        N, C, H, W, mirror, mean, inv_std, in_batch, in_strides, out_batch);
  } else {
    BatchedCropMirrorNormalizePermuteKernel<L, OUT, false><<<N, dim3(32, 32), 0, stream>>>(
        N, C, H, W, mirror, mean, inv_std, in_batch, in_strides, out_batch);
  }
  return NDLLSuccess;
}

/**
 * @brief Validates the parameters for 'BatchedCropMirrorNormalizePermute'
 * on host
 *
 * All parameters are host-side versions of the arguments to
 * 'BatchedCropMirrorNormalizePermute'. This method exists so that
 * the user can efficiently manage memory copies to the GPU, but stil
 * have a method for validating input arguments before calling the
 * batched function.
 *
 * Checks that...
 * - in_batch device pointers are not nullptr
 * - in_strides values are >= W*C
 * - N > 0, H > 0, W > 0, C == 1 || C == 3
 */
template <typename OUT>
NDLLError_t ValidateBatchedCropMirrorNormalizePermute(const uint8 * const *in_batch,
    const int *in_strides, int N, int H, int W, int C, const bool *mirror,
    const float *mean, const float *inv_std, OUT *out_batch) {
  NDLL_ASSERT(N > 0);
  NDLL_ASSERT(H > 0);
  NDLL_ASSERT(W > 0);
  NDLL_ASSERT(C == 1 || C == 3);
  NDLL_ASSERT(in_batch != nullptr);
  NDLL_ASSERT(in_strides != nullptr);
  for (int i = 0; i < N; ++i) {
    NDLL_ASSERT(in_batch[i] != nullptr);
    NDLL_ASSERT(in_strides[i] >= C*W);
  }
  return NDLLSuccess;
}

}  // namespace

template<>
template <typename OUT>
void CropMirrorNormalize<GPUBackend>::RunHelper(Workspace<GPUBackend> *ws, const int idx) {
  auto output = ws->Output<GPUBackend>(idx);
  if (output_layout_ == NDLL_NCHW) {
    NDLL_CALL((BatchedCropMirrorNormalizePermute<NDLL_NCHW, OUT>(
            input_ptrs_gpu_.template data<const uint8*>(),
            input_strides_gpu_.template data<int>(),
            batch_size_, crop_h_, crop_w_, C_, pad_,
            mirror_gpu_.template data<bool>(),
            mean_.template data<float>(),
            inv_std_.template data<float>(),
            output->template mutable_data<OUT>(),
            ws->stream())));
  } else {
    NDLL_CALL((BatchedCropMirrorNormalizePermute<NDLL_NHWC, OUT>(
            input_ptrs_gpu_.template data<const uint8*>(),
            input_strides_gpu_.template data<int>(),
            batch_size_, crop_h_, crop_w_, C_, pad_,
            mirror_gpu_.template data<bool>(),
            mean_.template data<float>(),
            inv_std_.template data<float>(),
            output->template mutable_data<OUT>(),
            ws->stream())));
  }
}

template<>
template <typename OUT>
void CropMirrorNormalize<GPUBackend>::ValidateHelper(TensorList<GPUBackend> *output) {
  // Validate parameters
  NDLL_CALL(ValidateBatchedCropMirrorNormalizePermute(
          input_ptrs_.template mutable_data<const uint8*>(),
          input_strides_.template mutable_data<int>(),
          batch_size_, crop_h_, crop_w_, C_,
          mirror_.template mutable_data<bool>(),
          mean_vec_.data(), inv_std_vec_.data(),
          output->template mutable_data<OUT>()));
}

template<>
void CropMirrorNormalize<GPUBackend>::DataDependentSetup(DeviceWorkspace *ws, const int idx) {
  auto &input = ws->Input<GPUBackend>(idx);
  auto output = ws->Output<GPUBackend>(idx);
  NDLL_ENFORCE(IsType<uint8>(input.type()),
      "Expected input data as uint8.");

  vector<Dims> output_shape(batch_size_);
  for (int i = 0; i < batch_size_; ++i) {
    vector<Index> input_shape = input.tensor_shape(i);
    NDLL_ENFORCE(input_shape.size() == 3,
        "Expects 3-dimensional image input.");

    int H = input_shape[0];
    int W = input_shape[1];

    NDLL_ENFORCE(H == per_sample_dimensions_[i].first &&
        W == per_sample_dimensions_[i].second,
        "Corresponding images in different input sets need to have the same height and width");

    int C = input_shape[2];

    NDLL_ENFORCE(C == C_,
        "Input channel dimension does not match "
        "the output image type. Expected input with "
        + to_string(C_) + " channels, got " + to_string(C) + ".");

    // retrieve already determined crop parameters
    int crop_y = per_sample_crop_[i].first;
    int crop_x = per_sample_crop_[i].second;

    // Save image stride & crop offset
    input_strides_.template mutable_data<int>()[i] = W*C_;
    crop_offsets_[i] = crop_y*W*C_ + crop_x*C_;

    // Pad to 4 channels
    int pad_C = pad_ ? 4 : C_;

    // Save the output shape of this image
    if (output_layout_ == NDLL_NCHW) {
      output_shape[i] = {pad_C, crop_h_, crop_w_};
    } else {
      output_shape[i] = {crop_h_, crop_w_, pad_C};
    }
  }

  // Resize the output data
  output->Resize(output_shape);

  // Copy strides and mirror data to gpu
  input_strides_gpu_.Copy(input_strides_, ws->stream());

  // Calculate input pointers and copy to gpu
  for (int i = 0; i < batch_size_; ++i) {
    input_ptrs_.template mutable_data<const uint8*>()[i] =
      input.template tensor<uint8>(i) + crop_offsets_[i];
  }
  input_ptrs_gpu_.Copy(input_ptrs_, ws->stream());

  // Validate
  if (output_type_ == NDLL_FLOAT) {
    ValidateHelper<float>(output);
  } else if (output_type_ == NDLL_FLOAT16) {
    ValidateHelper<float16>(output);
  } else {
    NDLL_FAIL("Unsupported output type.");
  }
}

template<>
void CropMirrorNormalize<GPUBackend>::RunImpl(DeviceWorkspace *ws, const int idx) {
  DataDependentSetup(ws, idx);
  if (output_type_ == NDLL_FLOAT) {
    RunHelper<float>(ws, idx);
  } else if (output_type_ == NDLL_FLOAT16) {
    RunHelper<float16>(ws, idx);
  } else {
    NDLL_FAIL("Unsupported output type.");
  }
}

template<>
void CropMirrorNormalize<GPUBackend>::SetupSharedSampleParams(DeviceWorkspace *ws) {
  for (int i = 0; i < batch_size_; ++i) {
    auto &input = ws->Input<GPUBackend>(0);
    NDLL_ENFORCE(IsType<uint8>(input.type()),
        "Expected input data as uint8.");
    vector<Index> input_shape = input.tensor_shape(i);
    NDLL_ENFORCE(input_shape.size() == 3,
        "Expects 3-dimensional image input.");

    int H = input_shape[0];
    int W = input_shape[1];

    per_sample_dimensions_[i] = std::make_pair(H, W);

    int C = input_shape[2];

    NDLL_ENFORCE(C == C_,
        "Input channel dimension does not match "
        "the output image type. Expected input with "
        + to_string(C_) + " channels, got " + to_string(C) + ".");


    // Random crop
    NDLL_ENFORCE(H >= crop_h_);
    NDLL_ENFORCE(W >= crop_w_);

    int crop_x, crop_y;
    if (random_crop_) {
      crop_y = std::uniform_int_distribution<>(0, H - crop_h_)(rand_gen_);
      crop_x = std::uniform_int_distribution<>(0, W - crop_w_)(rand_gen_);
    } else {
      crop_y = (H - crop_h_) / 2;
      crop_x = (W - crop_w_) / 2;
    }
    per_sample_crop_[i] = std::make_pair(crop_y, crop_x);

    // Set mirror parameters
    mirror_.template mutable_data<bool>()[i] =
      std::bernoulli_distribution(mirror_prob_)(rand_gen_);
  }
  mirror_gpu_.Copy(mirror_, ws->stream());
}

// Register operator
NDLL_REGISTER_OPERATOR(CropMirrorNormalize, CropMirrorNormalize<GPUBackend>, GPU);

}  // namespace ndll
