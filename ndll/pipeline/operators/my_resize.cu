#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.

#include <nppdefs.h>
#include <npp.h>
#include "ndll/pipeline/operators/my_resize.h"
#include "../../common.h"
#include "../../../../../../../usr/local/cuda-9.0/include/npp.h"

namespace ndll {

void DataDependentSetupCPU(const Tensor<CPUBackend> &input,
                            Tensor<CPUBackend> *output, const char *pOpName,
                            vector<const uint8 *> *inPtrs, vector<uint8 *> *outPtrs,
                            vector<NDLLSize> *pSizes, const NDLLSize *out_size) {
     NDLL_ENFORCE(input.ndim() == 3);
     NDLL_ENFORCE(IsType<uint8>(input.type()),
                  "Expects input data in uint8.");

     const vector <Index> &shape = input.shape();
     const int C = shape[2];
     NDLL_ENFORCE(C == 1 || C == 3,
                  string(pOpName? pOpName : "Operation") +
                  " supports only hwc rgb & grayscale inputs.");

     if (out_size)
         output->Resize({out_size->height, out_size->width, C});
     else
         output->Resize(shape);

     output->set_type(input.type());

     if (!inPtrs)
         return;

     (*inPtrs)[0] = input.template data<uint8>();
     if (outPtrs)
         (*outPtrs)[0] = static_cast<uint8*>(output->raw_mutable_data());

     if (pSizes) {
         (*pSizes)[0].height = shape[0];
         (*pSizes)[0].width = shape[1];
     }
 }

void DataDependentSetupGPU(const TensorList<GPUBackend> &input, TensorList<GPUBackend> *output,
                           size_t batch_size, bool reshapeBatch, vector<const uint8 *> *inPtrs,
                           vector<uint8 *> *outPtrs, vector<NDLLSize> *pSizes, ResizeAttr *pResize,
                           vector<NppiRect>  *pOutResize) {
    NDLL_ENFORCE(IsType<uint8>(input.type()),
                 "Expected input data stored in uint8.");

    vector<Dims> output_shape(batch_size);
    for (size_t i = 0; i < batch_size; ++i) {
        // Verify the inputs
        const auto &input_shape = input.tensor_shape(i);
        NDLL_ENFORCE(input_shape.size() == 3,
                     "Expects 3-dimensional image input.");

        NDLL_ENFORCE(input_shape[2] == 1 || input_shape[2] == 3,
                     "Not valid color type argument (1 or 3)");

        // Collect the output shapes
        if (pResize) {
            // We are resizing
            NDLLSize &out_size = pResize->size(output_t, i);
            pResize->SetSize(pResize->size(input_t, i), input_shape,
                pResize->newSizes(i), out_size);

            NppiRect &outResize = (*pOutResize)[i];
            outResize.height = out_size.height;
            outResize.width = out_size.width;

            const bool doingCrop = pResize->CropNeeded(out_size);
            if (doingCrop)
                pResize->DefineCrop(out_size, &outResize.x, &outResize.y);

            // Collect the output shapes
            output_shape[i] = {out_size.height, out_size.width, input_shape[2]};
        } else
            output_shape[i] = input_shape;

        if (pSizes) {
            (*pSizes)[i].height = input_shape[0];
            (*pSizes)[i].width = input_shape[1];
            if (reshapeBatch) {
                // When batch is reshaped: only one "image" will be used
                (*pSizes)[i].height *= batch_size;
                pSizes = NULL;
            }
        }
    }

    // Resize the output
    output->Resize(output_shape);
    output->set_type(input.type());

    CollectPointersForExecution(reshapeBatch? 1 : batch_size, input, inPtrs, output, outPtrs);
}


void CollectPointersForExecution(size_t batch_size,
                                 const TensorList<GPUBackend> &input, vector<const uint8 *> *inPtrs,
                                 TensorList<GPUBackend> *output, vector<uint8 *> *outPtrs) {
    if (!inPtrs || !outPtrs)
        return;

    // Collect the pointers for execution
    for (size_t i = 0; i < batch_size; ++i) {
        (*inPtrs)[i] = input.template tensor<uint8>(i);
        (*outPtrs)[i] = output->template mutable_tensor<uint8>(i);
    }
}

__constant__ ResizeGridParam resizeParam[3];

__global__ void BatchedCongenericResizeKernel(
                    int H0, int W0, const uint8 *img_in, int H, int W, uint8 *img_out,
                    int C, const ResizeMapping *pResizeMapping, const PixMapping *pPixMapping) {
    if (pResizeMapping && pPixMapping) {
        AUGMENT_RESIZE_GPU_CONGENERIC(H, W, C, img_in, img_out, RESIZE_N);
    } else {
        AUGMENT_RESIZE_GPU_CONGENERIC(H, W, C, img_in, img_out, RESIZE);
    }
}

NDLLError_t BatchedCongenericResize(int N, const dim3 &gridDim, hipStream_t stream, int C,
                          const NDLLSize &sizeIn, const uint8 *in_batch,
                          const NDLLSize &sizeOut, uint8 *out_batch,
                          const ResizeGridParam *pResizeParam, const ResizeMappingTable *pTbl) {
    if (pResizeParam) {
        // Copying the descriptor of operation into __constant__ memory
        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(resizeParam), pResizeParam, sizeof(resizeParam)));
    }

    BatchedCongenericResizeKernel<<<N, gridDim, 0, stream>>>
          (sizeIn.height, sizeIn.width, in_batch, sizeOut.height, sizeOut.width, out_batch, C,
           pTbl? pTbl->pResizeMapping[1] : NULL, pTbl? pTbl->pPixMapping[1] : NULL);

    return NDLLSuccess;
}

//  Greatest Common Factor
int gcf (int a, int b) {
    int t;
    if (b > a) {
        t = a;
        a = b;
        b = t;
    }

    while (b) {
        t = a % b;
        a = b;
        b = t;
    }

    return a;
}

#include <assert.h>

// Least Common Multiplier
int lcm (int a, int b) {
    return a / gcf (a, b) * b;
}

__global__ void BatchedResizeKernel(int C, const NppiRect *resizeDescr,
                                    const NDLLSize *in_sizes, const uint8 *const imgs_in[],
                                    const NDLLSize *out_sizes, uint8 *const imgs_out[]) {
    /*
    const int id = blockIdx.x;
    const uint8 *img_in = imgs_in[id];
    uint8 *img_out = imgs_out[id];
    const int H0 = in_sizes[id].height;
    const int W0 = in_sizes[id].width;
    const int H1 = resizeDescr[id].height;
    const int W1 = resizeDescr[id].width;
    const int H = out_sizes[id].height;
    const int W = out_sizes[id].width; */
/*
    ResizeGridParam resizeParam[3];
    const int lcmH = lcm(H0, H1);
    const int lcmW = lcm(W0, W1);
    resizeParam[0].Init(lcmW / W0, lcmH / H0);
    resizeParam[1].Init(lcmW / W1, lcmH / H1);
    resizeParam[2].Init(resizeDescr[id].x, resizeDescr[id].y); */
 /*


    const int sx0 = resizeParam[0].nX;     \
    const int sy0 = resizeParam[0].nY;     \
    const int sx1 = resizeParam[1].nX;     \
    const int sy1 = resizeParam[1].nY;     \
    const int cropX = resizeParam[2].nX;   \
    const int cropY = resizeParam[2].nY;   \
    const int area = sx1 * sy1;
    */
//    AUGMENT_RESIZE_GPU_GENERIC(H, W, C, img_in, img_out, RESIZE);
/*
    RESIZE_PREAMBLE(H, W, C);
    const int stepH = blockDim.y;
    const int startH = threadIdx.y;
    const int startW = threadIdx.x;
    const int stepW = blockDim.x;
    const int imgIdx = 0;
    const uint32_t offset = nYoffset(W, C);                         \
    const uint32_t shift = stepH * offset;                          \
    const uint8 *in = img_in + H0 *nYoffset(W0, C) * imgIdx;        \
    uint8 *out = img_out + (H * imgIdx + startH) * offset - shift;  \
    for (int y = startH; y < H; y += stepH) {                       \
        out += shift;                                               \
        for (int x = startW; x < W; x += stepW) {                   \
            ;//RESIZE_CORE(C);
        }                                                           \
    } */
}

NDLLError_t BatchedResize(int N, const dim3 &gridDim, hipStream_t stream, int C,
                          const vector<NDLLSize> &inImg, const vector<const uint8 *> *in_batch,
                          const vector<NDLLSize> &outImg, vector<uint8 *> *out_batch,
                          const vector<NppiRect> &resizeDescr) {
/*
    static int cntr;
    const NDLLSize *in_sizes = inImg.data();
    const NDLLSize *out_sizes = outImg.data();
    const NppiRect *resizeDescrData = resizeDescr.data();
    FILE *file = fopen("ccc1A.txt", cntr++? "a" : "w");
    for (int i = 0; i < N; i++) {
        const int id = i;
        const int H0 = in_sizes[id].height;
        const int W0 = in_sizes[id].width;
        const int H1 = resizeDescrData[id].height;
        const int W1 = resizeDescrData[id].width;
        const int H = out_sizes[id].height;
        const int W = out_sizes[id].width;

        ResizeGridParam resizeParam[3];
        const int lcmH = lcm(H0, H1);
        const int lcmW = lcm(W0, W1);
        resizeParam[0].Init(lcmW / W0, lcmH / H0);
        resizeParam[1].Init(lcmW / W1, lcmH / H1);
        resizeParam[2].Init(resizeDescrData[id].x, resizeDescrData[id].y);

        fprintf(file, "H0 = %3d,  W0 = %3d,  H1 = %3d  W1 = %3d  H = %2d  W = %3d cropXY = (%3d %3d)\n", H0,
                W0, H1, W1, H, W, resizeParam[2].nX, resizeParam[2].nY);
    }

    fclose(file);
*/
    BatchedResizeKernel<<<N, gridDim, 0, stream>>>(C, resizeDescr.data(), inImg.data(), in_batch->data(),
            outImg.data(), out_batch->data());

    return NDLLSuccess;
}
/*
void releaseCudaResizeMapingTable() {
    CUDA_FREE(pResizeMappingGPU_);
    CUDA_FREE(pPixMappingGPU_);
//    CUDA_FREE(resizeParamGPU_);
}
 */

#include <assert.h>

ResizeMappingTable::ResizeMappingTable(int H0, int W0, int H1, int W1, int C,
             uint16_t xSize, uint16_t ySize) {
    io_size[0] = {W0, H0};
    io_size[1] = {W1, H1};
    C_ = C;

    pResizeMapping[0] = new ResizeMapping [xSize * ySize];
    pResizeMapping[1] = NULL;

    tableLength = xSize * ySize * sizeof(pResizeMapping[0][0]);
    memset(pResizeMapping[0], 0, tableLength);
    pPixMapping[0] = pPixMapping[1] = NULL;
}

ResizeMappingTable::~ResizeMappingTable() {
    delete [] pPixMapping[0];
    delete [] pResizeMapping[0];
    releaseCudaResizeMapingTable();
}

bool ResizeMappingTable::IsValid(int H0, int W0, int H1, int W1) const {
    if (!pPixMapping[0] || !pResizeMapping[0])
        return false;

    return io_size[0].height == H0 && io_size[0].width == W0 &&
           io_size[1].height == H1 && io_size[1].width == W1;
}

void ResizeMappingTable::CopyCongenericResizeParam() {
    // Copying the descriptor of operation into __constant__ memory
    /*
    if (!resizeParamGPU_)
        CUDA_MALLOC(resizeParamGPU_, sizeof(resizeParam));

    CUDA_MEMCPY(resizeParamGPU_, resizeParam, sizeof(resizeParam));
*/
    releaseCudaResizeMapingTable();
    CUDA_MALLOC(pResizeMapping[1], getMappingTableLength());
    CUDA_MEMCPY(pResizeMapping[1], pResizeMapping[0], getMappingTableLength());
    CUDA_MALLOC(pPixMapping[1], pixMappingLen);
    CUDA_MEMCPY(pPixMapping[1], pPixMapping[0], pixMappingLen);
}

void ResizeMappingTable::releaseCudaResizeMapingTable() {
    CUDA_FREE(pResizeMapping[1]);
    CUDA_FREE(pPixMapping[1]);
}

class PixMappingHelper {
 public:
    PixMappingHelper(uint32_t len, ResizeMapping *pMapping, bool useClosest = false);
    void AddPixel(uint32_t addr, uint32_t area, int crdX, int crdY);
    void UpdateMapping(int shift, int centerX, int centerY);
    inline PixMapping *getPixMapping() const    { return pPixMapping_; }
    inline uint32_t numUsed() const             { return numPixMapUsed_; }
 private:
    inline float distance(float x, float y) const  { return x * x + y * y; }
    uint32_t numPixMapMax_;  // length of the allocated PixMapping array
    uint32_t numPixMapUsed_; // number of already used elements of pPixMapping
    PixMapping *pPixMapping_ = new PixMapping[numPixMapMax_];
    ResizeMapping *pMappingBase_;
    ResizeMapping *pMapping_;

    const uint32_t area_;
    const bool useClosest_;
    float closestDist_;
    float centerX_, centerY_;
};

PixMappingHelper::PixMappingHelper(uint32_t area, ResizeMapping *pMapping, bool useClosest) :
        area_(area), useClosest_(useClosest) {
    numPixMapUsed_ = 0;
    pPixMapping_ = new PixMapping[numPixMapMax_ = 2 * area];
    pMappingBase_ = pMapping;
}

void PixMappingHelper::AddPixel(uint32_t addr, uint32_t area, int crdX, int crdY) {
    if (numPixMapUsed_ == numPixMapMax_) {
        // Previously allocated array needs to be extended
        PixMapping *pPixMappingNew = new PixMapping[numPixMapMax_ <<= 1];
        memcpy(pPixMappingNew, pPixMapping_, numPixMapUsed_ * sizeof(pPixMappingNew[0]));
        pPixMapping_ = pPixMappingNew;
    }

    assert(area != 0);

    if (!useClosest_) {
        pMapping_->nPixels++;
        pPixMapping_[numPixMapUsed_++].Init(addr, area);
    } else {
       const float newDist = distance((crdX << 1) - centerX_, (crdY << 1) - centerY_);
       if (closestDist_ < 0) {
           pMapping_->nPixels++;
           pPixMapping_[numPixMapUsed_++].Init(addr, area_);
           closestDist_ = newDist;
       } else
       if (closestDist_ > newDist) {
           closestDist_ = newDist;
           pPixMapping_[numPixMapUsed_ - 1].Init(addr, area_);
       }
    }
}

void PixMappingHelper::UpdateMapping(int shift, int centerX, int centerY) {
    (pMapping_ = pMappingBase_ + shift)->intersectInfoAddr = numUsed();
    centerX_ = centerX;
    centerY_ = centerY;
    closestDist_ = -1;
}

#define RUN_CHECK_1     0

ResizeMappingTable *createResizeMappingTable(int H0, int W0, int H1, int W1, int C, bool useClosest)
{
    // The table, which contains the information about correspondence of pixels of the initial
    // image to the pixels of the resized one.

    // Resizing from (H0, W0) to (H1, W1)
    // Main equations are:
    // H0 * sy0 = H1 * sy1
    // W0 * sx0 = W1 * sx1
    const size_t lcmH = lcm(H0, H1);
    const size_t lcmW = lcm(W0, W1);

    const int sy0 = lcmH / H0;
    const int sy1 = lcmH / H1;
    const int sx0 = lcmW / W0;
    const int sx1 = lcmW / W1;

    ResizeMappingTable *pTable = new ResizeMappingTable(H0, W0, H1, W1, C, sx0, sy0);
    PixMappingHelper helper(sx0 * sy0, pTable->pResizeMapping[0], useClosest);

    // (x, y) pixel coordinate of PIX in resized image
    // 0 <= x < W1;  0 <= y < H1

    for (int y = 0; y < sy0; ++y) {
        for (int x = 0; x < sx0; ++x) {

            const int nX = x * sx1;
            const int nY = y * sy1;
            // The indices of the top-left pixel of the initial image, intersecting with PIX
            const int begIdx[2] = { nX / sx0, nY / sy0 };

            // The indices of the bottom-right pixel of the initial image, intersecting with PIX
            int endIdx[2] = { (nX + sx1) / sx0, (nY + sy1) / sy0 };

            // Intersection of the right (bottom) pixels with the PIX (could be equal to 0)
            const int extra[2] = { (nX + sx1) % sx0, (nY + sy1) % sy0 };

            // Length of the left (top) pixels intersecting with the PIX
            const int lenFirst[2] = { (sx0 - nX % sx0),   (sy0 - nY % sy0) };

            // Doubled (x,y) coordinates of the pixel's center
            const int lenX = endIdx[0] + begIdx[0] - (extra[0] ? 0 : 1);
            const int lenY = endIdx[1] + begIdx[1] - (extra[1] ? 0 : 1);

            // Relative address to the first intersecting pixels
            helper.UpdateMapping(((y * sy1) % sy0) * sx0 + (x * sx1) % sx0, lenX, lenY);

            endIdx[0] -= begIdx[0];
            endIdx[1] -= begIdx[1];
#if RUN_CHECK_1
            size_t check = 0;
#endif
            size_t rowMult = lenFirst[1];
            int y0 = 0;
            while (true) {
                int x0 = endIdx[0];

                // Relative address of the last pixel in row y0, intersecting with PIX
                uint32_t pixAddr = ((y0 * W0) + x0) * C;
                if (extra[0])
                    helper.AddPixel(pixAddr, extra[0] * rowMult, x0, y0);

                while (--x0 > 0) {
                    helper.AddPixel(pixAddr -= C, sx0 * rowMult, x0, y0);
                }

                helper.AddPixel(pixAddr -= C, lenFirst[0] * rowMult, x0, y0);

#if RUN_CHECK_1
                check += rowMult * (sx0 * (endIdx[0] - 1) + lenFirst[0] + extra[0]);
#endif
                if (++y0  < endIdx[1])
                    rowMult = sy0;
                else {
                    if (y0 > endIdx[1] || !(rowMult = extra[1]))
                        break;
                }
            }

#if RUN_CHECK_1
            assert(check == sx1 * sy1);
#endif
        }
    }

    pTable->pPixMapping[0] = helper.getPixMapping();
    pTable->pixMappingLen = helper.numUsed() * sizeof(pTable->pPixMapping[0][0]);
    return pTable;
}

}  // namespace ndll

