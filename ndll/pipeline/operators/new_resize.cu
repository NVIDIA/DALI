#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.

#include "ndll/pipeline/operators/new_resize.h"
#include <float.h>
#include <assert.h>
#include <string>
#include <vector>
#include <nppdefs.h>
#include <npp.h>

namespace ndll {

void DataDependentSetupCPU(const Tensor<CPUBackend> &input,
                            Tensor<CPUBackend> *output, const char *pOpName,
                            vector<const uint8 *> *inPtrs, vector<uint8 *> *outPtrs,
                            vector<NDLLSize> *pSizes, const NDLLSize *out_size) {
    NDLL_ENFORCE(input.ndim() == 3);
    NDLL_ENFORCE(IsType<uint8>(input.type()), "Expects input data in uint8.");

    const vector <Index> &shape = input.shape();
    const int C = shape[2];
    NDLL_ENFORCE(C == 1 || C == 3,
              string(pOpName? pOpName : "Operation") +
              " supports only hwc rgb & grayscale inputs.");

    if (out_size)
        output->Resize({out_size->height, out_size->width, C});
    else
        output->Resize(shape);

    output->set_type(input.type());

    if (!inPtrs)
        return;

    (*inPtrs)[0] = input.template data<uint8>();
    if (outPtrs)
        (*outPtrs)[0] = static_cast<uint8*>(output->raw_mutable_data());

    if (pSizes) {
        (*pSizes)[0].height = shape[0];
        (*pSizes)[0].width = shape[1];
    }
}

void DataDependentSetupGPU(const TensorList<GPUBackend> &input, TensorList<GPUBackend> *output,
                           size_t batch_size, bool reshapeBatch, vector<const uint8 *> *inPtrs,
                           vector<uint8 *> *outPtrs, vector<NDLLSize> *pSizes, ResizeAttr *pResize,
                           vector<NppiRect>  *pOutResize) {
    NDLL_ENFORCE(IsType<uint8>(input.type()),
                 "Expected input data stored in uint8.");

    vector<Dims> output_shape(batch_size);
    for (size_t i = 0; i < batch_size; ++i) {
        // Verify the inputs
        const auto &input_shape = input.tensor_shape(i);
        NDLL_ENFORCE(input_shape.size() == 3,
                     "Expects 3-dimensional image input.");

        NDLL_ENFORCE(input_shape[2] == 1 || input_shape[2] == 3,
                     "Not valid color type argument (1 or 3)");

        // Collect the output shapes
        if (pResize) {
            // We are resizing
            NDLLSize *out_size = pResize->size(output_t, i);
            pResize->SetSize(pResize->size(input_t, i), input_shape,
                pResize->newSizes(i), out_size);

            if (pOutResize) {
                NppiRect &outResize = (*pOutResize)[i];
                outResize.height = out_size->height;
                outResize.width = out_size->width;

                const bool doingCrop = pResize->CropNeeded(*out_size);
                if (doingCrop)
                    pResize->DefineCrop(out_size, &outResize.x, &outResize.y);
            }

            // Collect the output shapes
            output_shape[i] = {out_size->height, out_size->width, input_shape[2]};
        } else {
            output_shape[i] = input_shape;
        }

        if (pSizes) {
            (*pSizes)[i].height = input_shape[0];
            (*pSizes)[i].width = input_shape[1];
            if (reshapeBatch) {
                // When batch is reshaped: only one "image" will be used
                (*pSizes)[i].height *= batch_size;
                pSizes = NULL;
            }
        }
    }

    // Resize the output
    output->Resize(output_shape);
    output->set_type(input.type());

    CollectPointersForExecution(reshapeBatch? 1 : batch_size, input, inPtrs, output, outPtrs);
}


void CollectPointersForExecution(size_t batch_size,
                                 const TensorList<GPUBackend> &input, vector<const uint8 *> *inPtrs,
                                 TensorList<GPUBackend> *output, vector<uint8 *> *outPtrs) {
    if (!inPtrs || !outPtrs)
        return;

    // Collect the pointers for execution
    for (size_t i = 0; i < batch_size; ++i) {
        (*inPtrs)[i] = input.template tensor<uint8>(i);
        (*outPtrs)[i] = output->template mutable_tensor<uint8>(i);
    }
}

__global__ void BatchedCongenericResizeKernel(
                    int H0, int W0, const uint8 *img_in, int H, int W, uint8 *img_out,
                    int C, const ResizeGridParam *resizeParam,
                    const ResizeMapping *pResizeMapping, const PixMapping *pPixMapping) {
    if (pResizeMapping && pPixMapping) {
        AUGMENT_RESIZE_GPU_CONGENERIC(H, W, C, img_in, img_out, RESIZE_N);
    } else {
        AUGMENT_RESIZE_GPU_CONGENERIC(H, W, C, img_in, img_out, RESIZE);
    }
}

NDLLError_t BatchedCongenericResize(int N, const dim3 &gridDim, hipStream_t stream, int C,
                          const NDLLSize &sizeIn, const uint8 *in_batch,
                          const NDLLSize &sizeOut, uint8 *out_batch,
                          const ResizeGridParam *pResizeParam, const ResizeMappingTable *pTbl) {
    BatchedCongenericResizeKernel<<<N, gridDim, 0, stream>>>
          (sizeIn.height, sizeIn.width, in_batch, sizeOut.height, sizeOut.width, out_batch, C,
           pResizeParam, pTbl? pTbl->pResizeMapping[1] : NULL, pTbl? pTbl->pPixMapping[1] : NULL);

    return NDLLSuccess;
}

//  Greatest Common Factor
int __host__ __device__ gcf(int a, int b) {
    int t;
    if (b > a) {
        t = a;
        a = b;
        b = t;
    }

    while (b) {
        t = a % b;
        a = b;
        b = t;
    }

    return a;
}

// Least Common Multiplier
int __host__ __device__ lcm(int a, int b) {
    return a / gcf (a, b) * b;
}

__global__ void BatchedResizeKernel(int C, const NppiRect *resizeDescr,
                                    const NDLLSize *in_sizes, const uint8 *const imgs_in[],
                                    const NDLLSize *out_sizes, uint8 *const imgs_out[]) {
    const int H0 = in_sizes[blockIdx.x].height;
    const int W0 = in_sizes[blockIdx.x].width;

    const int H1 = resizeDescr[blockIdx.x].height;
    const int W1 = resizeDescr[blockIdx.x].width;
    const int H = out_sizes[blockIdx.x].height;
    const int W = out_sizes[blockIdx.x].width;

    const int lcmH = lcm(H0, H1);
    const int lcmW = lcm(W0, W1);
    ResizeGridParam resizeParam[3] = {
            {lcmW / W0, lcmH / H0},
            {lcmW / W1, lcmH / H1},
            {resizeDescr[blockIdx.x].x, resizeDescr[blockIdx.x].y}
    };

    AUGMENT_RESIZE_GPU_GENERIC(H, W, C, imgs_in[blockIdx.x], imgs_out[blockIdx.x], RESIZE);
}

NDLLError_t BatchedResize(int N, const dim3 &gridDim, hipStream_t stream, int C,
                          const NppiRect *resizeDescr,
                          const NDLLSize * const sizes[], uint8 ** const raster[]) {
    const uint8 * const* in = raster[input_t];
    uint8 * const *out = raster[output_t];

    BatchedResizeKernel<<<N, gridDim, 0, stream>>>(C, resizeDescr,
            sizes[input_t], in, sizes[output_t], out);

    return NDLLSuccess;
}

ResizeMappingTable::ResizeMappingTable(int H0, int W0, int H1, int W1, int C,
             uint16_t xSize, uint16_t ySize) {
    io_size[0] = {W0, H0};
    io_size[1] = {W1, H1};
    C_ = C;

    pResizeMapping[0] = new ResizeMapping[xSize * ySize];
    pResizeMapping[1] = NULL;

    tableLength = xSize * ySize * sizeof(pResizeMapping[0][0]);
    memset(pResizeMapping[0], 0, tableLength);
    pPixMapping[0] = pPixMapping[1] = NULL;
}

ResizeMappingTable::~ResizeMappingTable() {
    delete [] pPixMapping[0];
    delete [] pResizeMapping[0];
    releaseCudaResizeMapingTable();
}

bool ResizeMappingTable::IsValid(int H0, int W0, int H1, int W1) const {
    if (!pResizeMapping[0])
        return false;

    return io_size[0].height == H0 && io_size[0].width == W0 &&
           io_size[1].height == H1 && io_size[1].width == W1;
}

void ResizeMappingTable::CopyCongenericResizeParam() {
    releaseCudaResizeMapingTable();
    CUDA_MALLOC(pResizeMapping[1], getMappingTableLength());
    CUDA_MEMCPY(pResizeMapping[1], pResizeMapping[0], getMappingTableLength());
    if (pPixMapping[0]) {
        CUDA_MALLOC(pPixMapping[1], pixMappingLen);
        CUDA_MEMCPY(pPixMapping[1], pPixMapping[0], pixMappingLen);
    }
}

void ResizeMappingTable::releaseCudaResizeMapingTable() {
    CUDA_FREE(pResizeMapping[1]);
    CUDA_FREE(pPixMapping[1]);
}

class PixMappingHelper {
 public:
    PixMappingHelper(uint32_t len, ResizeMapping *pMapping, uint32_t resizedArea = 0);
    void AddPixel(uint32_t addr, uint32_t area, int crdX, int crdY);
    void UpdateMapping(int shift, int centerX, int centerY);
    inline PixMapping *getPixMapping() const        { return pPixMapping_; }
    inline uint32_t numUsed() const                 { return numPixMapUsed_; }
 private:
    inline float distance(float x, float y) const   { return x * x + y * y; }
    uint32_t numPixMapMax_;     // length of the allocated PixMapping array
    uint32_t numPixMapUsed_;    // number of already used elements of pPixMapping
    PixMapping *pPixMapping_;
    ResizeMapping *pMappingBase_;
    ResizeMapping *pMapping_;

    const uint32_t area_;
    const uint32_t resizedArea_;
    float closestDist_;
    float centerX_, centerY_;
};

PixMappingHelper::PixMappingHelper(uint32_t area, ResizeMapping *pMapping, uint32_t resizedArea) :
        area_(area), resizedArea_(resizedArea) {
    numPixMapMax_ = 1;
    numPixMapUsed_ = 0;
    pPixMapping_ = resizedArea == 0? new PixMapping[numPixMapMax_ = 2 * area] : NULL;
    pMappingBase_ = pMapping;
}

void PixMappingHelper::AddPixel(uint32_t addr, uint32_t area, int crdX, int crdY) {
    assert(area != 0);
    if (numPixMapUsed_ == numPixMapMax_) {
        // Previously allocated array needs to be extended
        PixMapping *pPixMappingNew = new PixMapping[numPixMapMax_ <<= 1];
        memcpy(pPixMappingNew, pPixMapping_, numPixMapUsed_ * sizeof(pPixMappingNew[0]));
        pPixMapping_ = pPixMappingNew;
    }

    if (resizedArea_ == 0) {
        pMapping_->nPixels++;
        pPixMapping_[numPixMapUsed_++].Init(addr, area);
    } else {
       const float newDist = distance((crdX << 1) - centerX_, (crdY << 1) - centerY_);
       if (closestDist_ > newDist) {
           closestDist_ = newDist;
           pMapping_->intersectInfoAddr = addr;
       }
    }
}

void PixMappingHelper::UpdateMapping(int shift, int centerX, int centerY) {
    pMapping_ = pMappingBase_ + shift;
    pMapping_->intersectInfoAddr = resizedArea_? 0 : numUsed();

    centerX_ = centerX;
    centerY_ = centerY;
    closestDist_ = FLT_MAX;
}

#define RUN_CHECK_1     0

ResizeMappingTable *createResizeMappingTable(int H0, int W0, int H1, int W1, int C, bool use_NN) {
    // The table, which contains the information about correspondence of pixels of the initial
    // image to the pixels of the resized one.

    // Resizing from (H0, W0) to (H1, W1)
    // Main equations are:
    // H0 * sy0 = H1 * sy1
    // W0 * sx0 = W1 * sx1
    const size_t lcmH = lcm(H0, H1);
    const size_t lcmW = lcm(W0, W1);

    const int sy0 = lcmH / H0;
    const int sy1 = lcmH / H1;
    const int sx0 = lcmW / W0;
    const int sx1 = lcmW / W1;

    ResizeMappingTable *pTable = new ResizeMappingTable(H0, W0, H1, W1, C, sx0, sy0);
    PixMappingHelper helper(sx0 * sy0, pTable->pResizeMapping[0], use_NN? sx1 * sy1 : 0);

    // (x, y) pixel coordinate of PIX in resized image
    // 0 <= x < W1;  0 <= y < H1

    for (int y = 0; y < sy0; ++y) {
        for (int x = 0; x < sx0; ++x) {
            const int nX = x * sx1;
            const int nY = y * sy1;
            // The indices of the top-left pixel of the initial image, intersecting with PIX
            const int begIdx[2] = { nX / sx0, nY / sy0 };

            // The indices of the bottom-right pixel of the initial image, intersecting with PIX
            int endIdx[2] = { (nX + sx1) / sx0, (nY + sy1) / sy0 };

            // Intersection of the right (bottom) pixels with the PIX (could be equal to 0)
            const int extra[2] = { (nX + sx1) % sx0, (nY + sy1) % sy0 };

            // Length of the left (top) pixels intersecting with the PIX
            const int lenFirst[2] = { (sx0 - nX % sx0),   (sy0 - nY % sy0) };

            // Doubled (x,y) coordinates of the pixel's center
            const int lenX = endIdx[0] + begIdx[0] - (extra[0] ? 0 : 1);
            const int lenY = endIdx[1] + begIdx[1] - (extra[1] ? 0 : 1);

            // Relative address to the first intersecting pixels
            helper.UpdateMapping(((y * sy1) % sy0) * sx0 + (x * sx1) % sx0, lenX, lenY);

            endIdx[0] -= begIdx[0];
            endIdx[1] -= begIdx[1];
#if RUN_CHECK_1
            size_t check = 0;
#endif
            size_t rowMult = lenFirst[1];
            int y0 = 0;
            while (true) {
                int x0 = endIdx[0];

                // Relative address of the last pixel in row y0, intersecting with PIX
                uint32_t pixAddr = ((y0 * W0) + x0) * C;
                if (extra[0])
                    helper.AddPixel(pixAddr, extra[0] * rowMult, x0, y0);

                while (--x0 > 0)
                    helper.AddPixel(pixAddr -= C, sx0 * rowMult, x0, y0);

                helper.AddPixel(pixAddr -= C, lenFirst[0] * rowMult, x0, y0);

#if RUN_CHECK_1
                check += rowMult * (sx0 * (endIdx[0] - 1) + lenFirst[0] + extra[0]);
#endif
                if (++y0  >= endIdx[1]) {
                    if (y0 > endIdx[1] || !(rowMult = extra[1]))
                        break;
                } else {
                    rowMult = sy0;
                }
            }

#if RUN_CHECK_1
            assert(check == sx1 * sy1);
#endif
        }
    }

    pTable->pPixMapping[0] = helper.getPixMapping();
    pTable->pixMappingLen = helper.numUsed() * sizeof(pTable->pPixMapping[0][0]);
    return pTable;
}

}  // namespace ndll

