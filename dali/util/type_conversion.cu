#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <cmath>
#include "dali/util/type_conversion.h"
#include "dali/core/cuda_utils.h"

namespace dali {

namespace {
template <typename IN, typename OUT>
__global__ void ConvertKernel(const IN *data, int n, OUT *out) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) {
    out[tid] = (OUT)data[tid];
  }
}

// Specialize the implementation of float16 for CUDA 8 which does
// not have builtin cast for float16.
// IN CASE there are MORE specializations of "Convert" down there,
// corresponding specializations of ConvertKernel should be added below
#if CUDART_VERSION < 9000
template<>
__global__ void ConvertKernel(const float16 *data, int n, double *out) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) {
    out[tid] = static_cast<double>(__half2float(data[tid]));
  }
}
#endif  // CUDART_VERSION < 9000

}  // namespace

template <typename IN, typename OUT>
DLL_PUBLIC void Convert(const IN *data, int n, OUT *out) {
  int block_size = 512;
  int blocks = ceil(static_cast<float>(n) / block_size);
  ConvertKernel<<<blocks, block_size, 0, 0>>>(data, n, out);
}

// Note: These are used in the test suite for output verification, we
// don't care if we do extra copy from T to T.
template DLL_PUBLIC void Convert<uint8, double>(const uint8*, int, double*);
template DLL_PUBLIC void Convert<int16, double>(const int16*, int, double*);
template DLL_PUBLIC void Convert<int, double>(const int*, int, double*);
template DLL_PUBLIC void Convert<int64, double>(const int64*, int, double*);
template DLL_PUBLIC void Convert<float16, double>(const float16*, int, double*);
template DLL_PUBLIC void Convert<float, double>(const float*, int, double*);
template DLL_PUBLIC void Convert<double, double>(const double*, int, double*);

}  // namespace dali
