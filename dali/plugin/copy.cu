// Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.


#include <hip/hip_runtime.h>

#include "dali/plugin/copy.h"
#include "dali/error_handling.h"
#include "dali/util/user_stream.h"
#include "dali/util/device_guard.h"

namespace dali {

template <typename T>
static void CopyToExternalTensorHelper(const dali::Buffer<T> &src, void *dst,
                                       device_type_t dst_type, size_t num) {}

template <>
void CopyToExternalTensorHelper<CPUBackend>(const dali::Buffer<CPUBackend> &src, void *dst,
                                            device_type_t dst_type, size_t num) {
  if (dst_type == CPU) {
    std::memcpy(dst, src.raw_data(), num);
  } else {
    DALI_FAIL("Coping from CPUBackend to device type " + to_string(dst_type));
  }
}

template <>
void CopyToExternalTensorHelper<GPUBackend>(const dali::Buffer<GPUBackend> &src, void *dst,
                                            device_type_t dst_type, size_t num) {
  DeviceGuard d(src.device_id());
  hipMemcpyKind direction;
  hipStream_t stream = UserStream::Get()->GetStream(src);
  if (dst_type == GPU) {
    direction = hipMemcpyDeviceToDevice;
  } else if (dst_type == CPU) {
    direction = hipMemcpyDeviceToHost;
  } else {
    DALI_FAIL("Coping from GPUBackend to device type " + to_string(dst_type));
  }
  CUDA_CALL(hipMemcpyAsync(dst,
                            src.raw_data(),
                            num,
                            direction,
                            stream));
  CUDA_CALL(hipStreamSynchronize(stream));
}

template <typename T>
static void CopyToExternalTensorListHelper(TensorList<T>* tl, void* ptr,
                                           device_type_t dst_type) {
  if (tl->IsDenseTensor()) {
    Tensor<T> t;
    t.ShareData(tl);
    CopyToExternalTensor(t, ptr, dst_type);
  } else {
    CopyToExternalTensorHelper<T>(*tl, ptr, dst_type, tl->nbytes());
  }
}

void CopyToExternalTensor(const Tensor<CPUBackend>& t, void* ptr,
                          device_type_t dst_type) {
  DALI_ENFORCE(t.ndim() > 0, "Can't copy empty Tensor!");
  CopyToExternalTensorHelper<CPUBackend>(t, ptr, dst_type,
                                         volume(t.shape()) * t.type().size());
}

void CopyToExternalTensor(const Tensor<GPUBackend>& t, void* ptr,
                          device_type_t dst_type) {
  DALI_ENFORCE(t.ndim() > 0, "Can't copy empty Tensor!");
  CopyToExternalTensorHelper<GPUBackend>(t, ptr, dst_type,
                                         volume(t.shape()) * t.type().size());
}
void CopyToExternalTensor(TensorList<CPUBackend>* tl, void* ptr,
                          device_type_t dst_type) {
  CopyToExternalTensorListHelper<CPUBackend>(tl, ptr, dst_type);
}

void CopyToExternalTensor(TensorList<GPUBackend>* tl, void* ptr,
                          device_type_t dst_type) {
  CopyToExternalTensorListHelper<GPUBackend>(tl, ptr, dst_type);
}

}  // namespace dali
