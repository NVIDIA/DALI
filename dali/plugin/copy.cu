// Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.


#include <hip/hip_runtime.h>

#include "dali/plugin/copy.h"
#include "dali/error_handling.h"
#include "dali/util/user_stream.h"
#include "dali/pipeline/util/device_guard.h"

namespace dali {

void CopyToExternalTensor(const Tensor<CPUBackend>& t, void* ptr) {
  DALI_ENFORCE(t.ndim() > 0, "Can't copy empty Tensor!");
  std::memcpy(ptr,
              t.raw_data(),
              Product(t.shape()) * t.type().size());
}

void CopyToExternalTensor(const Tensor<GPUBackend>& t, void* ptr) {
  DALI_ENFORCE(t.ndim() > 0, "Can't copy empty Tensor!");
  DeviceGuard d(t.device_id());
  hipStream_t stream = UserStream::Get()->GetStream(t);
  CUDA_CALL(hipMemcpyAsync(ptr,
                            t.raw_data(),
                            Product(t.shape()) * t.type().size(),
                            hipMemcpyDeviceToDevice,
                            stream));
  CUDA_CALL(hipStreamSynchronize(stream));
}

void CopyToExternalTensor(TensorList<CPUBackend>* tl, void* ptr) {
  Tensor<CPUBackend> t;
  t.ShareData(tl);
  CopyToExternalTensor(t, ptr);
}

void CopyToExternalTensor(TensorList<GPUBackend>* tl, void* ptr) {
  Tensor<GPUBackend> t;
  t.ShareData(tl);
  CopyToExternalTensor(t, ptr);
}

}  // namespace dali
