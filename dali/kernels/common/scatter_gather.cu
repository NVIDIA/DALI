#include "hip/hip_runtime.h"
// Copyright (c) 2019-2022, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>
#include <cassert>
#include <utility>
#include <vector>

#include "dali/core/cuda_error.h"
#include "dali/kernels/common/scatter_gather.h"
#include "dali/kernels/dynamic_scratchpad.h"

namespace dali {
namespace kernels {

namespace detail {
size_t Coalesce(span<CopyRange> ranges) {
  if (ranges.empty())
    return 0;
  std::sort(ranges.begin(), ranges.end(), [](const CopyRange &a, const CopyRange &b) {
    return a.src < b.src;
  });

  int start = 0;
  size_t n = ranges.size();
  bool changed = false;

  // merge
  for (size_t i = 1; i < n; i++) {
    if (ranges[i].src == ranges[start].src + ranges[start].size &&
        ranges[i].dst == ranges[start].dst + ranges[start].size) {
      ranges[start].size += ranges[i].size;
      ranges[i] = { nullptr, nullptr, 0 };
      changed = true;
    } else {
      start = i;
    }
  }

  if (changed) {
    // compact
    size_t new_size = 1;  // first item is guaranteed to be non-empty
    for (size_t i = 1; i < n; i++) {
      if (ranges[i].size > 0) {
        size_t j = new_size++;
        if (j != i)
          ranges[j] = ranges[i];
      }
    }
    return new_size;
  }
  return n;
}

}  // namespace detail

std::pair<size_t, size_t>
ScatterGatherBase::BlockCountAndSize(const std::vector<CopyRange> &ranges) const {
  size_t max_size = 0;

  for (auto &r : ranges) {
    if (r.size > max_size)
      max_size = r.size;
  }

  size_t size_per_block = std::min(max_size, max_size_per_block_);

  size_t num_blocks = 0;
  for (auto &r : ranges)
    num_blocks += div_ceil(r.size, size_per_block);

  return { num_blocks, size_per_block };
}


template <typename BlockCollection>
void ScatterGatherBase::MakeBlocks(BlockCollection &blocks,
                                   const std::vector<CopyRange> &ranges,
                                   size_t size_per_block) {
  auto nblocks = dali::size(blocks);
  decltype(nblocks) b = 0;

  for (auto &r : ranges) {
    for (size_t ofs = 0; ofs < r.size; ofs += size_per_block) {
      assert(b < nblocks);
      blocks[b++] = { r.src + ofs, r.dst + ofs, std::min(r.size - ofs, size_per_block) };
    }
  }
  assert(b == nblocks);
}

void ScatterGatherCPU::MakeBlocks(size_t blocks_lower_limit) {
  size_t heap_size = std::max(blocks_lower_limit, ranges_.size());

  heap_.clear();
  heap_.reserve(heap_size);
  heap_ = ranges_;

  auto cmp = [](CopyRange &left, CopyRange &right) { return left.size < right.size; };

  std::make_heap(heap_.begin(), heap_.end(), cmp);

  while (heap_.size() < blocks_lower_limit) {
    // Take out the larges
    std::pop_heap(heap_.begin(), heap_.end(), cmp);
    CopyRange largest = heap_.back();
    heap_.pop_back();

    // Split the range into halfs
    CopyRange first_half = {largest.src, largest.dst, largest.size / 2};
    CopyRange second_half = {largest.src + first_half.size, largest.dst + first_half.size,
                             largest.size - first_half.size};

    // Add back to the heap
    heap_.push_back(first_half);
    std::push_heap(heap_.begin(), heap_.end(), cmp);
    heap_.push_back(second_half);
    std::push_heap(heap_.begin(), heap_.end(), cmp);
  }

  size_t num_blocks, size_per_block;
  std::tie(num_blocks, size_per_block) = BlockCountAndSize(heap_);
  blocks_.resize(num_blocks);
  ScatterGatherBase::MakeBlocks(blocks_, heap_, size_per_block);
}

__global__ void BatchCopy(const ScatterGatherBase::CopyRange *ranges) {
  auto range = ranges[blockIdx.x];

  for (size_t i = threadIdx.x; i < range.size; i += blockDim.x) {
    range.dst[i] = range.src[i];
  }
}

constexpr int kMaxRangesByVal = 2048 / sizeof(ScatterGatherBase::CopyRange);

struct CopyRanges {
  ScatterGatherBase::CopyRange ranges[kMaxRangesByVal];
};

__global__ void BatchCopy(CopyRanges ranges) {
  auto range = ranges.ranges[blockIdx.x];

  for (size_t i = threadIdx.x; i < range.size; i += blockDim.x) {
    range.dst[i] = range.src[i];
  }
}

void ScatterGatherGPU::Run(hipStream_t stream, bool reset, ScatterGatherGPU::Method method,
                           hipMemcpyKind memcpyKind) {
  Coalesce();

  // TODO(michalz): Error handling

  bool use_memcpy = (method == ScatterGatherGPU::Method::Memcpy) ||
    (method == ScatterGatherGPU::Method::Default && ranges_.size() <= 2);

  if (use_memcpy) {
    for (auto &r : ranges_) {
      CUDA_CALL(hipMemcpyAsync(r.dst, r.src, r.size, memcpyKind, stream));
    }
  } else {
    size_t num_blocks, size_per_block;
    std::tie(num_blocks, size_per_block) = BlockCountAndSize(ranges_);
    if (num_blocks > kMaxRangesByVal) {
      kernels::DynamicScratchpad scratchpad({}, stream);
      auto *blocks_pinned = scratchpad.Allocate<mm::memory_kind::pinned, CopyRange>(num_blocks);
      auto blocks = make_span(blocks_pinned, num_blocks);
      ScatterGatherBase::MakeBlocks(blocks, ranges_, size_per_block);
      auto *blocks_dev = scratchpad.ToGPU(stream, blocks);

      dim3 grid(blocks.size());
      dim3 block(std::min<size_t>(size_per_block, 1024));
      BatchCopy<<<grid, block, 0, stream>>>(blocks_dev);
    } else {
      CopyRanges ranges = {};
      auto blocks = make_span(ranges.ranges, num_blocks);
      ScatterGatherBase::MakeBlocks(blocks, ranges_, size_per_block);

      dim3 grid(blocks.size());
      dim3 block(std::min<size_t>(size_per_block, 1024));
      BatchCopy<<<grid, block, 0, stream>>>(ranges);
    }
    CUDA_CALL(hipGetLastError());
  }

  if (reset)
    Reset();
}

}  // namespace kernels
}  // namespace dali
