#include "hip/hip_runtime.h"
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gtest/gtest.h>
#include <vector>
#include <tuple>
#include "dali/core/geom/mat.h"
#include "dali/kernels/algebra/linear_transformation_gpu.h"
#include "dali/kernels/scratch.h"
#include "dali/kernels/tensor_shape.h"
#include "dali/kernels/common/copy.h"
#include "dali/kernels/test/tensor_test_utils.h"
#include "dali/kernels/test/kernel_test_utils.h"

namespace dali {
namespace kernels {
namespace linear_transformation {
namespace test {


namespace {

/**
 * Rounding to nearest even (like GPU does it)
 */
template <class Out>
std::enable_if_t<std::is_integral<Out>::value, Out> custom_round(float val) {
  return static_cast<Out>(std::nearbyint(val));
}


template <class Out>
std::enable_if_t<!std::is_integral<Out>::value, Out> custom_round(float val) {
  return val;
}


constexpr int kNDims = 3;
constexpr int kNChannelsIn = 5;
constexpr int kNChannelsOut = 2;


}  // namespace

template <class InputOutputTypes>
class LinearTransformationGpuTest : public ::testing::Test {
  using In = typename InputOutputTypes::In;
  using Out = typename InputOutputTypes::Out;

 public:
  LinearTransformationGpuTest() {
    input_host_.resize(dataset_size(in_shapes_));
  }


  void SetUp() final {
    std::mt19937_64 rng;
    UniformRandomFill(input_host_, rng, 0., 10.);
    calc_output();
    CUDA_CALL(hipMalloc(&input_device_, sizeof(In) * dataset_size(in_shapes_)));
    CUDA_CALL(hipMemcpy(input_device_, input_host_.data(), input_host_.size() * sizeof(In),
                         hipMemcpyDefault));
    CUDA_CALL(hipMalloc(&output_, dataset_size(out_shapes_) * sizeof(Out)));
    hipDeviceSynchronize();
  }


  In *input_device_;
  Out *output_;
  std::vector<In> input_host_;
  std::vector<Out> ref_output_;
  std::vector<TensorShape<kNDims>> in_shapes_ = {{4, 3, kNChannelsIn}};
  std::vector<TensorShape<kNDims>> out_shapes_ = {{4, 3, kNChannelsOut}};
  mat<kNChannelsOut, kNChannelsIn, float> mat_{{{1, 2, 3, 4, 5}, {6, 7, 8, 9, 10}}};
  std::vector<mat<kNChannelsOut, kNChannelsIn, float>> vmat_ = {mat_};


  void calc_output() {
    for (size_t i = 0; i < input_host_.size(); i += kNChannelsIn) {
      for (size_t j = 0; j < kNChannelsOut; j++) {
        float res = 0;
        for (size_t k = 0; k < kNChannelsIn; k++) {
          res += input_host_[i + k] * mat_.at(j, k);
        }
        ref_output_.push_back(custom_round<Out>(res));
      }
    }
  }


  size_t dataset_size(const std::vector<TensorShape<kNDims>> &shapes) {
    int ret = 0;
    for (auto sh : shapes) {
      ret += volume(sh);
    }
    return ret;
  }
};

using TestTypes = std::tuple<uint8_t, int32_t>;
/* Cause the line below takes RIDICULOUSLY long time to compile */
// using TestTypes = std::tuple<uint8_t, int8_t, uint16_t, int16_t, int32_t, float>;

INPUT_OUTPUT_TYPED_TEST_SUITE(LinearTransformationGpuTest, TestTypes);

namespace {

template <class GtestTypeParam>
using TheKernel = LinearTransformationGpu<typename GtestTypeParam::Out, typename GtestTypeParam::In,
        kNChannelsOut, kNChannelsIn, kNDims - 1>;

}  // namespace


TYPED_TEST(LinearTransformationGpuTest, check_kernel) {
  check_kernel<TheKernel<TypeParam>>();
}


TYPED_TEST(LinearTransformationGpuTest, setup_test) {
  TheKernel<TypeParam> kernel;
  KernelContext ctx;
  InListGPU<typename TypeParam::In, kNDims> in(this->input_device_, this->in_shapes_);
  auto reqs = kernel.Setup(ctx, in, this->vmat_);
  ASSERT_EQ(this->out_shapes_.size(), static_cast<size_t>(reqs.output_shapes[0].num_samples()))
                        << "Kernel::Setup provides incorrect shape";
  for (size_t i = 0; i < this->out_shapes_.size(); i++) {
    EXPECT_EQ(this->out_shapes_[i], reqs.output_shapes[0][i])
                  << "Kernel::Setup provides incorrect shape";
  }
}


TYPED_TEST(LinearTransformationGpuTest, run_test) {
  TheKernel<TypeParam> kernel;
  KernelContext c;
  InListGPU<typename TypeParam::In, kNDims> in(this->input_device_, this->in_shapes_);
  OutListGPU<typename TypeParam::Out, kNDims> out(this->output_,
                                                  TensorListShape<kNDims>(this->out_shapes_));

  auto reqs = kernel.Setup(c, in, this->vmat_);

  ScratchpadAllocator sa;
  sa.Reserve(reqs.scratch_sizes);
  auto scratchpad = sa.GetScratchpad();
  c.scratchpad = &scratchpad;
  kernel.Run(c, out, in, this->vmat_);
  hipDeviceSynchronize();

  auto res = copy<AllocType::Host>(out[0]);
  ASSERT_EQ(static_cast<int>(this->ref_output_.size()), res.first.num_elements());
  for (size_t i = 0; i < this->ref_output_.size(); i++) {
    EXPECT_EQ(this->ref_output_[i], res.second.get()[i]) << "Failed for index " << i;
  }
}


namespace {

template <int ndims>
bool cmp_shapes(const TensorShape<ndims> &lhs, ivec<ndims - 1> rhs) {
  std::reverse(rhs.begin(), rhs.end());
  for (size_t i = 0; i < rhs.size(); i++) {
    if (lhs[i] != rhs[i]) return false;
  }
  return true;
}

}  // namespace

TYPED_TEST(LinearTransformationGpuTest, sample_descriptors) {
  using In = typename TypeParam::In;
  using Out = typename TypeParam::Out;

  InListGPU<In, kNDims> in(this->input_device_, this->in_shapes_);
  OutListGPU<Out, kNDims> out(this->output_, TensorListShape<3>(this->out_shapes_));

  auto res = detail::CreateSampleDescriptors
          <Out, In, kNChannelsOut, kNChannelsIn, kNDims - 1>(out, in, this->vmat_);

  EXPECT_EQ(this->input_device_, res[0].in);
  EXPECT_EQ(this->output_, res[0].out);
  EXPECT_TRUE(cmp_shapes<kNDims>(this->in_shapes_[0], res[0].in_size));
  EXPECT_TRUE(cmp_shapes<kNDims>(this->out_shapes_[0], res[0].out_size));
  EXPECT_EQ(this->in_shapes_[0].shape.back(), res[0].in_channels);
  EXPECT_EQ(this->out_shapes_[0].shape.back(), res[0].out_channels);
  EXPECT_EQ(this->vmat_[0], res[0].transformation_matrix);
}


}  // namespace test
}  // namespace linear_transformation
}  // namespace kernels
}  // namespace dali

