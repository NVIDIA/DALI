// Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gtest/gtest.h>
#include "dali/kernels/kernel.h"
#include "dali/core/static_switch.h"
#include "dali/kernels/type_tag.h"

namespace {
template <typename I, typename O, int C>
struct Functor {
  void operator()(int &calls, int i, int o, int c) {
    EXPECT_EQ(i, dali::TypeTag<I>::value);
    EXPECT_EQ(o, dali::TypeTag<O>::value);
    EXPECT_EQ(c, C);
    calls++;
  }
};

template <typename T>
struct StaticSwitch_NVCC : testing::Test {};
}  // namespace

typedef testing::Types<int, float, double> MyTypes;
TYPED_TEST_SUITE(StaticSwitch_NVCC, MyTypes);

TYPED_TEST(StaticSwitch_NVCC, TypeSwitch) {
  using T = gtest_TypeParam_;
  int tag = -1;
  TYPE_SWITCH(dali::TypeTag<T>::value, dali::TypeTag, IType, (int, float, double),
    (
      tag = dali::TypeTag<IType>::value;
      EXPECT_TRUE((std::is_same<IType, T>::value)) << "TypeSwitch type mismatch";
    ),  // NOLINT
    GTEST_FAIL() << "Invalid type";
  );  // NOLINT

  EXPECT_EQ(dali::TypeTag<T>::value, tag)
    << "Tag mismatch - did type switch actually call anything?";
}

TEST(StaticSwitch_NVCC, Nested) {
  int input_type = dali::TypeTag<float>();
  int output_type = dali::TypeTag<int64_t>();
  int calls = 0;
  for (int channels = 1; channels <= 4; channels++) {
    TYPE_SWITCH(input_type, dali::TypeTag, IType, (int, float, double, int64_t), (
        TYPE_SWITCH(output_type, dali::TypeTag, OType, (int, uint8_t, int64_t), (
            VALUE_SWITCH(channels, num_channels, (1, 2, 3, 4),
              (Functor<IType, OType, num_channels>()(calls, input_type, output_type, channels); ),
              FAIL() << "Unsupported value";
            )                                       // NOLINT
          ), FAIL() << "Unsupported output type";   // NOLINT
        )                                           // NOLINT
      ), FAIL() << "Unsupported input type";        // NOLINT
    )                                               // NOLINT
  }
  // check that the test functor was actually called
  EXPECT_EQ(calls, 4) << "Test functor was expected to be called 4 times";
}
