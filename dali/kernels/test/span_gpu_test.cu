#include "hip/hip_runtime.h"
// Copyright (c) 2019-2021, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gtest/gtest.h>
#include "dali/core/span.h"
#include "dali/core/mm/memory.h"
#include "dali/core/util.h"
#include "dali/core/cuda_error.h"

namespace dali {
namespace kernels {
namespace {

template <typename T, span_extent_t extent>
__global__ void TestSpanKernel(span<T, extent> span) {
  int x = threadIdx.x + blockIdx.x*blockDim.x;
  if (x < span.size()) {
    span[x] += 1 + x;
  }
}

}  // namespace

inline void Validate(span<const int> s) {
  int i = 1;
  for (auto a : s)
    EXPECT_EQ(a, i++);
  EXPECT_EQ(i, static_cast<int>(s.size() + 1));
}

TEST(Span, Convert) {
  int A[10];
  auto s = make_span(A);
  int i = 1;
  for (auto &a : s)
    a = i++;
  Validate(s);
}

TEST(TestGPUSpan, Test1) {
  const int N = 1000;
  int array[N], out[N];
  for (int i = 0; i < N; i++) {
    array[i] = i + 1;
  }

  dim3 block(32);
  dim3 grid = div_ceil(N, block.x);
  ASSERT_EQ(hipGetLastError(), hipSuccess);

  auto gpumem = mm::alloc_raw_unique<int, mm::memory_kind::device>(N);
  CUDA_CALL(hipMemcpy(gpumem.get(), array, sizeof(array), hipMemcpyHostToDevice));
  span<int> dyn_span = { gpumem.get(), N };
  TestSpanKernel<<<grid, block>>>(dyn_span);
  CUDA_CALL(hipMemcpy(out, gpumem.get(), sizeof(array), hipMemcpyDeviceToHost));
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  for (int i = 0; i < N; i++) {
    EXPECT_EQ(out[i], array[i] + 1 + i);
  }

  CUDA_CALL(hipMemcpy(gpumem.get(), array, sizeof(array), hipMemcpyHostToDevice));
  span<int, N> static_span = { gpumem.get() };
  TestSpanKernel<<<grid, block>>>(static_span);
  CUDA_CALL(hipMemcpy(out, gpumem.get(), sizeof(array), hipMemcpyDeviceToHost));
  ASSERT_EQ(hipGetLastError(), hipSuccess);

  for (int i = 0; i < N; i++) {
    EXPECT_EQ(out[i], array[i] + 1 + i);
  }
}

}  // namespace kernels
}  // namespace dali
