#include "hip/hip_runtime.h"
// Copyright (c) 2020-2022, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "dali/kernels/transpose/transpose_gpu_impl.cuh"   // NOLINT
#include "dali/kernels/transpose/transpose_gpu_setup.cuh"  // NOLINT
#include <gtest/gtest.h>
#include <algorithm>
#include <numeric>
#include <vector>
#include "dali/core/dev_buffer.h"
#include "dali/kernels/common/utils.h"
#include "dali/core/tensor_shape_print.h"
#include "dali/test/test_tensors.h"
#include "dali/core/cuda_event.h"
#include "dali/kernels/transpose/transpose_test.h"

namespace dali {
namespace kernels {

using namespace transpose_impl;  // NOLINT


TEST(SimplifyPermute, NoSimplification) {
  int64_t shape[] = { 2, 3, 4, 5 };
  int perm[] = { 0, 3, 2, 1 };
  TensorShape<> s_shape, ref_shape;
  SmallVector<int, 6> s_perm, ref_perm;
  SimplifyPermute(s_shape, s_perm, shape, perm, 4);
  ref_shape = { 2, 3, 4, 5 };
  ref_perm = { 0, 3, 2, 1 };
  EXPECT_EQ(s_shape, ref_shape);
  EXPECT_EQ(s_perm, ref_perm);
}

TEST(SimplifyPermute, CollapseUnitDims) {
  int64_t shape[] = { 2, 1, 3, 4, 1, 5 };
  int perm[] = { 0, 5, 1, 3, 2, 4 };
  TensorShape<> s_shape, ref_shape;
  SmallVector<int, 6> s_perm, ref_perm;
  SimplifyPermute(s_shape, s_perm, shape, perm, 6);
  ref_shape = { 2, 3, 4, 5 };
  ref_perm = { 0, 3, 2, 1 };
  EXPECT_EQ(s_shape, ref_shape);
  EXPECT_EQ(s_perm, ref_perm);
}

TEST(SimplifyPermute, Collapse) {
  int64_t shape[] = { 2, 1, 3, 4, 1, 5 };
  int perm[] = { 3, 4, 5, 0, 1, 2 };
  TensorShape<> s_shape, ref_shape;
  SmallVector<int, 6> s_perm, ref_perm;
  SimplifyPermute(s_shape, s_perm, shape, perm, 6);
  ref_shape = { 6, 20 };
  ref_perm = { 1, 0 };
  EXPECT_EQ(s_shape, ref_shape);
  EXPECT_EQ(s_perm, ref_perm);
}

TEST(TransposeGPU, GetTransposeMethod) {
  {
    TensorShape<> shape = { 640*480, 3 };
    int perm[] = { 1, 0 };
    EXPECT_EQ(GetTransposeMethod(shape.data(), perm, 2, sizeof(int)),
              TransposeMethod::Deinterleave);
  }
  {
    TensorShape<> shape = { 3, 640*480 };
    int perm[] = { 1, 0 };  // interleave
    EXPECT_EQ(GetTransposeMethod(shape.data(), perm, 2, sizeof(int)),
              TransposeMethod::Interleave);
  }
  {
    TensorShape<> shape = { 640, 480 };
    int perm[] = { 1, 0 };  // scalar tiled
    EXPECT_EQ(GetTransposeMethod(shape.data(), perm, 2, sizeof(int)),
              TransposeMethod::Tiled);
  }
  {
    TensorShape<> shape = { 20, 640, 480 };
    int perm[] = { 1, 2, 0 };  // scalar tiled
    EXPECT_EQ(GetTransposeMethod(shape.data(), perm, 3, sizeof(int)),
              TransposeMethod::Tiled);
  }
  {
    TensorShape<> shape = { 640, 480, 3 };
    int perm[] = { 1, 0, 2 };  // vectorized tiled
    EXPECT_EQ(GetTransposeMethod(shape.data(), perm, 3, sizeof(int)),
              TransposeMethod::Tiled);
  }
  {
    TensorShape<> shape = { 640, 3, 480 };
    int perm[] = { 1, 2, 0 };  // some mess
    EXPECT_EQ(GetTransposeMethod(shape.data(), perm, 3, sizeof(int)),
              TransposeMethod::Generic);
  }
  {
    TensorShape<> shape = { 640, 480, 50 };
    int perm[] = { 1, 0, 2 };  // generic stuff
    EXPECT_EQ(GetTransposeMethod(shape.data(), perm, 3, sizeof(int)),
              TransposeMethod::Generic);
  }
  {
    TensorShape<> shape = { 640*480 };
    int perm[] = { 0 };  // identity
    EXPECT_EQ(GetTransposeMethod(shape.data(), perm, 1, sizeof(int)),
              TransposeMethod::Copy);
  }
}

TEST(TransposeTiled, AllPerm4DInnermost) {
  TensorShape<> shape = { 19, 57, 37, 53 };  // a bunch of primes, just to make it harder
  int size = volume(shape);
  vector<int> in_cpu(size), out_cpu(size), ref(size);
  std::iota(in_cpu.begin(), in_cpu.end(), 0);
  DeviceBuffer<int> in_gpu, out_gpu;
  in_gpu.resize(size);
  out_gpu.resize(size);
  copyH2D(in_gpu.data(), in_cpu.data(), size);
  auto start = CUDAEvent::CreateWithFlags(0);
  auto end = CUDAEvent::CreateWithFlags(0);

  int grid_size = std::max(1, size / 512);
  ASSERT_LT(grid_size * 512, size) << "Weak test error: Grid too large to test grid loop";

  for (auto &perm : testing::Permutations4) {
    if (perm[3] == 3)
      continue;  // innermost dim must be permuted

    std::cerr << "Testing permutation "
      << perm[0] << " " << perm[1] << " " << perm[2] << " " << perm[3] << "\n";
    CUDA_CALL(hipMemset(out_gpu, 0xff, size*sizeof(*in_gpu.data())));

    TiledTransposeDesc<int> desc;
    memset(&desc, 0xCC, sizeof(desc));
    InitTiledTranspose(desc, shape, make_span(perm), out_gpu, in_gpu, grid_size);
    CUDA_CALL(hipEventRecord(start));
    TransposeTiledSingle<<<grid_size, dim3(32, 16), kTiledTransposeMaxSharedMem>>>(desc);
    CUDA_CALL(hipEventRecord(end));
    copyD2H(out_cpu.data(), out_gpu.data(), size);
    testing::RefTranspose(ref.data(), in_cpu.data(), shape.data(), perm, 4);

    float time;
    CUDA_CALL(hipEventElapsedTime(&time, start, end));
    time *= 1e+6;
    std::cerr << 2*size*sizeof(*in_gpu.data()) / time << " GB/s" << "\n";

    for (int i = 0; i < size; i++) {
      ASSERT_EQ(out_cpu[i], ref[i]) << " at " << i;
    }
  }
}

TEST(TransposeTiled, BuildDescVectorized) {
  TensorShape<> shape = { 57, 37, 53, 4 };  // a bunch of primes, just to make it harder
  int size = volume(shape);
  vector<int> in_cpu(size), out_cpu(size), ref(size);
  std::iota(in_cpu.begin(), in_cpu.end(), 0);
  DeviceBuffer<int> in_gpu, out_gpu;
  in_gpu.resize(size);
  out_gpu.resize(size);
  CUDA_CALL(hipMemset(out_gpu, 0xff, size*sizeof(*in_gpu.data())));
  copyH2D(in_gpu.data(), in_cpu.data(), size);

  SmallVector<int, 6> perm = { 1, 2, 0, 3 };

  int grid_size = 1024;
  TiledTransposeDesc<int> desc;
  memset(&desc, 0xCC, sizeof(desc));
  InitTiledTranspose(desc, shape, make_span(perm), out_gpu, in_gpu, grid_size);
  EXPECT_EQ(desc.lanes, 4) << "Lanes not detected";
  EXPECT_EQ(desc.ndim, 3) << "Number of dimensions should have shrunk in favor of lanes";
  TransposeTiledSingle<<<grid_size, dim3(32, 16), kTiledTransposeMaxSharedMem>>>(desc);
  copyD2H(out_cpu.data(), out_gpu.data(), size);
  testing::RefTranspose(ref.data(), in_cpu.data(), shape.data(), perm.data(), perm.size());

  for (int i = 0; i < size; i++) {
    ASSERT_EQ(out_cpu[i], ref[i]) << " at " << i;
  }
}

TEST(TransposeTiled, BuildDescAndForceMisalignment) {
  TensorShape<> shape = { 57, 37, 52, 4 };  // a bunch of primes, just to make it harder
  int size = volume(shape);
  vector<uint8> in_cpu(size + 4), out_cpu(size + 4);
  vector<uint8> ref(size + 4);

  DeviceBuffer<uint8> in_gpu, out_gpu;
  in_gpu.resize(size + 4);
  out_gpu.resize(size + 4);

  for (uintptr_t offset = 0; offset < 4; offset++) {
    std::iota(in_cpu.begin(), in_cpu.end(), 0);
    CUDA_CALL(hipMemset(out_gpu, 0xff, size*sizeof(*in_gpu.data())));

    copyH2D(in_gpu.data() + offset, in_cpu.data(), size);

    SmallVector<int, 6> perm = { 1, 2, 0, 3 };

    int grid_size = 1024;
    TiledTransposeDesc<uint8> desc;
    memset(&desc, 0xCC, sizeof(desc));
    InitTiledTranspose(desc, shape, make_span(perm), out_gpu.data() + offset,
                       in_gpu.data() + offset, grid_size);
    EXPECT_EQ(desc.lanes, 4) << "Lanes not detected";
    EXPECT_EQ(desc.ndim, 3) << "Number of dimensions should have shrunk in favor of lanes";

    TransposeTiledSingle<<<grid_size, dim3(32, 16), kTiledTransposeMaxSharedMem>>>(desc);
    copyD2H(out_cpu.data(), out_gpu.data() + offset, size);
    testing::RefTranspose(ref.data(), in_cpu.data(), shape.data(), perm.data(), perm.size());

    for (int i = 0; i < size; i++) {
      ASSERT_EQ(out_cpu[i], ref[i]) << " at " << i;
    }
  }
}

TEST(TransposeTiled, BuildDescVectorized16BitOpt) {
  TensorShape<> shape = { 57, 37, 53, 4 };  // a bunch of primes, just to make it harder
  int size = volume(shape);
  vector<uint16_t> in_cpu(size), out_cpu(size);
  vector<uint16_t> ref(size);
  std::iota(in_cpu.begin(), in_cpu.end(), 0);
  DeviceBuffer<uint16_t> in_gpu, out_gpu;
  in_gpu.resize(size);
  out_gpu.resize(size);
  CUDA_CALL(hipMemset(out_gpu, 0xff, size*sizeof(*in_gpu.data())));
  copyH2D(in_gpu.data(), in_cpu.data(), size);

  SmallVector<int, 6> perm = { 1, 2, 0, 3 };

  int grid_size = 1024;
  TiledTransposeDesc<uint16_t> desc;
  memset(&desc, 0xCC, sizeof(desc));
  InitTiledTranspose(desc, shape, make_span(perm), out_gpu, in_gpu, grid_size);
  EXPECT_EQ(desc.lanes, 4) << "Lanes not detected";
  EXPECT_EQ(desc.ndim, 3) << "Number of dimensions should have shrunk in favor of lanes";

  TransposeTiledSingle<<<grid_size, dim3(32, 16), kTiledTransposeMaxSharedMem>>>(desc);
  copyD2H(out_cpu.data(), out_gpu.data(), size);
  testing::RefTranspose(ref.data(), in_cpu.data(), shape.data(), perm.data(), perm.size());

  for (int i = 0; i < size; i++) {
    ASSERT_EQ(out_cpu[i], ref[i]) << " at " << i;
  }
}

TEST(TransposeTiled, HighDimensionTest) {
  TensorShape<> shape = {3, 3, 5, 7, 23, 3, 37, 4 };  // a bunch of primes, just to make it harder
  int size = volume(shape);
  vector<uint8> in_cpu(size), out_cpu(size);
  vector<uint8> ref(size);

  DeviceBuffer<uint8> in_gpu, out_gpu;
  in_gpu.resize(size);
  out_gpu.resize(size);

  for (int size_of_last_dim = 1; size_of_last_dim <= 4; size_of_last_dim++) {
    shape = { 3, 3, 5, 7, 23, 3, 37, size_of_last_dim };
    size = volume(shape);
    std::iota(in_cpu.begin(), in_cpu.end(), 0);
    CUDA_CALL(hipMemset(out_gpu, 0xff, size*sizeof(*in_gpu.data())));

    copyH2D(in_gpu.data(), in_cpu.data(), size);

    SmallVector<int, 8> perm = { 1, 0, 4, 2, 6, 3, 5, 7 };

    int grid_size = 1024;
    TiledTransposeDesc<uint8> desc;
    memset(&desc, 0xCC, sizeof(desc));
    InitTiledTranspose(desc, shape, make_span(perm), out_gpu.data(), in_gpu.data(), grid_size);

    TransposeTiledSingle<<<grid_size, dim3(32, 16), kTiledTransposeMaxSharedMem>>>(desc);
    copyD2H(out_cpu.data(), out_gpu.data(), size);
    testing::RefTranspose(ref.data(), in_cpu.data(), shape.data(), perm.data(), perm.size());

    for (int i = 0; i < size; i++) {
      ASSERT_EQ(out_cpu[i], ref[i]) << " at " << i;
    }
  }
}

TEST(TransposeDeinterleave, AllPerm4DInnermost) {
  int channels = 3;
  TensorShape<> shape = { 19, 157, 137, channels };  // small inner dimension
  int size = volume(shape);
  vector<int> in_cpu(size), out_cpu(size), ref(size);
  std::iota(in_cpu.begin(), in_cpu.end(), 0);
  DeviceBuffer<int> in_gpu, out_gpu;
  in_gpu.resize(size);
  out_gpu.resize(size);
  copyH2D(in_gpu.data(), in_cpu.data(), size);

  int block_size = 256;
  int grid_size = std::max(1, size / (block_size * channels));
  ASSERT_LT(grid_size * block_size * channels, size)
      << "Weak test error: Grid too large to test grid loop";

  auto start = CUDAEvent::CreateWithFlags(0);
  auto end = CUDAEvent::CreateWithFlags(0);

  for (auto &perm : testing::Permutations4) {
    if (perm[3] == 3)
      continue;  // innermost dim must be permuted

    std::cerr << "Testing permutation "
      << perm[0] << " " << perm[1] << " " << perm[2] << " " << perm[3] << "\n";
      CUDA_CALL(hipMemset(out_gpu, 0xff, size*sizeof(*in_gpu.data())));

    DeinterleaveDesc<int> desc;
    memset(&desc, 0xCC, sizeof(desc));
    InitDeinterleave(desc, shape, make_span(perm), out_gpu, in_gpu);
    CUDA_CALL(hipEventRecord(start));
    TransposeDeinterleaveSingle<<<grid_size, block_size>>>(desc);
    CUDA_CALL(hipEventRecord(end));
    copyD2H(out_cpu.data(), out_gpu.data(), size);
    testing::RefTranspose(ref.data(), in_cpu.data(), shape.data(), perm, 4);

    float time;
    CUDA_CALL(hipEventElapsedTime(&time, start, end));
    time *= 1e+6;
    std::cerr << 2*size*sizeof(*in_gpu.data()) / time << " GB/s" << "\n";


    for (int i = 0; i < size; i++) {
      ASSERT_EQ(out_cpu[i], ref[i]) << " at " << i;
    }
  }
}

TEST(TransposeGeneric, AllPerm4D) {
  TensorShape<> shape = { 31, 43, 53, 47 };
  int size = volume(shape);
  vector<int> in_cpu(size), out_cpu(size), ref(size);
  std::iota(in_cpu.begin(), in_cpu.end(), 0);
  DeviceBuffer<int> in_gpu, out_gpu;
  in_gpu.resize(size);
  out_gpu.resize(size);
  copyH2D(in_gpu.data(), in_cpu.data(), size);

  int grid_size = 2048;
  int block_size = 256;
  ASSERT_LT(grid_size * block_size, size) << "Weak test error: Grid too large to test grid loop";

  for (auto &perm : testing::Permutations4) {
    std::cerr << "Testing permutation "
      << perm[0] << " " << perm[1] << " " << perm[2] << " " << perm[3] << "  input shape "
      << shape << "\n";
      CUDA_CALL(hipMemset(out_gpu, 0xff, size*sizeof(*in_gpu.data())));

    GenericTransposeDesc<int> desc;
    memset(&desc, 0xCC, sizeof(desc));
    InitGenericTranspose(desc, shape, make_span(perm), out_gpu, in_gpu);
    TransposeGenericSingle<<<grid_size, block_size>>>(desc);
    copyD2H(out_cpu.data(), out_gpu.data(), size);

    testing::RefTranspose(ref.data(), in_cpu.data(), shape.data(), perm, 4);

    for (int i = 0; i < size; i++) {
      ASSERT_EQ(out_cpu[i], ref[i]) << " at " << i;
    }

    TensorShape<> simplified_shape;
    SmallVector<int, 6> simplified_perm;
    SimplifyPermute(simplified_shape, simplified_perm, shape.data(), perm, 4);

    if (simplified_shape == shape) {
      for (int i = 0; i < 4; i++) {
        ASSERT_EQ(simplified_perm[i], perm[i]);
      }
      // no simplification, don't repeat the test
      continue;
    }

    std::cerr << "Testing permutation ";
    for (auto i : simplified_perm)
      std::cerr << i << " ";
    std::cerr << " input shape " << simplified_shape << "\n";

    memset(&desc, 0xCC, sizeof(desc));
    CUDA_CALL(hipMemset(out_gpu, 0xff, size*sizeof(*in_gpu.data())));
    InitGenericTranspose(desc, simplified_shape, make_span(simplified_perm), out_gpu, in_gpu);
    TransposeGenericSingle<<<grid_size, block_size>>>(desc);
    copyD2H(out_cpu.data(), out_gpu.data(), size);

    for (int i = 0; i < size; i++) {
      ASSERT_EQ(out_cpu[i], ref[i]) << " at " << i;
    }
  }
}

}  // namespace kernels
}  // namespace dali
