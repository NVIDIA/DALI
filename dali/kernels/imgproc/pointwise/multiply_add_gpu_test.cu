#include "hip/hip_runtime.h"
// Copyright (c) 2019-2021, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gtest/gtest.h>
#include <vector>
#include <tuple>
#include "dali/kernels/scratch.h"
#include "dali/core/tensor_shape.h"
#include "dali/kernels/common/copy.h"
#include "dali/test/tensor_test_utils.h"
#include "dali/kernels/test/kernel_test_utils.h"
#include "dali/kernels/imgproc/pointwise/multiply_add_gpu.h"

namespace dali {
namespace kernels {
namespace multiply_add {
namespace test {

namespace {

constexpr size_t kNdims = 3;


/**
 * Rounding to nearest even (like GPU does it)
 */
template <class In, class Out>
std::enable_if_t<std::is_integral<Out>::value, Out> custom_round(float val) {
  return static_cast<Out>(std::nearbyint(val));
}


template <class In, class Out>
std::enable_if_t<!std::is_integral<Out>::value, Out> custom_round(float val) {
  return val;
}


}  // namespace

template <class InputOutputTypes>
class MultiplyAddGpuTest : public ::testing::Test {
  using In = typename InputOutputTypes::In;
  using Out = typename InputOutputTypes::Out;

 public:
  MultiplyAddGpuTest() {
    input_host_.resize(dataset_size());
  }


  void SetUp() final {
    std::mt19937_64 rng;
    UniformRandomFill(input_host_, rng, 0., 10.);
    calc_output(0);
    CUDA_CALL(hipMalloc(&input_device_, sizeof(In) * dataset_size()));
    CUDA_CALL(hipMemcpy(input_device_, input_host_.data(), input_host_.size() * sizeof(In),
                         hipMemcpyDefault));
    CUDA_CALL(hipMalloc(&output_, dataset_size() * sizeof(Out)));
    CUDA_CALL(hipDeviceSynchronize());

    verify_test();
  }


  In *input_device_;
  Out *output_;
  std::vector<In> input_host_;
  std::vector<Out> ref_output_;
  std::vector<TensorShape<kNdims>> shapes_ = {{480, 640, 3}};
  std::vector<float> addends_ = {4};
  std::vector<float> multipliers_ = {3};


  void verify_test() {
    assert(shapes_.size() == addends_.size());
    assert(addends_.size() == multipliers_.size());
    assert(dataset_size() == input_host_.size());
    assert(dataset_size() == ref_output_.size());
  }


  void calc_output(int idx) {
    for (auto in : input_host_) {
      ref_output_.push_back(custom_round<In, Out>(in * multipliers_[idx] + addends_[idx]));
    }
  }


  size_t dataset_size() {
    int ret = 0;
    for (auto sh : shapes_) {
      ret += volume(sh);
    }
    return ret;
  }
};

using TestTypes = std::tuple<int8_t, float>;
/* Cause the line below takes RIDICULOUSLY long time to compile */
// using TestTypes = std::tuple<uint8_t, int8_t, uint16_t, int16_t, int32_t, float>;

INPUT_OUTPUT_TYPED_TEST_SUITE(MultiplyAddGpuTest, TestTypes);

namespace {

template <class GtestTypeParam>
using TheKernel = MultiplyAddGpu
        <typename GtestTypeParam::Out, typename GtestTypeParam::In, kNdims>;

}  // namespace


TYPED_TEST(MultiplyAddGpuTest, check_kernel) {
  check_kernel<TheKernel<TypeParam>>();
}


TYPED_TEST(MultiplyAddGpuTest, setup_test) {
  TheKernel<TypeParam> kernel;
  KernelContext ctx;
  InListGPU<typename TypeParam::In, kNdims> in(this->input_device_, this->shapes_);
  auto reqs = kernel.Setup(ctx, in, this->addends_, this->multipliers_);
  ASSERT_EQ(this->shapes_.size(), static_cast<size_t>(reqs.output_shapes[0].num_samples()))
                        << "Kernel::Setup provides incorrect shape";
  for (size_t i = 0; i < this->shapes_.size(); i++) {
    EXPECT_EQ(this->shapes_[i], reqs.output_shapes[0][i])
                  << "Kernel::Setup provides incorrect shape";
  }
}


TYPED_TEST(MultiplyAddGpuTest, run_test) {
  TheKernel<TypeParam> kernel;
  KernelContext c;
  InListGPU<typename TypeParam::In, kNdims> in(this->input_device_, this->shapes_);
  OutListGPU<typename TypeParam::Out, kNdims> out(this->output_,
                                                  TensorListShape<kNdims>(this->shapes_));

  auto reqs = kernel.Setup(c, in, this->addends_, this->multipliers_);

  ScratchpadAllocator sa;
  sa.Reserve(reqs.scratch_sizes);
  auto scratchpad = sa.GetScratchpad();
  c.scratchpad = &scratchpad;
  kernel.Run(c, out, in, this->addends_, this->multipliers_);
  CUDA_CALL(hipDeviceSynchronize());

  auto res = copy<mm::memory_kind::host>(out[0]);
  ASSERT_EQ(static_cast<int>(this->ref_output_.size()), res.first.num_elements());
  for (size_t i = 0; i < this->ref_output_.size(); i++) {
    EXPECT_FLOAT_EQ(this->ref_output_[i], res.second.get()[i]) << "Failed for index " << i;
  }
}


TYPED_TEST(MultiplyAddGpuTest, sample_descriptors) {
  using InType = typename TypeParam::In;
  using OutType = typename TypeParam::Out;
  InListGPU<InType, kNdims> in(this->input_device_, this->shapes_);
  OutListGPU<OutType, kNdims> out(this->output_, TensorListShape<3>(this->shapes_));
  std::vector<SampleDescriptor<OutType, InType, kNdims-1>> res(in.num_samples());
  CreateSampleDescriptors(make_span(res), out, in, this->addends_, this->multipliers_);
  EXPECT_EQ(this->input_device_, res[0].in);
  EXPECT_EQ(this->output_, res[0].out);
  ivec<kNdims - 2> ref_pitch = {1920};
  EXPECT_EQ(ref_pitch, res[0].in_pitch);
  EXPECT_EQ(ref_pitch, res[0].out_pitch);
  EXPECT_EQ(this->addends_[0], res[0].addend);
  EXPECT_EQ(this->multipliers_[0], res[0].multiplier);
}


}  // namespace test
}  // namespace multiply_add
}  // namespace kernels
}  // namespace dali
