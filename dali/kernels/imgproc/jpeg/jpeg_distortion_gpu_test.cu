#include "hip/hip_runtime.h"
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime_api.h>
#include <gtest/gtest.h>
#include <cstdlib>
#include <opencv2/opencv.hpp>
#include <string>
#include <vector>
#include "dali/core/cuda_event.h"
#include "dali/core/hip_stream.h"
#include "dali/kernels/imgproc/jpeg/jpeg_distortion_gpu.cuh"
#include "dali/kernels/test/kernel_test_utils.h"
#include "dali/pipeline/data/tensor_list.h"
#include "dali/test/dali_test_config.h"
#include "dali/test/tensor_test_utils.h"
#include "dali/test/test_tensors.h"
#include "dali/util/image.h"

#define DEBUG_LOGS 0

namespace dali {
namespace kernels {
namespace test {

using KernelPtr = void(*)(const SampleDesc *, const kernels::BlockDesc<2> *);
using testing::dali_extra_path;

cv::Mat rgb2bgr(const cv::Mat& img) {
  cv::Mat bgr;
  cv::cvtColor(img, bgr, cv::COLOR_RGB2BGR);
  return bgr;
}

cv::Mat bgr2rgb(const cv::Mat& img) {
  return rgb2bgr(img);
}

template <typename GTestParams>
class JpegDistortionTestGPU : public ::testing::Test {
  using T = typename GTestParams::T;
  static constexpr bool vert_subsample = GTestParams::vert_subsample;
  static constexpr bool horz_subsample = GTestParams::horz_subsample;
  static constexpr bool use_real_images = true;
  static constexpr bool perf_run = false;
  static constexpr bool dump_images = false;

 public:
  void SetUp() final {
    if (use_real_images) {
      std::vector<std::string> paths = ImageList(
        testing::dali_extra_path() + "/db/single/bmp", {".bmp"}, 3);
      in_shapes_.resize(paths.size(), 3);
      std::vector<cv::Mat> images(paths.size());
      for (size_t i = 0; i < paths.size(); i++) {
        images[i] = bgr2rgb(cv::imread(paths[i]));
        TensorShape<> sh{images[i].rows, images[i].cols, images[i].channels()};
        in_shapes_.set_tensor_shape(i, sh);
      }
      in_.reshape(in_shapes_);
      auto in_cpu = in_.cpu();
      for (size_t i = 0; i < paths.size(); i++) {
        std::memcpy(in_cpu[i].data, images[i].data,
                    in_shapes_.tensor_size(i) * sizeof(uint8_t));
      }
    } else {
      if (perf_run) {
        in_shapes_ = uniform_list_shape(64, TensorShape<3>{600, 800, 3});
      } else {
        in_shapes_ = {{7, 9, 3}, {8, 16, 3}};
      }
      in_.reshape(in_shapes_);
      std::mt19937_64 rng;
      UniformRandomFill(in_.cpu(), rng, 0., 255.);
    }

    out_.reshape(in_shapes_);
  }

  void TearDown() final {
  }

  void TestKernel(KernelPtr kernel_fn) {
    CUDAStream stream = CUDAStream::Create(true);

    TensorListShape<2> chroma_shape(in_shapes_.size(), 2);
    std::vector<SampleDesc> samples_cpu;
    samples_cpu.resize(in_shapes_.size());

    auto in_view_gpu = in_.gpu(stream);
    auto out_view_gpu = out_.gpu(stream);
    out_.invalidate_cpu();
    CUDA_CALL(hipStreamSynchronize(stream));

    for (int i = 0; i < in_shapes_.size(); i++) {
      auto &sample_desc = samples_cpu[i];
      const auto& in_sh = in_shapes_[i];
      auto chroma_sh = chroma_shape.tensor_shape_span(i);
      auto shape_vol = volume(in_sh);
      auto width = in_sh[1];
      auto height = in_sh[0];
      auto chroma_width = div_ceil(width, 1 + horz_subsample);
      auto chroma_height = div_ceil(height, 1 + vert_subsample);
      // used to generate logical blocks (one thread per chroma pixel)
      chroma_sh[0] = chroma_height;
      chroma_sh[1] = chroma_width;
      chroma_sh[2] = 3;

      sample_desc.in = in_view_gpu[i].data;
      sample_desc.out = out_view_gpu[i].data;
      sample_desc.size.x = width;
      sample_desc.size.y = height;
      sample_desc.strides.x = 3;
      sample_desc.strides.y = width * 3;
      sample_desc.luma_Q_table = luma_table_;
      sample_desc.chroma_Q_table = chroma_table_;
    }

    block_setup_.SetBlockDim(dim3(32, 16, 1));
    int xblock = 64*(2-horz_subsample);
    int yblock = 128;
    block_setup_.SetDefaultBlockSize({xblock, yblock});
    block_setup_.SetupBlocks(chroma_shape, true);
    auto blocks_cpu = block_setup_.Blocks();

    SampleDesc *samples_gpu;
    CUDA_CALL(hipMalloc(&samples_gpu, sizeof(SampleDesc) * samples_cpu.size()));
    CUDA_CALL(hipMemcpy(samples_gpu, samples_cpu.data(),
                         sizeof(SampleDesc) * samples_cpu.size(),
                         hipMemcpyDefault));
    BlockDesc *blocks_gpu;
    CUDA_CALL(hipMalloc(&blocks_gpu, sizeof(BlockDesc) * blocks_cpu.size()));
    CUDA_CALL(hipMemcpy(blocks_gpu, blocks_cpu.data(), sizeof(BlockDesc) * blocks_cpu.size(),
                         hipMemcpyDefault));

    dim3 grid_dim = block_setup_.GridDim();
    dim3 block_dim = block_setup_.BlockDim();
#if DEBUG_LOGS
    std::cout << "\ngrid dim " << grid_dim.x << " " << grid_dim.y << " " << grid_dim.z
    << "\nblock_dim " << block_dim.x << " " << block_dim.y << " " << block_dim.z << "\n";
    for (size_t i = 0; i < blocks_cpu.size(); i++) {
      auto &blk = blocks_cpu[i];
      std::cout << "block " << i << " sample idx " << blk.sample_idx
                << " from " << blk.start << " to " << blk.end << "\n";
    }
#endif
    CUDAEvent start = CUDAEvent::CreateWithFlags(0);
    CUDAEvent end = CUDAEvent::CreateWithFlags(0);

    if (perf_run)  // warm up
      kernel_fn<<<grid_dim, block_dim, 0, stream>>>(samples_gpu, blocks_gpu);

    CUDA_CALL(hipEventRecord(start, stream));

    kernel_fn<<<grid_dim, block_dim, 0, stream>>>(samples_gpu, blocks_gpu);
    CUDA_CALL(hipGetLastError());

    CUDA_CALL(hipEventRecord(end, stream));
    CUDA_CALL(hipStreamSynchronize(stream));

    if (perf_run) {
      float time = 0;
      CUDA_CALL(hipEventElapsedTime(&time, start, end));
      time *= 1e+6;  // to nanoseconds
      int64_t size = 2 * in_shapes_.num_elements() * sizeof(uint8_t);
      std::cerr << "Throughput: " << size / time << " GB/s\n";
    }

    auto out_view_cpu = out_.cpu(stream);
    CUDA_CALL(hipStreamSynchronize(stream));
    auto in_view_cpu = in_.cpu();
    auto out_ref_cpu = out_ref_.cpu();

    for (int i = 0; i < in_shapes_.size(); i++) {
      auto sh = in_shapes_[i];
      cv::Mat in_mat(sh[0], sh[1], CV_8UC3, static_cast<void *>(in_view_cpu[i].data));
      cv::Mat out_mat(sh[0], sh[1], CV_8UC3, static_cast<void *>(out_view_cpu[i].data));
      cv::Mat out_ref(sh[0], sh[1], CV_8UC3, static_cast<void *>(out_ref_cpu[i].data));
      cv::Mat diff;
      cv::absdiff(out_mat, out_ref, diff);

#if DEBUG_LOGS
      std::cout << "input:\n" << in_mat << "\n";
      std::cout << "output:\n" << out_mat << "\n";
      std::cout << "reference:\n" << out_ref << "\n";
      std::cout << "abs(output - reference):\n" << diff << "\n";
#endif

      if (dump_images) {
        std::stringstream ss1, ss2, ss3, ss4;
        ss1 << "jpeg_distortion_test_" << i << "_in.bmp";
        ss2 << "jpeg_distortion_test_" << i << "_out_ref.jpg";
        ss3 << "jpeg_distortion_test_" << i << "_out.bmp";
        ss4 << "jpeg_distortion_test_" << i << "_diff.bmp";
        cv::imwrite(ss1.str(), rgb2bgr(in_mat));
        cv::imwrite(ss2.str(), rgb2bgr(out_ref));
        cv::imwrite(ss3.str(), rgb2bgr(out_mat));
        cv::imwrite(ss4.str(), rgb2bgr(diff));
      }

      // Sanity check. Checking that the maximum pixel difference is small enough.
      // It is always best to compare the diff image visually to look for artifacts.
      auto mean = cv::mean(diff);
      double min_val, max_val;
      cv::minMaxLoc(diff, &min_val, &max_val);
      EXPECT_LE(max_val, max_abs_error_);
      for (int d = 0; d < 3; d++)
        EXPECT_LE(mean[d], max_avg_error_);
    }

    CUDA_CALL(hipFree(blocks_gpu));
    CUDA_CALL(hipFree(samples_gpu));
  }

  void CalcOut_ChromaSubsampleDistortion() {
    // simplest implementation for test purposes. First convert to YCbCr, then subsample
    out_ref_.reshape(in_shapes_);
    auto out_ref_view = out_ref_.cpu();

    std::vector<T> tmp_y;
    std::vector<T> tmp_cb;
    std::vector<T> tmp_cr;
    auto in_view = in_.cpu();
    for (int sample = 0; sample < in_shapes_.size(); sample++) {
      auto sh = in_shapes_[sample];
      int64_t npixels = sh[0] * sh[1];
      tmp_y.resize(npixels);
      tmp_cb.resize(npixels);
      tmp_cr.resize(npixels);
      for (int64_t i = 0; i < npixels; i++) {
        uint8_t r = in_view[sample].data[3*i];
        uint8_t g = in_view[sample].data[3*i + 1];
        uint8_t b = in_view[sample].data[3*i + 2];
        tmp_y[i]  =  0.29900000f * r + 0.58700000f * g + 0.11400000f * b;
        tmp_cb[i] = -0.16873589f * r - 0.33126411f * g + 0.50000000f * b + 128.0f;
        tmp_cr[i] =  0.50000000f * r - 0.41868759f * g - 0.08131241f * b + 128.0f;
      }

      auto subsample_f = [&](std::vector<T> &component) {
        for (int64_t y = 0; y < sh[0]; y+=(1 << vert_subsample)) {
          const bool edge_y = vert_subsample && (y == (sh[0] - 1));
          for (int64_t x = 0; x < sh[1]; x+=(1 << horz_subsample)) {
            const bool edge_x = horz_subsample && (x == (sh[1] - 1));
            size_t in_offset_1 = y * sh[1] + x;
            size_t in_offset_2 = !edge_x ? y * sh[1] + x + 1 : in_offset_1;
            size_t in_offset_3 = !edge_y ? (y + 1) * sh[1] + x : in_offset_1;
            size_t in_offset_4;
            if (edge_x && edge_y)  in_offset_4 = in_offset_1;
            else if (edge_x)       in_offset_4 = in_offset_3;
            else if (edge_y)       in_offset_4 = in_offset_2;
            else                   in_offset_4 = (y + 1) * sh[1] + x + 1;

            if (horz_subsample && vert_subsample) {
              T avg = ConvertSat<T>(0.25f * (component[in_offset_1] + component[in_offset_2] +
                                             component[in_offset_3] + component[in_offset_4]));
              component[in_offset_1] = avg;
              component[in_offset_2] = avg;
              component[in_offset_3] = avg;
              component[in_offset_4] = avg;
            } else if (horz_subsample) {
              T avg = ConvertSat<T>(0.5f * (component[in_offset_1] + component[in_offset_2]));
              component[in_offset_1] = avg;
              component[in_offset_2] = avg;
            } else if (vert_subsample) {
              T avg = ConvertSat<T>(0.5f * (component[in_offset_1] + component[in_offset_3]));
              component[in_offset_1] = avg;
              component[in_offset_3] = avg;
            }
          }
        }
      };
      if (horz_subsample || vert_subsample) {
        subsample_f(tmp_cb);
        subsample_f(tmp_cr);
      }

      auto* sample_data = out_ref_view[sample].data;
      for (int64_t i = 0; i < npixels; i++) {
        float y = static_cast<float>(tmp_y[i]);
        float cb = static_cast<float>(tmp_cb[i]) - 128.0f;
        float cr = static_cast<float>(tmp_cr[i]) - 128.0f;
        auto r = ConvertSat<T>(y + 1.402f * cr);
        auto g = ConvertSat<T>(y - 0.34413629f * cb - 0.71413629f * cr);
        auto b = ConvertSat<T>(y + 1.772f * cb);
        sample_data[3*i]   = r;
        sample_data[3*i+1] = g;
        sample_data[3*i+2] = b;
      }
    }
  }

  void CalcOut_JpegCompressionDistortion() {
    out_ref_.reshape(in_shapes_);
    auto out_ref_view = out_ref_.cpu();
    auto in_view_cpu = in_.cpu();
    for (int i = 0; i < in_shapes_.size(); i++) {
      auto sh = in_shapes_[i];
      cv::Mat in_mat(sh[0], sh[1], CV_8UC3, static_cast<void *>(in_view_cpu[i].data));

      std::vector<uint8_t> encoded;
      cv::imencode(".jpg", rgb2bgr(in_mat), encoded,
                   {cv::IMWRITE_JPEG_QUALITY, ConvertSat<int>(quality_factor)});

      cv::Mat encoded_mat(1, encoded.size(), CV_8UC1, encoded.data());
      auto out_ref = bgr2rgb(cv::imdecode(encoded_mat, cv::IMREAD_COLOR));
      std::memcpy(out_ref_view[i].data, out_ref.data,
                  in_shapes_.tensor_size(i) * sizeof(uint8_t));
    }
  }

  void TestJpegCompressionDistortion(int quality) {
    quality_factor = quality;
    max_abs_error_ = vert_subsample && horz_subsample ? 80 : 128;
    max_avg_error_ = vert_subsample && horz_subsample ? 3 : 10;
    luma_table_ = GetLumaQuantizationTable(quality_factor);
    chroma_table_ = GetChromaQuantizationTable(quality_factor);
    CalcOut_JpegCompressionDistortion();
    TestKernel(JpegCompressionDistortion<horz_subsample, vert_subsample>);
  }

  void TestJpegCompressionDistortion_NoQuantization() {
    // Chroma subsampling + DCT + IDCT (no quantization step)
    CalcOut_ChromaSubsampleDistortion();
    TestKernel(JpegCompressionDistortion<horz_subsample, vert_subsample, false>);
  }

  void TestChromaSubsampleDistortion() {
    CalcOut_ChromaSubsampleDistortion();
    TestKernel(ChromaSubsampleDistortion<horz_subsample, vert_subsample>);
  }

  CUDAStream stream_;
  TensorListShape<> in_shapes_;
  TestTensorList<uint8_t> in_;
  TestTensorList<uint8_t>out_;
  TestTensorList<uint8_t> out_ref_;

  using BlkSetup = BlockSetup<2, -1>;
  BlkSetup block_setup_;
  using BlockDesc = BlkSetup::BlockDesc;

  float quality_factor = 20.0f;
  mat<8, 8, uint8_t> luma_table_;
  mat<8, 8, uint8_t> chroma_table_;

  int max_abs_error_ = 5;
  int max_avg_error_ = 3;
};

template <typename OutType, bool v, bool h>
struct jpeg_distortion_params_t {
  using T = OutType;
  static constexpr bool vert_subsample = v;
  static constexpr bool horz_subsample = h;
};

using TestParams = ::testing::Types<
  jpeg_distortion_params_t<uint8_t, true, true>,
  jpeg_distortion_params_t<uint8_t, false, true>,
  jpeg_distortion_params_t<uint8_t, true, false>,
  jpeg_distortion_params_t<uint8_t, false, false>
>;

TYPED_TEST_SUITE_P(JpegDistortionTestGPU);

TYPED_TEST_P(JpegDistortionTestGPU, ChromaSubsampleDistortion) {
  this->TestChromaSubsampleDistortion();
}

TYPED_TEST_P(JpegDistortionTestGPU, JpegCompressionDistortion_LowQuality) {
  this->TestJpegCompressionDistortion(5);
}

TYPED_TEST_P(JpegDistortionTestGPU, JpegCompressionDistortion_HighQuality) {
  this->TestJpegCompressionDistortion(95);
}

TYPED_TEST_P(JpegDistortionTestGPU, JpegCompressionDistortion_NoQuantization) {
  this->TestJpegCompressionDistortion_NoQuantization();
}

REGISTER_TYPED_TEST_SUITE_P(JpegDistortionTestGPU, ChromaSubsampleDistortion,
                                                   JpegCompressionDistortion_LowQuality,
                                                   JpegCompressionDistortion_HighQuality,
                                                   JpegCompressionDistortion_NoQuantization);
INSTANTIATE_TYPED_TEST_SUITE_P(JpegDistortionSuite, JpegDistortionTestGPU, TestParams);

}  // namespace test
}  // namespace kernels
}  // namespace dali

