// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime_api.h>
#include <gtest/gtest.h>
#include <opencv2/opencv.hpp>
#include <vector>
#include <cstdlib>
#include "dali/core/cuda_event.h"
#include "dali/core/hip_stream.h"
#include "dali/pipeline/data/tensor_list.h"
#include "dali/test/tensor_test_utils.h"
#include "dali/test/test_tensors.h"
#include "dali/test/dali_test_config.h"
#include "dali/kernels/test/kernel_test_utils.h"
#include "dali/kernels/imgproc/jpeg/jpeg_distortion_gpu.cuh"

#define DEBUG_LOGS 0

namespace dali {
namespace kernels {
namespace test {

using KernelPtr = void(*)(const SampleDesc *, const kernels::BlockDesc<2> *);
using testing::dali_extra_path;

template <typename GTestParams>
class JpegDistortionTestGPU : public ::testing::Test {
  using T = typename GTestParams::T;
  static constexpr bool vert_subsample = GTestParams::vert_subsample;
  static constexpr bool horz_subsample = GTestParams::horz_subsample;
  static constexpr bool use_real_images = true;
  static constexpr bool perf_run = false;
  static constexpr bool dump_images = false;

 public:
  void SetUp() final {
    if (use_real_images) {
      std::vector<std::string> paths{
        dali_extra_path() + "/db/single/bmp/0/cat-111793_640_palette_8bit.bmp",
        dali_extra_path() + "/db/single/bmp/0/cat-1046544_640.bmp",
        dali_extra_path() + "/db/single/bmp/0/cat-3591348_640.bmp",
      };
      in_shapes_.resize(paths.size(), 3);
      std::vector<cv::Mat> images(paths.size());
      for (size_t i = 0; i < paths.size(); i++) {
        images[i] = cv::imread(paths[i]);
        cv::cvtColor(images[i], images[i], cv::COLOR_BGR2RGB);
        TensorShape<> sh{images[i].rows, images[i].cols, images[i].channels()};
        in_shapes_.set_tensor_shape(i, sh);
      }
      in_.reshape(in_shapes_);
      auto in_cpu = in_.cpu();
      for (size_t i = 0; i < paths.size(); i++) {
        std::memcpy(in_cpu[i].data, images[i].data,
                    in_shapes_.tensor_size(i) * sizeof(uint8_t));
      }
    } else {
      if (perf_run) {
        in_shapes_ = uniform_list_shape(64, TensorShape<3>{600, 800, 3});
      } else {
        // TODO(janton): Fix blocks not aligned to 16 length
        in_shapes_ = {{16, 16, 3}, {16, 16, 3}};
      }
      in_.reshape(in_shapes_);
      std::mt19937_64 rng;
      UniformRandomFill(in_.cpu(), rng, 0., 255.);
    }

    out_.reshape(in_shapes_);
  }

  void TearDown() final {
  }

  using KernelLaunchFn = std::function<
    void(dim3, dim3, hipStream_t, const SampleDesc*, const kernels::BlockDesc<2>*)>;

  void TestKernel(KernelLaunchFn kernel_launch_fn) {
    CUDAStream stream = CUDAStream::Create(true);

    TensorListShape<2> chroma_shape(in_shapes_.size(), 2);
    std::vector<SampleDesc> samples_cpu;
    samples_cpu.resize(in_shapes_.size());

    auto in_view_gpu = in_.gpu(stream);
    auto out_view_gpu = out_.gpu(stream);
    out_.invalidate_cpu();
    CUDA_CALL(hipStreamSynchronize(stream));

    for (size_t i = 0; i < in_shapes_.size(); i++) {
      auto &sample_desc = samples_cpu[i];
      const auto& in_sh = in_shapes_[i];
      auto chroma_sh = chroma_shape.tensor_shape_span(i);
      auto shape_vol = volume(in_sh);
      auto width = in_sh[1];
      auto height = in_sh[0];
      auto chroma_width = div_ceil(width, 1 + horz_subsample);
      auto chroma_height = div_ceil(height, 1 + vert_subsample);
      // used to generate logical blocks (one thread per chroma pixel)
      chroma_sh[0] = chroma_height;
      chroma_sh[1] = chroma_width;
      chroma_sh[2] = 3;

      sample_desc.in = in_view_gpu[i].data;
      sample_desc.out = out_view_gpu[i].data;
      sample_desc.size.x = width;
      sample_desc.size.y = height;
      sample_desc.strides.x = 3;
      sample_desc.strides.y = width * 3;
    }

    block_setup_.SetBlockDim(dim3(32, 16, 1));
    int xblock = 64*(2-horz_subsample);
    int yblock = 128;
    block_setup_.SetDefaultBlockSize({xblock, yblock});
    block_setup_.SetupBlocks(chroma_shape, true);
    auto blocks_cpu = block_setup_.Blocks();

    SampleDesc *samples_gpu;
    CUDA_CALL(hipMalloc(&samples_gpu, sizeof(SampleDesc) * samples_cpu.size()));
    CUDA_CALL(hipMemcpy(samples_gpu, samples_cpu.data(),
                         sizeof(SampleDesc) * samples_cpu.size(),
                         hipMemcpyDefault));
    BlockDesc *blocks_gpu;
    CUDA_CALL(hipMalloc(&blocks_gpu, sizeof(BlockDesc) * blocks_cpu.size()));
    CUDA_CALL(hipMemcpy(blocks_gpu, blocks_cpu.data(), sizeof(BlockDesc) * blocks_cpu.size(),
                         hipMemcpyDefault));

    dim3 grid_dim = block_setup_.GridDim();
    dim3 block_dim = block_setup_.BlockDim();
#if DEBUG_LOGS
    std::cout << "\ngrid dim " << grid_dim.x << " " << grid_dim.y << " " << grid_dim.z
    << "\nblock_dim " << block_dim.x << " " << block_dim.y << " " << block_dim.z << "\n";
    for (size_t i = 0; i < blocks_cpu.size(); i++) {
      auto &blk = blocks_cpu[i];
      std::cout << "block " << i << " sample idx " << blk.sample_idx
                << " from " << blk.start << " to " << blk.end << "\n";
    }
#endif
    CUDAEvent start = CUDAEvent::CreateWithFlags(0);
    CUDAEvent end = CUDAEvent::CreateWithFlags(0);

    if (perf_run)  // warm up
      kernel_launch_fn(grid_dim, block_dim, stream, samples_gpu, blocks_gpu);

    CUDA_CALL(hipEventRecord(start, stream));

    kernel_launch_fn(grid_dim, block_dim, stream, samples_gpu, blocks_gpu);
    CUDA_CALL(hipGetLastError());

    CUDA_CALL(hipEventRecord(end, stream));
    CUDA_CALL(hipStreamSynchronize(stream));

    if (perf_run) {
      float time = 0;
      CUDA_CALL(hipEventElapsedTime(&time, start, end));
      time *= 1e+6;  // to nanoseconds
      int64_t size = 2 * in_shapes_.num_elements() * sizeof(uint8_t);
      std::cerr << "Throughput: " << size / time << " GB/s\n";
    }

    auto out_view_cpu = out_.cpu(stream);
    CUDA_CALL(hipStreamSynchronize(stream));
    auto in_view_cpu = in_.cpu();
    auto out_ref_cpu = out_ref_.cpu();

#if DEBUG_LOGS
    auto print_out = [&](TensorListView<StorageCPU, uint8_t, 3> view) {
      int off = 0;
      auto shape = view.shape;
      auto nsamples = shape.size();
      for (size_t i = 0; i < nsamples; i++) {
        auto sample_view = view[i];
        auto sample_sh = sample_view.shape;
        auto width = sample_sh[1];
        auto height = sample_sh[0];
        assert(sample_sh[2] == 3);
        std::cout << "\nSample " << i << "\n";
        for (size_t k = 0; k < width * height; k++) {
          if (k > 0 && k % width == 0) std::cout << "\n";
          std::cout << " " << static_cast<int>(sample_view.data[off++])
                    << "-" << static_cast<int>(sample_view.data[off++])
                    << "-" << static_cast<int>(sample_view.data[off++]);
        }
        std::cout << "\n";
      }
    };

    std::cout << "\nInput:";
    print_out(in_view_cpu);
    std::cout << "\nReference:";
    print_out(out_ref_cpu);
    std::cout << "\nOutput:";
    print_out(out_view_cpu);
#endif

    // In the kernel we average the RGB values, then converto to YCbCr
    // while here we are first converting and then averaging
    // Check(out_view_cpu, out_ref_cpu, EqualEps(40));

    for (size_t i = 0; i < in_shapes_.size(); i++) {
      auto sh = in_shapes_[i];
      cv::Mat in_mat(sh[0], sh[1], CV_8UC3, (void *) in_view_cpu[i].data);
      cv::Mat out_mat(sh[0], sh[1], CV_8UC3, (void *) out_view_cpu[i].data);
      cv::Mat out_ref(sh[0], sh[1], CV_8UC3, (void *) out_ref_cpu[i].data);
      cv::Mat diff;
      cv::absdiff(out_mat, out_ref, diff);

      // Sanity check. Checking that the average pixel difference is small
      // It is always best to compare the diff image visually to look for
      // artifacts.
      auto mean = cv::mean(diff);
      // different subsampling -> bigger error
      int avg_diff_max = horz_subsample && vert_subsample ? 2 : 6;
      for (int d = 0; d < 3; d++)
        ASSERT_LT(mean[d], avg_diff_max);

      if (dump_images) {
        std::stringstream ss1, ss2, ss3, ss4;
        ss1 << "jpeg_distortion_test_" << i << "_in.bmp";
        ss2 << "jpeg_distortion_test_" << i << "_out_ref.jpg";
        ss3 << "jpeg_distortion_test_" << i << "_out.bmp";
        ss4 << "jpeg_distortion_test_" << i << "_diff.bmp";
        cv::cvtColor(in_mat, in_mat, cv::COLOR_RGB2BGR);
        cv::imwrite(ss1.str(), in_mat);
        cv::cvtColor(in_mat, in_mat, cv::COLOR_BGR2RGB);

        cv::cvtColor(out_ref, out_ref, cv::COLOR_RGB2BGR);
        cv::imwrite(ss2.str(), out_ref);
        cv::cvtColor(out_ref, out_ref, cv::COLOR_BGR2RGB);

        cv::cvtColor(out_mat, out_mat, cv::COLOR_RGB2BGR);
        cv::imwrite(ss3.str(), out_mat);
        cv::cvtColor(out_mat, out_mat, cv::COLOR_BGR2RGB);

        cv::cvtColor(diff, diff, cv::COLOR_RGB2BGR);
        cv::imwrite(ss4.str(), diff);
        cv::cvtColor(diff, diff, cv::COLOR_BGR2RGB);
      }
    }

    CUDA_CALL(hipFree(blocks_gpu));
    CUDA_CALL(hipFree(samples_gpu));
  }

  void CalcOut_ChromaSubsampleDistortion() {
    // simplest implementation for test purposes. First convert to YCbCr, then subsample
    out_ref_.reshape(in_shapes_);
    auto out_ref_view = out_ref_.cpu();

    std::vector<T> tmp_y;
    std::vector<T> tmp_cb;
    std::vector<T> tmp_cr;
    auto in_view = in_.cpu();
    for (size_t sample = 0; sample < in_shapes_.size(); sample++) {
      auto sh = in_shapes_[sample];
      int64_t npixels = sh[0] * sh[1];
      tmp_y.resize(npixels);
      tmp_cb.resize(npixels);
      tmp_cr.resize(npixels);
      for (int64_t i = 0; i < npixels; i++) {
        uint8_t r = in_view[sample].data[3*i];
        uint8_t g = in_view[sample].data[3*i + 1];
        uint8_t b = in_view[sample].data[3*i + 2];
        tmp_y[i]  =  0.29900000f * r + 0.58700000f * g + 0.11400000f * b;
        tmp_cb[i] = -0.16873589f * r - 0.33126411f * g + 0.50000000f * b + 128.0f;
        tmp_cr[i] =  0.50000000f * r - 0.41868759f * g - 0.08131241f * b + 128.0f;
      }

#if DEBUG_LOGS
      std::cout << "\nYCbCr original ref:\n";
      for (int64_t i = 0; i < npixels; i++) {
        if (i > 0 && i % sh[1] == 0) std::cout << "\n";
        std::cout << " " << static_cast<int>(tmp_y[i])
                  << "-" << static_cast<int>(tmp_cb[i])
                  << "-" << static_cast<int>(tmp_cr[i]);
      }
#endif

      auto subsample_f = [&](std::vector<T> &component) {
        for (int64_t y = 0; y < sh[0]; y+=(1 << vert_subsample)) {
          for (int64_t x = 0; x < sh[1]; x+=(1 << horz_subsample)) {
            auto in_offset_1 = y * sh[1] + x;
            auto in_offset_2 = y * sh[1] + x + 1;
            auto in_offset_3 = (y + 1) * sh[1] + x;
            auto in_offset_4 = (y + 1) * sh[1] + x + 1;
            if (horz_subsample && vert_subsample) {
              T avg = ConvertSat<T>(0.25f * (component[in_offset_1] + component[in_offset_2] +
                                             component[in_offset_3] + component[in_offset_4]));
              component[in_offset_1] = avg;
              component[in_offset_2] = avg;
              component[in_offset_3] = avg;
              component[in_offset_4] = avg;
            } else if (horz_subsample) {
              T avg = ConvertSat<T>(0.5f * (component[in_offset_1] + component[in_offset_2]));
              component[in_offset_1] = avg;
              component[in_offset_2] = avg;
            } else if (vert_subsample) {
              T avg = ConvertSat<T>(0.5f * (component[in_offset_1] + component[in_offset_3]));
              component[in_offset_1] = avg;
              component[in_offset_3] = avg;
            }
          }
        }
      };
      if (horz_subsample || vert_subsample) {
        subsample_f(tmp_cb);
        subsample_f(tmp_cr);
      }

#if DEBUG_LOGS
      std::cout << "\nYCbCr subsampled ref:\n";
      for (int64_t i = 0; i < npixels; i++) {
        if (i > 0 && i % sh[1] == 0) std::cout << "\n";
        std::cout << " " << static_cast<int>(tmp_y[i])
                  << "-" << static_cast<int>(tmp_cb[i])
                  << "-" << static_cast<int>(tmp_cr[i]);
      }
      std::cout << "\n";
#endif

      auto* sample_data = out_ref_view[sample].data;
      for (int64_t i = 0; i < npixels; i++) {
        float y = static_cast<float>(tmp_y[i]);
        float cb = static_cast<float>(tmp_cb[i]) - 128.0f;
        float cr = static_cast<float>(tmp_cr[i]) - 128.0f;
        auto r = ConvertSat<T>(y + 1.402f * cr);
        auto g = ConvertSat<T>(y - 0.34413629f * cb - 0.71413629f * cr);
        auto b = ConvertSat<T>(y + 1.772f * cb);
        sample_data[3*i]   = r;
        sample_data[3*i+1] = g;
        sample_data[3*i+2] = b;
      }
    }
  }

  void CalcOut_JpegCompressionDistortion() {
    out_ref_.reshape(in_shapes_);
    auto out_ref_view = out_ref_.cpu();
    auto in_view_cpu = in_.cpu();
    for (size_t i = 0; i < in_shapes_.size(); i++) {
      auto sh = in_shapes_[i];
      cv::Mat in_mat(sh[0], sh[1], CV_8UC3, (void *) in_view_cpu[i].data);

      std::vector<uint8_t> encoded;

      cv::cvtColor(in_mat, in_mat, cv::COLOR_RGB2BGR);
      cv::imencode(".jpg", in_mat, encoded, {cv::IMWRITE_JPEG_QUALITY, ConvertSat<int>(quality_factor)});
      cv::cvtColor(in_mat, in_mat, cv::COLOR_BGR2RGB);

      cv::Mat encoded_mat(1, encoded.size(), CV_8UC1, encoded.data());
      auto out_ref = cv::imdecode(encoded_mat, cv::IMREAD_COLOR);
      cv::cvtColor(out_ref, out_ref, cv::COLOR_BGR2RGB);
      std::memcpy(out_ref_view[i].data, out_ref.data,
                  in_shapes_.tensor_size(i) * sizeof(uint8_t));
    }
  }

  void TestJpegCompressionDistortion() {
    quality_factor = 5;
    luma_table = GetLumaQuantizationTable(quality_factor);
    chroma_table = GetChromaQuantizationTable(quality_factor);
    CalcOut_JpegCompressionDistortion();
    TestKernel(
      [this](dim3 gridDim, dim3 blockDim, hipStream_t stream,
             const SampleDesc* samples, const kernels::BlockDesc<2>* blocks) {
        JpegCompressionDistortion<horz_subsample, vert_subsample>
          <<<gridDim, blockDim, 0, stream>>>(samples, blocks, luma_table, chroma_table);
      });
  }

  void TestChromaSubsampleDistortion() {
    quality_factor = 99;
    CalcOut_ChromaSubsampleDistortion();
    TestKernel(
      [](dim3 gridDim, dim3 blockDim, hipStream_t stream,
         const SampleDesc* samples, const kernels::BlockDesc<2>* blocks) {
        ChromaSubsampleDistortion<horz_subsample, vert_subsample>
          <<<gridDim, blockDim, 0, stream>>>(samples, blocks);
      });
  }

  CUDAStream stream_;
  TensorListShape<> in_shapes_;
  TestTensorList<uint8_t> in_;
  TestTensorList<uint8_t>out_;
  TestTensorList<uint8_t> out_ref_;

  using BlkSetup = BlockSetup<2, -1>;
  BlkSetup block_setup_;
  using BlockDesc = BlkSetup::BlockDesc;

  float quality_factor = 20.0f;
  DeviceArray<uint8_t, 64> luma_table;
  DeviceArray<uint8_t, 64> chroma_table;
};

template <typename OutType, bool v, bool h>
struct jpeg_distortion_params_t {
  using T = OutType;
  static constexpr bool vert_subsample = v;
  static constexpr bool horz_subsample = h;
};

using TestParams = ::testing::Types<
  jpeg_distortion_params_t<uint8_t, true, true>,
  jpeg_distortion_params_t<uint8_t, false, true>,
  jpeg_distortion_params_t<uint8_t, true, false>,
  jpeg_distortion_params_t<uint8_t, false, false>
>;

TYPED_TEST_SUITE_P(JpegDistortionTestGPU);

TYPED_TEST_P(JpegDistortionTestGPU, ChromaSubsampleDistortion) {
  this->TestChromaSubsampleDistortion();
}

TYPED_TEST_P(JpegDistortionTestGPU, JpegCompressionDistortion) {
  this->TestJpegCompressionDistortion();
}

REGISTER_TYPED_TEST_SUITE_P(JpegDistortionTestGPU, ChromaSubsampleDistortion,
                                                   JpegCompressionDistortion);
INSTANTIATE_TYPED_TEST_SUITE_P(JpegDistortionSuite, JpegDistortionTestGPU, TestParams);

}  // namespace test
}  // namespace kernels
}  // namespace dali

