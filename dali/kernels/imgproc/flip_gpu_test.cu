// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "dali/kernels/imgproc/flip_gpu.cuh"
#include <gtest/gtest.h>
#include <random>
#include <vector>
#include "dali/kernels/imgproc/flip_test.h"
#include "dali/test/test_tensors.h"
#include "dali/test/tensor_test_utils.h"

namespace dali {
namespace kernels {

class FlipGpuTest: public testing::TestWithParam<std::array<Index, sample_ndim>> {
 public:
  FlipGpuTest()
  : tensor_shape_(GetParam())
  , volume_(volume(tensor_shape_))
  , shape_({tensor_shape_, tensor_shape_, tensor_shape_, tensor_shape_,
           tensor_shape_, tensor_shape_, tensor_shape_, tensor_shape_}) {}

  void SetUp() final {
    ttl_in_.reshape(shape_);
    auto tlv = ttl_in_.cpu(nullptr);
    std::mt19937_64 rng;
    UniformRandomFill(tlv, rng, 0., 10.);
  }

 protected:
  std::vector<int> flip_x_{0, 0, 1, 1, 0, 0, 1, 1};
  std::vector<int> flip_y_{0, 1, 0, 1, 0, 1, 0, 1};
  std::vector<int> flip_z_{0, 0, 0, 0, 1, 1, 1, 1};
  TensorShape<sample_ndim> tensor_shape_;
  size_t volume_;
  TensorListShape<sample_ndim> shape_;
  TestTensorList<float, sample_ndim> ttl_in_;
  TestTensorList<float, sample_ndim> ttl_out_;
};

TEST_P(FlipGpuTest, ImplTest) {
  KernelContext ctx;
  FlipGPU<float> kernel;
  auto in_view = ttl_in_.gpu(nullptr);
  ttl_in_.invalidate_cpu();
  KernelRequirements reqs = kernel.Setup(ctx, in_view);
  ttl_out_.reshape(reqs.output_shapes[0].to_static<sample_ndim>());
  auto out_view = ttl_out_.gpu();
  for (int i = 0; i < in_view.num_samples(); ++i) {
    detail::gpu::FlipImpl(
        out_view.tensor_data(i), in_view.tensor_data(i),
        tensor_shape_, flip_z_[i], flip_y_[i], flip_z_[i], nullptr);
  }
  kernel.Run(ctx, out_view, in_view, flip_z_, flip_y_, flip_x_);
  auto out_view_cpu = ttl_out_.cpu(nullptr);
  auto in_view_cpu = ttl_in_.cpu(nullptr);
  for (int i = 0; i < out_view_cpu.num_samples(); ++i) {
    ASSERT_TRUE(is_flipped(out_view_cpu.tensor_data(i),
                           in_view_cpu.tensor_data(i),
                           shape_[i][0], shape_[i][1], shape_[i][2], shape_[i][3], shape_[i][4],
                           flip_z_[i], flip_y_[i], flip_x_[i]));
  }
}

TEST_P(FlipGpuTest, KernelTest) {
  KernelContext ctx;
  FlipGPU<float> kernel;
  auto in_view = ttl_in_.gpu(nullptr);
  ttl_in_.invalidate_cpu();
  KernelRequirements reqs = kernel.Setup(ctx, in_view);
  ttl_out_.reshape(reqs.output_shapes[0].to_static<sample_ndim>());
  auto out_view = ttl_out_.gpu();
  kernel.Run(ctx, out_view, in_view, flip_z_, flip_y_, flip_x_);
  auto out_view_cpu = ttl_out_.cpu(nullptr);
  auto in_view_cpu = ttl_in_.cpu(nullptr);
  for (int i = 0; i < out_view_cpu.num_samples(); ++i) {
    ASSERT_TRUE(is_flipped(out_view_cpu.tensor_data(i),
                           in_view_cpu.tensor_data(i),
                           shape_[i][0], shape_[i][1], shape_[i][2], shape_[i][3], shape_[i][4],
                           flip_z_[i], flip_y_[i], flip_x_[i]));
  }
}

INSTANTIATE_TEST_SUITE_P(FlipGpuTest, FlipGpuTest,
    ::testing::ValuesIn({
        std::array<Index, sample_ndim>{4, 1, 2, 2, 10},
        std::array<Index, sample_ndim>{4, 1, 2, 2, 2},
        std::array<Index, sample_ndim>{1, 4, 9, 18, 3},
        std::array<Index, sample_ndim>{1, 3, 18, 9, 4}}));

}  // namespace kernels
}  // namespace dali
