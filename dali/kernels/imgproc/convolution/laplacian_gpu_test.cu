#include "hip/hip_runtime.h"
// Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gtest/gtest.h>
#include <array>
#include <cmath>
#include <vector>

#include "dali/kernels/common/utils.h"
#include "dali/kernels/imgproc/convolution/laplacian_cpu.h"
#include "dali/kernels/imgproc/convolution/laplacian_gpu.cuh"
#include "dali/kernels/imgproc/convolution/laplacian_test.h"
#include "dali/kernels/dynamic_scratchpad.h"
#include "dali/test/tensor_test_utils.h"
#include "dali/test/test_tensors.h"

namespace dali {
namespace kernels {

template <typename Out_, typename In_, int axes_, bool has_channels_, bool is_sequence_,
          bool use_smoothing_>
struct test_laplacian {
  static constexpr int axes = axes_;
  static constexpr bool has_channels = has_channels_;
  static constexpr bool is_sequence = is_sequence_;
  static constexpr bool use_smoothing = use_smoothing_;
  using Out = Out_;
  using In = In_;
};

/**
 * @brief Compares GPU implementation against CPU implementation.
 */
template <typename T>
struct LaplacianGpuTest : public ::testing::Test {
  static constexpr int max_window_size = 23;
  static constexpr int max_sample_dim = 5;
  static constexpr int max_axes = 3;
  static constexpr bool has_channels = T::has_channels;
  static constexpr bool is_sequence = T::is_sequence;
  static constexpr bool use_smoothing = T::use_smoothing;
  static constexpr int axes = T::axes;
  static constexpr int sample_ndim = axes + static_cast<int>(has_channels);
  static constexpr int ndim = sample_ndim + static_cast<int>(is_sequence);
  using Out = typename T::Out;
  using In = typename T::In;
  using W = float;
  using KernelCpu = LaplacianCpu<Out, In, W, axes, has_channels>;
  using KernelGpu = LaplacianGpu<Out, In, W, axes, has_channels, is_sequence>;

  static TensorListShape<ndim> GetShape() {
    static const TensorListShape<> shapes = {
        {7, 29, 145, 128}, {3, 64, 64, 64},  {4, 164, 164, 164}, {11, 12, 12, 12},
        {4, 4, 200, 180},  {1, 200, 4, 180}, {1, 75, 75, 75},    {2, 16, 256, 256}};
    static const TensorListShape<> channels = {{3}, {3}, {1}, {5}, {7}, {3}, {5}, {1}};
    if (!has_channels) {
      return shapes.template first<ndim>();
    } else {
      auto shape = shapes.template first<ndim - 1>();
      TensorListShape<ndim> result(shapes.num_samples());
      for (int i = 0; i < shapes.num_samples(); i++) {
        result.set_tensor_shape(i, shape_cat(shape[i], channels[i]));
      }
      return result;
    }
  }

  static TensorListShape<axes> GetWindowSize() {
    static const TensorListShape<> window_sizes = {{3, 5, 7}, {7, 5, 3},   {5, 5, 5},
                                                   {3, 3, 3}, {11, 11, 5}, {13, 15, 13},
                                                   {7, 9, 7}, {23, 19, 17}};
    return window_sizes.template last<axes>();
  }

  static TensorListShape<axes> GetSmoothingSize() {
    // use 1 as the middle window size in a whole batch to test if optimization that removes
    // unnecessary smoothing convolutions on per partial derivative basis gives correct results
    static const TensorListShape<> window_sizes = {{3, 1, 1}, {5, 1, 9},   {7, 1, 7}, {1, 1, 7},
                                                   {7, 1, 5}, {13, 1, 13}, {7, 1, 9}, {23, 1, 17}};
    return window_sizes.template last<axes>();
  }

  void FillWindows() {
    // get per sample x per axis window sizes
    auto deriv_sizes = GetWindowSize();
    int nsamples = deriv_sizes.num_samples();
    auto smoothing_sizes =
        use_smoothing ? GetSmoothingSize() : uniform_list_shape(nsamples, uniform_array<axes>(1));
    // flatten window sizes
    TensorListShape<1> flat_deriv_sizes;
    TensorListShape<1> flat_smoothing_sizes;
    flat_deriv_sizes.resize(nsamples * axes);
    flat_smoothing_sizes.resize(nsamples * axes);
    for (int i = 0; i < nsamples; i++) {
      for (int axis = 0; axis < axes; axis++) {
        flat_deriv_sizes.set_tensor_shape(i * axes + axis, {deriv_sizes[i][axis]});
        flat_smoothing_sizes.set_tensor_shape(i * axes + axis, {smoothing_sizes[i][axis]});
      }
    }
    deriv_windows_.reshape(flat_deriv_sizes);
    smoothing_windows_.reshape(flat_smoothing_sizes);
    deriv_win_ = deriv_windows_.cpu();
    smoothing_win_ = smoothing_windows_.cpu();
    for (int i = 0; i < deriv_win_.num_samples(); i++) {
      FillSobelWindow(make_span(deriv_win_[i].data, deriv_win_[i].num_elements()), 2);
    }
    for (int i = 0; i < smoothing_win_.num_samples(); i++) {
      FillSobelWindow(make_span(smoothing_win_[i].data, smoothing_win_[i].num_elements()), 0);
    }
  }

  void SetUp() override {
    FillWindows();

    auto shapes = GetShape();
    input_.reshape(shapes);
    baseline_in_ = input_.cpu();

    std::mt19937 rng;
    UniformRandomFill(baseline_in_, rng, 0, 64);
    in_ = input_.gpu();

    output_.reshape(shapes);
    baseline_output_.reshape(shapes);

    int nsamples = shapes.size();
    for (int i = 0; i < axes; i++) {
      for (int j = 0; j < axes; j++) {
        win_sizes_[i][j].resize(nsamples);
        windows_[i][j].resize(nsamples);
      }
      scales_[i].resize(nsamples);
      scale_spans_[i] = make_span(scales_[i]);
    }
    for (int sample_idx = 0; sample_idx < nsamples; sample_idx++) {
      for (int i = 0; i < axes; i++) {
        int win_size_sum = -axes - 2;
        for (int j = 0; j < axes; j++) {
          auto& windows = i == j ? deriv_win_ : smoothing_win_;
          auto window = windows[sample_idx * axes + j];
          win_size_sum += window.shape.num_elements();
          win_sizes_[i][j].set_tensor_shape(sample_idx, window.shape);
          windows_[i][j].data[sample_idx] = window.data;
          windows_[i][j].shape.set_tensor_shape(sample_idx, window.shape);
        }
        scales_[i][sample_idx] = std::exp2f(-win_size_sum);
      }
    }
  }

  void RunTest() {
    KernelContext ctx_cpu = {}, ctx_gpu = {};
    ctx_gpu.gpu.stream = 0;
    KernelCpu kernel_cpu;
    KernelGpu kernel_gpu;
    int nsamples = in_.shape.size();
    baseline_out_ = baseline_output_.cpu();
    out_ = output_.gpu();

    std::array<bool, axes> has_smoothing = uniform_array<axes>(false);
    for (int sample_idx = 0; sample_idx < nsamples; sample_idx++) {
      std::array<std::array<int, axes>, axes> window_size;
      std::array<std::array<TensorView<StorageCPU, const W, 1>, axes>, axes> windows;
      std::array<float, axes> scales;
      for (int i = 0; i < axes; i++) {
        for (int j = 0; j < axes; j++) {
          if (i != j && win_sizes_[i][j][sample_idx].num_elements() > 1) {
            has_smoothing[i] = true;
          }
          window_size[i][j] = win_sizes_[i][j][sample_idx].num_elements();
          windows[i][j] = windows_[i][j][sample_idx];
        }
        scales[i] = scales_[i][sample_idx];
      }
      auto elem_shape = baseline_in_.shape[sample_idx].template last<sample_ndim>();
      auto req = kernel_cpu.Setup(ctx_cpu, elem_shape, window_size);

      const auto& shape = baseline_in_.shape[sample_idx];
      auto elem_volume = volume(shape.begin() + static_cast<int>(is_sequence), shape.end());
      int seq_elements = volume(shape.begin(), shape.begin() + static_cast<int>(is_sequence));
      int64_t stride = elem_volume;

      for (int elem_idx = 0; elem_idx < seq_elements; elem_idx++) {
        auto in_view = TensorView<StorageCPU, const In, sample_ndim>{
            baseline_in_[sample_idx].data + stride * elem_idx, elem_shape};
        auto out_view = TensorView<StorageCPU, Out, sample_ndim>{
            baseline_out_[sample_idx].data + stride * elem_idx, elem_shape};
        // Copy context so that the kernel instance can modify scratchpad
        DynamicScratchpad scratchpad;
        ctx_cpu.scratchpad = &scratchpad;
        kernel_cpu.Run(ctx_cpu, out_view, in_view, windows, scales);
      }
    }

    for (int i = 0; i < axes; i++) {
      if (!has_smoothing[i]) {
        for (int j = 0; j < axes; j++) {
          if (i != j) {
            win_sizes_[i][j].resize(0);
            windows_[i][j].resize(0);
          }
        }
      }
    }

    auto req = kernel_gpu.Setup(ctx_gpu, in_.shape, win_sizes_);

    DynamicScratchpad scratchpad;
    ctx_gpu.scratchpad = &scratchpad;
    kernel_gpu.Run(ctx_gpu, out_, in_, windows_, scale_spans_);

    auto out_cpu_ = output_.cpu();

    double eps = std::is_integral<Out>::value ? 1 : 0.01;
    Check(out_cpu_, baseline_out_, EqualEps(eps));
  }

  TestTensorList<W, 1> deriv_windows_;
  TestTensorList<W, 1> smoothing_windows_;
  TestTensorList<In, ndim> input_;
  TestTensorList<Out, ndim> output_;
  TestTensorList<Out, ndim> baseline_output_;

  TensorListView<StorageCPU, W, 1> deriv_win_;
  TensorListView<StorageCPU, W, 1> smoothing_win_;
  TensorListView<StorageGPU, In, ndim> in_;
  TensorListView<StorageGPU, Out, ndim> out_;
  TensorListView<StorageCPU, In, ndim> baseline_in_;
  TensorListView<StorageCPU, Out, ndim> baseline_out_;

  std::array<std::array<TensorListShape<1>, axes>, axes> win_sizes_;
  std::array<std::array<TensorListView<StorageCPU, const float, 1>, axes>, axes> windows_;
  std::array<std::vector<float>, axes> scales_;
  std::array<span<const float>, axes> scale_spans_;
};

TYPED_TEST_SUITE_P(LaplacianGpuTest);

using LaplacianTestValues =
    ::testing::Types<test_laplacian<float, float, 1, true, true, false>,
                     test_laplacian<float, float, 1, true, false, false>,
                     test_laplacian<float, float, 1, false, true, false>,
                     test_laplacian<float, float, 1, false, false, false>,
                     test_laplacian<float, float, 2, true, true, false>,
                     test_laplacian<float, float, 2, true, false, false>,
                     test_laplacian<float, float, 2, false, true, false>,
                     test_laplacian<float, float, 2, false, false, false>,
                     test_laplacian<float, float, 2, true, true, true>,
                     test_laplacian<float, float, 2, true, false, true>,
                     test_laplacian<float, float, 2, false, true, true>,
                     test_laplacian<float, float, 2, false, false, true>,
                     test_laplacian<float, float, 3, true, true, false>,
                     test_laplacian<float, float, 3, true, false, false>,
                     test_laplacian<float, float, 3, false, true, false>,
                     test_laplacian<float, float, 3, false, false, false>,
                     test_laplacian<float, float, 3, true, true, true>,
                     test_laplacian<float, float, 3, true, false, true>,
                     test_laplacian<float, float, 3, false, true, true>,
                     test_laplacian<float, float, 3, false, false, true>,

                     test_laplacian<uint8_t, uint8_t, 1, true, true, true>,
                     test_laplacian<uint8_t, uint8_t, 2, true, true, true>,
                     test_laplacian<uint8_t, uint8_t, 3, true, true, true>,
                     test_laplacian<uint8_t, uint8_t, 1, true, true, false>,
                     test_laplacian<uint8_t, uint8_t, 2, true, true, false>,
                     test_laplacian<uint8_t, uint8_t, 3, true, true, false>>;

TYPED_TEST_P(LaplacianGpuTest, DoLaplacian) {
  this->RunTest();
}

REGISTER_TYPED_TEST_SUITE_P(LaplacianGpuTest, DoLaplacian);
INSTANTIATE_TYPED_TEST_SUITE_P(LaplacianGpuKernel, LaplacianGpuTest, LaplacianTestValues);

}  // namespace kernels
}  // namespace dali
