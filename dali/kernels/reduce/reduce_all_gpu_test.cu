#include "hip/hip_runtime.h"
// Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gtest/gtest.h>
#include <random>
#include <vector>
#include "dali/kernels/reduce/reduce_all_gpu_impl.cuh"
#include "dali/core/util.h"
#include "dali/kernels/alloc.h"
#include "dali/core/span.h"
#include "dali/core/cuda_event.h"

namespace dali {
namespace kernels {

template <typename T>
double RefSum(span<T> in) {
  switch (in.size()) {
    case 0:
      return 0;
    case 1:
      return in[0];
    default: {
      int m = in.size() / 2;
      int n = in.size() - m;
      return RefSum(make_span(in.data(), m)) + RefSum(make_span(in.data() + m, n));
    }
  }
}

TEST(ReduceGPU, ReduceAllKernel) {
  std::mt19937_64 rng(1234);
  std::uniform_real_distribution<float> dist(0, 1);

  int n_in = 1<<24;  // 16M numbers
  dim3 block(32, 32);
  int nblock = 1024;
  int n_out = std::min<int>(div_ceil(n_in, nblock), 1024);
  auto in_data = memory::alloc_unique<float>(AllocType::GPU, n_in);
  auto out_data = memory::alloc_unique<float>(AllocType::GPU, n_out);
  std::vector<float> in_cpu(n_in), out_cpu(n_out);
  for (auto &x : in_cpu)
    x = dist(rng);
  double ref_sum = RefSum(make_cspan(in_cpu));

  hipMemcpy(in_data.get(), in_cpu.data(), n_in * sizeof(*in_data), hipMemcpyHostToDevice);

  dim3 grid = n_out;
  ReduceAllKernel<<<1, block>>>(out_data.get(), in_data.get(), n_in);
  hipDeviceSynchronize();
  auto start = CUDAEvent::CreateWithFlags(0);
  auto end =   CUDAEvent::CreateWithFlags(0);
  hipEventRecord(start);
  ReduceAllKernel<<<grid, block>>>(out_data.get(), in_data.get(), n_in);
  hipEventRecord(end);
  hipMemcpy(out_cpu.data(), out_data.get(), n_out * sizeof(*out_data), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  float t = 0;
  hipEventElapsedTime(&t, start, end);
  double out_sum = RefSum(make_cspan(out_cpu));
  EXPECT_NEAR(out_sum, ref_sum, ref_sum * 1e-7 + 1e-7);

  t /= 1000;
  std::cout << n_in * sizeof(*in_data) / t * 1e-9 << " GB/s" << std::endl;
}

TEST(ReduceGPU, ReduceAllBatchedKernel) {
  std::mt19937_64 rng(1234);
  std::uniform_real_distribution<float> dist(0, 1);
  std::uniform_int_distribution<int> size_dist(10000, 1000000);

  int64_t n_in = 0;
  int samples = 35;
  std::vector<int64_t> sizes(samples);
  for (auto &size : sizes) {
    size = size_dist(rng);
    n_in += size;
  }

  dim3 block(32, 32);
  int nout_per_sample = 32;
  int n_out = samples * nout_per_sample;
  dim3 grid(nout_per_sample, samples);
  auto in_data = memory::alloc_unique<float>(AllocType::GPU, n_in);
  auto out_data = memory::alloc_unique<float>(AllocType::GPU, n_out);
  std::vector<float> in_cpu(n_in), out_cpu(n_out);
  for (auto &x : in_cpu)
    x = dist(rng);

  auto gpu_dev_ptrs = memory::alloc_unique<const float*>(AllocType::GPU, samples);
  auto gpu_sizes = memory::alloc_unique<int64_t>(AllocType::GPU, samples);
  vector<const float *> host_ptrs(samples);
  vector<const float *> cpu_dev_ptrs(samples);
  int64_t offset = 0;
  for (int i = 0; i < samples; i++) {
    host_ptrs[i] = in_cpu.data() + offset;
    cpu_dev_ptrs[i] = in_data.get() + offset;
    offset += sizes[i];
  }

  // data
  hipMemcpy(in_data.get(), in_cpu.data(), n_in * sizeof(*in_data), hipMemcpyHostToDevice);
  // pointers to sample data
  hipMemcpy(gpu_dev_ptrs.get(), cpu_dev_ptrs.data(), samples * sizeof(*gpu_dev_ptrs),
             hipMemcpyHostToDevice);
  // sample sizes
  hipMemcpy(gpu_sizes.get(), sizes.data(), samples * sizeof(*gpu_sizes), hipMemcpyHostToDevice);

  // warm-up
  ReduceAllBatchedKernel<<<1, block>>>(out_data.get(), gpu_dev_ptrs.get(), gpu_sizes.get());
  hipDeviceSynchronize();
  auto start = CUDAEvent::CreateWithFlags(0);
  auto end =   CUDAEvent::CreateWithFlags(0);
  hipEventRecord(start);
  ReduceAllBatchedKernel<<<grid, block>>>(out_data.get(), gpu_dev_ptrs.get(), gpu_sizes.get());
  hipEventRecord(end);
  hipMemcpy(out_cpu.data(), out_data.get(), n_out * sizeof(*out_data), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  float t = 0;
  hipEventElapsedTime(&t, start, end);

  offset = 0;
  for (int i = 0; i < samples; i++) {
    double ref_sum = RefSum(make_span(host_ptrs[i], sizes[i]));
    double out_sum = RefSum(make_span(&out_cpu[i*nout_per_sample], nout_per_sample));
    EXPECT_NEAR(ref_sum, out_sum, ref_sum * 1e-7 + 1e-7);
    offset += sizes[i];
  }

  t /= 1000;
  std::cout << n_in * sizeof(*in_data) / t * 1e-9 << " GB/s" << std::endl;
}

}  // namespace kernels
}  // namespace dali
