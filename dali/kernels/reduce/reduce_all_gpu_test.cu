#include "hip/hip_runtime.h"
// Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gtest/gtest.h>
#include <random>
#include <vector>
#include "dali/kernels/reduce/reduce_all_gpu_impl.cuh"
#include "dali/core/util.h"
#include "dali/kernels/alloc.h"
#include "dali/core/span.h"
#include "dali/core/cuda_event.h"

namespace dali {
namespace kernels {

template <typename Out, typename Reduction, typename T>
Out RefReduce(span<T> in, const Reduction &R) {
  switch (in.size()) {
    case 0:
      return R.template neutral<Out>();
    case 1:
      return in[0];
    default: {
      if (in.size() <= 128) {
        double acc = R.template neutral<Out>();
        for (auto &x : in)
          R(acc, x);
        return acc;
      }
      int m = in.size() / 2;
      int n = in.size() - m;
      Out out = RefReduce<Out>(make_span(in.data(), m), R);
      R(out, RefReduce<Out>(make_span(in.data() + m, n), R));
      return out;
    }
  }
}

using ReductionTestTypes = ::testing::Types<reductions::sum, reductions::min, reductions::max>;

inline bool IsAccurate(const reductions::min &) { return true; }
inline bool IsAccurate(const reductions::max &) { return true; }
template <typename Reduction>
inline bool IsAccurate(const Reduction &) { return false; }

template <typename Reduction>
class ReduceAllGPUTest : public ::testing::Test {
 public:
  void TestReduceAll();
  void TestReduceBatched();

  template <typename T>
  inline auto ref_reduce(span<T> in) const {
    return RefReduce<double>(in, R);
  }

  Reduction R;
};

TYPED_TEST_SUITE(ReduceAllGPUTest, ReductionTestTypes);

template <typename Reduction>
void ReduceAllGPUTest<Reduction>::TestReduceAll() {
  std::mt19937_64 rng(1234);
  std::uniform_real_distribution<float> dist(0, 1);

  int n_in = 1<<23;  // 8M numbers
  dim3 block(32, 32);
  int nblock = 1024;
  int n_out0 = std::min<int>(div_ceil(n_in, nblock), 1024);
  int n_out = n_out0 + 1;
  auto in_data = memory::alloc_unique<float>(AllocType::GPU, n_in);
  auto out_data = memory::alloc_unique<float>(AllocType::GPU, n_out);
  std::vector<float> in_cpu(n_in), out_cpu(n_out);
  for (auto &x : in_cpu)
    x = dist(rng);
  double ref_value = ref_reduce(make_cspan(in_cpu));

  hipMemcpy(in_data.get(), in_cpu.data(), n_in * sizeof(*in_data), hipMemcpyHostToDevice);

  dim3 grid = n_out0;
  ReduceAllKernel<<<1, block>>>(out_data.get(), in_data.get(), n_in);
  hipDeviceSynchronize();
  auto start = CUDAEvent::CreateWithFlags(0);
  auto end =   CUDAEvent::CreateWithFlags(0);
  hipEventRecord(start);
  ReduceAllKernel<<<grid, block>>>(out_data.get() + 1, in_data.get(), n_in, R);
  ReduceAllKernel<<<1, block>>>(out_data.get(), out_data.get() + 1, n_out0, R);
  hipEventRecord(end);
  hipMemcpy(out_cpu.data(), out_data.get(), n_out * sizeof(*out_data), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  float t = 0;
  hipEventElapsedTime(&t, start, end);
  double out_value = out_cpu[0];
  double out_partial = ref_reduce(make_cspan(&out_cpu[1], n_out0));
  if (IsAccurate(R)) {
    EXPECT_EQ(out_value, ref_value);
    EXPECT_EQ(out_partial, ref_value);
  } else {
    double eps = ref_value * 1e-7 + 1e-7;
    EXPECT_NEAR(out_value, ref_value, eps);
    EXPECT_NEAR(out_partial, ref_value, eps);
  }

  t /= 1000;  // convert to seconds
  std::cout << n_in * sizeof(*in_data) / t * 1e-9 << " GB/s" << std::endl;
}

TYPED_TEST(ReduceAllGPUTest, ReduceAllKernel) {
  this->TestReduceAll();
}


template <typename Reduction>
void ReduceAllGPUTest<Reduction>::TestReduceBatched() {
  std::mt19937_64 rng(1234);
  std::uniform_real_distribution<float> dist(0, 1);
  std::uniform_int_distribution<int> size_dist(10000, 1000000);

  int64_t n_in = 0;
  int samples = 35;
  std::vector<int64_t> sizes(samples);
  for (auto &size : sizes) {
    size = size_dist(rng);
    n_in += size;
  }

  dim3 block(32, 32);
  int n_out_per_sample = 32;
  int n_out0 = samples * n_out_per_sample;
  int n_out = n_out0 + samples;
  dim3 grid(n_out_per_sample, samples);
  auto in_data = memory::alloc_unique<float>(AllocType::GPU, n_in);
  auto out_data = memory::alloc_unique<float>(AllocType::GPU, n_out);
  std::vector<float> in_cpu(n_in), out_cpu(n_out);
  for (auto &x : in_cpu)
    x = dist(rng);

  auto gpu_dev_ptrs = memory::alloc_unique<const float*>(AllocType::GPU, samples);
  auto gpu_sizes = memory::alloc_unique<int64_t>(AllocType::GPU, samples);
  vector<const float *> host_ptrs(samples);
  vector<const float *> cpu_dev_ptrs(samples);
  int64_t offset = 0;
  for (int i = 0; i < samples; i++) {
    host_ptrs[i] = in_cpu.data() + offset;
    cpu_dev_ptrs[i] = in_data.get() + offset;
    offset += sizes[i];
  }

  // data
  hipMemcpy(in_data.get(), in_cpu.data(), n_in * sizeof(*in_data), hipMemcpyHostToDevice);
  // pointers to sample data
  hipMemcpy(gpu_dev_ptrs.get(), cpu_dev_ptrs.data(), samples * sizeof(*gpu_dev_ptrs),
             hipMemcpyHostToDevice);
  // sample sizes
  hipMemcpy(gpu_sizes.get(), sizes.data(), samples * sizeof(*gpu_sizes), hipMemcpyHostToDevice);

  // warm-up
  ReduceAllBatchedKernel<<<1, block>>>(out_data.get(), gpu_dev_ptrs.get(), gpu_sizes.get(), R);
  hipDeviceSynchronize();
  auto start = CUDAEvent::CreateWithFlags(0);
  auto end =   CUDAEvent::CreateWithFlags(0);
  hipEventRecord(start);
  ReduceAllBatchedKernel<<<grid, block>>>(out_data.get() + samples,
                                          gpu_dev_ptrs.get(), gpu_sizes.get(), R);

  dim3 grid2(1, samples);
  ReduceAllBlockwiseKernel<<<grid2, block>>>(out_data.get(),
                                             out_data.get() + samples, n_out_per_sample,
                                             R);
  hipEventRecord(end);
  hipMemcpy(out_cpu.data(), out_data.get(), n_out * sizeof(*out_data), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  float t = 0;
  hipEventElapsedTime(&t, start, end);

  offset = 0;
  for (int i = 0; i < samples; i++) {
    double ref_value = ref_reduce(make_cspan(host_ptrs[i], sizes[i]));
    double out_value = out_cpu[i];
    auto partial_values = make_cspan(&out_cpu[samples + i * n_out_per_sample], n_out_per_sample);
    double out_partial = ref_reduce(partial_values);
    if (IsAccurate(R)) {
      EXPECT_EQ(out_value, ref_value);
      EXPECT_EQ(out_partial, ref_value);
    } else {
      double eps = ref_value * 1e-7 + 1e-7;
      EXPECT_NEAR(out_value, ref_value, eps);
      EXPECT_NEAR(out_partial, ref_value, eps);
    }
    offset += sizes[i];
  }

  t /= 1000;  // convert to seconds
  std::cout << n_in * sizeof(*in_data) / t * 1e-9 << " GB/s" << std::endl;
}

TYPED_TEST(ReduceAllGPUTest, ReduceAllBatchedKernel) {
  this->TestReduceBatched();
}

}  // namespace kernels
}  // namespace dali
