// Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gtest/gtest.h>
#include <cmath>
#include <complex>
#include <tuple>
#include <vector>

#include "dali/kernels/common/utils.h"
#include "dali/kernels/erase/erase_gpu.h"
#include "dali/kernels/scratch.h"
#include "dali/pipeline/data/tensor_list.h"
#include "dali/test/tensor_test_utils.h"
#include "dali/test/test_tensors.h"

#include "dali/kernels/erase/erase_cpu.h"

#include "dali/core/cuda_event.h"

namespace dali {
namespace kernels {

template <int ndim>
void debug_print(TensorListView<StorageCPU, uint8_t, ndim> tlv, int height, int width) {
  for (int y = 0; y < height; y++) {
    for (int x = 0; x < width; x++) {
      std::cout << std::setw(2) << int(*tlv[0](y, x)) % 100;
    }
    std::cout << endl;
  }
}

void verify_regions(ivec<4> region_shape, ivec<4> sample_shape, ivec<4> expected_cover) {
  auto cover = div_ceil(sample_shape, region_shape);
  EXPECT_EQ(cover, expected_cover);
  int idx = 0;
  for (int d0 = 0; d0 < expected_cover[0]; d0++) {
    for (int d1 = 0; d1 < expected_cover[1]; d1++) {
      for (int d2 = 0; d2 < expected_cover[2]; d2++) {
        for (int d3 = 0; d3 < expected_cover[3]; d3++) {
          auto regions_start = get_region_start(idx, region_shape, sample_shape);
          ivec<4> region_start_expected = {d0 * region_shape[0], d1 * region_shape[1],
                                           d2 * region_shape[2], d3 * region_shape[3]};
          EXPECT_EQ(regions_start, region_start_expected);
          idx++;
        }
      }
    }
  }
}

TEST(EraseGpuKernelTest, CheckUtils) {
  {
    ivec<4> region_shape = {2, 2, 32, 32};
    ivec<4> sample_shape = {16, 8, 64, 64};
    ivec<4> expected_cover = {8, 4, 2, 2};
    verify_regions(region_shape, sample_shape, expected_cover);
  }
  {
    ivec<4> region_shape = {2, 2, 32, 32};
    ivec<4> sample_shape = {16, 8, 32, 64};
    ivec<4> expected_cover = {8, 4, 1, 2};
    verify_regions(region_shape, sample_shape, expected_cover);
  }
  {
    ivec<4> region_shape = {2, 2, 32, 32};
    ivec<4> sample_shape = {16, 8, 64, 32};
    ivec<4> expected_cover = {8, 4, 2, 1};
    verify_regions(region_shape, sample_shape, expected_cover);
  }
  {
    ivec<4> region_shape = {2, 2, 32, 32};
    ivec<4> sample_shape = {16, 8, 32, 32};
    ivec<4> expected_cover = {8, 4, 1, 1};
    verify_regions(region_shape, sample_shape, expected_cover);
  }
}

enum class RegionGen {
  NO_ERASE,  ///< only copy, no erase
  FULL_ERASE,  ///< full, 1-element cover, only erase
  RANDOM_ERASE  ///< randomly generated cover
};

enum class FillType {
  MAGIC_42,  ///< use single value `42` for erase
  CHANNEL_CONSECUTIVE,  ///< use consecutive values to erase channels
  DEFAULT  ///< do not pass a value, use default `0`
};

template <int ndim>
struct EraseTestParams {
  int max_erase_regions;
  RegionGen region_generation;
  FillType fill_type;
  TensorShape<ndim> shape;
};


std::ostream& operator<<(std::ostream& os, RegionGen p) {
  switch (p) {
    case RegionGen::NO_ERASE:
      os << "RegionGen::NO_ERASE";
      break;
    case RegionGen::FULL_ERASE:
      os << "RegionGen::FULL_ERASE";
      break;
    case RegionGen::RANDOM_ERASE:
      os << "RegionGen::RANDOM_ERASE";
      break;
  }
  return os;
}


std::ostream& operator<<(std::ostream& os, FillType p) {
  switch (p) {
    case FillType::MAGIC_42:
      os << "FillType::MAGIC_42";
      break;
    case FillType::CHANNEL_CONSECUTIVE:
      os << "FillType::CHANNEL_CONSECUTIVE";
      break;
    case FillType::DEFAULT:
      os << "FillType::DEFAULT";
      break;
  }
  return os;
}

template <int ndim>
std::ostream& operator<<(std::ostream& os, const EraseTestParams<ndim>& p) {
  os << "Num erase regions: " << p.max_erase_regions << ", region generation: "
     << p.region_generation << ", fill type: " << p.fill_type << ", shape: " << p.shape;
  return os;
}

template <typename T, int ndim, int channel_dim = -1>
struct EraseGpuKernelTest :
    public testing::TestWithParam<EraseTestParams<ndim>> {
  void SetUp() override {
    auto params = this->GetParam();
    max_erase_regions_ = params.max_erase_regions;
    region_generation_ = params.region_generation;
    fill_type_ = params.fill_type;
    shape_ = params.shape;
    test_shape_ = uniform_list_shape<ndim>(batch_size_, shape_);

    input_.reshape(test_shape_);
    output_.reshape(test_shape_);
    baseline_.reshape(test_shape_);
    auto cpu_input_view = input_.cpu();
    SequentialFill(cpu_input_view);
    if (fill_type_ == FillType::DEFAULT) {
      fill_values_.resize(0);
    } else if (fill_type_ == FillType::CHANNEL_CONSECUTIVE) {
      fill_values_.resize(shape_[channel_dim]);
      int value = 0;
      for (auto &elem : fill_values_) {
        elem = value++;
      }
    } else if (fill_type_ == FillType::MAGIC_42) {
      fill_values_.resize(1);
      fill_values_[0] = 42;
    }
  }

  void RunTest() {
    if (region_generation_ == RegionGen::NO_ERASE) {
      std::cerr << ">> No cover" << std::endl;
    } else if (region_generation_ == RegionGen::FULL_ERASE) {
      std::cerr << ">> Full cover" << std::endl;
    } else if (region_generation_ == RegionGen::RANDOM_ERASE) {
      std::cerr << ">> Random cover of size: " << max_erase_regions_ << std::endl;
    }
    EraseGpu<T, ndim, channel_dim> kernel;
    KernelContext ctx;
    ctx.gpu.stream = 0;

    CreateRegions();

    auto regions_gpu = regions_.gpu();

    auto in_view = input_.gpu();

    auto req = kernel.Setup(ctx, in_view, regions_gpu, make_span(fill_values_));

    auto out_view = output_.gpu();

    ScratchpadAllocator scratch_alloc;
    scratch_alloc.Reserve(req.scratch_sizes);
    auto scratchpad = scratch_alloc.GetScratchpad();
    ctx.scratchpad = &scratchpad;

    kernel.Run(ctx, out_view, in_view, regions_gpu, make_span(fill_values_));

    CUDA_CALL(hipDeviceSynchronize());
    CUDA_CALL(hipGetLastError());

    RepackAndCalcCpu();
    Verify();
  }

  void Verify() {
    auto cpu_out_view = output_.cpu();
    auto cpu_baseline_view = baseline_.cpu();
    Check(cpu_out_view, cpu_baseline_view);
  }

  void RepackAndCalcCpu() {
    auto input_tlv = input_.cpu();
    auto baseline_tlv = baseline_.cpu();
    auto regions_tlv = regions_.cpu();
    for (int i = 0; i < batch_size_; i++) {
      auto baseline_tv = baseline_tlv[i];
      auto input_tv = input_tlv[i];

      EraseArgs<T, ndim> args;
      auto n_regions = regions_tlv[i].num_elements();
      args.rois.resize(n_regions);
      for (int j = 0; j < n_regions; j++) {
        for (int d = 0; d < ndim; d++) {
          args.rois[j].anchor[d] = regions_tlv[i](j)->lo[d];
          args.rois[j].shape[d] = regions_tlv[i](j)->hi[d] - regions_tlv[i](j)->lo[d];
          args.rois[j].fill_values = fill_values_;
          args.rois[j].channels_dim = channel_dim;
        }
      }

      EraseCpu<T, ndim> cpu_kernel;
      KernelContext ctx;
      cpu_kernel.Run(ctx, baseline_tv, input_tv, args);
    }
  }

  void CreateRegions() {
    TensorListShape<1> region_list_shape(batch_size_);
    std::mt19937 gen(0);
    if (region_generation_ == RegionGen::NO_ERASE) {
      // no cover
      region_list_shape = uniform_list_shape<1>(batch_size_, {0});
    } else if (region_generation_ == RegionGen::FULL_ERASE) {
      // full cover
      region_list_shape = uniform_list_shape<1>(batch_size_, {1});
    } else {
      std::uniform_int_distribution<> n_regions(0, max_erase_regions_);
      for (int i = 0; i < batch_size_; ++i) {
        region_list_shape.set_tensor_shape(i, {n_regions(gen)});
      }
    }
    regions_.reshape(region_list_shape);
    auto regions_cpu = regions_.cpu();
    if (region_generation_ == RegionGen::FULL_ERASE) {
      // full cover
      for (int i = 0; i < batch_size_; i++) {
        auto regions_tv = regions_cpu[i];
        *regions_tv(0) = ibox<ndim>({0}, to_ivec(shape_));
      }
    } else if (region_generation_ == RegionGen::RANDOM_ERASE) {
      for (int i = 0; i < batch_size_; i++) {
        auto regions_tv = regions_cpu[i];
        for (int j = 0; j < regions_tv.shape[0]; j ++) {
          ibox<ndim> region_box;
          for (int d = 0; d < ndim; d++) {
            std::uniform_int_distribution<>  start_dim(0, shape_[d] - 1);
            region_box.lo[d] = start_dim(gen);
            std::uniform_int_distribution<>  end_dim(region_box.lo[d] + 1, shape_[d]);
            region_box.hi[d] = end_dim(gen);
          }
          *regions_tv(j) = region_box;
        }
      }
    }
  }

  int max_erase_regions_;
  RegionGen region_generation_;
  FillType fill_type_;
  std::vector<T> fill_values_;
  TensorShape<ndim> shape_;
  TensorListShape<ndim> test_shape_;
  constexpr static int batch_size_ = 16;
  TestTensorList<T, ndim> input_, output_, baseline_;
  TestTensorList<ibox<ndim>, 1> regions_;
};

using EraseGpuKernel1fTest = EraseGpuKernelTest<float, 1>;
using EraseGpuKernel2fTest = EraseGpuKernelTest<float, 2>;
using EraseGpuKernel2NCfTest = EraseGpuKernelTest<float, 2, 1>;
using EraseGpuKernel3fTest = EraseGpuKernelTest<float, 3>;
using EraseGpuKernel3fHWCTest = EraseGpuKernelTest<float, 3, 2>;
using EraseGpuKernel3fCHWTest = EraseGpuKernelTest<float, 3, 0>;
using EraseGpuKernel4fDHCWTest = EraseGpuKernelTest<float, 4, 2>;
using EraseGpuKernel4fDHWCTest = EraseGpuKernelTest<float, 4, 3>;
using EraseGpuKernel5fTest = EraseGpuKernelTest<float, 5>;

#define ERASE_TEST_P(TEST) \
  TEST_P(TEST, RunAndVerify) { \
    this->RunTest(); \
  }

ERASE_TEST_P(EraseGpuKernel1fTest)
ERASE_TEST_P(EraseGpuKernel2fTest)
ERASE_TEST_P(EraseGpuKernel2NCfTest)
ERASE_TEST_P(EraseGpuKernel3fTest)
ERASE_TEST_P(EraseGpuKernel3fHWCTest)
ERASE_TEST_P(EraseGpuKernel3fCHWTest)
ERASE_TEST_P(EraseGpuKernel4fDHCWTest)
ERASE_TEST_P(EraseGpuKernel4fDHWCTest)
ERASE_TEST_P(EraseGpuKernel5fTest)

// Parameters for tests are:
// <number of erase regions>, <generation scheme>, <fill_type>, <shape>

std::vector<EraseTestParams<1>> values_1 = {
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {512 * 1024}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {512 * 1024}},
    {1, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {512 * 1024}},
    {10, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {512 * 1024}},
    {100, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {512 * 1024}},
    {0, RegionGen::NO_ERASE, FillType::DEFAULT, {512 * 1024}},
    {1, RegionGen::FULL_ERASE, FillType::DEFAULT, {512 * 1024}},
    {1, RegionGen::RANDOM_ERASE, FillType::DEFAULT, {512 * 1024}},
    {10, RegionGen::RANDOM_ERASE, FillType::DEFAULT, {512 * 1024}},
    {100, RegionGen::RANDOM_ERASE, FillType::DEFAULT, {512 * 1024}},
};

std::vector<EraseTestParams<2>> values_2 = {
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {512, 1024}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {512, 1024}},
    {1, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {512, 1024}},
    {10, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {512, 1024}},
    {100, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {512, 1024}},
};

std::vector<EraseTestParams<2>> values_2NC = {
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {512 * 1024, 3}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {512 * 1024, 3}},
    {1, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {512 * 1024, 3}},
    {10, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {512 * 1024, 3}},
    {100, RegionGen::RANDOM_ERASE, FillType::DEFAULT, {512 * 1024, 3}},
};

std::vector<EraseTestParams<3>> values_3 = {
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {16, 256, 256}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {16, 256, 256}},
    {1, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {16, 256, 256}},
    {10, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {16, 256, 256}},
    {100, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {16, 256, 256}},
    {1000, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {16, 256, 256}},
};

std::vector<EraseTestParams<3>> values_3HWC = {
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {256, 256, 1}},
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {256, 256, 3}},
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {256, 256, 4}},
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {256, 256, 8}},
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {256, 256, 16}},
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {256, 256, 64}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {256, 256, 1}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {256, 256, 3}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {256, 256, 4}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {256, 256, 8}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {256, 256, 16}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {256, 256, 64}},

    {0, RegionGen::NO_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 1}},
    {0, RegionGen::NO_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 3}},
    {0, RegionGen::NO_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 4}},
    {0, RegionGen::NO_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 8}},
    {0, RegionGen::NO_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 16}},
    {0, RegionGen::NO_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 64}},
    {1, RegionGen::FULL_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 1}},
    {1, RegionGen::FULL_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 3}},
    {1, RegionGen::FULL_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 4}},
    {1, RegionGen::FULL_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 8}},
    {1, RegionGen::FULL_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 16}},
    {1, RegionGen::FULL_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 64}},

    {0, RegionGen::NO_ERASE, FillType::DEFAULT, {256, 256, 1}},
    {0, RegionGen::NO_ERASE, FillType::DEFAULT, {256, 256, 3}},
    {0, RegionGen::NO_ERASE, FillType::DEFAULT, {256, 256, 4}},
    {0, RegionGen::NO_ERASE, FillType::DEFAULT, {256, 256, 8}},
    {0, RegionGen::NO_ERASE, FillType::DEFAULT, {256, 256, 16}},
    {0, RegionGen::NO_ERASE, FillType::DEFAULT, {256, 256, 64}},
    {1, RegionGen::FULL_ERASE, FillType::DEFAULT, {256, 256, 1}},
    {1, RegionGen::FULL_ERASE, FillType::DEFAULT, {256, 256, 3}},
    {1, RegionGen::FULL_ERASE, FillType::DEFAULT, {256, 256, 4}},
    {1, RegionGen::FULL_ERASE, FillType::DEFAULT, {256, 256, 8}},
    {1, RegionGen::FULL_ERASE, FillType::DEFAULT, {256, 256, 16}},
    {1, RegionGen::FULL_ERASE, FillType::DEFAULT, {256, 256, 64}},

    {1, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 1}},
    {10, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 1}},
    {100, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 1}},
    {1000, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 1}},
    {1, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 3}},
    {10, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 3}},
    {100, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 3}},
    {1000, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 3}},
    {1, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 16}},
    {10, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 16}},
    {100, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 16}},
    {1000, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {256, 256, 16}},
};

std::vector<EraseTestParams<3>> values_3CHW = {
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {3, 256, 256}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {3, 256, 256}},
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {16, 256, 256}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {16, 256, 256}},
    {1, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {3, 256, 256}},
    {10, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {3, 256, 256}},
    {100, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {3, 256, 256}},
    {1000, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {3, 256, 256}},
};

std::vector<EraseTestParams<4>> values_4DHCW = {
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {64, 64, 3, 64}},
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {64, 64, 4, 64}},
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {64, 64, 8, 64}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {64, 64, 3, 64}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {64, 64, 4, 64}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {64, 64, 8, 64}},
    {0, RegionGen::NO_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 3, 64}},
    {0, RegionGen::NO_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 4, 64}},
    {0, RegionGen::NO_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 8, 64}},
    {1, RegionGen::FULL_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 3, 64}},
    {1, RegionGen::FULL_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 4, 64}},
    {1, RegionGen::FULL_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 8, 64}},
    {1, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 3, 64}},
    {10, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 3, 64}},
    {100, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 3, 64}},
    {100, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {16, 128, 3, 256}},
    {1000, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 3, 64}},
    {1000, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {16, 128, 3, 256}},
};

std::vector<EraseTestParams<4>> values_4DHWC = {
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {64, 64, 64, 3}},
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {64, 64, 64, 4}},
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {64, 64, 64, 8}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {64, 64, 64, 3}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {64, 64, 64, 4}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {64, 64, 64, 8}},
    {0, RegionGen::NO_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 64, 3}},
    {0, RegionGen::NO_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 64, 4}},
    {0, RegionGen::NO_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 64, 8}},
    {1, RegionGen::FULL_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 64, 3}},
    {1, RegionGen::FULL_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 64, 4}},
    {1, RegionGen::FULL_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 64, 8}},
    {1, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 64, 3}},
    {10, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 64, 3}},
    {100, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 64, 3}},
    {100, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {16, 128, 256, 3}},
    {1000, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {64, 64, 64, 3}},
    {1000, RegionGen::RANDOM_ERASE, FillType::CHANNEL_CONSECUTIVE, {16, 128, 256, 3}},
};

std::vector<EraseTestParams<5>> values_5 = {
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {4, 6, 5, 64, 64}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {4, 6, 5, 64, 64}},
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {2, 3, 3, 256, 256}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {2, 3, 3, 256, 256}},
    {1, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {4, 6, 32, 32, 32}},
    {1, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {4, 6, 5, 64, 64}},
    {0, RegionGen::NO_ERASE, FillType::MAGIC_42, {2, 3, 3, 256, 16}},
    {1, RegionGen::FULL_ERASE, FillType::MAGIC_42, {2, 3, 3, 256, 16}},
    {1, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {2, 3, 3, 256, 16}},
    {10, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {2, 3, 3, 256, 16}},
    {100, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {2, 3, 3, 256, 16}},
    {1000, RegionGen::RANDOM_ERASE, FillType::MAGIC_42, {2, 3, 3, 256, 16}},
};

#define INSTANTIATE_ERASE_SUITE(TEST, VALUES) \
  INSTANTIATE_TEST_SUITE_P(TEST, TEST ## Test, testing::ValuesIn(VALUES));

INSTANTIATE_ERASE_SUITE(EraseGpuKernel1f, values_1);
INSTANTIATE_ERASE_SUITE(EraseGpuKernel2f, values_2);
INSTANTIATE_ERASE_SUITE(EraseGpuKernel2NCf, values_2NC);
INSTANTIATE_ERASE_SUITE(EraseGpuKernel3f, values_3);
INSTANTIATE_ERASE_SUITE(EraseGpuKernel3fHWC, values_3HWC);
INSTANTIATE_ERASE_SUITE(EraseGpuKernel3fCHW, values_3CHW);
INSTANTIATE_ERASE_SUITE(EraseGpuKernel4fDHCW, values_4DHCW);
INSTANTIATE_ERASE_SUITE(EraseGpuKernel4fDHWC, values_4DHWC);
INSTANTIATE_ERASE_SUITE(EraseGpuKernel5f, values_5);


}  // namespace kernels
}  // namespace dali
