#include "hip/hip_runtime.h"
// Copyright (c) 2022, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include "dali/core/dev_buffer.h"
#include "dali/core/mm/memory.h"
#include "dali/core/static_switch.h"
#include "dali/kernels/dynamic_scratchpad.h"
#include "dali/kernels/kernel.h"
#include "dali/kernels/signal/resampling_gpu.cuh"
#include "dali/kernels/signal/resampling_gpu.h"

namespace dali {
namespace kernels {
namespace signal {

namespace resampling {

template <typename Out, typename In>
void ResamplerGPU<Out, In>::Initialize(int lobes, int lookup_size) {
  windowed_sinc(window_cpu_, lookup_size, lobes);
  window_gpu_storage_.from_host(window_cpu_.storage);
  window_gpu_ = window_cpu_;
  window_gpu_.lookup = window_gpu_storage_.data();
  CUDA_CALL(hipStreamSynchronize(0));
}

template <typename Out, typename In>
KernelRequirements ResamplerGPU<Out, In>::Setup(KernelContext &context, const InListGPU<In> &in,
                                                span<const Args> args) {
  KernelRequirements req;
  auto out_shape = in.shape;
  for (int i = 0; i < in.num_samples(); i++) {
    auto in_sh = in.shape.tensor_shape_span(i);
    auto out_sh = out_shape.tensor_shape_span(i);
    auto &arg = args[i];
    if (arg.out_begin > 0 || arg.out_end > 0) {
      out_sh[0] = arg.out_end - arg.out_begin;
    } else {
      out_sh[0] = resampled_length(in_sh[0], arg.in_rate, arg.out_rate);
    }
  }
  req.output_shapes = {out_shape};
  return req;
}

template <typename Out, typename In>
void ResamplerGPU<Out, In>::Run(KernelContext &context, const OutListGPU<Out> &out,
                                const InListGPU<In> &in, span<const Args> args) {
  if (window_gpu_storage_.empty())
    Initialize();

  assert(context.scratchpad);
  auto &scratch = *context.scratchpad;

  int nsamples = in.num_samples();
  auto samples_cpu =
      make_span(scratch.Allocate<mm::memory_kind::pinned, SampleDesc>(nsamples), nsamples);

  bool any_multichannel = false;
  for (int i = 0; i < nsamples; i++) {
    auto &desc = samples_cpu[i];
    auto in_sample = in[i];
    auto out_sample = out[i];
    desc.in = in_sample.data;
    desc.out = out_sample.data;
    desc.window = window_gpu_;
    const auto &in_sh = in_sample.shape;
    desc.in_len = in_sh[0];
    auto &arg = args[i];
    desc.out_begin = arg.out_begin > 0 ? arg.out_begin : 0;
    desc.out_end =
        arg.out_end > 0 ? arg.out_end : resampled_length(in_sh[0], arg.in_rate, arg.out_rate);
    assert((desc.out_end - desc.out_begin) == out_sample.shape[0]);
    desc.nchannels = in_sh.sample_dim() > 1 ? in_sh[1] : 1;
    desc.scale = arg.in_rate / arg.out_rate;
    any_multichannel |= desc.nchannels > 1;
  }

  auto samples_gpu = scratch.ToGPU(context.gpu.stream, samples_cpu);

  dim3 block(256, 1);
  int blocks_per_sample = std::max(32, 1024 / nsamples);
  dim3 grid(blocks_per_sample, nsamples);

  // window coefficients and temporary per channel out values
  size_t shm_size = (window_gpu_storage_.size() + (SHM_NCHANNELS + 1) * block.x) * sizeof(float);

  BOOL_SWITCH(!any_multichannel, SingleChannel,
              (ResampleGPUKernel<Out, In, SingleChannel>
               <<<grid, block, shm_size, context.gpu.stream>>>(samples_gpu);));  // NOLINT
  CUDA_CALL(hipGetLastError());
}

DALI_INSTANTIATE_RESAMPLER_GPU();

}  // namespace resampling
}  // namespace signal
}  // namespace kernels
}  // namespace dali
