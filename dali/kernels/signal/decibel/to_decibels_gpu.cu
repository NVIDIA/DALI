#include "hip/hip_runtime.h"
// Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "dali/kernels/signal/decibel/to_decibels_gpu.h"
#include <cmath>
#include <complex>
#include <vector>
#include "dali/core/common.h"
#include "dali/core/error_handling.h"
#include "dali/core/format.h"
#include "dali/kernels/kernel.h"
#include "dali/kernels/signal/decibel/decibel_calculator.h"

namespace dali {
namespace kernels {
namespace signal {

template <typename T>
struct SampleDesc {
  const T *in = nullptr;
  T *out = nullptr;
  int64_t size = 0;
};

template <typename T>
__global__ void ToDecibelsKernel(const SampleDesc<T>* sample_data,
                                 ToDecibelsArgs<T> args,
                                 const T* max_values = nullptr) {
  const int64_t block_size = blockDim.y * blockDim.x;
  const int64_t grid_size = gridDim.x * block_size;
  const int sample_idx = blockIdx.y;
  const auto sample = sample_data[sample_idx];
  const int64_t offset = block_size * blockIdx.x;
  const int64_t tid = threadIdx.y * blockDim.x + threadIdx.x;
  assert(args.ref_max == (max_values != nullptr));
  T s_ref = args.ref_max ? max_values[sample_idx] : args.s_ref;
  MagnitudeToDecibel<T> dB(args.multiplier, s_ref, args.min_ratio);
  for (int64_t idx = offset + tid; idx < sample.size; idx += grid_size) {
    sample.out[idx] = dB(sample.in[idx]);
  }
}

template <typename T, int Dims>
ToDecibelsGpu<T, Dims>::~ToDecibelsGpu() = default;

template <typename T, int Dims>
KernelRequirements ToDecibelsGpu<T, Dims>::Setup(KernelContext &context,
                                                 const InListGPU<T, Dims> &in) {
  auto out_shape = in.shape;
  const size_t num_samples = in.size();
  ScratchpadEstimator se;
  se.add<SampleDesc<T>>(AllocType::Host, num_samples);
  se.add<SampleDesc<T>>(AllocType::GPU, num_samples);
  KernelRequirements req;
  req.scratch_sizes = se.sizes;
  req.output_shapes = {out_shape};
  return req;
}

template <typename T, int Dims>
void ToDecibelsGpu<T, Dims>::Run(KernelContext &context, const OutListGPU<T, Dims> &out,
                                 const InListGPU<T, Dims> &in, const ToDecibelsArgs<T> &args,
                                 InTensorGPU<T, 1> max_values) {
  auto num_samples = in.size();
  auto* sample_data = context.scratchpad->Allocate<SampleDesc<T>>(AllocType::Host, num_samples);

  for (int i = 0; i < num_samples; i++) {
    auto &sample = sample_data[i];
    sample.out = out.tensor_data(i);
    sample.in = in.tensor_data(i);
    sample.size = volume(in.tensor_shape(i));
    assert(sample.size == volume(out.tensor_shape(i)));
  }

  auto* sample_data_gpu = context.scratchpad->Allocate<SampleDesc<T>>(AllocType::GPU, num_samples);
  CUDA_CALL(
    hipMemcpyAsync(sample_data_gpu, sample_data, num_samples * sizeof(SampleDesc<T>),
                    hipMemcpyHostToDevice, context.gpu.stream));

  dim3 block(32, 32);
  auto blocks_per_sample = std::max(32, 1024 / num_samples);
  dim3 grid(blocks_per_sample, num_samples);
  ToDecibelsKernel<T><<<grid, block, 0, context.gpu.stream>>>(
      sample_data_gpu, args, max_values.data);
}

template class ToDecibelsGpu<float, 1>;
template class ToDecibelsGpu<float, 2>;
template class ToDecibelsGpu<float, 3>;
template class ToDecibelsGpu<float, 4>;

template class ToDecibelsGpu<double, 1>;
template class ToDecibelsGpu<double, 2>;
template class ToDecibelsGpu<double, 3>;
template class ToDecibelsGpu<double, 4>;

}  // namespace signal
}  // namespace kernels
}  // namespace dali
