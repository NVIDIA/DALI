// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gtest/gtest.h>
#include <memory>
#include <vector>
#include "dali/core/util.h"
#include "dali/test/test_tensors.h"
#include "dali/kernels/signal/window/extract_windows_gpu.cuh"
#include "dali/kernels/scratch.h"
#include "dali/kernels/signal/window/window_functions.h"

namespace dali {
namespace kernels {
namespace signal {

TEST(ExtractWindowsGpu, NonBatchedKernel) {
  float *in_gpu, *out_gpu;
  int winlen = 60;
  int outwinlen = 63;
  int windows = 80;
  int stride = windows;
  int step = 10;
  int length = windows * step - 100;;
  int center = 5;
  bool reflect = false;
  hipMalloc(&in_gpu, sizeof(float)*length);
  hipMalloc(&out_gpu, sizeof(float)*windows*outwinlen);
  std::vector<float> in(length), out(windows*outwinlen);

  for (int i = 0; i < length; i++) {
    in[i] = i + 1000;
  }

  hipMemcpy(in_gpu, in.data(), sizeof(float)*length, hipMemcpyHostToDevice);
  hipMemset(out_gpu, 0xff, sizeof(float)*windows*outwinlen);
  int xblocks = div_ceil(length, 32);
  int yblocks = div_ceil(winlen, 32);
  window::ExtractVerticalWindowsKernel<<<dim3(xblocks, yblocks), dim3(32, 32)>>>(
    out_gpu, windows, stride, in_gpu, length, nullptr, winlen, outwinlen, center, step, reflect);
  hipMemcpy(out.data(), out_gpu, sizeof(float)*winlen*windows, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  for (int w = 0; w < windows; w++) {
    for (int i = 0; i < winlen; i++) {
      int idx = w * step + i - center;
      if (reflect)
        idx = boundary::idx_reflect_101(idx, 0, length);

      float ref = idx >= 0 && idx < length ? in[idx] : 0;
      EXPECT_EQ(out[w + i*stride], ref)
        << "@ window = " << w << ", index = " << i;
    }
    for (int i = winlen; i < outwinlen; i++) {
      EXPECT_EQ(out[w + i*stride], 0)
        << "padding @ window = " << w << ", index = " << i;
    }
  }

  if (HasFailure()) {
    std::cout << "Debug: Extract window actual output:\n";
    for (int i = 0; i < outwinlen; i++) {
      for (int j = 0; j < windows; j++) {
        std::cout << out[i*stride+j] << " ";
      }
      std::cout << "\n";
    }
    std::cout << std::flush;
  }

  hipFree(in_gpu);
  hipFree(out_gpu);
}


void TestBatchedExtract(
    const TensorListShape<1> &lengths,
    bool concatenate,
    Padding padding,
    bool vertical,
    span<const float> window,
    int out_win_len = -1) {
  std::unique_ptr<ExtractWindowsGpuImpl<float, float>> extract;
  if (vertical)
    extract = std::make_unique<ExtractVerticalWindowsGpuImpl<float, float>>();
  else
    extract = std::make_unique<ExtractHorizontalWindowsGpuImpl<float, float>>();

  int N = lengths.num_samples();

  ptrdiff_t total_length = 0;
  for (int i = 0; i < N; i++) {
    total_length += lengths[i][0];
  }

  TestTensorList<float, 1> in_list;
  in_list.reshape(lengths);
  auto in_cpu = in_list.cpu();
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < lengths[i][0]; j++)
      in_cpu[i].data[j] = 1000*(i+1)+j;
  }

  ExtractWindowsArgs args;
  args.window_length = window.empty() ? 55 : window.size();
  args.window_center = window.empty() ? 21 : window.size()/2;
  args.window_step = 2;
  args.padding = padding;

  int out_win_len_actual = out_win_len < 0 ? args.window_length : out_win_len;


  KernelContext ctx;
  ScratchpadAllocator sa;

  auto in_gpu = in_list.gpu(0);

  auto req = extract->Setup(ctx, lengths, args, concatenate, out_win_len);
  ASSERT_EQ(req.output_shapes.size(), 1u);
  ASSERT_EQ(req.output_shapes[0].num_samples(), concatenate ? 1 : N);

  sa.Reserve(req.scratch_sizes);
  auto scratchpad = sa.GetScratchpad();
  ctx.scratchpad = &scratchpad;

  TestTensorList<float, 2> out;
  memory::KernelUniquePtr<float> gpu_win;
  if (!window.empty()) {
    gpu_win = memory::alloc_unique<float>(AllocType::GPU, window.size());
    hipMemcpy(gpu_win.get(), window.data(), sizeof(float)*window.size(), hipMemcpyHostToDevice);
  }
  auto window_gpu = make_tensor_gpu<1>(gpu_win.get(), { window.size() });
  out.reshape(req.output_shapes[0].to_static<2>());
  auto out_gpu = out.gpu(0);
  hipMemset(out_gpu.data[0], 0xff, sizeof(float)*out_gpu.shape.num_elements());
  extract->Run(ctx, out_gpu, in_gpu, window_gpu);
  auto out_cpu = out.cpu();

  ptrdiff_t ofs = 0;

  for (int sample = 0; sample < N; sample++) {
    ptrdiff_t length = lengths[sample][0];
    int nwnd = args.num_windows(length);

    int out_sample = 0;
    if (!concatenate) {
      ofs = 0;
      out_sample = sample;
    }
    ptrdiff_t sample_stride = vertical ? out_cpu.shape[out_sample][1] : 1;
    ptrdiff_t window_stride = vertical ? 1 : out_cpu.shape[out_sample][1];

    for (int w = 0; w < nwnd; w++, ofs += window_stride) {
        int i = 0;
        for (; i < args.window_length; i++) {
        ptrdiff_t idx = w * args.window_step + i - args.window_center;
        if (args.padding == Padding::Reflect) {
          idx = boundary::idx_reflect_101(idx, length);
        }
        float ref = idx >= 0 && idx < length ? in_cpu.data[sample][idx] : 0;
        if (!window.empty())
          ref *= window[i];
        ASSERT_EQ(out_cpu.data[out_sample][ofs + i*sample_stride], ref)
          << "@ sample = " << sample
          << ", window = " << w << ", index = " << i;
      }
      for (; i < out_win_len_actual; i++) {
        ASSERT_EQ(out_cpu.data[out_sample][ofs + i*sample_stride], 0)
          << "padding @ sample = " << sample
          << ", window = " << w << ", index = " << i;
      }
    }
  }
}

void TestBatchedExtract(
    bool concatenate,
    Padding padding,
    bool vertical,
    span<const float> window,
    int out_win_len = -1) {
  TensorListShape<1> lengths({ TensorShape<1>{5}, TensorShape<1>{305}, TensorShape<1>{157} });
  TestBatchedExtract(lengths, concatenate, padding, vertical, window, out_win_len);
}

TEST(ExtractVerticalWindowsGpu, BatchedConcat) {
  TestBatchedExtract(true, Padding::Reflect, true, {});
}

TEST(ExtractVerticalWindowsGpu, BatchedSeparate) {
  TestBatchedExtract(false, Padding::Zero, true, {});
}

TEST(ExtractVerticalWindowsGpu, BatchedConcatWindowFunc) {
  vector<float> window(60);
  HannWindow(make_span(window));
  TestBatchedExtract(true, Padding::Zero, true, make_cspan(window));
}

TEST(ExtractVerticalWindowsGpu, BatchedSeparateWindowFunc) {
  vector<float> window(60);
  HammingWindow(make_span(window));
  TestBatchedExtract(false, Padding::Reflect, true, make_cspan(window));
}

TEST(ExtractVerticalWindowsGpu, BatchedSeparateWindowFuncPad) {
  vector<float> window(60);
  HammingWindow(make_span(window));
  TestBatchedExtract(true, Padding::Reflect, false, make_cspan(window), 72);
}

TEST(ExtractHorizontalWindowsGpu, BatchedConcat) {
  TestBatchedExtract(true, Padding::Reflect, false, {});
}

TEST(ExtractHorizontalWindowsGpu, BatchedSeparate) {
  TestBatchedExtract(false, Padding::Zero, false, {});
}

TEST(ExtractHorizontalWindowsGpu, BatchedConcatWindowFunc) {
  vector<float> window(60);
  HannWindow(make_span(window));
  TestBatchedExtract(true, Padding::Zero, false, make_cspan(window));
}

TEST(ExtractHorizontalWindowsGpu, BatchedSeparateWindowFunc) {
  vector<float> window(60);
  HammingWindow(make_span(window));
  TestBatchedExtract(false, Padding::Reflect, false, make_cspan(window));
}

TEST(ExtractHorizontalWindowsGpu, BatchedSeparateWindowFuncPad) {
  vector<float> window(60);
  HammingWindow(make_span(window));
  TestBatchedExtract(false, Padding::Reflect, false, make_cspan(window), 72);
}

TEST(ExtractHorizontalWindowsGpu, BatchedConcatWindowFuncPad) {
  vector<float> window(60);
  HammingWindow(make_span(window));
  TestBatchedExtract(false, Padding::Reflect, true, make_cspan(window), 72);
}

TEST(ExtractHorizontalWindowsGpu, SizeSweep) {
  int max_size = 2048;
  std::vector<TensorShape<1>> lengths;
  int step = 1;
  for (int s = 1; s <= max_size; s+=step) {
    if ((s&255) == 0) {
      if (step > 1)  // add 2^n-1
        lengths.push_back({s-1});
      step += step;
    }
    lengths.push_back({s});
  }
  TensorListShape<1> shape(lengths);
  vector<float> window(60);
  HammingWindow(make_span(window));
  TestBatchedExtract(shape, false, Padding::Reflect, false, make_cspan(window));
}



}  // namespace signal
}  // namespace kernels
}  // namespace dali
