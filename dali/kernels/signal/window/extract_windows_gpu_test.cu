// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gtest/gtest.h>
#include <vector>
#include "dali/core/util.h"
#include "dali/test/test_tensors.h"
#include "dali/kernels/signal/window/extract_windows_gpu.cuh"
#include "dali/kernels/scratch.h"
#include "dali/kernels/signal/window/window_functions.h"

namespace dali {
namespace kernels {
namespace signal {

TEST(ExtractWindowsGPU, NonBatchedKernel) {
  float *in_gpu, *out_gpu;
  int winlen = 60;
  int windows = 80;
  int stride = windows;
  int step = 10;
  int length = windows * step - 100;;
  int center = 5;
  bool reflect = false;
  hipMalloc(&in_gpu, sizeof(float)*length);
  hipMalloc(&out_gpu, sizeof(float)*windows*winlen);
  std::vector<float> in(length), out(windows*winlen);

  for (int i = 0; i < length; i++) {
    in[i] = i + 1000;
  }

  hipMemcpy(in_gpu, in.data(), sizeof(float)*length, hipMemcpyHostToDevice);
  hipMemset(out_gpu, 0, sizeof(float)*windows*winlen);
  int xblocks = div_ceil(length, 32);
  int yblocks = div_ceil(winlen, 32);
  window::ExtractWindowsKernel<<<dim3(xblocks, yblocks), dim3(32, 32)>>>(
    out_gpu, windows, stride, in_gpu, length, nullptr, winlen, center, step, reflect);
  hipMemcpy(out.data(), out_gpu, sizeof(float)*winlen*windows, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  for (int w = 0; w < windows; w++) {
    for (int i = 0; i < winlen; i++) {
      int idx = w * step + i - center;
      if (reflect) {
        for (;;) {
          if (idx < 0)
            idx = -idx;
          else if (idx >= length)
            idx = 2*(length-1) - idx;
          else
            break;
        }
      }
      float ref = idx >= 0 && idx < length ? in[idx] : 0;
      EXPECT_EQ(out[w + i*stride], ref)
        << "@ window = " << w << ", index = " << i;
    }
  }

  if (HasFailure()) {
    std::cout << "Debug: Extract window actual output:\n";
    for (int i = 0; i < winlen; i++) {
      for (int j = 0; j < windows; j++) {
        std::cout << out[i*stride+j] << " ";
      }
      std::cout << "\n";
    }
    std::cout << std::flush;
  }

  hipFree(in_gpu);
  hipFree(out_gpu);
}



void TestBatchedExtract(bool concatenate, Padding padding, span<const float> window) {
  ExtractWindowsGPUImpl<float, float> extract;

  TensorListShape<1> lengths({ TensorShape<1>{5}, TensorShape<1>{305}, TensorShape<1>{157} });
  int N = lengths.num_samples();

  ptrdiff_t total_length = 0;
  for (int i = 0; i < N; i++) {
    total_length += lengths[i][0];
  }

  TestTensorList<float, 1> in_list;
  in_list.reshape(lengths);
  auto in_cpu = in_list.cpu();
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < lengths[j][0]; j++)
      in_cpu[i].data[j] = 1000*(i+1)+j;
  }

  ExtractWindowsArgs args;
  args.window_length = window.empty() ? 55 : window.size();
  args.window_center = window.empty() ? 21 : window.size()/2;
  args.window_step = 2;
  args.padding = padding;

  KernelContext ctx;
  ScratchpadAllocator sa;

  auto in_gpu = in_list.gpu(0);

  auto req = extract.Setup(ctx, in_gpu, args, concatenate);
  ASSERT_EQ(req.output_shapes.size(), 1u);
  ASSERT_EQ(req.output_shapes[0].num_samples(), concatenate ? 1 : N);

  sa.Reserve(req.scratch_sizes);
  auto scratchpad = sa.GetScratchpad();
  ctx.scratchpad = &scratchpad;

  TestTensorList<float, 2> out;
  memory::KernelUniquePtr<float> gpu_win;
  if (!window.empty()) {
    gpu_win = memory::alloc_unique<float>(AllocType::GPU, window.size());
    hipMemcpy(gpu_win.get(), window.data(), sizeof(float)*window.size(), hipMemcpyHostToDevice);
  }
  auto window_gpu = make_tensor_gpu<1>(gpu_win.get(), { window.size() });
  out.reshape(req.output_shapes[0].to_static<2>());
  extract.Run(ctx, out.gpu(0), in_gpu, window_gpu);
  auto out_cpu = out.cpu();

  int ofs = 0;

  for (int sample = 0; sample < N; sample++) {
    ptrdiff_t length = lengths[sample][0];
    int nwnd = args.num_windows(length);

    int out_sample = 0;
    if (!concatenate) {
      ofs = 0;
      out_sample = sample;
    }
    ptrdiff_t stride = out_cpu.shape[out_sample][1];

    for (int w = 0; w < nwnd; w++, ofs++) {
        for (int i = 0; i < args.window_length; i++) {
        ptrdiff_t idx = w * args.window_step + i - args.window_center;
        if (args.padding == Padding::Reflect) {
          for (;;) {
            if (idx < 0)
              idx = -idx;
            else if (idx >= length)
              idx = 2*(length-1) - idx;
            else
              break;
          }
        }
        float ref = idx >= 0 && idx < length ? in_cpu.data[sample][idx] : 0;
        if (!window.empty())
          ref *= window[i];
        ASSERT_EQ(out_cpu.data[out_sample][ofs + i*stride], ref)
          << "@ sample = " << sample
          << ", window = " << w << ", index = " << i;
      }
    }
  }
}

TEST(ExtractWindowsGPU, BatchedConcat) {
  TestBatchedExtract(true, Padding::Reflect, {});
}

TEST(ExtractWindowsGPU, BatchedSeparate) {
  TestBatchedExtract(false, Padding::Zero, {});
}

TEST(ExtractWindowsGPU, BatchedConcatWindowFunc) {
  vector<float> window(60);
  HannWindow(make_span(window));
  TestBatchedExtract(true, Padding::Zero, make_cspan(window));
}

TEST(ExtractWindowsGPU, BatchedSeparateWindowFunc) {
  vector<float> window(60);
  HammingWindow(make_span(window));
  TestBatchedExtract(false, Padding::Reflect, make_cspan(window));
}


}  // namespace signal
}  // namespace kernels
}  // namespace dali
