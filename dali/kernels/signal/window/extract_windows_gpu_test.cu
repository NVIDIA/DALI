// Copyright (c) 2019-2021, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gtest/gtest.h>
#include <memory>
#include <vector>
#include "dali/core/util.h"
#include "dali/test/test_tensors.h"
#include "dali/kernels/signal/window/extract_windows_gpu.cuh"
#include "dali/kernels/scratch.h"
#include "dali/kernels/signal/window/window_functions.h"

namespace dali {
namespace kernels {
namespace signal {

TEST(ExtractWindowsGPU, NonBatchedKernel) {
  float *in_gpu, *out_gpu;
  int winlen = 60;
  int outwinlen = 63;
  int windows = 80;
  int stride = windows;
  int step = 10;
  int length = windows * step - 100;
  int center = 5;
  CUDA_CALL(hipMalloc(&in_gpu, sizeof(float)*length));
  CUDA_CALL(hipMalloc(&out_gpu, sizeof(float)*windows*outwinlen));
  std::vector<float> in(length), out(windows*outwinlen);

  for (int i = 0; i < length; i++) {
    in[i] = i + 1000;
  }

  for (bool reflect : {true, false}) {
    CUDA_CALL(hipMemcpy(in_gpu, in.data(), sizeof(float)*length, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemset(out_gpu, 0xff, sizeof(float)*windows*outwinlen));
    int xblocks = div_ceil(length, 32);
    int yblocks = div_ceil(winlen, 32);
    int win_data_start = (outwinlen - winlen) / 2;

    window::ExtractVerticalWindowsKernel<<<dim3(xblocks, yblocks), dim3(32, 32)>>>(
        out_gpu, windows, stride, in_gpu, length, nullptr, outwinlen, win_data_start, winlen,
        center, step, reflect);
    CUDA_CALL(hipMemcpy(out.data(), out_gpu, sizeof(float) * outwinlen * windows,
                         hipMemcpyDeviceToHost));
    CUDA_CALL(hipDeviceSynchronize());

    for (int w = 0; w < windows; w++) {
      int i = 0;
      for (; i < win_data_start; i++) {
        EXPECT_EQ(out[w + i*stride], 0)
          << "padding @ window = " << w << ", index = " << i;
      }
      for (int j = 0; j < winlen; j++, i++) {
        int idx = w * step + j - center;
        if (reflect)
          idx = boundary::idx_reflect_101(idx, 0, length);

        float ref = idx >= 0 && idx < length ? in[idx] : 0;
        EXPECT_EQ(out[w + i*stride], ref)
          << "@ window = " << w << ", index = " << i;
      }
      for (; i < outwinlen; i++) {
        EXPECT_EQ(out[w + i*stride], 0)
          << "padding @ window = " << w << ", index = " << i;
      }
    }

    if (HasFailure()) {
      std::cout << "Debug: Extract window actual output:\n";
      for (int i = 0; i < outwinlen; i++) {
        for (int j = 0; j < windows; j++) {
          std::cout << out[i*stride+j] << " ";
        }
        std::cout << "\n";
      }
      std::cout << std::flush;
    }
  }

  hipFree(in_gpu);
  hipFree(out_gpu);
}


void TestBatchedExtract(
    ExtractWindowsImplGPU<float, float> *extract,
    const TensorListShape<1> &lengths,
    bool concatenate,
    Padding padding,
    span<const float> window,
    int out_win_len = -1) {
  bool vertical = extract->IsVertical();

  ScratchpadAllocator sa;

  int N = lengths.num_samples();

  ptrdiff_t total_length = 0;
  for (int i = 0; i < N; i++) {
    total_length += lengths[i][0];
  }

  TestTensorList<float, 1> in_list;
  in_list.reshape(lengths);
  auto in_cpu = in_list.cpu();
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < lengths[i][0]; j++)
      in_cpu[i].data[j] = 1000*(i+1)+j;
  }

  ExtractWindowsArgs args;
  args.window_length = window.empty() ? 55 : window.size();
  args.window_center = window.empty() ? 21 : window.size()/2;
  args.window_step = 2;
  args.padding = padding;

  int out_win_len_actual = out_win_len < 0 ? args.window_length : out_win_len;
  int in_win_start = (out_win_len_actual - args.window_length) / 2;

  KernelContext ctx;

  auto in_gpu = in_list.gpu(0);

  auto req = extract->Setup(ctx, make_span(lengths.shapes), args, concatenate, out_win_len);
  ASSERT_EQ(req.output_shapes.size(), 1u);
  ASSERT_EQ(req.output_shapes[0].num_samples(), concatenate ? 1 : N);

  sa.Reserve(req.scratch_sizes);
  auto scratchpad = sa.GetScratchpad();
  ctx.scratchpad = &scratchpad;

  TestTensorList<float, 2> out;
  mm::uptr<float> gpu_win;
  if (!window.empty()) {
    gpu_win = mm::alloc_raw_unique<float, mm::memory_kind::device>(window.size());
    CUDA_CALL(
      hipMemcpy(gpu_win.get(), window.data(), sizeof(float)*window.size(),
                 hipMemcpyHostToDevice));
  }
  auto window_gpu = make_tensor_gpu<1>(gpu_win.get(), { window.size() });
  out.reshape(req.output_shapes[0].to_static<2>());
  auto out_gpu = out.gpu(0);
  CUDA_CALL(hipMemset(out_gpu.data[0], 0xff, sizeof(float)*out_gpu.shape.num_elements()));
  extract->Run(ctx, out_gpu, in_gpu, window_gpu);
  auto out_cpu = out.cpu();

  ptrdiff_t ofs = 0;

  for (int sample = 0; sample < N; sample++) {
    ptrdiff_t length = lengths[sample][0];
    int nwnd = args.num_windows(length);

    int out_sample = 0;
    if (!concatenate) {
      ofs = 0;
      out_sample = sample;
    }
    ptrdiff_t sample_stride = vertical ? out_cpu.shape[out_sample][1] : 1;
    ptrdiff_t window_stride = vertical ? 1 : out_cpu.shape[out_sample][1];

    for (int w = 0; w < nwnd; w++, ofs += window_stride) {
      int i = 0;
      for (; i < in_win_start; i++) {
        ASSERT_EQ(out_cpu.data[out_sample][ofs + i * sample_stride], 0)
            << "padding @ sample = " << sample << ", window = " << w << ", index = " << i;
      }
      for (int j = 0 ; j < args.window_length; j++, i++) {
        ptrdiff_t idx = w * args.window_step + j - args.window_center;
        if (args.padding == Padding::Reflect) {
          idx = boundary::idx_reflect_101(idx, length);
        }
        float ref = idx >= 0 && idx < length ? in_cpu.data[sample][idx] : 0.0f;
        if (!window.empty())
          ref *= window[j];
        ASSERT_EQ(out_cpu.data[out_sample][ofs + i * sample_stride], ref)
            << "@ sample = " << sample << ", window = " << w << ", index = " << i;
      }
      for (; i < out_win_len_actual; i++) {
        ASSERT_EQ(out_cpu.data[out_sample][ofs + i * sample_stride], 0)
            << "padding @ sample = " << sample << ", window = " << w << ", index = " << i;
      }
    }
  }
}

void TestBatchedExtract(
    const TensorListShape<1> &lengths,
    bool concatenate,
    Padding padding,
    bool vertical,
    span<const float> window,
    int out_win_len = -1) {
  std::unique_ptr<ExtractWindowsImplGPU<float, float>> extract;
  if (vertical)
    extract = std::make_unique<ExtractVerticalWindowsImplGPU<float, float>>();
  else
    extract = std::make_unique<ExtractHorizontalWindowsImplGPU<float, float>>();

  TestBatchedExtract(extract.get(), lengths, concatenate, padding, window, out_win_len);
}

void TestBatchedExtract(
    bool concatenate,
    Padding padding,
    bool vertical,
    span<const float> window,
    int out_win_len = -1) {
  std::unique_ptr<ExtractWindowsImplGPU<float, float>> extract;
  if (vertical)
    extract = std::make_unique<ExtractVerticalWindowsImplGPU<float, float>>();
  else
    extract = std::make_unique<ExtractHorizontalWindowsImplGPU<float, float>>();

  TensorListShape<1> lengths = {{ 5, 305, 157 }};
  TestBatchedExtract(extract.get(), lengths, concatenate, padding, window, out_win_len);

  if (vertical)
    extract = std::make_unique<ExtractVerticalWindowsImplGPU<float, float>>();
  else
    extract = std::make_unique<ExtractHorizontalWindowsImplGPU<float, float>>();
  lengths = {{ 137, 203, 150, 12 }};
  TestBatchedExtract(extract.get(), lengths, concatenate, padding, window, out_win_len);
}

TEST(ExtractVerticalWindowsGPU, BatchedConcat) {
  TestBatchedExtract(true, Padding::Reflect, true, {});
}

TEST(ExtractVerticalWindowsGPU, BatchedSeparate) {
  TestBatchedExtract(false, Padding::Zero, true, {});
}

TEST(ExtractVerticalWindowsGPU, BatchedConcatWindowFunc) {
  vector<float> window(60);
  HannWindow(make_span(window));
  TestBatchedExtract(true, Padding::Zero, true, make_cspan(window));
}

TEST(ExtractVerticalWindowsGPU, BatchedSeparateWindowFunc) {
  vector<float> window(60);
  HammingWindow(make_span(window));
  TestBatchedExtract(false, Padding::Reflect, true, make_cspan(window));
}

TEST(ExtractVerticalWindowsGPU, BatchedSeparateWindowFuncPad) {
  vector<float> window(60);
  HammingWindow(make_span(window));
  TestBatchedExtract(true, Padding::Reflect, true, make_cspan(window), 72);
}

TEST(ExtractHorizontalWindowsGPU, BatchedConcat) {
  TestBatchedExtract(true, Padding::Reflect, false, {});
}

TEST(ExtractHorizontalWindowsGPU, BatchedSeparate) {
  TestBatchedExtract(false, Padding::Zero, false, {});
}

TEST(ExtractHorizontalWindowsGPU, BatchedConcatWindowFunc) {
  vector<float> window(60);
  HannWindow(make_span(window));
  TestBatchedExtract(true, Padding::Zero, false, make_cspan(window));
}

TEST(ExtractHorizontalWindowsGPU, BatchedSeparateWindowFunc) {
  vector<float> window(60);
  HammingWindow(make_span(window));
  TestBatchedExtract(false, Padding::Reflect, false, make_cspan(window));
}

TEST(ExtractHorizontalWindowsGPU, BatchedSeparateWindowFuncPad) {
  vector<float> window(60);
  HammingWindow(make_span(window));
  TestBatchedExtract(false, Padding::Reflect, false, make_cspan(window), 72);
}

TEST(ExtractHorizontalWindowsGPU, BatchedConcatWindowFuncPad) {
  vector<float> window(60);
  HammingWindow(make_span(window));
  TestBatchedExtract(false, Padding::Reflect, true, make_cspan(window), 72);
}

TEST(ExtractHorizontalWindowsGPU, SizeSweep) {
  int max_size = 2048;
  std::vector<TensorShape<1>> lengths;
  int step = 1;
  for (int s = 1; s <= max_size; s+=step) {
    if ((s&255) == 0) {
      if (step > 1)  // add 2^n-1
        lengths.push_back({s-1});
      step += step;
    }
    lengths.push_back({s});
  }
  TensorListShape<1> shape(lengths);
  vector<float> window(60);
  HammingWindow(make_span(window));
  TestBatchedExtract(shape, false, Padding::Reflect, false, make_cspan(window));
}

}  // namespace signal
}  // namespace kernels
}  // namespace dali
