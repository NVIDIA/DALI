// Copyright (c) 2022, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gtest/gtest.h>
#include <vector>
#include <numeric>
#include "dali/kernels/signal/resampling_gpu.h"
#include "dali/kernels/signal/resampling_test.h"

namespace dali {
namespace kernels {
namespace signal {
namespace resampling {
namespace test {

class ResamplingGPUTest : public ResamplingTest {
 public:
  void RunResampling(span<const float> in_rates, span<const float> out_rates) override {
    ResamplerGPU<float> R;
    R.Initialize(16);

    KernelContext ctx;
    ctx.gpu.stream = 0;

    auto req = R.Setup(ctx, ttl_in_.gpu(), in_rates, out_rates);
    auto outref_sh = ttl_outref_.cpu().shape;
    auto in_batch_sh = ttl_in_.cpu().shape;
    for (int s = 0; s < outref_sh.size(); s++) {
      auto sh = req.output_shapes[0].tensor_shape_span(s);
      auto expected_sh = outref_sh.tensor_shape_span(s);
      ASSERT_EQ(sh, expected_sh);
    }

    R.Run(ctx, ttl_out_.gpu(), ttl_in_.gpu(), in_rates, out_rates);

    CUDA_CALL(hipStreamSynchronize(ctx.gpu.stream));
  }
};

TEST_F(ResamplingGPUTest, SingleChannel) {
  this->RunTest(8, 1);
}

TEST_F(ResamplingGPUTest, TwoChannel) {
  this->RunTest(3, 2);
}

TEST_F(ResamplingGPUTest, EightChannel) {
  this->RunTest(3, 8);
}

}  // namespace test
}  // namespace resampling
}  // namespace signal
}  // namespace kernels
}  // namespace dali
