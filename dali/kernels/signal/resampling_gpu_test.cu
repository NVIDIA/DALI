// Copyright (c) 2022, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gtest/gtest.h>
#include <vector>
#include <numeric>
#include "dali/kernels/signal/resampling_gpu.h"
#include "dali/kernels/signal/resampling_test.h"
#include "dali/core/cuda_event.h"

namespace dali {
namespace kernels {
namespace signal {
namespace resampling {
namespace test {

class ResamplingGPUTest : public ResamplingTest {
 public:
  void RunResampling(span<const Args> args) override {
    ResamplerGPU<float> R;
    R.Initialize(16);

    KernelContext ctx;
    ctx.gpu.stream = 0;
    DynamicScratchpad dyn_scratchpad({}, AccessOrder(ctx.gpu.stream));
    ctx.scratchpad = &dyn_scratchpad;

    auto req = R.Setup(ctx, ttl_in_.gpu(), args);
    auto outref_sh = ttl_outref_.cpu().shape;
    auto in_batch_sh = ttl_in_.cpu().shape;
    for (int s = 0; s < outref_sh.size(); s++) {
      auto sh = req.output_shapes[0].tensor_shape_span(s);
      auto expected_sh = outref_sh.tensor_shape_span(s);
      ASSERT_EQ(sh, expected_sh);
    }

    R.Run(ctx, ttl_out_.gpu(), ttl_in_.gpu(), args);

    CUDA_CALL(hipStreamSynchronize(ctx.gpu.stream));
  }

  void RunPerfTest(int batch_size, int nchannels, int n_iters = 1000) {
    std::vector<Args> args_v(batch_size, {22050.0f, 16000.0f});
    auto args = make_cspan(args_v);
    int nsec = 30;

    this->PrepareData(batch_size, nchannels, args, nsec);

    ResamplerGPU<float> R;
    R.Initialize(16);

    KernelContext ctx;
    ctx.gpu.stream = 0;
    auto req = R.Setup(ctx, ttl_in_.gpu(), args);
    ASSERT_EQ(ttl_out_.cpu().shape, req.output_shapes[0]);

    CUDAEvent start = CUDAEvent::CreateWithFlags(0);
    CUDAEvent end = CUDAEvent::CreateWithFlags(0);
    double total_time_ms = 0;
    int64_t in_elems = ttl_in_.cpu().shape.num_elements();
    int64_t out_elems = ttl_out_.cpu().shape.num_elements();
    int64_t out_bytes = out_elems * sizeof(float);
    std::cout << "Resampling GPU Perf test.\n"
              << "Input contains " << in_elems << " floats.\n"
              << "Output contains " << out_elems << " floats.\n";

    for (int i = 0; i < n_iters; ++i) {
      CUDA_CALL(hipDeviceSynchronize());

      DynamicScratchpad dyn_scratchpad({}, AccessOrder(ctx.gpu.stream));
      ctx.scratchpad = &dyn_scratchpad;

      CUDA_CALL(hipEventRecord(start));
      R.Run(ctx, ttl_out_.gpu(), ttl_in_.gpu(), args);
      CUDA_CALL(hipEventRecord(end));
      CUDA_CALL(hipDeviceSynchronize());
      float time_ms;
      CUDA_CALL(hipEventElapsedTime(&time_ms, start, end));
      total_time_ms += time_ms;
    }
    std::cout << "Processed " << n_iters * out_bytes / (total_time_ms * 1e6) << " GBs/sec"
              << std::endl;
  }
};

TEST_F(ResamplingGPUTest, SingleChannel) {
  this->RunTest(8, 1);
}

TEST_F(ResamplingGPUTest, TwoChannel) {
  this->RunTest(3, 2);
}

TEST_F(ResamplingGPUTest, EightChannel) {
  this->RunTest(3, 8);
}

TEST_F(ResamplingGPUTest, HundredChannel) {
  this->RunTest(3, 100);
}

TEST_F(ResamplingGPUTest, OutBeginEnd) {
  this->RunTest(3, 1, true);
}

TEST_F(ResamplingGPUTest, EightChannelOutBeginEnd) {
  this->RunTest(3, 8, true);
}

TEST_F(ResamplingGPUTest, DISABLED_PerfTest) {
  this->RunPerfTest(64, 1, 1000);
}

}  // namespace test
}  // namespace resampling
}  // namespace signal
}  // namespace kernels
}  // namespace dali
