#include "hip/hip_runtime.h"
// Copyright (c) 2020, 2022, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gtest/gtest.h>
#include <random>
#include "dali/kernels/common/copy.h"
#include "dali/test/test_tensors.h"
#include "dali/test/tensor_test_utils.h"
#include "dali/kernels/dynamic_scratchpad.h"
#include "dali/kernels/signal/fft/fft_postprocess.cuh"

namespace dali {


template <>
bool EqualEps::operator()<float2, float2>(const float2 &a, const float2 &b) const {
  return std::abs(b.x - a.x) <= eps && std::abs(b.y - a.y) <= eps;
}

template <typename StorageBackend, int ndim, typename RandomGenerator>
void UniformRandomFill(const TensorListView<StorageBackend, float2, ndim> &tlv,
                       RandomGenerator &rng, float lo, float hi) {
  static_assert(is_cpu_accessible<StorageBackend>::value,
                "Function available only for CPU-accessible TensorListView backend");
  auto dist = uniform_distribution(lo, hi);
  auto gen = [&]() {
    return float2{ dist(rng), dist(rng) };
  };
  Fill(tlv, gen);
}

namespace kernels {
namespace signal {
namespace fft_postprocess {

TEST(FFTPostprocess, Norm2) {
  norm2 f;
  EXPECT_EQ(f(float2{0, 0}), 0);
  EXPECT_NEAR(f(float2{1, 1}), sqrt(2), 1e-5f);
  EXPECT_NEAR(f(float2{-1, 1}), sqrt(2), 1e-5f);
  EXPECT_NEAR(f(float2{-3, -4}), 5, 1e-5f);
}

TEST(FFTPostprocess, Norm2Square) {
  norm2square f;
  EXPECT_EQ(f(float2{0, 0}), 0.0f);
  EXPECT_EQ(f(float2{1, 1}), 2.0f);
  EXPECT_EQ(f(float2{-1, 1}), 2.0f);
  EXPECT_EQ(f(float2{-3, -4}), 25.0f);
}

TEST(FFTPostprocess, Power_dB) {
  power_dB f;
  EXPECT_NEAR(f(float2{0, 0}), -80, 1e-5f);
  EXPECT_EQ(f(float2{1, 0}), 0);
  EXPECT_EQ(f(float2{0, -1}), 0);

  EXPECT_NEAR(f(float2{1, 1}), 3.01029995664, 1e-5f);
  EXPECT_NEAR(f(float2{2, 0}), 6.02059991328, 1e-5f);
  EXPECT_NEAR(f(float2{2, 2}), 9.03089986992, 1e-5f);
  EXPECT_NEAR(f(float2{1e-3, 0}), -60, 1e-5f);

  f = power_dB(-40);
  EXPECT_NEAR(f(float2{0, 0}), -40, 1e-5f);
  EXPECT_NEAR(f(float2{1e-1, 0}), -20, 1e-5f);
  EXPECT_NEAR(f(float2{1e-2, 0}), -40, 1e-5f);
}

template <typename Out, typename In, typename Convert = identity>
struct FFTPostprocessArgs {};

template <typename TestArgs>
class FFTPostprocessTest;

template <typename Out, typename In, typename Convert>
class FFTPostprocessTest<FFTPostprocessArgs<Out, In, Convert>> : public ::testing::Test {
 public:
  void ToFreqMajor() {
    std::mt19937_64 rng;
    TensorListShape<2> in_shape, out_shape;
    std::uniform_int_distribution<int> dist(1, 500);
    int N = 10;
    int fft = 200;  // deliberately not a multiple of 32
    in_shape.resize(N);
    out_shape.resize(N);
    for (int i = 0; i < N; i++) {
      int len = dist(rng);
      in_shape.set_tensor_shape(i, { len, fft });
      out_shape.set_tensor_shape(i, { fft, len });
    }
    TestTensorList<In, 2> in;
    TestTensorList<Out, 2> out, ref;
    in.reshape(in_shape);
    auto cpu_in = in.cpu();
    UniformRandomFill(cpu_in, rng, -1, 1);

    ToFreqMajorSpectrum<Out, In, Convert> tr;
    KernelContext ctx;
    ctx.gpu.stream = 0;
    KernelRequirements req = tr.Setup(ctx, in_shape);
    ASSERT_EQ(req.output_shapes.size(), 1u);
    ASSERT_EQ(req.output_shapes[0], out_shape);
    DynamicScratchpad scratchpad;
    ctx.scratchpad = &scratchpad;
    out.reshape(out_shape);
    tr.Run(ctx, out.gpu(), in.gpu());
    CUDA_CALL(hipGetLastError());

    auto cpu_out = out.cpu();
    ref.reshape(out_shape);
    auto cpu_ref = ref.cpu();

    Convert convert;
    for (int i = 0; i < N; i++) {
      TensorView<StorageCPU, In, 2> in_tv = cpu_in[i];
      TensorView<StorageCPU, Out, 2> ref_tv = cpu_ref[i];
      for (int y = 0; y < in_tv.shape[0]; y++)
        for (int x = 0; x < in_tv.shape[1]; x++)
          *ref_tv(x, y) = convert(*in_tv(y, x));
    }

    double eps = std::is_same<Convert, identity>::value ? 0 : 1e-5;

    Check(cpu_out, cpu_ref, EqualEps(eps));
  }

  void ConvertTimeMajorInPlace() {
    std::mt19937_64 rng;
    TensorListShape<2> in_shape, out_shape;
    std::uniform_int_distribution<int> dist(1, 500);
    int N = 10;
    int fft = 200;  // deliberately not a multiple of 32
    in_shape.resize(N);
    out_shape.resize(N);
    for (int i = 0; i < N; i++) {
      int len = dist(rng);
      int ratio = sizeof(In) / sizeof(Out);
      in_shape.set_tensor_shape(i, { len, fft });
      out_shape.set_tensor_shape(i, { len, fft * ratio });
    }
    TestTensorList<In, 2> in;
    TestTensorList<Out, 2> out, ref;
    TensorListView<StorageGPU, Out, 2> out_gpu;
    in.reshape(in_shape);
    auto cpu_in = in.cpu();
    UniformRandomFill(cpu_in, rng, -1, 1);

    ref.reshape(out_shape);
    auto cpu_ref = ref.cpu();

    Convert convert;
    for (int i = 0; i < N; i++) {
      TensorView<StorageCPU, In, 2> in_tv = cpu_in[i];
      TensorView<StorageCPU, Out, 2> ref_tv = cpu_ref[i];
      for (int i = 0; i < in_tv.shape[0]; i++)
        for (int j = 0; j < in_tv.shape[1]; j++)
          *ref_tv(i, j) = convert(*in_tv(i, j));
    }

    out_gpu = make_tensor_list_gpu(reinterpret_cast<Out*>(in.gpu().data[0]), out_shape);

    ConvertTimeMajorSpectrum<Out, In, Convert> tr;
    KernelContext ctx;
    ctx.gpu.stream = 0;
    tr.Setup(ctx, in_shape);
    tr.Run(ctx, out_gpu, in.gpu());
    CUDA_CALL(hipGetLastError());
    out.reshape(out_shape);
    auto cpu_out = out.cpu();

    for (int i = 0; i < N; i++)
      copy(cpu_out[i], out_gpu[i], ctx.gpu.stream);
    CUDA_CALL(hipStreamSynchronize(ctx.gpu.stream));

    Compare(cpu_out, cpu_ref, in_shape);
  }

  void ConvertTimeMajorPadded() {
    std::mt19937_64 rng;
    TensorListShape<2> shape, padded_shape;
    std::uniform_int_distribution<int> dist(1, 500);
    int N = 10;
    int padded_nfft = 256;
    int nfft = 200;  // deliberately not a multiple of 32
    shape.resize(N);
    padded_shape.resize(N);

    for (int i = 0; i < N; i++) {
      int len = dist(rng);
      shape.set_tensor_shape(i, { len, nfft });
      padded_shape.set_tensor_shape(i, { len, padded_nfft });
    }
    TestTensorList<In, 2> in;
    TestTensorList<Out, 2> out, ref;
    in.reshape(padded_shape);
    out.reshape(shape);
    auto cpu_in = in.cpu();
    UniformRandomFill(cpu_in, rng, -1, 1);

    ref.reshape(shape);
    auto cpu_ref = ref.cpu();

    Convert convert;
    for (int s = 0; s < N; s++) {
      TensorView<StorageCPU, In, 2> in_tv = cpu_in[s];
      TensorView<StorageCPU, Out, 2> ref_tv = cpu_ref[s];
      auto sh = shape.tensor_shape_span(s);
      for (int i = 0; i < sh[0]; i++)
        for (int j = 0; j < sh[1]; j++)
          *ref_tv(i, j) = convert(*in_tv(i, j));
    }

    ConvertTimeMajorSpectrum<Out, In, Convert> tr;
    KernelContext ctx;
    ctx.gpu.stream = 0;
    tr.Setup(ctx, padded_shape);
    tr.Run(ctx, out.gpu(), in.gpu());
    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipStreamSynchronize(ctx.gpu.stream));

    Compare(out.cpu(), cpu_ref, shape);
  }

  /**
   * @brief Compares tensor list views excluding the padding area
   */
  void Compare(const TensorListView<StorageCPU, Out, 2> &cpu_out,
               const TensorListView<StorageCPU, Out, 2> &cpu_ref,
               const TensorListShape<>& tlv_sh) {
    int nsamples = cpu_out.size();
    // data requires padding - clear it so we don't make the comparison fail
    for (int s = 0; s < nsamples; s++) {
      TensorView<StorageCPU, Out, 2> ref_tv = cpu_ref[s];
      TensorView<StorageCPU, Out, 2> out_tv = cpu_out[s];
      auto sh = tlv_sh[s];
      for (int i = 0; i < ref_tv.shape[0]; i++) {
        for (int j = 0; j < ref_tv.shape[1]; j++) {
          if (i >= sh[0] || j >= sh[1]) {
            *ref_tv(i, j) = {};
            *out_tv(i, j) = {};
          }
        }
      }
    }
    double eps = std::is_same<Convert, identity>::value ? 0 : 1e-5;
    Check(cpu_out, cpu_ref, EqualEps(eps));
  }
};

using FFTPosprocessTestTypes = ::testing::Types<
  FFTPostprocessArgs<float, float,   identity>,
  FFTPostprocessArgs<float2, float2, identity>,
  FFTPostprocessArgs<float, float2,  norm2>,
  FFTPostprocessArgs<float, float2,  norm2square>,
  FFTPostprocessArgs<float, float2,  power_dB>
>;

TYPED_TEST_SUITE(FFTPostprocessTest, FFTPosprocessTestTypes);


TYPED_TEST(FFTPostprocessTest, ToFreqMajor) {
  this->ToFreqMajor();
}

TYPED_TEST(FFTPostprocessTest, ConvertTimeMajorInPlace) {
  this->ConvertTimeMajorInPlace();
}

TYPED_TEST(FFTPostprocessTest, ConvertTimeMajorPadded) {
  this->ConvertTimeMajorPadded();
}


}  // namespace fft_postprocess
}  // namespace signal
}  // namespace kernels
}  // namespace dali
