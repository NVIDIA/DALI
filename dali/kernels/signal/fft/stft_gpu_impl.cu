#include "hip/hip_runtime.h"
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "dali/kernels/signal/fft/stft_gpu_impl.cuh"

namespace dali {
namespace kernels {
namespace signal {
namespace fft {

KernelRequirements StftGpuImpl::Setup(span<const int64_t> lengths, const StftArgs &args) {
  if (args != args_) {
    plans_.clear();
    args_ = args;
  }
  int64_t windows = 0;
  TensorListShape<> shape;
  shape.resize(lengths.size(), 2);
  int N = lengths.size();
  for (int i = 0; i < N; i++) {
    int64_t l = lengths[i];
    int64_t n = args_.num_windows(l);
    windows += n;
    TensorShape<2> ts = { n, (args.window_length + 2) / 2 };
    shape.set_tensor_shape(i, ts);
  }
  KernelRequirements req;
  ScratchpadEstimator se;

  CreatePlans(windows);
  ReserveTempStorage(se, windows, args.window_length);

  req.output_shapes = { shape };
  return req;
}


void StftGpuImpl::CreatePlans(int64_t nwindows) {
  int64_t max_windows = kMaxSize;

  while (max_windows * transform_size() > kMaxSize)
    max_windows >>= 1;

  while (max_windows > nwindows)
    max_windows >>= 1;

  if (max_windows == 0)
    max_windows = 1;

  max_windows_ = max_windows;
  min_windows_ = std::min(max_windows_, next_pow2(kMinSize / transform_size()));

  int n[1] = { transform_size() };
  for (int w = max_windows_; w >= min_windows_; w >>= 1) {
    auto &plan = plans_[w];
    if (!plan.handle) {
      hipfftHandle handle;
      CUDA_CALL(hipfftCreate(&handle));
      plan.handle.reset(handle);
      CUDA_CALL(hipfftSetAutoAllocation(handle, 0));
      plan.work_size = 0;
      CUDA_CALL(hipfftMakePlanMany(handle, 1, n, 0, 0, 0, 0, 0, 0, HIPFFT_R2C, w, &plan.work_size));
      total_work_size_ += plan.work_size;
    }
  }

  CreateStreams(plans_.size());
}

void StftGpuImpl::CreateStreams(int new_num_streams) {
  int num_streams = streams_.size();

  if (num_streams < new_num_streams) {
    streams_.resize(new_num_streams);
    for (int i = num_streams; i < new_num_streams; i++)
      streams_[i] = CUDAStream::Create(true);
  }
}

void StftGpuImpl::ReserveTempStorage(ScratchpadEstimator &se, int64_t nwindows, int window_length) {
  assert(is_pow2(min_windows_));
  int64_t wnd = align_up(nwindows, min_windows_);
  se.add<float>(AllocType::GPU, wnd * window_length, 8);
  se.add<float>(AllocType::GPU, total_work_size_);
}


void StftGpuImpl::RunR2C(KernelContext &ctx,
                         const OutListGPU<complexf, 2> &out,
                         const InListGPU<float, 1> &in) {
  int N = in.num_samples();
  assert(out.num_samples() == N);
  int nout = (args_.window_length + 2) / 2;
  for (int i = 0; i < N; i++) {
    auto length = in.shape[i][0];
    assert(out.shape[i] == (TensorShape<2>{ args_.num_windows(length), nout }));
  }


}


}  // namespace fft
}  // namespace signal
}  // namespace kernels
}  // namespace dali
