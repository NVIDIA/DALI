// Copyright (c) 2020-2022, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "dali/operators/audio/mfcc/mfcc.h"
#include <vector>
#include "dali/kernels/signal/dct/dct_gpu.h"
#include "dali/core/static_switch.h"

namespace dali {

namespace detail {
template <>
DLL_PUBLIC  void LifterCoeffs<GPUBackend>::Calculate(int64_t target_length, float lifter,
                                                     hipStream_t stream)  {
  // If different lifter argument, clear previous coefficients
  if (lifter_ != lifter) {
    coeffs_.clear();
    lifter_ = lifter;
  }

  // 0 means no liftering
  if (lifter_ == 0.0f)
    return;

  // Calculate remaining coefficients (if necessary)
  if (static_cast<int64_t>(coeffs_.size()) < target_length) {
    int start_idx = coeffs_.size();
    int added_length = target_length - start_idx;
    coeffs_.resize(target_length, stream);
    std::vector<float> new_coeffs(added_length);
    CalculateCoeffs(new_coeffs.data(), start_idx, added_length);
    CUDA_CALL(
      hipMemcpyAsync(&coeffs_.data()[start_idx], new_coeffs.data(), added_length * sizeof(float),
                      hipMemcpyHostToDevice, stream));
  }
}

template <typename T>
std::vector<OutputDesc> SetupKernel(kernels::KernelManager &kmgr, kernels::KernelContext &ctx,
                                    const TensorList<GPUBackend> &input,
                                    span<const MFCC<GPUBackend>::DctArgs> args, int axis) {
  using Kernel = kernels::signal::dct::Dct1DGpu<T>;
  kmgr.Resize<Kernel>(1);
  auto in_view = view<const T>(input);
  auto &req = kmgr.Setup<Kernel>(0, ctx, in_view, args, axis);
  return {{req.output_shapes[0], input.type()}};
}

}  // namespace detail

template<>
bool MFCC<GPUBackend>::SetupImpl(std::vector<OutputDesc> &output_desc,
                                 const Workspace &ws) {
  GetArguments(ws);
  ctx_.gpu.stream = ws.stream();
  auto &input = ws.Input<GPUBackend>(0);

  auto in_shape = input.shape();
  int ndim = in_shape.sample_dim();
  DALI_ENFORCE(axis_ >= 0 && axis_ < ndim,
               make_string("Axis ", axis_, " is out of bounds [0,", ndim, ")"));

  TYPE_SWITCH(input.type(), type2id, T, MFCC_SUPPORTED_TYPES, (
    output_desc = detail::SetupKernel<T>(kmgr_, ctx_, input, make_cspan(args_), axis_);
  ), DALI_FAIL(make_string("Unsupported data type: ", input.type())));  // NOLINT
  int64_t max_ndct = 0;
  for (int i = 0; i < output_desc[0].shape.num_samples(); ++i) {
    int64_t ndct = output_desc[0].shape[i][axis_];
    if (ndct > max_ndct)
      max_ndct = ndct;
  }
  lifter_coeffs_.Calculate(max_ndct, lifter_, ws.stream());
  return true;
}

template<>
void MFCC<GPUBackend>::RunImpl(Workspace &ws) {
  auto &input = ws.Input<GPUBackend>(0);
  TYPE_SWITCH(input.type(), type2id, T, MFCC_SUPPORTED_TYPES, (
    using Kernel = kernels::signal::dct::Dct1DGpu<T>;
    auto in_view = view<const T>(input);
    auto out_view = view<T>(ws.Output<GPUBackend>(0));
    auto lifter_view = make_tensor_gpu<1>(lifter_coeffs_.data(),
                                          {static_cast<int64_t>(lifter_coeffs_.size())});
    kmgr_.Run<Kernel>(0, ctx_, out_view, in_view, lifter_view);
  ), DALI_FAIL(make_string("Unsupported data type: ", input.type())));  // NOLINT
}

DALI_REGISTER_OPERATOR(MFCC, MFCC<GPUBackend>, GPU);

}  // namespace dali
