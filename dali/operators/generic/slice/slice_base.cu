// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <vector>
#include "dali/image/transform.h"
#include "dali/kernels/slice/slice_gpu.cuh"
#include "dali/core/static_switch.h"
#include "dali/operators/generic/slice/slice_base.h"
#include "dali/pipeline/data/views.h"

namespace dali {
namespace detail {

template <typename OutputType, typename InputType>
void RunHelper(TensorList<GPUBackend>& output,
               const TensorList<GPUBackend>& input,
               const std::vector<std::vector<int64_t>>& slice_anchors,
               const std::vector<std::vector<int64_t>>& slice_shapes,
               hipStream_t stream,
               kernels::ScratchpadAllocator &scratch_alloc) {
  std::size_t number_of_dims = input.tensor_shape(0).size();
  VALUE_SWITCH(number_of_dims, NumDims, (3, 4), (
    kernels::SliceGPU<OutputType, InputType, NumDims> kernel;

    kernels::KernelContext ctx;
    ctx.gpu.stream = stream;
    auto in_view = view<const InputType, NumDims>(input);

    std::vector<kernels::SliceArgs<NumDims>> slice_args;
    slice_args.reserve(slice_anchors.size());
    for (std::size_t i = 0; i < slice_anchors.size(); i++) {
      std::array<int64_t, NumDims> anchor, shape;
      const auto& slice_anchor = slice_anchors[i];
      const auto& slice_shape = slice_shapes[i];
      for (std::size_t d = 0; d < NumDims; d++) {
        anchor[d] = slice_anchor[d];
        shape[d] = slice_shape[d];
      }
      slice_args.push_back({anchor, shape});
    }

    kernels::KernelRequirements req = kernel.Setup(ctx, in_view, slice_args);

    output.set_type(TypeInfo::Create<OutputType>());
    output.Resize(req.output_shapes[0]);

    scratch_alloc.Reserve(req.scratch_sizes);
    auto scratchpad = scratch_alloc.GetScratchpad();
    ctx.scratchpad = &scratchpad;

    auto out_view = view<OutputType, NumDims>(output);
    kernel.Run(ctx, out_view, in_view, slice_args);
  ),  // NOLINT
  (
    DALI_FAIL("Not supported number of dimensions: " + std::to_string(number_of_dims));
  ));  // NOLINT
}

}  // namespace detail


template <>
void SliceBase<GPUBackend>::RunImpl(DeviceWorkspace &ws) {
  this->DataDependentSetup(ws);
  const auto &input = ws.Input<GPUBackend>(0);
  auto &output = ws.Output<GPUBackend>(0);

  TYPE_SWITCH(input_type_, type2id, InputType, SLICE_TYPES, (
    if (input_type_ == output_type_) {
      detail::RunHelper<InputType, InputType>(
        output, input, slice_anchors_, slice_shapes_, ws.stream(), scratch_alloc_);
    } else {
      TYPE_SWITCH(output_type_, type2id, OutputType, (float, float16, uint8_t), (
        detail::RunHelper<OutputType, InputType>(
          output, input, slice_anchors_, slice_shapes_, ws.stream(), scratch_alloc_);
      ), DALI_FAIL(make_string("Not supported output type:", output_type_));); // NOLINT
    }
  ), DALI_FAIL(make_string("Not supported input type:", input_type_));); // NOLINT

  output.SetLayout(InputLayout(ws, 0));
}

}  // namespace dali
