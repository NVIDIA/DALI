/******************************************************************************
MIT License

Copyright (c) 2016 Antti-Pekka Hynninen
Copyright (c) 2016 Oak Ridge National Laboratory (UT-Batelle)

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*******************************************************************************/
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.


#include <stdio.h>
#ifdef ENABLE_NVTOOLS
#include <nvToolsExtCuda.h>
#endif

#include "dali/operators/generic/transpose/cutt/CudaUtils.h"

#include "dali/core/error_handling.h"

//----------------------------------------------------------------------------------------

void set_device_array_async_T(void *data, int value, const size_t ndata, hipStream_t stream, const size_t sizeofT) {
  CUDA_CALL(hipMemsetAsync(data, value, sizeofT*ndata, stream));
}

void set_device_array_T(void *data, int value, const size_t ndata, const size_t sizeofT) {
  CUDA_CALL(hipMemset(data, value, sizeofT*ndata));
}

//----------------------------------------------------------------------------------------
//
// Allocate gpu memory
// pp = memory pointer
// len = length of the array
//
void allocate_device_T(void **pp, const size_t len, const size_t sizeofT) {
  CUDA_CALL(hipMalloc(pp, sizeofT*len));
}

//----------------------------------------------------------------------------------------
//
// Deallocate gpu memory
// pp = memory pointer
//
void deallocate_device_T(void **pp) {

  if (*pp != NULL) {
    CUDA_CALL(hipFree((void *)(*pp)));
    *pp = NULL;
  }

}

//----------------------------------------------------------------------------------------
//
// Copies memory Host -> Device
//
void copy_HtoD_async_T(const void *h_array, void *d_array, size_t array_len, hipStream_t stream,
           const size_t sizeofT) {
  CUDA_CALL(hipMemcpyAsync(d_array, h_array, sizeofT*array_len, hipMemcpyHostToDevice, stream));
}

void copy_HtoD_T(const void *h_array, void *d_array, size_t array_len,
     const size_t sizeofT) {
  CUDA_CALL(hipMemcpy(d_array, h_array, sizeofT*array_len, hipMemcpyHostToDevice));
}

//----------------------------------------------------------------------------------------
//
// Copies memory Device -> Host
//
void copy_DtoH_async_T(const void *d_array, void *h_array, const size_t array_len, hipStream_t stream,
           const size_t sizeofT) {
  CUDA_CALL(hipMemcpyAsync(h_array, d_array, sizeofT*array_len, hipMemcpyDeviceToHost, stream));
}

void copy_DtoH_T(const void *d_array, void *h_array, const size_t array_len, const size_t sizeofT) {
  CUDA_CALL(hipMemcpy(h_array, d_array, sizeofT*array_len, hipMemcpyDeviceToHost));
}

//----------------------------------------------------------------------------------------
#ifdef ENABLE_NVTOOLS
void gpuRangeStart(const char *range_name) {
  static int color_id=0;
  nvtxEventAttributes_t att;
  att.version = NVTX_VERSION;
  att.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
  att.colorType = NVTX_COLOR_ARGB;
  if (color_id == 0) {
    att.color = 0xFFFF0000;
  } else if (color_id == 1) {
    att.color = 0xFF00FF00;
  } else if (color_id == 2) {
    att.color = 0xFF0000FF;
  } else if (color_id == 3) {
    att.color = 0xFFFF00FF;
  }
  color_id++;
  if (color_id > 3) color_id = 0;
  att.messageType = NVTX_MESSAGE_TYPE_ASCII;
  att.message.ascii = range_name;
  nvtxRangePushEx(&att);
}

void gpuRangeStop() {
  nvtxRangePop();
}
#endif
