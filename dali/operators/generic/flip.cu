// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "dali/operators/generic/flip.h"
#include <hip/hip_runtime_api.h>
#include <vector>
#include "dali/kernels/imgproc/flip_gpu.cuh"
#include "dali/operators/generic/flip_util.h"

namespace dali {

template <>
Flip<GPUBackend>::Flip(const OpSpec &spec) : Operator<GPUBackend>(spec) {}

void RunKernel(TensorList<GPUBackend> &output, const TensorList<GPUBackend> &input,
               const std::vector<int32> &depthwise, const std::vector<int32> &horizontal,
               const std::vector<int32> &vertical, hipStream_t stream) {
  DALI_TYPE_SWITCH(
      input.type().id(), DType,
      auto in_shape = TransformShapes(input.shape(), input.GetLayout());
      kernels::InListGPU<DType, flip_ndim> in_view(input.data<DType>(), in_shape);
      kernels::KernelContext ctx;
      ctx.gpu.stream = stream;
      kernels::FlipGPU<DType> kernel;
      auto reqs = kernel.Setup(ctx, in_view);
      kernels::OutListGPU<DType, flip_ndim> out_view(output.mutable_data<DType>(),
                                             reqs.output_shapes[0].to_static<flip_ndim>());
      kernel.Run(ctx, out_view, in_view, depthwise, vertical, horizontal);
  )
}

template <>
void Flip<GPUBackend>::RunImpl(Workspace<GPUBackend> &ws) {
  const auto &input = ws.Input<GPUBackend>(0);
  auto &output = ws.Output<GPUBackend>(0);
  output.SetLayout(input.GetLayout());
  output.set_type(input.type());
  output.ResizeLike(input);
  auto horizontal = GetHorizontal(ws);
  auto vertical = GetVertical(ws);
  auto depthwise = GetDepthwise(ws);
  RunKernel(output, input, depthwise, horizontal, vertical, ws.stream());
}

DALI_REGISTER_OPERATOR(Flip, Flip<GPUBackend>, GPU);

}  // namespace dali
