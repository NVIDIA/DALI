#include "hip/hip_runtime.h"
// Copyright (c) 2020-2021, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "dali/operators/generic/constant.h"

#include <hip/hip_runtime.h>
#include <vector>
#include "dali/core/convert.h"
#include "dali/core/static_switch.h"
#include "dali/pipeline/data/views.h"
#include "dali/kernels/common/scatter_gather.h"

namespace dali {

namespace {

template <size_t size, size_t alignment>
struct alignas(alignment) Placeholder {
  char payload[size];  // NOLINT(runtime/arrays)
};

template <typename T>
inline auto opaque(const T &value) {
  Placeholder<sizeof(T), alignof(T)> placeholder;
  memcpy(placeholder.payload, &value, sizeof(T));
  return placeholder;
}

template <size_t size, size_t alignment>
__global__ void Fill(void *data, size_t count, Placeholder<size, alignment> value) {
  auto i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  if (i < count)
    static_cast<Placeholder<size, alignment>*>(data)[i] = value;
}

// TODO(klecki): [Conditional] - replace by sharing repeated sample
template <typename Dst, typename Src>
void FillTensorList(
      TensorList<GPUBackend> &dst, const TensorListShape<> &shape, const std::vector<Src> &src,
      hipStream_t stream) {
  dst.Resize(shape);
  if (shape.num_samples() == 0)
    return;


  int64_t sample_size = shape[0].num_elements();

  if (src.size() == 1) {
    int64_t threads = 1024;
    int64_t blocks = div_ceil(sample_size, threads);
    Dst *data = dst.mutable_tensor<Dst>(0);

    Fill<<<dim3(blocks), dim3(threads), 0, stream>>>(data, sample_size,
                                                     opaque(ConvertSat<Dst>(src[0])));
  } else {
    SmallVector<Dst, 64> tmp;
    assert(static_cast<int>(src.size()) == sample_size);
    tmp.resize(src.size());
    for (size_t i = 0; i < tmp.size(); i++)
      tmp[i] = ConvertSat<Dst>(src[i]);

    int n = tmp.size() * sizeof(Dst);
    CUDA_CALL(
      hipMemcpyAsync(dst.mutable_tensor<Dst>(0), tmp.data(), n, hipMemcpyHostToDevice, stream));
  }

  kernels::ScatterGatherGPU scatter_gather;

  for (int i = 1; i < shape.num_samples(); i++) {
    scatter_gather.AddCopy(dst.mutable_tensor<Dst>(i), dst.mutable_tensor<Dst>(0),
                            sample_size * sizeof(Dst));
  }
  scatter_gather.Run(stream);
}

}  // namespace

template <>
void Constant<GPUBackend>::RunImpl(DeviceWorkspace &ws) {
  if (output_.ntensor() == 0) {
    TYPE_SWITCH(output_type_, type2id, type, CONSTANT_OP_SUPPORTED_TYPES,
      (
        if (!fdata_.empty()) {
          FillTensorList<type>(output_, max_output_shape_, fdata_, ws.stream());
        } else {
          FillTensorList<type>(output_, max_output_shape_, idata_, ws.stream());
        }
      ), (DALI_FAIL(make_string("Unsupported type: ", output_type_))));  // NOLINT
  }
  auto &out = ws.OutputRef<GPUBackend>(0);

  out.Reset();
  out.ShareData(&output_);
  out.Resize(output_shape_);
  int N = output_shape_.num_samples();
  for (int i = 0; i < N; i++) {
    assert(out.raw_tensor(i) == output_.raw_tensor(i));
  }
  out.SetLayout(layout_);
}

DALI_REGISTER_OPERATOR(Constant, Constant<GPUBackend>, GPU);

}  // namespace dali
