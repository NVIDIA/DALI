#include "hip/hip_runtime.h"
// Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <vector>
#include "dali/operators/generic/one_hot.h"
#include "dali/operators/generic/one_hot.cuh"

namespace dali {

class OneHotGPU : public OneHot<GPUBackend> {
 public:
  explicit OneHotGPU(const OpSpec &spec) : OneHot<GPUBackend>(spec) {
    scratch_mem_.set_type(TypeTable::GetTypeInfo(DALI_UINT8));
  }

  ~OneHotGPU() override = default;

  USE_OPERATOR_MEMBERS();

 protected:
  void RunImpl(workspace_t<GPUBackend> &ws) override;
  bool SetupImpl(std::vector<OutputDesc> &output_desc, const workspace_t<GPUBackend> &ws) override;

  template<typename OutputType, typename InputType>
  void RunImplTyped(workspace_t<GPUBackend> &ws, int placement_axis);

 private:
  std::vector<detail::SampleDesc> sample_descs_;
  Tensor<GPUBackend> scratch_mem_;
  int recent_n_samples_ = 0;
};

bool OneHotGPU::SetupImpl(std::vector<OutputDesc> &output_desc, const workspace_t<GPUBackend> &ws) {
  const auto &input = ws.template InputRef<GPUBackend>(0);
  int num_samples = input.shape().num_samples();
  if (num_samples != recent_n_samples_) {
    recent_n_samples_ = num_samples;
    int64_t samples_size = num_samples * sizeof(detail::SampleDesc);
    scratch_mem_.Resize({samples_size});
  }
  sample_descs_.clear();
  sample_descs_.reserve(num_samples);
  return OneHot<GPUBackend>::SetupImpl(output_desc, ws);
}

void OneHotGPU::RunImpl(workspace_t<GPUBackend> &ws) {
  const auto &input = ws.InputRef<GPUBackend>(0);
  auto &output = ws.OutputRef<GPUBackend>(0);
  int output_sample_dim = output.shape().sample_dim();
  int placement_axis = get_placement_axis(output_sample_dim);
  output.SetLayout(GetOutputLayout(ws, placement_axis, output_sample_dim));
  TYPE_SWITCH(input.type().id(), type2id, InputType, ONE_HOT_TYPES, (
    TYPE_SWITCH(output_type_, type2id, OutputType, ONE_HOT_TYPES, (
      RunImplTyped<OutputType, InputType>(ws, placement_axis);
    ), DALI_FAIL(make_string("Unsupported output type: ", output_type_)); );       // NOLINT
  ), DALI_FAIL(make_string("Unsupported input type: ", input.type().id())); );     // NOLINT
}

template <typename OutputType, typename InputType>
void OneHotGPU::RunImplTyped(workspace_t<GPUBackend> &ws, int axis) {
  const auto &input = ws.InputRef<GPUBackend>(0);
  auto &output = ws.OutputRef<GPUBackend>(0);
  int num_samples = input.shape().num_samples();

  uint64_t max_out_vol = 1;
  const auto &shape = output.shape();
  for (int sample_id = 0; sample_id < num_samples; ++sample_id) {
    detail::SampleDesc sample;
    auto output_shape = shape.tensor_shape_span(sample_id);
    auto outer_vol = volume(output_shape.begin(), output_shape.begin() + axis);
    sample.inner_vol = volume(output_shape.begin() + axis + 1, output_shape.end());
    sample.inner_vol_classes = sample.inner_vol * num_classes_;
    sample.output_vol = outer_vol * sample.inner_vol_classes;
    sample.out = output.mutable_tensor<OutputType>(sample_id);
    sample.in = input.tensor<InputType>(sample_id);
    sample_descs_.push_back(sample);
    max_out_vol = std::max(max_out_vol, sample.output_vol);
  }

  auto stream = ws.stream();

  scratch_mem_.Copy(sample_descs_, stream);
  const auto *scratch_mem_gpu = scratch_mem_.data<detail::SampleDesc>();

  const int block = 256;
  auto grid = detail::gridHelper(max_out_vol, num_samples, block);

  detail::PopulateOneHot<OutputType, InputType><<<grid, block, 0, stream>>>(
    on_value_, off_value_, scratch_mem_gpu);
}

DALI_REGISTER_OPERATOR(OneHot, OneHotGPU, GPU);

}  // namespace dali
