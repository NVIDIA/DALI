// Copyright (c) 2019-2022, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "dali/operators/math/expressions/arithmetic.h"
#include <vector>
#include "dali/kernels/type_tag.h"

namespace dali {
namespace expr {

template <>
void ArithmeticGenericOp<GPUBackend>::RunImpl(Workspace &ws) {
  PrepareSamplesPerTask<GPUBackend>(samples_per_task_, exec_order_, ws, constant_storage_, spec_);
  ws.Output<GPUBackend>(0).SetLayout(result_layout_);
  std::tie(tile_cover_, tile_range_) = GetTiledCover(result_shape_, kTileSize, kTaskSize);
  assert(tile_range_.size() == 1 && "Expected to cover whole GPU execution by 1 task");
  auto tiles = make_cspan(tile_cover_);
  for (size_t i = 0; i < exec_order_.size(); i++) {
    // call impl for whole batch
    exec_order_[i].impl->Execute(exec_order_[i].ctx, make_cspan(samples_per_task_[i]), tiles);
  }
}

}  // namespace expr

DALI_REGISTER_OPERATOR(ArithmeticGenericOp, expr::ArithmeticGenericOp<GPUBackend>, GPU);

}  // namespace dali
