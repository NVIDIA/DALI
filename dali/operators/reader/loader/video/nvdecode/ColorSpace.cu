#include "hip/hip_runtime.h"
/*
* Copyright 2017-2021 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

#include "ColorSpace.h"

__constant__ float matYuv2Rgb[3][3];
__constant__ float matRgb2Yuv[3][3];


void inline GetConstants(int iMatrix, float &wr, float &wb, int &black, int &white, int &max) {
    black = 16; white = 235;
    max = 255;

    switch (iMatrix)
    {
    case ColorSpaceStandard_BT709:
    default:
        wr = 0.2126f; wb = 0.0722f;
        break;

    case ColorSpaceStandard_FCC:
        wr = 0.30f; wb = 0.11f;
        break;

    case ColorSpaceStandard_BT470:
    case ColorSpaceStandard_BT601:
        wr = 0.2990f; wb = 0.1140f;
        break;

    case ColorSpaceStandard_SMPTE240M:
        wr = 0.212f; wb = 0.087f;
        break;

    case ColorSpaceStandard_BT2020:
    case ColorSpaceStandard_BT2020C:
        wr = 0.2627f; wb = 0.0593f;
        // 10-bit only
        black = 64 << 6; white = 940 << 6;
        max = (1 << 16) - 1;
        break;
    }
}

void SetMatYuv2Rgb(int iMatrix) {
    float wr, wb;
    int black, white, max;
    GetConstants(iMatrix, wr, wb, black, white, max);
    float mat[3][3] = {
        1.0f, 0.0f, (1.0f - wr) / 0.5f,
        1.0f, -wb * (1.0f - wb) / 0.5f / (1 - wb - wr), -wr * (1 - wr) / 0.5f / (1 - wb - wr),
        1.0f, (1.0f - wb) / 0.5f, 0.0f,
    };
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            mat[i][j] = (float)(1.0 * max / (white - black) * mat[i][j]);
        }
    }
    hipMemcpyToSymbol(HIP_SYMBOL(matYuv2Rgb), mat, sizeof(mat));
}

void SetMatRgb2Yuv(int iMatrix) {
    float wr, wb;
    int black, white, max;
    GetConstants(iMatrix, wr, wb, black, white, max);
    float mat[3][3] = {
        wr, 1.0f - wb - wr, wb,
        -0.5f * wr / (1.0f - wb), -0.5f * (1 - wb - wr) / (1.0f - wb), 0.5f,
        0.5f, -0.5f * (1.0f - wb - wr) / (1.0f - wr), -0.5f * wb / (1.0f - wr),
    };
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            mat[i][j] = (float)(1.0 * (white - black) / max * mat[i][j]);
        }
    }
    hipMemcpyToSymbol(HIP_SYMBOL(matRgb2Yuv), mat, sizeof(mat));
}

template<class T>
__device__ static T Clamp(T x, T lower, T upper) {
    return x < lower ? lower : (x > upper ? upper : x);
}

template<class Rgb, class YuvUnit>
__device__ inline Rgb YuvToRgbForPixel(YuvUnit y, YuvUnit u, YuvUnit v) {
    const int 
        low = 1 << (sizeof(YuvUnit) * 8 - 4),
        mid = 1 << (sizeof(YuvUnit) * 8 - 1);
    float fy = (int)y - low, fu = (int)u - mid, fv = (int)v - mid;
    const float maxf = (1 << sizeof(YuvUnit) * 8) - 1.0f;
    YuvUnit 
        r = (YuvUnit)Clamp(matYuv2Rgb[0][0] * fy + matYuv2Rgb[0][1] * fu + matYuv2Rgb[0][2] * fv, 0.0f, maxf),
        g = (YuvUnit)Clamp(matYuv2Rgb[1][0] * fy + matYuv2Rgb[1][1] * fu + matYuv2Rgb[1][2] * fv, 0.0f, maxf),
        b = (YuvUnit)Clamp(matYuv2Rgb[2][0] * fy + matYuv2Rgb[2][1] * fu + matYuv2Rgb[2][2] * fv, 0.0f, maxf);
    
    Rgb rgb{};
    const int nShift = abs((int)sizeof(YuvUnit) - (int)sizeof(rgb.c.r)) * 8;
    if (sizeof(YuvUnit) >= sizeof(rgb.c.r)) {
        rgb.c.r = r >> nShift;
        rgb.c.g = g >> nShift;
        rgb.c.b = b >> nShift;
    } else {
        rgb.c.r = r << nShift;
        rgb.c.g = g << nShift;
        rgb.c.b = b << nShift;
    }
    return rgb;
}

template<class YuvUnitx2, class Rgb, class RgbIntx2>
__global__ static void YuvToRgbKernel(uint8_t *pYuv, int nYuvPitch, uint8_t *pRgb, int nRgbPitch, int nWidth, int nHeight) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y) * 2;
    if (x + 1 >= nWidth || y + 1 >= nHeight) {
        return;
    }

    uint8_t *pSrc = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;
    uint8_t *pDst = pRgb + x * sizeof(Rgb) + y * nRgbPitch;

    YuvUnitx2 l0 = *(YuvUnitx2 *)pSrc;
    YuvUnitx2 l1 = *(YuvUnitx2 *)(pSrc + nYuvPitch);
    YuvUnitx2 ch = *(YuvUnitx2 *)(pSrc + (nHeight - y / 2) * nYuvPitch);

    *(RgbIntx2 *)pDst = RgbIntx2 {
        YuvToRgbForPixel<Rgb>(l0.x, ch.x, ch.y).d,
        YuvToRgbForPixel<Rgb>(l0.y, ch.x, ch.y).d,
    };
    *(RgbIntx2 *)(pDst + nRgbPitch) = RgbIntx2 {
        YuvToRgbForPixel<Rgb>(l1.x, ch.x, ch.y).d, 
        YuvToRgbForPixel<Rgb>(l1.y, ch.x, ch.y).d,
    };
}

template<class YuvUnitx2, class Rgb, class RgbIntx2>
__global__ static void Yuv444ToRgbKernel(uint8_t *pYuv, int nYuvPitch, uint8_t *pRgb, int nRgbPitch, int nWidth, int nHeight) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y);
    if (x + 1 >= nWidth || y  >= nHeight) {
        return;
    }

    uint8_t *pSrc = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;
    uint8_t *pDst = pRgb + x * sizeof(Rgb) + y * nRgbPitch;

    YuvUnitx2 l0 = *(YuvUnitx2 *)pSrc;
    YuvUnitx2 ch1 = *(YuvUnitx2 *)(pSrc + (nHeight * nYuvPitch));
    YuvUnitx2 ch2 = *(YuvUnitx2 *)(pSrc + (2 * nHeight * nYuvPitch));

    *(RgbIntx2 *)pDst = RgbIntx2{
        YuvToRgbForPixel<Rgb>(l0.x, ch1.x, ch2.x).d,
        YuvToRgbForPixel<Rgb>(l0.y, ch1.y, ch2.y).d,
    };
}

template<class YuvUnitx2, class Rgb, class RgbUnitx2>
__global__ static void YuvToRgbPlanarKernel(uint8_t *pYuv, int nYuvPitch, uint8_t *pRgbp, int nRgbpPitch, int nWidth, int nHeight) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y) * 2;
    if (x + 1 >= nWidth || y + 1 >= nHeight) {
        return;
    }

    uint8_t *pSrc = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;

    YuvUnitx2 l0 = *(YuvUnitx2 *)pSrc;
    YuvUnitx2 l1 = *(YuvUnitx2 *)(pSrc + nYuvPitch);
    YuvUnitx2 ch = *(YuvUnitx2 *)(pSrc + (nHeight - y / 2) * nYuvPitch);

    Rgb rgb0 = YuvToRgbForPixel<Rgb>(l0.x, ch.x, ch.y),
        rgb1 = YuvToRgbForPixel<Rgb>(l0.y, ch.x, ch.y),
        rgb2 = YuvToRgbForPixel<Rgb>(l1.x, ch.x, ch.y),
        rgb3 = YuvToRgbForPixel<Rgb>(l1.y, ch.x, ch.y);

    uint8_t *pDst = pRgbp + x * sizeof(RgbUnitx2) / 2 + y * nRgbpPitch;
    *(RgbUnitx2 *)pDst = RgbUnitx2 {rgb0.v.x, rgb1.v.x};
    *(RgbUnitx2 *)(pDst + nRgbpPitch) = RgbUnitx2 {rgb2.v.x, rgb3.v.x};
    pDst += nRgbpPitch * nHeight;
    *(RgbUnitx2 *)pDst = RgbUnitx2 {rgb0.v.y, rgb1.v.y};
    *(RgbUnitx2 *)(pDst + nRgbpPitch) = RgbUnitx2 {rgb2.v.y, rgb3.v.y};
    pDst += nRgbpPitch * nHeight;
    *(RgbUnitx2 *)pDst = RgbUnitx2 {rgb0.v.z, rgb1.v.z};
    *(RgbUnitx2 *)(pDst + nRgbpPitch) = RgbUnitx2 {rgb2.v.z, rgb3.v.z};
}

template<class YuvUnitx2, class Rgb, class RgbUnitx2>
__global__ static void Yuv444ToRgbPlanarKernel(uint8_t *pYuv, int nYuvPitch, uint8_t *pRgbp, int nRgbpPitch, int nWidth, int nHeight) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y);
    if (x + 1 >= nWidth || y >= nHeight) {
        return;
    }

    uint8_t *pSrc = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;

    YuvUnitx2 l0 = *(YuvUnitx2 *)pSrc;
    YuvUnitx2 ch1 = *(YuvUnitx2 *)(pSrc + (nHeight * nYuvPitch));
    YuvUnitx2 ch2 = *(YuvUnitx2 *)(pSrc + (2 * nHeight * nYuvPitch));

    Rgb rgb0 = YuvToRgbForPixel<Rgb>(l0.x, ch1.x, ch2.x),
        rgb1 = YuvToRgbForPixel<Rgb>(l0.y, ch1.y, ch2.y);


    uint8_t *pDst = pRgbp + x * sizeof(RgbUnitx2) / 2 + y * nRgbpPitch;
    *(RgbUnitx2 *)pDst = RgbUnitx2{ rgb0.v.x, rgb1.v.x };

    pDst += nRgbpPitch * nHeight;
    *(RgbUnitx2 *)pDst = RgbUnitx2{ rgb0.v.y, rgb1.v.y };

    pDst += nRgbpPitch * nHeight;
    *(RgbUnitx2 *)pDst = RgbUnitx2{ rgb0.v.z, rgb1.v.z };
}

template <class COLOR32>
void Nv12ToColor32(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbKernel<uchar2, COLOR32, uint2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpNv12, nNv12Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR64>
void Nv12ToColor64(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbKernel<uchar2, COLOR64, ulonglong2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpNv12, nNv12Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR32>
void YUV444ToColor32(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    Yuv444ToRgbKernel<uchar2, COLOR32, uint2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2), dim3(32, 2) >>>
        (dpYUV444, nPitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR64>
void YUV444ToColor64(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    Yuv444ToRgbKernel<uchar2, COLOR64, ulonglong2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2), dim3(32, 2) >>>
        (dpYUV444, nPitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR32>
void P016ToColor32(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbKernel<ushort2, COLOR32, uint2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpP016, nP016Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR64>
void P016ToColor64(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbKernel<ushort2, COLOR64, ulonglong2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpP016, nP016Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR32>
void YUV444P16ToColor32(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    Yuv444ToRgbKernel<ushort2, COLOR32, uint2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2), dim3(32, 2) >>>
        (dpYUV444, nPitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR64>
void YUV444P16ToColor64(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    Yuv444ToRgbKernel<ushort2, COLOR64, ulonglong2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2), dim3(32, 2) >>>
        (dpYUV444, nPitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR32>
void Nv12ToColorPlanar(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbPlanarKernel<uchar2, COLOR32, uchar2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpNv12, nNv12Pitch, dpBgrp, nBgrpPitch, nWidth, nHeight);
}

template <class COLOR32>
void P016ToColorPlanar(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbPlanarKernel<ushort2, COLOR32, uchar2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpP016, nP016Pitch, dpBgrp, nBgrpPitch, nWidth, nHeight);
}

template <class COLOR32>
void YUV444ToColorPlanar(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    Yuv444ToRgbPlanarKernel<uchar2, COLOR32, uchar2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2), dim3(32, 2) >>>
        (dpYUV444, nPitch, dpBgrp, nBgrpPitch, nWidth, nHeight);
}

template <class COLOR32>
void YUV444P16ToColorPlanar(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    Yuv444ToRgbPlanarKernel<ushort2, COLOR32, uchar2>
        << <dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2), dim3(32, 2) >> >
        (dpYUV444, nPitch, dpBgrp, nBgrpPitch, nWidth, nHeight);
}

// Explicit Instantiation
template void Nv12ToColor32<BGRA32>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void Nv12ToColor32<RGBA32>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void Nv12ToColor64<BGRA64>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void Nv12ToColor64<RGBA64>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444ToColor32<BGRA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444ToColor32<RGBA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444ToColor64<BGRA64>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444ToColor64<RGBA64>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void P016ToColor32<BGRA32>(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void P016ToColor32<RGBA32>(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void P016ToColor64<BGRA64>(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void P016ToColor64<RGBA64>(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444P16ToColor32<BGRA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444P16ToColor32<RGBA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444P16ToColor64<BGRA64>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444P16ToColor64<RGBA64>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void Nv12ToColorPlanar<BGRA32>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void Nv12ToColorPlanar<RGBA32>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void P016ToColorPlanar<BGRA32>(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void P016ToColorPlanar<RGBA32>(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444ToColorPlanar<BGRA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444ToColorPlanar<RGBA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444P16ToColorPlanar<BGRA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444P16ToColorPlanar<RGBA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);

template<class YuvUnit, class RgbUnit>
__device__ inline YuvUnit RgbToY(RgbUnit r, RgbUnit g, RgbUnit b) {
    const YuvUnit low = 1 << (sizeof(YuvUnit) * 8 - 4);
    return matRgb2Yuv[0][0] * r + matRgb2Yuv[0][1] * g + matRgb2Yuv[0][2] * b + low;
}

template<class YuvUnit, class RgbUnit>
__device__ inline YuvUnit RgbToU(RgbUnit r, RgbUnit g, RgbUnit b) {
    const YuvUnit mid = 1 << (sizeof(YuvUnit) * 8 - 1);
    return matRgb2Yuv[1][0] * r + matRgb2Yuv[1][1] * g + matRgb2Yuv[1][2] * b + mid;
}

template<class YuvUnit, class RgbUnit>
__device__ inline YuvUnit RgbToV(RgbUnit r, RgbUnit g, RgbUnit b) {
    const YuvUnit mid = 1 << (sizeof(YuvUnit) * 8 - 1);
    return matRgb2Yuv[2][0] * r + matRgb2Yuv[2][1] * g + matRgb2Yuv[2][2] * b + mid;
}

template<class YuvUnitx2, class Rgb, class RgbIntx2>
__global__ static void RgbToYuvKernel(uint8_t *pRgb, int nRgbPitch, uint8_t *pYuv, int nYuvPitch, int nWidth, int nHeight) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y) * 2;
    if (x + 1 >= nWidth || y + 1 >= nHeight) {
        return;
    }

    uint8_t *pSrc = pRgb + x * sizeof(Rgb) + y * nRgbPitch;
    RgbIntx2 int2a = *(RgbIntx2 *)pSrc;
    RgbIntx2 int2b = *(RgbIntx2 *)(pSrc + nRgbPitch);

    Rgb rgb[4] = {int2a.x, int2a.y, int2b.x, int2b.y};
    decltype(Rgb::c.r)
        r = (rgb[0].c.r + rgb[1].c.r + rgb[2].c.r + rgb[3].c.r) / 4,
        g = (rgb[0].c.g + rgb[1].c.g + rgb[2].c.g + rgb[3].c.g) / 4,
        b = (rgb[0].c.b + rgb[1].c.b + rgb[2].c.b + rgb[3].c.b) / 4;

    uint8_t *pDst = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;
    *(YuvUnitx2 *)pDst = YuvUnitx2 {
        RgbToY<decltype(YuvUnitx2::x)>(rgb[0].c.r, rgb[0].c.g, rgb[0].c.b),
        RgbToY<decltype(YuvUnitx2::x)>(rgb[1].c.r, rgb[1].c.g, rgb[1].c.b),
    };
    *(YuvUnitx2 *)(pDst + nYuvPitch) = YuvUnitx2 {
        RgbToY<decltype(YuvUnitx2::x)>(rgb[2].c.r, rgb[2].c.g, rgb[2].c.b),
        RgbToY<decltype(YuvUnitx2::x)>(rgb[3].c.r, rgb[3].c.g, rgb[3].c.b),
    };
    *(YuvUnitx2 *)(pDst + (nHeight - y / 2) * nYuvPitch) = YuvUnitx2 {
        RgbToU<decltype(YuvUnitx2::x)>(r, g, b), 
        RgbToV<decltype(YuvUnitx2::x)>(r, g, b),
    };
}

void Bgra64ToP016(uint8_t *dpBgra, int nBgraPitch, uint8_t *dpP016, int nP016Pitch, int nWidth, int nHeight, int iMatrix) {
    SetMatRgb2Yuv(iMatrix);
    RgbToYuvKernel<ushort2, BGRA64, ulonglong2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpBgra, nBgraPitch, dpP016, nP016Pitch, nWidth, nHeight);
}
