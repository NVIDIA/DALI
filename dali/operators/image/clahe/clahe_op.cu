// Copyright (c) 2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>

#ifndef CUDA_CHECK
#define CUDA_CHECK(expr)                                                                  \
  do {                                                                                    \
    hipError_t __err = (expr);                                                           \
    if (__err != hipSuccess) {                                                           \
      printf("CUDA error %d at %s:%d: %s\n", static_cast<int>(__err), __FILE__, __LINE__, \
             hipGetErrorString(__err));                                                  \
    }                                                                                     \
  } while (0)
#endif

static inline __host__ __device__ int div_up(int a, int b) {
  return (a + b - 1) / b;
}

// -------------------------------------------------------------------------------------
// Helper functions for RGB ↔ LAB conversion (match OpenCV exactly)
// -------------------------------------------------------------------------------------
__device__ float srgb_to_linear(float c) {
  // OpenCV's exact gamma correction
  return (c > 0.04045f) ? powf((c + 0.055f) / 1.055f, 2.4f) : c / 12.92f;
}

__device__ float linear_to_srgb(float c) {
  // OpenCV's exact inverse gamma correction
  return (c > 0.0031308f) ? 1.055f * powf(c, 1.0f / 2.4f) - 0.055f : 12.92f * c;
}

__device__ float xyz_to_lab_f(float t) {
  // OpenCV uses these exact thresholds and constants
  const float delta = 6.0f / 29.0f;
  const float delta_cube = delta * delta * delta;
  return (t > delta_cube) ? cbrtf(t) : (t / (3.0f * delta * delta) + 4.0f / 29.0f);
}

__device__ float lab_f_to_xyz(float t) {
  // OpenCV's exact inverse transformation
  const float delta = 6.0f / 29.0f;
  return (t > delta) ? (t * t * t) : (3.0f * delta * delta * (t - 4.0f / 29.0f));
}

__device__ void rgb_to_lab(uint8_t r, uint8_t g, uint8_t b, float *L, float *a_out, float *b_out) {
  // Normalize to [0,1]
  float rf = r / 255.0f;
  float gf = g / 255.0f;
  float bf = b / 255.0f;

  // sRGB to linear RGB
  rf = srgb_to_linear(rf);
  gf = srgb_to_linear(gf);
  bf = srgb_to_linear(bf);

  // Linear RGB to XYZ using OpenCV's exact matrix (sRGB D65)
  float x = 0.412453f * rf + 0.357580f * gf + 0.180423f * bf;
  float y = 0.212671f * rf + 0.715160f * gf + 0.072169f * bf;
  float z = 0.019334f * rf + 0.119193f * gf + 0.950227f * bf;

  // Normalize by D65 white point (OpenCV values)
  x = x / 0.950456f;
  y = y / 1.000000f;
  z = z / 1.088754f;

  // XYZ to LAB
  float fx = xyz_to_lab_f(x);
  float fy = xyz_to_lab_f(y);
  float fz = xyz_to_lab_f(z);

  *L = 116.0f * fy - 16.0f;
  *a_out = 500.0f * (fx - fy);
  *b_out = 200.0f * (fy - fz);
}

__device__ void lab_to_rgb(float L, float a, float b, uint8_t *r, uint8_t *g, uint8_t *b_out) {
  // LAB to XYZ
  float fy = (L + 16.0f) / 116.0f;
  float fx = a / 500.0f + fy;
  float fz = fy - b / 200.0f;

  // Convert using OpenCV's D65 white point
  float x = lab_f_to_xyz(fx) * 0.950456f;
  float y = lab_f_to_xyz(fy) * 1.000000f;
  float z = lab_f_to_xyz(fz) * 1.088754f;

  // XYZ to linear RGB using OpenCV's exact inverse matrix
  float rf = 3.240479f * x - 1.537150f * y - 0.498535f * z;
  float gf = -0.969256f * x + 1.875991f * y + 0.041556f * z;
  float bf = 0.055648f * x - 0.204043f * y + 1.057311f * z;

  // Linear RGB to sRGB
  rf = linear_to_srgb(rf);
  gf = linear_to_srgb(gf);
  bf = linear_to_srgb(bf);

  // Clamp and convert to uint8
  *r = (uint8_t)lrintf(fminf(fmaxf(rf * 255.0f, 0.f), 255.f));
  *g = (uint8_t)lrintf(fminf(fmaxf(gf * 255.0f, 0.f), 255.f));
  *b_out = (uint8_t)lrintf(fminf(fmaxf(bf * 255.0f, 0.f), 255.f));
}

// -------------------------------------------------------------------------------------
// Kernel 1: RGB -> Y (uint8). NHWC input (uint8), Y in [0..255] as uint8.
// BT.601 luma: Y = 0.299 R + 0.587 G + 0.114 B
// -------------------------------------------------------------------------------------
__global__ void rgb_to_y_u8_nhwc_kernel(const uint8_t *__restrict__ rgb,
                                        uint8_t *__restrict__ y_out, int H, int W) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int N = H * W;
  if (idx >= N)
    return;

  int c0 = 3 * idx;
  float r = rgb[c0 + 0];
  float g = rgb[c0 + 1];
  float b = rgb[c0 + 2];

  float y = 0.299f * r + 0.587f * g + 0.114f * b;
  int yi = static_cast<int>(lrintf(fminf(fmaxf(y, 0.f), 255.f)));
  y_out[idx] = static_cast<uint8_t>(yi);
}

// Vectorized version for better memory bandwidth (processes 4 pixels at once)
__global__ void rgb_to_y_u8_nhwc_vectorized_kernel(const uint8_t *__restrict__ rgb,
                                                   uint8_t *__restrict__ y_out, int H, int W) {
  int base_idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
  int N = H * W;

  // Process 4 pixels per thread for better memory coalescing
  for (int i = 0; i < 4 && (base_idx + i) < N; ++i) {
    int idx = base_idx + i;
    int c0 = 3 * idx;

    float r = rgb[c0 + 0];
    float g = rgb[c0 + 1];
    float b = rgb[c0 + 2];

    float y = 0.299f * r + 0.587f * g + 0.114f * b;
    int yi = static_cast<int>(lrintf(fminf(fmaxf(y, 0.f), 255.f)));
    y_out[idx] = static_cast<uint8_t>(yi);
  }
}

extern "C" void LaunchRGBToYUint8NHWC(const uint8_t *in_rgb, uint8_t *y_plane, int H, int W,
                                      hipStream_t stream) {
  int N = H * W;

  // Optimized occupancy settings for different image sizes
  if (N >= 4096) {                        // Use vectorized version for larger images
    int threads = 256;                    // Better occupancy on modern GPUs
    int blocks = div_up(N, threads * 4);  // Each thread processes 4 pixels
    rgb_to_y_u8_nhwc_vectorized_kernel<<<blocks, threads, 0, stream>>>(in_rgb, y_plane, H, W);
  } else {
    int threads = 256;
    int blocks = div_up(N, threads);
    rgb_to_y_u8_nhwc_kernel<<<blocks, threads, 0, stream>>>(in_rgb, y_plane, H, W);
  }
}

// -------------------------------------------------------------------------------------
// Fused Kernel: RGB to Y + Histogram per tile (optimized)
// Combines RGB->Y conversion with histogram computation to reduce memory round-trips
// Each block handles one tile and builds histogram from RGB data directly
// -------------------------------------------------------------------------------------
__global__ void fused_rgb_to_y_hist_kernel(const uint8_t *__restrict__ rgb,
                                           uint8_t *__restrict__ y_out, int H, int W, int tiles_x,
                                           int tiles_y, unsigned int *__restrict__ histograms) {
  extern __shared__ unsigned int shist[];  // 256 bins
  const int bins = 256;

  int tx = blockIdx.x;  // tile x
  int ty = blockIdx.y;  // tile y
  if (tx >= tiles_x || ty >= tiles_y)
    return;

  // Zero shared histogram
  for (int i = threadIdx.x; i < bins; i += blockDim.x)
    shist[i] = 0u;
  __syncthreads();

  // Compute tile bounds
  int tile_w = div_up(W, tiles_x);
  int tile_h = div_up(H, tiles_y);
  int x0 = tx * tile_w;
  int y0 = ty * tile_h;
  int x1 = min(x0 + tile_w, W);
  int y1 = min(y0 + tile_h, H);

  // Loop over tile pixels - fused RGB->Y + histogram
  int area = (x1 - x0) * (y1 - y0);
  for (int i = threadIdx.x; i < area; i += blockDim.x) {
    int dy = i / (x1 - x0);
    int dx = i - dy * (x1 - x0);
    int x = x0 + dx;
    int y = y0 + dy;

    int pixel_idx = y * W + x;
    int rgb_idx = 3 * pixel_idx;

    // RGB to LAB L* conversion (match OpenCV exactly)
    // First convert to normalized RGB [0,1]
    float r_val = rgb[rgb_idx + 0];
    float g_val = rgb[rgb_idx + 1];
    float b_val = rgb[rgb_idx + 2];

    float rf = r_val / 255.0f;
    float gf = g_val / 255.0f;
    float bf = b_val / 255.0f;

    // Apply gamma correction (sRGB to linear RGB)
    rf = srgb_to_linear(rf);
    gf = srgb_to_linear(gf);
    bf = srgb_to_linear(bf);

    // Convert to CIE XYZ using OpenCV's exact transformation matrix
    // From OpenCV source: cv::COLOR_RGB2Lab
    float x_xyz = 0.412453f * rf + 0.357580f * gf + 0.180423f * bf;
    float y_xyz = 0.212671f * rf + 0.715160f * gf + 0.072169f * bf;
    float z_xyz = 0.019334f * rf + 0.119193f * gf + 0.950227f * bf;

    // Normalize by D65 white point (OpenCV values)
    x_xyz = x_xyz / 0.950456f;
    y_xyz = y_xyz / 1.000000f;
    z_xyz = z_xyz / 1.088754f;

    // Convert Y to LAB L* using OpenCV's threshold and constants
    float fy = (y_xyz > 0.008856f) ? cbrtf(y_xyz) : (7.787f * y_xyz + 16.0f / 116.0f);
    float L = 116.0f * fy - 16.0f;

    // Scale L [0,100] to [0,255] for histogram (OpenCV LAB L* is [0,100])
    uint8_t y_u8 = (uint8_t)lrintf(fminf(fmaxf(L * 255.0f / 100.0f, 0.f), 255.f));  // Store Y value
    y_out[pixel_idx] = y_u8;

    // Add to histogram
    atomicAdd(&shist[static_cast<int>(y_u8)], 1u);
  }
  __syncthreads();

  // Write back histogram to global memory
  unsigned int *g_hist = histograms + (ty * tiles_x + tx) * bins;
  for (int i = threadIdx.x; i < bins; i += blockDim.x) {
    g_hist[i] = shist[i];
  }
}

extern "C" void LaunchFusedRGBToYHist(const uint8_t *rgb, uint8_t *y_plane, int H, int W,
                                      int tiles_x, int tiles_y, unsigned int *histograms,
                                      hipStream_t stream) {
  dim3 grid(tiles_x, tiles_y, 1);
  int threads = 512;  // Optimized for both compute and shared memory
  size_t shmem = 256 * sizeof(unsigned int);
  fused_rgb_to_y_hist_kernel<<<grid, threads, shmem, stream>>>(rgb, y_plane, H, W, tiles_x, tiles_y,
                                                               histograms);
}

// -------------------------------------------------------------------------------------
// Optimized Kernel: Histograms per tile with warp-privatized reduction (256 bins, uint32)
// Uses per-warp histograms to reduce atomic contention, then merges to shared memory
// -------------------------------------------------------------------------------------
__global__ void hist_per_tile_256_warp_optimized_kernel(const uint8_t *__restrict__ y_plane, int H,
                                                        int W, int tiles_x, int tiles_y,
                                                        unsigned int *__restrict__ histograms) {
  extern __shared__ unsigned int shist[];  // 256 bins
  const int bins = 256;
  const int warp_size = 32;
  const int warps_per_block = blockDim.x / warp_size;

  int tx = blockIdx.x;  // tile x
  int ty = blockIdx.y;  // tile y
  if (tx >= tiles_x || ty >= tiles_y)
    return;

  int warp_id = threadIdx.x / warp_size;
  int lane_id = threadIdx.x % warp_size;

  // Per-warp private histograms (warps_per_block * 256 bins)
  // This reduces atomic contention significantly
  unsigned int *warp_hist = shist + warp_id * bins;
  unsigned int *global_hist = shist + warps_per_block * bins;  // Final merged histogram

  // Zero per-warp histogram
  for (int i = lane_id; i < bins; i += warp_size)
    warp_hist[i] = 0u;

  // Zero global histogram (only first warp)
  if (warp_id == 0) {
    for (int i = lane_id; i < bins; i += warp_size)
      global_hist[i] = 0u;
  }
  __syncthreads();

  // Compute tile bounds
  int tile_w = div_up(W, tiles_x);
  int tile_h = div_up(H, tiles_y);
  int x0 = tx * tile_w;
  int y0 = ty * tile_h;
  int x1 = min(x0 + tile_w, W);
  int y1 = min(y0 + tile_h, H);

  // Each warp processes its portion of the tile
  int area = (x1 - x0) * (y1 - y0);
  for (int i = threadIdx.x; i < area; i += blockDim.x) {
    int dy = i / (x1 - x0);
    int dx = i - dy * (x1 - x0);
    int x = x0 + dx;
    int y = y0 + dy;
    uint8_t v = y_plane[y * W + x];

    // Atomic to warp-private histogram (much less contention)
    atomicAdd(&warp_hist[static_cast<int>(v)], 1u);
  }
  __syncthreads();

  // Merge warp histograms to final histogram
  for (int bin = lane_id; bin < bins; bin += warp_size) {
    unsigned int sum = 0u;
    for (int w = 0; w < warps_per_block; ++w) {
      sum += shist[w * bins + bin];
    }
    global_hist[bin] = sum;
  }
  __syncthreads();

  // Write back to global memory
  unsigned int *g_hist = histograms + (ty * tiles_x + tx) * bins;
  for (int i = threadIdx.x; i < bins; i += blockDim.x) {
    g_hist[i] = global_hist[i];
  }
}

extern "C" void LaunchHistPerTile256WarpOptimized(const uint8_t *y_plane, int H, int W, int tiles_x,
                                                  int tiles_y, unsigned int *histograms,
                                                  hipStream_t stream) {
  dim3 grid(tiles_x, tiles_y, 1);
  int threads = 512;  // 16 warps per block
  int warps_per_block = threads / 32;
  // Shared memory: warps_per_block * 256 (private) + 256 (final) = (warps_per_block + 1) * 256
  size_t shmem = (warps_per_block + 1) * 256 * sizeof(unsigned int);
  hist_per_tile_256_warp_optimized_kernel<<<grid, threads, shmem, stream>>>(y_plane, H, W, tiles_x,
                                                                            tiles_y, histograms);
}

// Original version kept for fallback
__global__ void hist_per_tile_256_kernel(const uint8_t *__restrict__ y_plane, int H, int W,
                                         int tiles_x, int tiles_y,
                                         unsigned int *__restrict__ histograms) {
  extern __shared__ unsigned int shist[];  // 256 bins
  const int bins = 256;

  int tx = blockIdx.x;  // tile x
  int ty = blockIdx.y;  // tile y
  if (tx >= tiles_x || ty >= tiles_y)
    return;

  // Zero shared histogram
  for (int i = threadIdx.x; i < bins; i += blockDim.x)
    shist[i] = 0u;
  __syncthreads();

  // Compute tile bounds
  int tile_w = div_up(W, tiles_x);
  int tile_h = div_up(H, tiles_y);
  int x0 = tx * tile_w;
  int y0 = ty * tile_h;
  int x1 = min(x0 + tile_w, W);
  int y1 = min(y0 + tile_h, H);

  // Loop over tile pixels
  int area = (x1 - x0) * (y1 - y0);
  for (int i = threadIdx.x; i < area; i += blockDim.x) {
    int dy = i / (x1 - x0);
    int dx = i - dy * (x1 - x0);
    int x = x0 + dx;
    int y = y0 + dy;
    uint8_t v = y_plane[y * W + x];
    atomicAdd(&shist[static_cast<int>(v)], 1u);
  }
  __syncthreads();

  // Write back to global memory
  unsigned int *g_hist = histograms + (ty * tiles_x + tx) * bins;
  for (int i = threadIdx.x; i < bins; i += blockDim.x) {
    g_hist[i] = shist[i];
  }
}

extern "C" void LaunchHistPerTile256(const uint8_t *y_plane, int H, int W, int tiles_x, int tiles_y,
                                     unsigned int *histograms, hipStream_t stream) {
  // Use warp-optimized version for larger tiles (where contention is higher)
  int tile_area = div_up(W, tiles_x) * div_up(H, tiles_y);
  if (tile_area >= 1024) {  // Threshold where warp optimization pays off
    LaunchHistPerTile256WarpOptimized(y_plane, H, W, tiles_x, tiles_y, histograms, stream);
  } else {
    // Use original version for small tiles
    dim3 grid(tiles_x, tiles_y, 1);
    int threads = 512;
    size_t shmem = 256 * sizeof(unsigned int);
    hist_per_tile_256_kernel<<<grid, threads, shmem, stream>>>(y_plane, H, W, tiles_x, tiles_y,
                                                               histograms);
  }
}

// -------------------------------------------------------------------------------------
// Kernel 3: Clip + CDF -> LUT per tile (uint8 LUT).
// clip_limit_rel: relative multiplier of the average bin count per tile
//   limit = clip_limit_rel * (tile_area / bins)
// Excess is redistributed uniformly.
// LUT[v] = round( (cdf[v] - cdf_min) / (tile_area - cdf_min) * 255 )
// -------------------------------------------------------------------------------------
__global__ void clip_cdf_lut_256_kernel(unsigned int *__restrict__ histograms, int tiles_x,
                                        int tiles_y, int tile_w,
                                        int tile_h,  // nominal, last tiles smaller
                                        int W, int H, float clip_limit_rel,
                                        uint8_t *__restrict__ luts) {
  const int bins = 256;
  int tid = threadIdx.x;

  int tx = blockIdx.x;
  int ty = blockIdx.y;
  if (tx >= tiles_x || ty >= tiles_y)
    return;

  // Actual tile bounds (handle edges)
  int x0 = tx * tile_w;
  int y0 = ty * tile_h;
  int x1 = min(x0 + tile_w, W);
  int y1 = min(y0 + tile_h, H);
  int area = max(1, (x1 - x0) * (y1 - y0));

  unsigned int *hist = histograms + (ty * tiles_x + tx) * bins;
  __shared__ unsigned int h[256];
  __shared__ unsigned int cdf[256];

  // Load histogram
  for (int i = tid; i < bins; i += blockDim.x)
    h[i] = hist[i];
  __syncthreads();

  // Compute clip limit (match OpenCV exactly)
  float clip_limit_f =
      clip_limit_rel * area / bins;  // OpenCV: clipLimit * tileSizeTotal / histSize
  unsigned int limit = static_cast<unsigned int>(clip_limit_f);
  limit = max(limit, 1u);  // OpenCV: std::max(clipLimit, 1)

  // Clip and accumulate excess
  __shared__ unsigned int excess;
  if (tid == 0)
    excess = 0u;
  __syncthreads();

  for (int i = tid; i < bins; i += blockDim.x) {
    unsigned int v = h[i];
    if (v > limit) {
      unsigned int over = v - limit;
      h[i] = limit;
      atomicAdd(&excess, over);
    }
  }
  __syncthreads();

  // Redistribute excess using OpenCV's exact algorithm
  unsigned int redistBatch = excess / bins;  // OpenCV: redistBatch = clipped / histSize
  unsigned int residual = excess % bins;     // OpenCV: residual = clipped - redistBatch * histSize

  for (int i = tid; i < bins; i += blockDim.x) {
    h[i] += redistBatch;  // OpenCV: tileHist[i] += redistBatch
  }
  __syncthreads();

  // Distribute residual using OpenCV's exact step pattern
  if (tid == 0 && residual > 0) {
    unsigned int residualStep = max(bins / residual, 1u);  // OpenCV: MAX(histSize / residual, 1)
    for (unsigned int i = 0; i < bins && residual > 0; i += residualStep, residual--) {
      h[i]++;  // OpenCV: tileHist[i]++
    }
  }
  __syncthreads();

  // Prefix-sum (CDF)
  if (tid == 0) {
    unsigned int acc = 0u;
    for (int i = 0; i < bins; ++i) {
      acc += h[i];
      cdf[i] = acc;
    }
  }
  __syncthreads();

  // Build LUT using OpenCV's exact scaling
  uint8_t *lut = luts + (ty * tiles_x + tx) * bins;
  float lutScale = static_cast<float>(bins - 1) /
                   static_cast<float>(area);  // OpenCV: (histSize - 1) / tileSizeTotal

  for (int i = tid; i < bins; i += blockDim.x) {
    float val = static_cast<float>(cdf[i]) * lutScale;  // OpenCV: sum * lutScale
    lut[i] = static_cast<uint8_t>(lrintf(fminf(fmaxf(val, 0.f), 255.f)));
  }
}

extern "C" void LaunchClipCdfToLut256(unsigned int *histograms, int H, int W, int tiles_x,
                                      int tiles_y, float clip_limit_rel, uint8_t *luts,
                                      hipStream_t stream) {
  int tile_w = div_up(W, tiles_x);
  int tile_h = div_up(H, tiles_y);
  dim3 grid(tiles_x, tiles_y, 1);

  // Optimize thread count for better occupancy on modern GPUs
  // 256 threads allows more blocks per SM, improving overall throughput
  int threads = 256;  // Changed from 512 for better occupancy
  clip_cdf_lut_256_kernel<<<grid, threads, 0, stream>>>(histograms, tiles_x, tiles_y, tile_w,
                                                        tile_h, W, H, clip_limit_rel, luts);
}

// -------------------------------------------------------------------------------------
// Optimized Vectorized Kernel: Apply LUT with bilinear interpolation for GRAYSCALE output.
// Uses float4 vectorized loads for better memory coalescing (4 pixels per load)
// -------------------------------------------------------------------------------------
__global__ void apply_lut_bilinear_gray_vectorized_kernel(const uint8_t *__restrict__ src_y,
                                                          uint8_t *__restrict__ dst_y, int H, int W,
                                                          int tiles_x, int tiles_y,
                                                          const uint8_t *__restrict__ luts) {
  int base_idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
  int N = H * W;

// Process 4 pixels per thread for better memory coalescing
#pragma unroll
  for (int i = 0; i < 4; ++i) {
    int idx = base_idx + i;
    if (idx >= N)
      return;

    int y = idx / W;
    int x = idx - y * W;

    // Tile geometry - use same calculation as histogram kernel for consistency
    int tile_w = div_up(W, tiles_x);
    int tile_h = div_up(H, tiles_y);

    // Tile coordinates
    float gx = (x + 0.5f) / tile_w - 0.5f;  // tile-space x
    float gy = (y + 0.5f) / tile_h - 0.5f;  // tile-space y
    int tx = static_cast<int>(floorf(gx));
    int ty = static_cast<int>(floorf(gy));
    float fx = gx - tx;
    float fy = gy - ty;

    // Handle border cases properly
    int tx0, ty0, tx1, ty1;

    if (tx < 0) {
      tx0 = tx1 = 0;
      fx = 0.f;
    } else if (tx >= tiles_x - 1) {
      tx0 = tx1 = tiles_x - 1;
      fx = 0.f;
    } else {
      tx0 = tx;
      tx1 = tx + 1;
      fx = fminf(fmaxf(fx, 0.f), 1.f);
    }

    if (ty < 0) {
      ty0 = ty1 = 0;
      fy = 0.f;
    } else if (ty >= tiles_y - 1) {
      ty0 = ty1 = tiles_y - 1;
      fy = 0.f;
    } else {
      ty0 = ty;
      ty1 = ty + 1;
      fy = fminf(fmaxf(fy, 0.f), 1.f);
    }

    int bins = 256;
    const uint8_t *lut_tl = luts + (ty0 * tiles_x + tx0) * bins;
    const uint8_t *lut_tr = luts + (ty0 * tiles_x + tx1) * bins;
    const uint8_t *lut_bl = luts + (ty1 * tiles_x + tx0) * bins;
    const uint8_t *lut_br = luts + (ty1 * tiles_x + tx1) * bins;

    uint8_t v = src_y[idx];
    float v_tl = lut_tl[v];
    float v_tr = lut_tr[v];
    float v_bl = lut_bl[v];
    float v_br = lut_br[v];

    // Bilinear blend
    float v_top = v_tl * (1.f - fx) + v_tr * fx;
    float v_bot = v_bl * (1.f - fx) + v_br * fx;
    float v_out = v_top * (1.f - fy) + v_bot * fy;

    int outi = static_cast<int>(lrintf(fminf(fmaxf(v_out, 0.f), 255.f)));
    dst_y[idx] = (uint8_t)outi;
  }
}

// Original single-pixel version
__global__ void apply_lut_bilinear_gray_kernel(const uint8_t *__restrict__ src_y,
                                               uint8_t *__restrict__ dst_y, int H, int W,
                                               int tiles_x, int tiles_y,
                                               const uint8_t *__restrict__ luts) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int N = H * W;
  if (idx >= N)
    return;

  int y = idx / W;
  int x = idx - y * W;

  // Tile geometry - use same calculation as histogram kernel for consistency
  int tile_w = div_up(W, tiles_x);
  int tile_h = div_up(H, tiles_y);

  // Tile coordinates
  float gx = (x + 0.5f) / tile_w - 0.5f;  // tile-space x
  float gy = (y + 0.5f) / tile_h - 0.5f;  // tile-space y
  int tx = static_cast<int>(floorf(gx));
  int ty = static_cast<int>(floorf(gy));
  float fx = gx - tx;
  float fy = gy - ty;

  // Handle border cases properly
  // For pixels outside tile boundaries, use border extrapolation
  int tx0, ty0, tx1, ty1;

  if (tx < 0) {
    tx0 = tx1 = 0;
    fx = 0.f;
  } else if (tx >= tiles_x - 1) {
    tx0 = tx1 = tiles_x - 1;
    fx = 0.f;
  } else {
    tx0 = tx;
    tx1 = tx + 1;
    fx = fminf(fmaxf(fx, 0.f), 1.f);
  }

  if (ty < 0) {
    ty0 = ty1 = 0;
    fy = 0.f;
  } else if (ty >= tiles_y - 1) {
    ty0 = ty1 = tiles_y - 1;
    fy = 0.f;
  } else {
    ty0 = ty;
    ty1 = ty + 1;
    fy = fminf(fmaxf(fy, 0.f), 1.f);
  }

  int bins = 256;
  const uint8_t *lut_tl = luts + (ty0 * tiles_x + tx0) * bins;
  const uint8_t *lut_tr = luts + (ty0 * tiles_x + tx1) * bins;
  const uint8_t *lut_bl = luts + (ty1 * tiles_x + tx0) * bins;
  const uint8_t *lut_br = luts + (ty1 * tiles_x + tx1) * bins;

  uint8_t v = src_y[idx];
  float v_tl = lut_tl[v];
  float v_tr = lut_tr[v];
  float v_bl = lut_bl[v];
  float v_br = lut_br[v];

  // Bilinear blend
  float v_top = v_tl * (1.f - fx) + v_tr * fx;
  float v_bot = v_bl * (1.f - fx) + v_br * fx;
  float v_out = v_top * (1.f - fy) + v_bot * fy;

  int outi = static_cast<int>(lrintf(fminf(fmaxf(v_out, 0.f), 255.f)));
  dst_y[idx] = (uint8_t)outi;
}

// -------------------------------------------------------------------------------------
// Simplified Texture-Optimized Kernel: Use 1D texture for LUT lookups
// -------------------------------------------------------------------------------------
__global__ void apply_lut_bilinear_gray_texture_kernel(const uint8_t *__restrict__ src_y,
                                                       uint8_t *__restrict__ dst_y, int H, int W,
                                                       int tiles_x, int tiles_y,
                                                       hipTextureObject_t lut_texture, int bins) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int N = H * W;
  if (idx >= N)
    return;

  int y = idx / W;
  int x = idx - y * W;

  // Tile geometry
  int tile_w = div_up(W, tiles_x);
  int tile_h = div_up(H, tiles_y);

  // Tile coordinates
  float gx = (x + 0.5f) / tile_w - 0.5f;
  float gy = (y + 0.5f) / tile_h - 0.5f;
  int tx = static_cast<int>(floorf(gx));
  int ty = static_cast<int>(floorf(gy));
  float fx = gx - tx;
  float fy = gy - ty;

  // Handle border cases
  int tx0, ty0, tx1, ty1;

  if (tx < 0) {
    tx0 = tx1 = 0;
    fx = 0.f;
  } else if (tx >= tiles_x - 1) {
    tx0 = tx1 = tiles_x - 1;
    fx = 0.f;
  } else {
    tx0 = tx;
    tx1 = tx + 1;
    fx = fminf(fmaxf(fx, 0.f), 1.f);
  }

  if (ty < 0) {
    ty0 = ty1 = 0;
    fy = 0.f;
  } else if (ty >= tiles_y - 1) {
    ty0 = ty1 = tiles_y - 1;
    fy = 0.f;
  } else {
    ty0 = ty;
    ty1 = ty + 1;
    fy = fminf(fmaxf(fy, 0.f), 1.f);
  }

  uint8_t v = src_y[idx];

  // Use texture memory for LUT lookups with 1D indexing
  int tile_tl = ty0 * tiles_x + tx0;
  int tile_tr = ty0 * tiles_x + tx1;
  int tile_bl = ty1 * tiles_x + tx0;
  int tile_br = ty1 * tiles_x + tx1;

  float v_tl = tex1Dfetch<uint8_t>(lut_texture, tile_tl * bins + v);
  float v_tr = tex1Dfetch<uint8_t>(lut_texture, tile_tr * bins + v);
  float v_bl = tex1Dfetch<uint8_t>(lut_texture, tile_bl * bins + v);
  float v_br = tex1Dfetch<uint8_t>(lut_texture, tile_br * bins + v);

  // Bilinear blend
  float v_top = v_tl * (1.f - fx) + v_tr * fx;
  float v_bot = v_bl * (1.f - fx) + v_br * fx;
  float v_out = v_top * (1.f - fy) + v_bot * fy;

  int outi = static_cast<int>(lrintf(fminf(fmaxf(v_out, 0.f), 255.f)));
  dst_y[idx] = (uint8_t)outi;
}

extern "C" void LaunchApplyLUTBilinearToGrayTexture(const uint8_t *src_gray, uint8_t *dst_gray,
                                                    int H, int W, int tiles_x, int tiles_y,
                                                    const uint8_t *luts, hipStream_t stream) {
  // Create 1D texture object for LUT array
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeLinear;
  resDesc.res.linear.devPtr = const_cast<uint8_t *>(luts);
  resDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
  resDesc.res.linear.desc.x = 8;  // 8-bit uint8
  resDesc.res.linear.sizeInBytes = tiles_x * tiles_y * 256 * sizeof(uint8_t);

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  hipTextureObject_t lutTexture = 0;
  hipCreateTextureObject(&lutTexture, &resDesc, &texDesc, nullptr);

  int N = H * W;
  int threads = 256;
  int blocks = div_up(N, threads);
  apply_lut_bilinear_gray_texture_kernel<<<blocks, threads, 0, stream>>>(
      src_gray, dst_gray, H, W, tiles_x, tiles_y, lutTexture, 256);

  // Clean up texture object
  hipDestroyTextureObject(lutTexture);
}

// Update the main launcher to optionally use texture version
extern "C" void LaunchApplyLUTBilinearToGray(const uint8_t *src_gray, uint8_t *dst_gray, int H,
                                             int W, int tiles_x, int tiles_y, const uint8_t *luts,
                                             hipStream_t stream) {
  int N = H * W;
  int total_tiles = tiles_x * tiles_y;

  // Use texture version for larger tile counts where cache benefits are significant
  if (total_tiles >= 32 && N >= 16384) {  // Threshold for texture memory benefit
    LaunchApplyLUTBilinearToGrayTexture(src_gray, dst_gray, H, W, tiles_x, tiles_y, luts, stream);
  } else if (N >= 8192) {  // Use vectorized version for medium images
    int threads = 256;
    int blocks = div_up(N, threads * 4);
    apply_lut_bilinear_gray_vectorized_kernel<<<blocks, threads, 0, stream>>>(
        src_gray, dst_gray, H, W, tiles_x, tiles_y, luts);
  } else {
    // Use original version for smaller images
    int threads = 512;
    int blocks = div_up(N, threads);
    apply_lut_bilinear_gray_kernel<<<blocks, threads, 0, stream>>>(src_gray, dst_gray, H, W,
                                                                   tiles_x, tiles_y, luts);
  }
}

// -------------------------------------------------------------------------------------
// Optimized Vectorized Kernel: Apply LUT for RGB using vectorized memory access
// Uses uchar4 loads for RGB data and processes multiple pixels per thread
// -------------------------------------------------------------------------------------
__global__ void apply_lut_bilinear_rgb_vectorized_kernel(const uint8_t *__restrict__ src_rgb,
                                                         const uint8_t *__restrict__ src_y,
                                                         uint8_t *__restrict__ dst_rgb, int H,
                                                         int W, int tiles_x, int tiles_y,
                                                         const uint8_t *__restrict__ luts) {
  int base_idx = (blockIdx.x * blockDim.x + threadIdx.x) * 2;  // Process 2 pixels per thread
  int N = H * W;

#pragma unroll
  for (int i = 0; i < 2; ++i) {
    int idx = base_idx + i;
    if (idx >= N)
      return;

    int y = idx / W;
    int x = idx - y * W;

    // Tile geometry calculations (same as before)
    float inv_tw = static_cast<float>(tiles_x) / static_cast<float>(W);
    float inv_th = static_cast<float>(tiles_y) / static_cast<float>(H);

    float txf = x * inv_tw - 0.5f;
    float tyf = y * inv_th - 0.5f;

    int tx = static_cast<int>(floorf(txf));
    int ty = static_cast<int>(floorf(tyf));
    float fx = txf - tx;
    float fy = tyf - ty;

    // Handle border cases
    int tx0, ty0, tx1, ty1;

    if (tx < 0) {
      tx0 = tx1 = 0;
      fx = 0.f;
    } else if (tx >= tiles_x - 1) {
      tx0 = tx1 = tiles_x - 1;
      fx = 0.f;
    } else {
      tx0 = tx;
      tx1 = tx + 1;
      fx = fminf(fmaxf(fx, 0.f), 1.f);
    }

    if (ty < 0) {
      ty0 = ty1 = 0;
      fy = 0.f;
    } else if (ty >= tiles_y - 1) {
      ty0 = ty1 = tiles_y - 1;
      fy = 0.f;
    } else {
      ty0 = ty;
      ty1 = ty + 1;
      fy = fminf(fmaxf(fy, 0.f), 1.f);
    }

    int bins = 256;
    const uint8_t *lut_tl = luts + (ty0 * tiles_x + tx0) * bins;
    const uint8_t *lut_tr = luts + (ty0 * tiles_x + tx1) * bins;
    const uint8_t *lut_bl = luts + (ty1 * tiles_x + tx0) * bins;
    const uint8_t *lut_br = luts + (ty1 * tiles_x + tx1) * bins;

    uint8_t orig_L_u8 = src_y[idx];
    float v_tl = lut_tl[orig_L_u8];
    float v_tr = lut_tr[orig_L_u8];
    float v_bl = lut_bl[orig_L_u8];
    float v_br = lut_br[orig_L_u8];

    float v_top = v_tl * (1.f - fx) + v_tr * fx;
    float v_bot = v_bl * (1.f - fx) + v_br * fx;
    float enhanced_L_u8 = v_top * (1.f - fy) + v_bot * fy;

    // Convert original RGB to LAB
    int base = 3 * idx;
    uint8_t orig_r = src_rgb[base + 0];
    uint8_t orig_g = src_rgb[base + 1];
    uint8_t orig_b = src_rgb[base + 2];

    float orig_L, orig_a, orig_b_lab;
    rgb_to_lab(orig_r, orig_g, orig_b, &orig_L, &orig_a, &orig_b_lab);

    // Replace L* with enhanced version, keep a* and b* unchanged
    float enhanced_L = enhanced_L_u8 * 100.0f / 255.0f;

    // Convert LAB back to RGB
    uint8_t new_r, new_g, new_b;
    lab_to_rgb(enhanced_L, orig_a, orig_b_lab, &new_r, &new_g, &new_b);

    dst_rgb[base + 0] = new_r;
    dst_rgb[base + 1] = new_g;
    dst_rgb[base + 2] = new_b;
  }
}

// Original single-pixel RGB version
__global__ void apply_lut_bilinear_rgb_kernel(const uint8_t *__restrict__ src_rgb,
                                              const uint8_t *__restrict__ src_y,  // original L*
                                              uint8_t *__restrict__ dst_rgb, int H, int W,
                                              int tiles_x, int tiles_y,
                                              const uint8_t *__restrict__ luts) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int N = H * W;
  if (idx >= N)
    return;

  int y = idx / W;
  int x = idx - y * W;

  // Tile geometry - match OpenCV exactly
  // OpenCV: tileSize = Size(src.width / tilesX, src.height / tilesY)
  float inv_tw =
      static_cast<float>(tiles_x) / static_cast<float>(W);  // OpenCV: 1.0f / tileSize.width
  float inv_th =
      static_cast<float>(tiles_y) / static_cast<float>(H);  // OpenCV: 1.0f / tileSize.height

  // Tile coordinates (match OpenCV exactly)
  float txf = x * inv_tw - 0.5f;  // OpenCV: x * inv_tw - 0.5f
  float tyf = y * inv_th - 0.5f;  // OpenCV: y * inv_th - 0.5f

  int tx = static_cast<int>(floorf(txf));  // OpenCV: cvFloor(txf)
  int ty = static_cast<int>(floorf(tyf));  // OpenCV: cvFloor(tyf)
  float fx = txf - tx;                     // OpenCV: xa = txf - tx1
  float fy = tyf - ty;                     // OpenCV: ya = tyf - ty1

  // Handle border cases properly
  // For pixels outside tile boundaries, use border extrapolation
  int tx0, ty0, tx1, ty1;

  if (tx < 0) {
    tx0 = tx1 = 0;
    fx = 0.f;
  } else if (tx >= tiles_x - 1) {
    tx0 = tx1 = tiles_x - 1;
    fx = 0.f;
  } else {
    tx0 = tx;
    tx1 = tx + 1;
    fx = fminf(fmaxf(fx, 0.f), 1.f);
  }

  if (ty < 0) {
    ty0 = ty1 = 0;
    fy = 0.f;
  } else if (ty >= tiles_y - 1) {
    ty0 = ty1 = tiles_y - 1;
    fy = 0.f;
  } else {
    ty0 = ty;
    ty1 = ty + 1;
    fy = fminf(fmaxf(fy, 0.f), 1.f);
  }

  int bins = 256;
  const uint8_t *lut_tl = luts + (ty0 * tiles_x + tx0) * bins;
  const uint8_t *lut_tr = luts + (ty0 * tiles_x + tx1) * bins;
  const uint8_t *lut_bl = luts + (ty1 * tiles_x + tx0) * bins;
  const uint8_t *lut_br = luts + (ty1 * tiles_x + tx1) * bins;

  uint8_t orig_L_u8 = src_y[idx];  // Original L* value scaled [0,255]
  float v_tl = lut_tl[orig_L_u8];
  float v_tr = lut_tr[orig_L_u8];
  float v_bl = lut_bl[orig_L_u8];
  float v_br = lut_br[orig_L_u8];

  float v_top = v_tl * (1.f - fx) + v_tr * fx;
  float v_bot = v_bl * (1.f - fx) + v_br * fx;
  float enhanced_L_u8 = v_top * (1.f - fy) + v_bot * fy;

  // Convert original RGB to LAB
  int base = 3 * idx;
  uint8_t orig_r = src_rgb[base + 0];
  uint8_t orig_g = src_rgb[base + 1];
  uint8_t orig_b = src_rgb[base + 2];

  float orig_L, orig_a, orig_b_lab;
  rgb_to_lab(orig_r, orig_g, orig_b, &orig_L, &orig_a, &orig_b_lab);

  // Replace L* with enhanced version, keep a* and b* unchanged
  float enhanced_L = enhanced_L_u8 * 100.0f / 255.0f;  // Scale back to [0,100] range

  // Convert LAB back to RGB
  uint8_t new_r, new_g, new_b;
  lab_to_rgb(enhanced_L, orig_a, orig_b_lab, &new_r, &new_g, &new_b);

  dst_rgb[base + 0] = new_r;
  dst_rgb[base + 1] = new_g;
  dst_rgb[base + 2] = new_b;
}

extern "C" void LaunchApplyLUTBilinearToRGB(const uint8_t *src_rgb, const uint8_t *src_y,
                                            uint8_t *dst_rgb, int H, int W, int tiles_x,
                                            int tiles_y, const uint8_t *luts, hipStream_t stream) {
  int N = H * W;

  // Use vectorized version for larger images
  if (N >= 8192) {                        // Threshold for vectorized processing
    int threads = 256;                    // Better occupancy with complex RGB processing
    int blocks = div_up(N, threads * 2);  // Each thread processes 2 pixels
    apply_lut_bilinear_rgb_vectorized_kernel<<<blocks, threads, 0, stream>>>(
        src_rgb, src_y, dst_rgb, H, W, tiles_x, tiles_y, luts);
  } else {
    // Use original version for smaller images
    int threads = 512;
    int blocks = div_up(N, threads);
    apply_lut_bilinear_rgb_kernel<<<blocks, threads, 0, stream>>>(src_rgb, src_y, dst_rgb, H, W,
                                                                  tiles_x, tiles_y, luts);
  }
}

// -------------------------------------------------------------------------------------
// Mega-Fused Kernel: Histogram + Clip + CDF + LUT generation in one pass
// Eliminates multiple kernel launches and global memory round-trips
// Each block handles one tile and computes everything from histogram to final LUT
// -------------------------------------------------------------------------------------
__global__ void mega_fused_hist_clip_cdf_lut_kernel(const uint8_t *__restrict__ y_plane, int H,
                                                    int W, int tiles_x, int tiles_y, int tile_w,
                                                    int tile_h, float clip_limit_rel,
                                                    uint8_t *__restrict__ luts) {
  extern __shared__ unsigned int sdata[];  // Dynamic shared memory
  const int bins = 256;
  const int warp_size = 32;
  const int warps_per_block = blockDim.x / warp_size;

  // Shared memory layout:
  // [0...warps_per_block*256) = per-warp histograms
  // [warps_per_block*256...warps_per_block*256+256) = final histogram
  // [warps_per_block*256+256...warps_per_block*256+512) = CDF
  unsigned int *warp_hist = sdata;
  unsigned int *hist = sdata + warps_per_block * bins;
  unsigned int *cdf = hist + bins;

  int tx = blockIdx.x;  // tile x
  int ty = blockIdx.y;  // tile y
  if (tx >= tiles_x || ty >= tiles_y)
    return;

  int warp_id = threadIdx.x / warp_size;
  int lane_id = threadIdx.x % warp_size;

  // Initialize shared memory
  unsigned int *my_warp_hist = warp_hist + warp_id * bins;
  for (int i = lane_id; i < bins; i += warp_size)
    my_warp_hist[i] = 0u;

  if (warp_id == 0) {
    for (int i = lane_id; i < bins; i += warp_size) {
      hist[i] = 0u;
      cdf[i] = 0u;
    }
  }
  __syncthreads();

  // Compute actual tile bounds
  int x0 = tx * tile_w;
  int y0 = ty * tile_h;
  int x1 = min(x0 + tile_w, W);
  int y1 = min(y0 + tile_h, H);
  int area = max(1, (x1 - x0) * (y1 - y0));

  // Build per-warp histograms
  int tile_area = (x1 - x0) * (y1 - y0);
  for (int i = threadIdx.x; i < tile_area; i += blockDim.x) {
    int dy = i / (x1 - x0);
    int dx = i - dy * (x1 - x0);
    int x = x0 + dx;
    int y = y0 + dy;
    uint8_t v = y_plane[y * W + x];
    atomicAdd(&my_warp_hist[static_cast<int>(v)], 1u);
  }
  __syncthreads();

  // Merge warp histograms
  for (int bin = lane_id; bin < bins; bin += warp_size) {
    unsigned int sum = 0u;
    for (int w = 0; w < warps_per_block; ++w) {
      sum += warp_hist[w * bins + bin];
    }
    hist[bin] = sum;
  }
  __syncthreads();

  // Clip histogram and redistribute excess
  float clip_limit_f = clip_limit_rel * area / bins;
  unsigned int limit = max(static_cast<unsigned int>(clip_limit_f), 1u);

  __shared__ unsigned int excess;
  if (threadIdx.x == 0)
    excess = 0u;
  __syncthreads();

  for (int i = threadIdx.x; i < bins; i += blockDim.x) {
    unsigned int v = hist[i];
    if (v > limit) {
      unsigned int over = v - limit;
      hist[i] = limit;
      atomicAdd(&excess, over);
    }
  }
  __syncthreads();

  // Redistribute excess
  unsigned int redistBatch = excess / bins;
  unsigned int residual = excess % bins;

  for (int i = threadIdx.x; i < bins; i += blockDim.x) {
    hist[i] += redistBatch;
  }
  __syncthreads();

  // Distribute residual (single thread)
  if (threadIdx.x == 0 && residual > 0) {
    unsigned int residualStep = max(bins / residual, 1u);
    for (unsigned int i = 0; i < bins && residual > 0; i += residualStep, residual--) {
      hist[i]++;
    }
  }
  __syncthreads();

  // Compute CDF (prefix sum)
  if (threadIdx.x == 0) {
    unsigned int acc = 0u;
    for (int i = 0; i < bins; ++i) {
      acc += hist[i];
      cdf[i] = acc;
    }
  }
  __syncthreads();

  // Generate LUT
  uint8_t *lut = luts + (ty * tiles_x + tx) * bins;
  float lutScale = static_cast<float>(bins - 1) / static_cast<float>(area);

  for (int i = threadIdx.x; i < bins; i += blockDim.x) {
    float val = static_cast<float>(cdf[i]) * lutScale;
    lut[i] = static_cast<uint8_t>(lrintf(fminf(fmaxf(val, 0.f), 255.f)));
  }
}

extern "C" void LaunchMegaFusedHistClipCdfLut(const uint8_t *y_plane, int H, int W, int tiles_x,
                                              int tiles_y, float clip_limit_rel, uint8_t *luts,
                                              hipStream_t stream) {
  int tile_w = div_up(W, tiles_x);
  int tile_h = div_up(H, tiles_y);
  dim3 grid(tiles_x, tiles_y, 1);
  int threads = 256;  // Optimized for occupancy

  // Calculate shared memory needed
  int warps_per_block = threads / 32;
  size_t shmem = (warps_per_block + 2) * 256 * sizeof(unsigned int);  // warp_hists + hist + cdf

  mega_fused_hist_clip_cdf_lut_kernel<<<grid, threads, shmem, stream>>>(
      y_plane, H, W, tiles_x, tiles_y, tile_w, tile_h, clip_limit_rel, luts);
}
extern "C" void LaunchCLAHE_Grayscale_U8_NHWC(const uint8_t *src_gray, uint8_t *dst_gray, int H,
                                              int W, int tiles_x, int tiles_y, float clip_limit_rel,
                                              unsigned int *tmp_histograms,  // tiles*bins
                                              uint8_t *tmp_luts,             // tiles*bins
                                              hipStream_t stream) {
  // Use mega-fused version for larger images where the fusion overhead pays off
  int total_tiles = tiles_x * tiles_y;
  if (total_tiles >= 16) {  // Threshold where fusion is beneficial
    LaunchMegaFusedHistClipCdfLut(src_gray, H, W, tiles_x, tiles_y, clip_limit_rel, tmp_luts,
                                  stream);
  } else {
    // Use traditional 3-kernel approach for smaller tile counts
    LaunchHistPerTile256(src_gray, H, W, tiles_x, tiles_y, tmp_histograms, stream);
    LaunchClipCdfToLut256(tmp_histograms, H, W, tiles_x, tiles_y, clip_limit_rel, tmp_luts, stream);
  }
  LaunchApplyLUTBilinearToGray(src_gray, dst_gray, H, W, tiles_x, tiles_y, tmp_luts, stream);
}

extern "C" void LaunchCLAHE_RGB_U8_NHWC(const uint8_t *src_rgb, uint8_t *dst_rgb,
                                        uint8_t *y_plane,  // [H*W]
                                        int H, int W, int tiles_x, int tiles_y,
                                        float clip_limit_rel,
                                        unsigned int *tmp_histograms,  // tiles*bins
                                        uint8_t *tmp_luts,             // tiles*bins
                                        hipStream_t stream) {
  LaunchRGBToYUint8NHWC(src_rgb, y_plane, H, W, stream);
  LaunchHistPerTile256(y_plane, H, W, tiles_x, tiles_y, tmp_histograms, stream);
  LaunchClipCdfToLut256(tmp_histograms, H, W, tiles_x, tiles_y, clip_limit_rel, tmp_luts, stream);
  LaunchApplyLUTBilinearToRGB(src_rgb, y_plane, dst_rgb, H, W, tiles_x, tiles_y, tmp_luts, stream);
  CUDA_CHECK(hipGetLastError());
}

// Optimized version using fused RGB->Y + histogram kernel
extern "C" void LaunchCLAHE_RGB_U8_NHWC_Optimized(const uint8_t *src_rgb, uint8_t *dst_rgb,
                                                  uint8_t *y_plane,  // [H*W]
                                                  int H, int W, int tiles_x, int tiles_y,
                                                  float clip_limit_rel,
                                                  unsigned int *tmp_histograms,  // tiles*bins
                                                  uint8_t *tmp_luts,             // tiles*bins
                                                  hipStream_t stream) {
  // Fused RGB->Y conversion + histogram computation (saves one kernel launch + memory round-trip)
  LaunchFusedRGBToYHist(src_rgb, y_plane, H, W, tiles_x, tiles_y, tmp_histograms, stream);
  LaunchClipCdfToLut256(tmp_histograms, H, W, tiles_x, tiles_y, clip_limit_rel, tmp_luts, stream);
  LaunchApplyLUTBilinearToRGB(src_rgb, y_plane, dst_rgb, H, W, tiles_x, tiles_y, tmp_luts, stream);
  CUDA_CHECK(hipGetLastError());
}
