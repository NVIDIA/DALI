#include "hip/hip_runtime.h"
// Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "dali/operators/image/remap/warp_affine_params.h"
#include <hip/hip_runtime.h>

namespace dali {
namespace {

template <int ndims>
__global__ void InvertTransformsKernel(WarpAffineParams<ndims> *output,
                                       const WarpAffineParams<ndims> *input, int count) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x)
    output[i] = input[i].inv();
}

template <int ndims>
void InvertTransforms(WarpAffineParams<ndims> *output, const WarpAffineParams<ndims> *input,
                      int count, hipStream_t stream) {
  int blocks = div_ceil(count, 512);
  int threads = std::min(count, 512);
  InvertTransformsKernel<ndims><<<blocks, threads, 0, stream>>>(output, input, count);
}

}  // namespace


template <>
void InvertTransformsGPU<2>(WarpAffineParams<2> *output, const WarpAffineParams<2> *input,
                            int count, hipStream_t stream) {
  InvertTransforms<2>(output, input, count, stream);
}

template <>
void InvertTransformsGPU<3>(WarpAffineParams<3> *output, const WarpAffineParams<3> *input,
                            int count, hipStream_t stream) {
  InvertTransforms<3>(output, input, count, stream);
}

}  // namespace dali
