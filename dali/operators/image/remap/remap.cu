// Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "dali/operators/image/remap/remap.h"
#include "dali/operators/image/remap/remap.cuh"
#include "dali/kernels/kernel_manager.h"

namespace dali {
namespace remap {

class RemapGpu : public Remap<GPUBackend> {
  using B = GPUBackend;

 public:
  explicit RemapGpu(const OpSpec &spec) : Remap<B>(spec) {}


  void RunImpl(Workspace &ws) override {
    const auto &input = ws.template Input<B>(0);
    TYPE_SWITCH(input.type(), type2id, InputType, REMAP_SUPPORTED_TYPES, (
    {
      RunImplTyped<InputType>(ws);
    }
    ), DALI_FAIL(make_string("Unsupported input type: ", input.type())))  // NOLINT
  }


 private:
  template<typename InputType>
  void RunImplTyped(Workspace &ws) {
    using Kernel = kernels::remap::NppRemapKernel<StorageGPU, InputType>;
    const auto &input = ws.template Input<B>(0);
    const auto &mapx = ws.template Input<B>(1);
    const auto &mapy = ws.template Input<B>(2);
    auto &output = ws.template Output<B>(0);
    km_.Resize<Kernel>(1, spec_.template GetArgument<int>("device_id"));
    kernels::KernelContext ctx;
    ctx.gpu.stream = ws.stream();
    dali::kernels::DynamicScratchpad scratchpad({}, ws.stream());
    ctx.scratchpad = &scratchpad;

    TensorList<B> mapx_shifted, mapy_shifted;
    mapx_shifted.set_order(ws.stream());
    mapy_shifted.set_order(ws.stream());
    if (shift_pixels_) {
      mapx_shifted.Copy(mapx);
      mapy_shifted.Copy(mapy);
      detail::ShiftPixelOrigin(view<float>(mapx_shifted), shift_value_, scratchpad, ws.stream());
      detail::ShiftPixelOrigin(view<float>(mapy_shifted), shift_value_, scratchpad, ws.stream());
    }
    km_.Run<Kernel>(0, ctx, view<InputType, 3>(output), view<const InputType, 3>(input),
                    view<const float, 2>(shift_pixels_ ? mapx_shifted : mapx),
                    view<const float, 2>(shift_pixels_ ? mapy_shifted : mapy),
                    span<const kernels::Roi<2>>{}, span<const kernels::Roi<2>>{},
                    make_span(interps_));
  }


  kernels::KernelManager km_;
};

DALI_REGISTER_OPERATOR(experimental__Remap, RemapGpu, GPU);

}  // namespace remap
}  // namespace dali
