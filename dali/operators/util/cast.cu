#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "dali/operators/util/cast.h"
#include "dali/core/error_handling.h"
#include "dali/core/cuda_utils.h"
#include "dali/core/convert.h"

namespace dali {

template <typename OType, typename IType>
__global__ void
BatchedCastKernel(OType * output, const IType * in, size_t N) {
  size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid < N) {
    output[tid] = ConvertSat<OType>(in[tid]);
  }
}

template <typename OType, typename IType>
DALIError_t BatchedCast(OType * output,
                        const IType * input,
                        size_t N,
                        hipStream_t stream) {
  DALI_ASSERT(output != nullptr);
  DALI_ASSERT(input != nullptr);
  const int threads = 512;
  const int blocks = (N + threads - 1)/threads;
  BatchedCastKernel<<<blocks, threads, 0, stream>>>(output, input, N);
  return DALISuccess;
}

template<>
void Cast<GPUBackend>::RunImpl(DeviceWorkspace &ws) {
  const auto &input = ws.Input<GPUBackend>(0);
  auto &output = ws.Output<GPUBackend>(0);

  DALIDataType itype = input.type().id();

  DALI_TYPE_SWITCH_WITH_FP16(output_type_, OType,
      output.mutable_data<OType>();
      output.ResizeLike(input);
      DALI_TYPE_SWITCH_WITH_FP16(itype, IType,
        DALI_CALL(BatchedCast(
            output.mutable_data<OType>(),
            input.data<IType>(),
            input.size(),
            ws.stream()));););
}

DALI_REGISTER_OPERATOR(Cast, Cast<GPUBackend>, GPU);

}  // namespace dali
