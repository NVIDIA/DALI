// Copyright (c) 2017-2022, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime_api.h>
#include "dali/test/plugins/dummy/dummy.h"

namespace other_ns {

template<>
void Dummy<::dali::GPUBackend>::RunImpl(::dali::Workspace &ws) {
  const auto &input = ws.Input<::dali::GPUBackend>(0);
  const auto &shape = input.shape();
  auto &output = ws.Output<::dali::GPUBackend>(0);
  for (int sample_idx = 0; sample_idx < shape.num_samples(); sample_idx++) {
    CUDA_CALL(hipMemcpyAsync(
            output.raw_mutable_tensor(sample_idx),
            input.raw_tensor(sample_idx),
            shape[sample_idx].num_elements() * input.type_info().size(),
            hipMemcpyDeviceToDevice,
            ws.stream()));
  }
}

}  // namespace other_ns

DALI_REGISTER_OPERATOR(CustomDummy, ::other_ns::Dummy<::dali::GPUBackend>, ::dali::GPU);


