#include "hip/hip_runtime.h"
// Copyright (c) 2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gtest/gtest.h>
#include <chrono>
#include <cmath>
#include <iostream>
#include <random>
#include <sstream>
#include <string>
#include <thread>
#include <vector>
#include "dali/core/mm/default_resources.h"
#include "dali/core/spinlock.h"
#include "dali/core/hip_stream.h"
#include "dali/core/cuda_stream_pool.h"
#include "dali/core/cuda_event.h"
#include "dali/core/cuda_error.h"
#include "dali/core/device_guard.h"

namespace dali {
namespace mm {
namespace test {

using perf_timer = std::chrono::high_resolution_clock;

inline void print_time(std::ostream &os, double seconds) {
  if (seconds < 1e-6) {
    os << seconds * 1e+9 << " ns";
  } else if (seconds < 1e-3) {
    os << seconds * 1e+6 << " µs";
  } else if (seconds < 1.0) {
    os << seconds * 1e+3 << " ms";
  } else {
    os << seconds << " s";
  }
}

template <typename Rep, typename Period>
double seconds(std::chrono::duration<Rep, Period> time) {
  return std::chrono::duration_cast<std::chrono::duration<double>>(time).count();
}

template <typename Rep, typename Period>
void print_time(std::ostream &os, std::chrono::duration<Rep, Period> time) {
  return format_time(seconds(time));
}

inline std::string format_time(double seconds) {
  std::stringstream ss;
  print_time(ss, seconds);
  return ss.str();
}

template <typename Rep, typename Period>
std::string format_time(std::chrono::duration<Rep, Period> time) {
  return format_time(seconds(time));
}

void RunBenchmark(mm::async_memory_resource<mm::memory_kind::device> *res,
                  int num_threads,
                  int num_streams,
                  double test_time) {
  std::vector<CUDAStreamLease> streams;
  streams.reserve(num_streams);
  for (int i = 0; i < num_streams; i++)
    streams.push_back(CUDAStreamPool::instance().Get());
  struct Alloc {
    void *ptr;
    size_t size, alignment;
    AccessOrder order = AccessOrder::host();
  };
  std::vector<Alloc> allocs;


  allocs.reserve(100000);
  spinlock lock;

  perf_timer::duration total_alloc_time = {};
  perf_timer::duration total_dealloc_time = {};
  perf_timer::duration total_async_alloc_time = {};
  perf_timer::duration total_async_dealloc_time = {};
  int64_t total_num_allocs = 0, total_num_deallocs = 0;
  int64_t total_num_async_allocs = 0, total_num_async_deallocs = 0;

  std::vector<std::thread> threads;
  for (int tid = 0; tid < num_threads; tid++) {
    threads.emplace_back([&, tid]() {
      std::mt19937_64 rng;
      std::uniform_int_distribution<int> stream_dist(-1, num_streams - 1);
      std::uniform_real_distribution<float> size_log_dist(4, 28);
      std::bernoulli_distribution action_dist(0.5);
      auto test_start = perf_timer::now();

      perf_timer::duration alloc_time = {};
      perf_timer::duration dealloc_time = {};
      perf_timer::duration async_alloc_time = {};
      perf_timer::duration async_dealloc_time = {};
      int64_t num_allocs = 0, num_deallocs = 0;
      int64_t num_async_allocs = 0, num_async_deallocs = 0;

      // while (seconds(perf_timer::now() - test_start) < test_time) {
      for (int iter = 0; iter < 10000; iter++) {
        bool is_free = action_dist(rng);
        hipDeviceSynchronize();
        if (is_free) {
          Alloc alloc;

          {
            // Get an allocation and quickly remove it from `allocs` by swapping it with
            // the last element.

            std::lock_guard g(lock);
            if (allocs.empty())
              continue;
            int idx = std::uniform_int_distribution<int>(0, allocs.size() - 1)(rng);
            alloc = allocs[idx];
            std::swap(allocs[idx], allocs.back());
            allocs.pop_back();
          }
          int stream_idx = stream_dist(rng);
          if (stream_idx < 0) {
            auto start = perf_timer::now();
            res->deallocate(alloc.ptr, alloc.size, alloc.alignment);
            auto end = perf_timer::now();
            dealloc_time += (end-start);
            num_deallocs++;
          } else {
            assert(stream_idx >= 0 && stream_idx < streams.size());
            hipStream_t stream = streams[stream_idx].get();
            auto start = perf_timer::now();
            res->deallocate_async(alloc.ptr, alloc.size, alloc.alignment, stream);
            auto end = perf_timer::now();
            async_dealloc_time += (end-start);
            num_async_deallocs++;
          }
        } else {  // allocate
          int stream_idx = stream_dist(rng);

          Alloc alloc = {};
          alloc.size = static_cast<int>(powf(2, size_log_dist(rng)));
          alloc.alignment = 256;

          if (stream_idx < 0) {
            alloc.order = AccessOrder::host();
            auto start = perf_timer::now();
            alloc.ptr = res->allocate(alloc.size, alloc.alignment);
            auto end = perf_timer::now();
            alloc_time += (end-start);
            num_allocs++;
          } else {
            assert(stream_idx >= 0 && stream_idx < streams.size());
            hipStream_t stream = streams[stream_idx].get();
            alloc.order = stream;
            auto start = perf_timer::now();
            alloc.ptr = res->allocate_async(alloc.size, alloc.alignment, stream);
            auto end = perf_timer::now();
            async_alloc_time += (end-start);
            num_async_allocs++;
          }
          {
            std::lock_guard g(lock);
            allocs.push_back(alloc);
          }
        }
      }

      {
        std::lock_guard g(lock);
        total_alloc_time += alloc_time;
        total_dealloc_time += dealloc_time;
        total_num_allocs += num_allocs;
        total_num_deallocs += num_deallocs;

        total_async_alloc_time += async_alloc_time;
        total_async_dealloc_time += async_dealloc_time;
        total_num_async_allocs += num_async_allocs;
        total_num_async_deallocs += num_async_deallocs;
      }
    });
  }

  for (auto &t : threads)
    t.join();

  for (auto &alloc : allocs) {
    res->deallocate(alloc.ptr, alloc.size, alloc.alignment);
  }

  print(std::cout,
    "# allocations:           ", total_num_allocs, "\n"
    "# deallocations:         ", total_num_deallocs, "\n"
    "# async allocations:     ", total_num_async_allocs, "\n"
    "# async deallocations:   ", total_num_async_deallocs, "\n"
    "Allocation time:         ", format_time(seconds(total_alloc_time) / total_num_allocs), "\n"
    "Dellocation time:        ", format_time(seconds(total_dealloc_time) / total_num_allocs), "\n"
    "Async allocation time:   ", format_time(seconds(total_async_alloc_time) / total_num_allocs),
    "\n"
    "Async deallocation time: ", format_time(seconds(total_async_dealloc_time) / total_num_allocs),
    "\n");
}

class cuda_malloc_async_resource : public mm::async_memory_resource<mm::memory_kind::device> {
 public:
  cuda_malloc_async_resource() {
    fake_stream_ = CUDAStreamPool::instance().Get();
  }

 private:
  void *do_allocate(size_t size, size_t alignment) override {
    void *ptr;
    CUDA_CALL(hipMallocAsync(&ptr, size, fake_stream_.get()));
    CUDA_CALL(hipStreamSynchronize(fake_stream_));
    return ptr;
  }

  void do_deallocate(void *ptr, size_t size, size_t alignment) override  {
    CUDA_CALL(hipFreeAsync(ptr, fake_stream_.get()));
  }

  void *do_allocate_async(size_t size, size_t alignment, stream_view stream) override  {
    void *ptr;
    CUDA_CALL(hipMallocAsync(&ptr, size, stream.get()));
    return ptr;
  }

  void do_deallocate_async(void *ptr, size_t size, size_t alignment, stream_view stream) override {
    CUDA_CALL(hipFreeAsync(ptr, stream.get()));
  }

  CUDAStreamLease fake_stream_;
};

TEST(MMPerfTest, DefaultGPUAlloc) {
  auto *res = mm::GetDefaultDeviceResource(0);

  RunBenchmark(res, 1, 1, 1);
}

TEST(MMPerfTest, CudaMallocAsync) {
  cuda_malloc_async_resource res;

  RunBenchmark(&res, 1, 1, 1);
}

}  // namespace test
}  // namespace mm
}  // namespace dali

