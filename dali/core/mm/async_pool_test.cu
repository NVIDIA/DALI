#include "hip/hip_runtime.h"
// Copyright (c) 2021-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <random>
#include <vector>
#include "dali/core/mm/async_pool.h"
#include "dali/core/dev_buffer.h"
#include "dali/core/mm/mm_test_utils.h"
#include "dali/core/hip_stream.h"
#include "dali/core/mm/cuda_vm_resource.h"

namespace dali {
namespace mm {

struct GPUHog {
  ~GPUHog() {
    if (mem) {
      CUDA_DTOR_CALL(hipFree(mem));
      mem = nullptr;
    }
  }

  void init() {
    if (!mem)
      CUDA_CALL(hipMalloc(&mem, size));
  }

  void run(hipStream_t stream, int count = 1) {
    for (int i = 0; i < count; i++) {
      CUDA_CALL(hipMemsetAsync(mem, i+1, size, stream));
    }
  }

  uint8_t *mem = nullptr;
  size_t size = 16<<20;
};

TEST(MMAsyncPool, SingleStreamReuse) {
  GPUHog hog;
  hog.init();

  CUDAStream stream = CUDAStream::Create(true);
  test::test_device_resource upstream;

  async_pool_resource<memory_kind::device> pool(&upstream);
  stream_view sv(stream);
  int size1 = 1<<20;
  void *ptr = pool.allocate_async(size1, sv);
  hog.run(stream, 2);
  pool.deallocate_async(ptr, size1, sv);
  void *p2 = pool.allocate_async(size1, sv);
  CUDA_CALL(hipStreamSynchronize(stream));
  EXPECT_EQ(ptr, p2);
}

TEST(MMAsyncPool, TwoStream) {
  mm::test::test_device_resource upstream;
  CUDAStream s1 = CUDAStream::Create(true);
  CUDAStream s2 = CUDAStream::Create(true);
  stream_view sv1(s1);
  stream_view sv2(s2);

  GPUHog hog;
  hog.init();
  const int min_success = 10;
  const int max_not_busy = 100;
  int stream_not_busy = 0;
  int success = 0;
  while (success < min_success) {
    async_pool_resource<memory_kind::device> pool(&upstream);
    void *p1 = pool.allocate_async(1000, sv1);
    hog.run(s1);
    pool.deallocate_async(p1, 1000, sv1);
    void *p2 = pool.allocate_async(1000, sv2);
    void *p3 = pool.allocate_async(1000, sv1);
    hipError_t e = hipStreamQuery(s1);
    if (e != hipErrorNotReady) {
      std::cerr << "Stream s1 finished before attempt to allocate on s2 was made - retrying\n";
      CUDA_CALL(hipGetLastError());
      if (++stream_not_busy > max_not_busy) {
        FAIL() << "Stream s1 finished - test unreliable.";
      }
      continue;
    }
    stream_not_busy = 0;
    ASSERT_NE(p1, p2);
    ASSERT_EQ(p1, p3);
    CUDA_CALL(hipStreamSynchronize(s1));
    success++;
    CUDA_CALL(hipStreamSynchronize(s2));
  }
  std::cerr << "Peak consumption:     " << upstream.get_peak_size() << " bytes\n";
  std::cerr << "Upstream allocations: " << upstream.get_num_allocs() << std::endl;
  upstream.check_leaks();
}

namespace {

__global__ void Check(const void *ptr, size_t size, uint8_t fill, int *failures) {
  size_t idx = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  if (idx < size) {
    if (static_cast<const uint8_t*>(ptr)[idx] != fill)
      atomicAdd(failures, 1);
  }
}

struct block {
  void *ptr;
  size_t size;
  uint8_t fill;
  hipStream_t stream;
};

template <typename Pool, typename Mutex>
void AsyncPoolTest(Pool &pool, vector<block> &blocks, Mutex &mtx, CUDAStream &stream,
                   int max_iters = 20000, bool use_hog = false) {
  stream_view sv(stream);
  std::mt19937_64 rng(12345);
  std::poisson_distribution<> size_dist(1024);
  const int max_size = 1 << 20;
  std::uniform_int_distribution<> sync_dist(10, 10);
  std::bernoulli_distribution action_dist;
  std::bernoulli_distribution hog_dist(0.05f);
  std::uniform_int_distribution<> fill_dist(1, 255);
  DeviceBuffer<int> failure_buf;
  int failures = 0;
  failure_buf.from_host(&failures, 1, sv.get());
  GPUHog hog;
  if (use_hog)
    hog.init();
  int hogs = 0;
  int max_hogs = sync_dist(rng);
  CUDAEvent event = CUDAEvent::Create();
  for (int i = 0; i < max_iters; i++) {
    if (i == max_iters / 2)
      pool.release_unused();

    if (use_hog && hog_dist(rng)) {
      if (hogs++ > max_hogs) {
        CUDA_CALL(hipStreamSynchronize(stream));
        max_hogs = sync_dist(rng);
      }
      hog.run(stream);
    }
    if (action_dist(rng) || blocks.empty()) {
      size_t size;
      do {
        size = size_dist(rng);
      } while (size > max_size);
      uint8_t fill = fill_dist(rng);
      void *ptr = stream ? pool.allocate_async(size, sv) : pool.allocate(size);
      CUDA_CALL(hipMemsetAsync(ptr, fill, size, stream));
      {
        std::lock_guard<Mutex> guard(mtx);
        (void)guard;  // for dummy mutexes
        blocks.push_back({ ptr, size, fill, stream });
      }
    } else {
      block blk;
      {
        std::lock_guard<Mutex> guard(mtx);
        (void)guard;  // for dummy mutexes
        if (blocks.empty())
          continue;
        int i = std::uniform_int_distribution<>(0, blocks.size()-1)(rng);
        std::swap(blocks[i], blocks.back());
        blk = blocks.back();
        blocks.pop_back();
      }
      if (blk.stream != stream) {
        if (stream) {
          CUDA_CALL(hipEventRecord(event, blk.stream));
          CUDA_CALL(hipStreamWaitEvent(stream, event, 0));
        } else {
          CUDA_CALL(hipStreamSynchronize(blk.stream));
        }
      }
      Check<<<div_ceil(blk.size, 1024), 1024, 0, stream>>>(
            blk.ptr, blk.size, blk.fill, failure_buf);
      if (stream) {
        pool.deallocate_async(blk.ptr, blk.size, sv);
      } else {
        CUDA_CALL(hipStreamSynchronize(stream));
        pool.deallocate(blk.ptr, blk.size);
      }
    }
  }
  copyD2H<int>(&failures, failure_buf, 1, AccessOrder(stream));
  CUDA_CALL(hipStreamSynchronize(stream));
  ASSERT_EQ(failures, 0);
}

}  // namespace

TEST(MMAsyncPool, SingleStreamRandom) {
  CUDAStream stream = CUDAStream::Create(true);
  test::test_device_resource upstream;

  {
    async_pool_resource<memory_kind::device> pool(&upstream);
    vector<block> blocks;
    detail::dummy_lock mtx;
    AsyncPoolTest(pool, blocks, mtx, stream);
  }

  CUDA_CALL(hipStreamSynchronize(stream));
  std::cerr << "Peak consumption:     " << upstream.get_peak_size() << " bytes\n";
  std::cerr << "Upstream allocations: " << upstream.get_num_allocs() << std::endl;
  upstream.check_leaks();
}

TEST(MMAsyncPool, MultiThreadedSingleStreamRandom) {
  CUDAStream stream = CUDAStream::Create(true);
  mm::test::test_device_resource upstream;
  {
    vector<block> blocks;
    std::mutex mtx;

    async_pool_resource<memory_kind::device> pool(&upstream);

    vector<std::thread> threads;

    for (int t = 0; t < 10; t++) {
      threads.push_back(std::thread([&]() {
        AsyncPoolTest(pool, blocks, mtx, stream);
      }));
    }
    for (auto &t : threads)
      t.join();
  }
  CUDA_CALL(hipStreamSynchronize(stream));
  std::cerr << "Peak consumption:     " << upstream.get_peak_size() << " bytes\n";
  std::cerr << "Upstream allocations: " << upstream.get_num_allocs() << std::endl;
  upstream.check_leaks();
}

TEST(MMAsyncPool, MultiThreadedMultiStreamRandom) {
  mm::test::test_device_resource upstream;
  {
    async_pool_resource<memory_kind::device> pool(&upstream);

    vector<std::thread> threads;

    for (int t = 0; t < 10; t++) {
      threads.push_back(std::thread([&]() {
        CUDAStream stream = CUDAStream::Create(true);
        vector<block> blocks;
        detail::dummy_lock mtx;
        AsyncPoolTest(pool, blocks, mtx, stream);
        CUDA_CALL(hipStreamSynchronize(stream));
      }));
    }
    for (auto &t : threads)
      t.join();
  }
  std::cerr << "Peak consumption:     " << upstream.get_peak_size() << " bytes\n";
  std::cerr << "Upstream allocations: " << upstream.get_num_allocs() << std::endl;
  upstream.check_leaks();
}

TEST(MMAsyncPool, MultiStreamRandomWithGPUHogs) {
  mm::test::test_device_resource upstream;
  {
    async_pool_resource<memory_kind::device> pool(&upstream, false);

    vector<std::thread> threads;

    for (int t = 0; t < 10; t++) {
      threads.push_back(std::thread([&]() {
        // 0-th thread uses null stream, which triggers non-async API usage
        CUDAStream stream = t ? CUDAStream::Create(true) : CUDAStream();
        vector<block> blocks;
        detail::dummy_lock mtx;
        AsyncPoolTest(pool, blocks, mtx, stream, 20000, true);
        CUDA_CALL(hipStreamSynchronize(stream));
      }));
    }
    for (auto &t : threads)
      t.join();
  }
  std::cerr << "Peak consumption:     " << upstream.get_peak_size() << " bytes\n";
  std::cerr << "Upstream allocations: " << upstream.get_num_allocs() << std::endl;
  upstream.check_leaks();
}


TEST(MMAsyncPool, CrossStream) {
  mm::test::test_device_resource upstream;
  {
    async_pool_resource<memory_kind::device> pool(&upstream, false);

    vector<std::thread> threads;
    vector<CUDAStream> streams;

    vector<block> blocks;
    std::mutex mtx;

    const int N = 10;
    streams.resize(N);
    for (int t = 0; t < N; t++) {
      if (t != 0)  // keep empty stream at index 0 to mix sync/async allocations
        streams[t] = CUDAStream::Create(true);
      threads.push_back(std::thread([&, t]() {
        AsyncPoolTest(pool, blocks, mtx, streams[t]);
        CUDA_CALL(hipStreamSynchronize(streams[t]));
      }));
    }
    for (auto &t : threads)
      t.join();
  }
  std::cerr << "Peak consumption:     " << upstream.get_peak_size() << " bytes\n";
  std::cerr << "Upstream allocations: " << upstream.get_num_allocs() << std::endl;
  upstream.check_leaks();
}

TEST(MMAsyncPool, CrossStreamWithHogs) {
  mm::test::test_device_resource upstream;
  {
    async_pool_resource<memory_kind::device> pool(&upstream);

    vector<std::thread> threads;
    vector<CUDAStream> streams;

    vector<block> blocks;
    std::mutex mtx;

    const int N = 10;
    streams.resize(N);
    for (int t = 0; t < N; t++) {
      if (t != 0)  // keep empty stream at index 0 to mix sync/async allocations
        streams[t] = CUDAStream::Create(true);
      threads.push_back(std::thread([&, t]() {
        AsyncPoolTest(pool, blocks, mtx, streams[t], 10000, true);
        CUDA_CALL(hipStreamSynchronize(streams[t]));
      }));
    }
    for (auto &t : threads)
      t.join();
  }
  std::cerr << "Peak consumption:     " << upstream.get_peak_size() << " bytes\n";
  std::cerr << "Upstream allocations: " << upstream.get_num_allocs() << std::endl;
  upstream.check_leaks();
}

#if DALI_USE_CUDA_VM_MAP

TEST(MM_VMAsyncPool, MultiThreadedSingleStreamRandom) {
  if (!cuvm::IsSupported())
    GTEST_SKIP() << "Virtual memory management API is not supported on this machine.";

  CUDAStream stream = CUDAStream::Create(true);
  {
    vector<block> blocks;
    std::mutex mtx;

    async_pool_resource<memory_kind::device, cuda_vm_resource> pool;

    vector<std::thread> threads;

    for (int t = 0; t < 10; t++) {
      threads.push_back(std::thread([&]() {
        AsyncPoolTest(pool, blocks, mtx, stream);
      }));
    }
    for (auto &t : threads)
      t.join();
  }
}

TEST(MM_VMAsyncPool, MultiThreadedMultiStreamRandom) {
  if (!cuvm::IsSupported())
    GTEST_SKIP() << "Virtual memory management API is not supported on this machine.";

  async_pool_resource<memory_kind::device, cuda_vm_resource> pool;

  vector<std::thread> threads;

  for (int t = 0; t < 10; t++) {
    threads.push_back(std::thread([&]() {
      CUDAStream stream = CUDAStream::Create(true);
      vector<block> blocks;
      detail::dummy_lock mtx;
      AsyncPoolTest(pool, blocks, mtx, stream);
      CUDA_CALL(hipStreamSynchronize(stream));
    }));
  }
  for (auto &t : threads)
    t.join();
}

TEST(MM_VMAsyncPool, MultiStreamRandomWithGPUHogs) {
  if (!cuvm::IsSupported())
    GTEST_SKIP() << "Virtual memory management API is not supported on this machine.";

  async_pool_resource<memory_kind::device, cuda_vm_resource> pool;

  vector<std::thread> threads;

  for (int t = 0; t < 10; t++) {
    threads.push_back(std::thread([&]() {
      // 0-th thread uses null stream, which triggers non-async API usage
      CUDAStream stream = t ? CUDAStream::Create(true) : CUDAStream();
      vector<block> blocks;
      detail::dummy_lock mtx;
      AsyncPoolTest(pool, blocks, mtx, stream, 20000, true);
      CUDA_CALL(hipStreamSynchronize(stream));
    }));
  }
  for (auto &t : threads)
    t.join();
}

TEST(MM_VMAsyncPool, CrossStream) {
  if (!cuvm::IsSupported())
    GTEST_SKIP() << "Virtual memory management API is not supported on this machine.";

  async_pool_resource<memory_kind::device, cuda_vm_resource> pool;

  vector<std::thread> threads;
  vector<CUDAStream> streams;

  vector<block> blocks;
  std::mutex mtx;

  const int N = 10;
  streams.resize(N);
  for (int t = 0; t < N; t++) {
    if (t != 0)  // keep empty stream at index 0 to mix sync/async allocations
      streams[t] = CUDAStream::Create(true);
    threads.push_back(std::thread([&, t]() {
      AsyncPoolTest(pool, blocks, mtx, streams[t]);
      CUDA_CALL(hipStreamSynchronize(streams[t]));
    }));
  }
  for (auto &t : threads)
    t.join();
}

TEST(MM_VMAsyncPool, CrossStreamWithHogs) {
  if (!cuvm::IsSupported())
    GTEST_SKIP() << "Virtual memory management API is not supported on this machine.";

  async_pool_resource<memory_kind::device, cuda_vm_resource> pool;

  vector<std::thread> threads;
  vector<CUDAStream> streams;

  vector<block> blocks;
  std::mutex mtx;

  const int N = 10;
  streams.resize(N);
  for (int t = 0; t < N; t++) {
    if (t != 0)  // keep empty stream at index 0 to mix sync/async allocations
      streams[t] = CUDAStream::Create(true);
    threads.push_back(std::thread([&, t]() {
      AsyncPoolTest(pool, blocks, mtx, streams[t], 10000, true);
      CUDA_CALL(hipStreamSynchronize(streams[t]));
    }));
  }
  for (auto &t : threads)
    t.join();
}

#endif

}  // namespace mm
}  // namespace dali
