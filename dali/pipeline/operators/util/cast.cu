#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
#include "dali/pipeline/operators/util/cast.h"
#include "dali/error_handling.h"

namespace dali {

template <typename IType, typename OType>
__global__ void
BatchedCastKernel(OType * output, const IType * in, size_t N) {
  size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid < N) {
    if (std::is_same<IType, dali::float16>::value) {
      output[tid] = static_cast<OType>(static_cast<float>(in[tid]));
    } else {
      output[tid] = static_cast<OType>(in[tid]);
    }
  }
}

template <typename IType, typename OType>
DALIError_t BatchedCast(OType * output,
                        const IType * input,
                        size_t N,
                        hipStream_t stream) {
  DALI_ASSERT(output != nullptr);
  DALI_ASSERT(input != nullptr);
  const int threads = 512;
  const int blocks = (N + threads - 1)/threads;
  BatchedCastKernel<<<blocks, threads, 0, stream>>>(output, input, N);
  return DALISuccess;
}

template<>
void Cast<GPUBackend>::RunImpl(DeviceWorkspace *ws, int idx) {
  const auto &input = ws->Input<GPUBackend>(idx);
  auto *output = ws->Output<GPUBackend>(idx);

  DALIDataType itype = input.type().id();

  DALI_TYPE_SWITCH(output_type_, OType,
      output->mutable_data<OType>();
      output->ResizeLike(input);
      DALI_TYPE_SWITCH(itype, IType,
        DALI_CALL(BatchedCast(
            output->mutable_data<OType>(),
            input.data<IType>(),
            input.size(),
            ws->stream()));););
}

DALI_REGISTER_OPERATOR(Cast, Cast<GPUBackend>, GPU);

}  // namespace dali
