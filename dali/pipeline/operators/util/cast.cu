#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "dali/pipeline/operators/util/cast.h"
#include "dali/error_handling.h"
#include "dali/util/cuda_utils.h"

namespace dali {

template <typename IType, typename OType>
__global__ void
BatchedCastKernel(OType * output, const IType * in, size_t N) {
  size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid < N) {
    if (std::is_same<IType, dali::float16>::value) {
      output[tid] = static_cast<OType>(static_cast<float>(in[tid]));
    } else {
      output[tid] = static_cast<OType>(in[tid]);
    }
  }
}

template <typename IType, typename OType>
DALIError_t BatchedCast(OType * output,
                        const IType * input,
                        size_t N,
                        hipStream_t stream) {
  DALI_ASSERT(output != nullptr);
  DALI_ASSERT(input != nullptr);
  const int threads = 512;
  const int blocks = (N + threads - 1)/threads;
  BatchedCastKernel<<<blocks, threads, 0, stream>>>(output, input, N);
  return DALISuccess;
}

template<>
void Cast<GPUBackend>::RunImpl(DeviceWorkspace *ws, int idx) {
  const auto &input = ws->Input<GPUBackend>(idx);
  auto &output = ws->Output<GPUBackend>(idx);

  DALIDataType itype = input.type().id();

  DALI_TYPE_SWITCH(output_type_, OType,
      output.mutable_data<OType>();
      output.ResizeLike(input);
      DALI_TYPE_SWITCH(itype, IType,
        DALI_CALL(BatchedCast(
            output.mutable_data<OType>(),
            input.data<IType>(),
            input.size(),
            ws->stream()));););
}

DALI_REGISTER_OPERATOR(Cast, Cast<GPUBackend>, GPU);

}  // namespace dali
