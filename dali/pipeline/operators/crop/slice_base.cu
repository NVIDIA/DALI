// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <vector>
#include "dali/image/transform.h"
#include "dali/kernels/slice/slice_gpu.cuh"
#include "dali/core/static_switch.h"
#include "dali/pipeline/operators/crop/slice_base.h"
#include "dali/pipeline/data/views.h"

namespace dali {
namespace detail {

template <typename OutputType, typename InputType>
void RunHelper(TensorList<GPUBackend>& output,
               const TensorList<GPUBackend>& input,
               const std::vector<std::vector<int64_t>>& slice_anchors,
               const std::vector<std::vector<int64_t>>& slice_shapes,
               hipStream_t stream,
               kernels::ScratchpadAllocator &scratch_alloc) {
  std::size_t number_of_dims = input.tensor_shape(0).size();
  VALUE_SWITCH(number_of_dims, NumDims, (3, 4), (
    kernels::SliceGPU<OutputType, InputType, NumDims> kernel;

    kernels::KernelContext ctx;
    ctx.gpu.stream = stream;
    auto in_view = view<const InputType, NumDims>(input);

    std::vector<kernels::SliceArgs<NumDims>> slice_args;
    slice_args.reserve(slice_anchors.size());
    for (std::size_t i = 0; i < slice_anchors.size(); i++) {
      std::array<int64_t, NumDims> anchor, shape;
      const auto& slice_anchor = slice_anchors[i];
      const auto& slice_shape = slice_shapes[i];
      for (std::size_t d = 0; d < NumDims; d++) {
        anchor[d] = slice_anchor[d];
        shape[d] = slice_shape[d];
      }
      slice_args.push_back({anchor, shape});
    }

    kernels::KernelRequirements req = kernel.Setup(ctx, in_view, slice_args);

    output.set_type(TypeInfo::Create<OutputType>());
    output.SetLayout(input.GetLayout());
    output.Resize(req.output_shapes[0]);

    scratch_alloc.Reserve(req.scratch_sizes);
    auto scratchpad = scratch_alloc.GetScratchpad();
    ctx.scratchpad = &scratchpad;

    auto out_view = view<OutputType, NumDims>(output);
    kernel.Run(ctx, out_view, in_view, slice_args);
  ),  // NOLINT
  (
    DALI_FAIL("Not supported number of dimensions: " + std::to_string(number_of_dims));
  ));  // NOLINT
}

}  // namespace detail


template <>
void SliceBase<GPUBackend>::RunImpl(DeviceWorkspace *ws, const int idx) {
  this->DataDependentSetup(ws, idx);
  const auto &input = ws->Input<GPUBackend>(idx);
  auto &output = ws->Output<GPUBackend>(idx);

  DALI_TYPE_SWITCH_WITH_FP16_GPU(input_type_, InputType,
    DALI_TYPE_SWITCH_WITH_FP16_GPU(output_type_, OutputType,
      detail::RunHelper<OutputType, InputType>(
        output, input, slice_anchors_, slice_shapes_, ws->stream(), scratch_alloc_);
    )
  )
}

}  // namespace dali
