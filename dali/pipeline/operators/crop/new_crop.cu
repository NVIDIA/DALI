// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <vector>
#include "dali/image/transform.h"
#include "dali/kernels/slice/slice_gpu.cuh"
#include "dali/core/static_switch.h"
#include "dali/pipeline/operators/crop/new_crop.h"
#include "dali/pipeline/data/views.h"

namespace dali {

namespace detail {

  template <typename InputType, typename OutputType, std::size_t D>
  void RunHelper(TensorList<GPUBackend>& output,
                 const TensorList<GPUBackend>& input,
                 const std::vector<std::array<int64_t, D>>& slice_anchors,
                 const std::vector<std::array<int64_t, D>>& slice_shapes,
                 hipStream_t stream) {
    kernels::SliceGPU<OutputType, InputType, D> kernel;

    kernels::KernelContext ctx;
    ctx.gpu.stream = stream;
    auto in_view = view<const InputType, D>(input);

    std::vector<kernels::SliceArgs<D>> slice_args;
    slice_args.reserve(slice_anchors.size());
    for (std::size_t i = 0; i < slice_anchors.size(); i++) {
      slice_args.push_back({slice_anchors[i], slice_shapes[i]});
    }

    kernels::KernelRequirements kernel_req = kernel.Setup(ctx, in_view, slice_args);

    auto out_view = view<OutputType, D>(output);
    kernel.Run(ctx, out_view, in_view, slice_args);
  }
}  // namespace detail

template <>
void NewCrop<GPUBackend>::SetupSharedSampleParams(DeviceWorkspace *ws) {
  CropAttr::ProcessArguments(ws);
  const auto &input = ws->Input<GPUBackend>(0);
  input_type_ = input.type().id();
  if (output_type_ == DALI_NO_TYPE)
    output_type_ = input_type_;
}

template <>
void NewCrop<GPUBackend>::DataDependentSetup(DeviceWorkspace *ws, const int idx) {
  const auto &input = ws->Input<GPUBackend>(idx);

  const DALITensorLayout in_layout = input.GetLayout();
  // TODO(janton) : support other layouts
  DALI_ENFORCE(in_layout == DALI_NHWC || in_layout == DALI_NCHW,
    "Unexpected data layout");
  DALITensorLayout out_layout = in_layout;

  std::vector<Dims> output_shape(batch_size_);
  for (int i = 0; i < batch_size_; ++i) {
    DataDependentSetup(i, in_layout, input.tensor_shape(i));
    auto &slice_shape = slice_shapes_[i];
    output_shape[i] = { slice_shape[0], slice_shape[1], slice_shape[2] };
  }
  auto &output = ws->Output<GPUBackend>(idx);
  output.Resize(output_shape);
  output.SetLayout(out_layout);
}

template <>
void NewCrop<GPUBackend>::RunImpl(DeviceWorkspace *ws, const int idx) {
  DataDependentSetup(ws, idx);
  const auto &input = ws->Input<GPUBackend>(idx);
  auto &output = ws->Output<GPUBackend>(idx);

  if (input_type_ == DALI_FLOAT16 || output_type_ == DALI_FLOAT16) {
    DALI_ENFORCE(input_type_ == output_type_,
      "type conversion is not supported for half precision floats");
    detail::RunHelper<float16, float16, 3>(
      output, input, slice_anchors_, slice_shapes_, ws->stream());
    return;
  }

  DALI_TYPE_SWITCH(input_type_, InputType,
    DALI_TYPE_SWITCH(output_type_, OutputType,
      detail::RunHelper<OutputType, InputType, 3>(
        output, input, slice_anchors_, slice_shapes_, ws->stream());
    )
  )
}

// Register operator
DALI_REGISTER_OPERATOR(NewCrop, NewCrop<GPUBackend>, GPU);

}  // namespace dali
