#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "dali/pipeline/operators/reader/nvdecoder/imgproc.h"

#include <hip/hip_fp16.h>

namespace dali {

namespace {

// using math from https://msdn.microsoft.com/en-us/library/windows/desktop/dd206750(v=vs.85).aspx

template<typename T>
struct yuv {
    T y, u, v;
};

// https://docs.microsoft.com/en-gb/windows/desktop/medfound/recommended-8-bit-yuv-formats-for-video-rendering#converting-8-bit-yuv-to-rgb888
__constant__ float yuv2rgb_mat_norm[9] = {
    1.164383f,  0.0f,       1.596027f,
    1.164383f, -0.391762f, -0.812968f,
    1.164383f,  2.017232f,  0.0f
};

// not normalized need *255
__constant__ float yuv2rgb_mat[9] = {
    1.164383f * 255.f,  0.0f,       1.596027f * 255.f,
    1.164383f * 255.f, -0.391762f * 255.f, -0.812968f * 255.f,
    1.164383f * 255.f,  2.017232f * 255.f,  0.0f
};

__device__ float clip(float x, float max) {
    return fminf(fmaxf(x, 0.0f), max);
}

template<typename T>
__device__ T convert(const float x) {
    return static_cast<T>(x);
}

template<>
__device__ half convert<half>(const float x) {
    return __float2half(x);
}

template<>
__device__ uint8_t convert<uint8_t>(const float x) {
    return static_cast<uint8_t>(roundf(x));
}

template<typename YUV_T, typename RGB_T, bool Normalized = false>
__device__ void yuv2rgb(const yuv<YUV_T>& yuv, RGB_T* rgb,
                        size_t stride) {
    auto y = (static_cast<float>(yuv.y) - 16.0f/255);
    auto u = (static_cast<float>(yuv.u) - 128.0f/255);
    auto v = (static_cast<float>(yuv.v) - 128.0f/255);


    float r, g, b;
    if (Normalized) {
        auto& m = yuv2rgb_mat_norm;
        r = clip(y*m[0] + u*m[1] + v*m[2], 1.0);
        g = clip(y*m[3] + u*m[4] + v*m[5], 1.0);
        b = clip(y*m[6] + u*m[7] + v*m[8], 1.0);
    } else {
        auto& m = yuv2rgb_mat;
        r = clip(y*m[0] + u*m[1] + v*m[2], 255.0);
        g = clip(y*m[3] + u*m[4] + v*m[5], 255.0);
        b = clip(y*m[6] + u*m[7] + v*m[8], 255.0);
    }

    rgb[0] = convert<RGB_T>(r);
    rgb[stride] = convert<RGB_T>(g);
    rgb[stride*2] = convert<RGB_T>(b);
}

template<typename T, bool Normalized = false>
__global__ void process_frame_kernel(
    hipTextureObject_t luma, hipTextureObject_t chroma,
    T* dst, int index,
    float fx, float fy,
    int dst_width, int dst_height, int c) {

    const int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int dst_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (dst_x >= dst_width || dst_y >= dst_height)
        return;

    auto src_x = 0.0f;
    src_x = static_cast<float>(dst_x) * fx;
    auto src_y = static_cast<float>(dst_y) * fy;




    // TODO(spanev) something less hacky here, why 4:2:0 fails on this edge?
    float shift = (dst_x == dst_width - 1) ? 0 : 0.5f;

    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#tex2d-object
    yuv<float> yuv;
    yuv.y = tex2D<float>(luma, src_x + shift, src_y + shift);
    auto uv = tex2D<float2>(chroma, (src_x / 2) + shift, (src_y / 2) + shift);
    yuv.u = uv.x;
    yuv.v = uv.y;

    auto* out = &dst[(dst_x + dst_y * dst_width) * c];

    size_t stride = 1;
    yuv2rgb<float, float, Normalized>(yuv, out, stride);
}

inline constexpr int divUp(int total, int grain) {
    return (total + grain - 1) / grain;
}

}  // namespace

template<typename T>
void process_frame(
    hipTextureObject_t chroma, hipTextureObject_t luma,
    SequenceWrapper& output, int index, hipStream_t stream,
    uint16_t input_width, uint16_t input_height) {
    auto scale_width = input_width;
    auto scale_height = input_height;

    auto fx = static_cast<float>(input_width) / scale_width;
    auto fy = static_cast<float>(input_height) / scale_height;

    dim3 block(32, 8);
    dim3 grid(divUp(output.width, block.x), divUp(output.height, block.y));

    int frame_stride = index * output.height * output.width * output.channels;
    LOG_LINE << "Processing frame " << index
             << " (frame_stride=" << frame_stride << ")" << std::endl;
    auto* tensor_out = output.sequence.mutable_data<T>() + frame_stride;

    // TODO(spanev) Handle normalized and YCbCr versions
    constexpr bool normalized = false;
    process_frame_kernel<T, normalized><<<grid, block, 0, stream>>>
            (luma, chroma, tensor_out, index, fx, fy, output.width, output.height, output.channels);
}

template
void process_frame<float>(
    hipTextureObject_t chroma, hipTextureObject_t luma,
    SequenceWrapper& output, int index, hipStream_t stream,
    uint16_t input_width, uint16_t input_height);

}  // namespace dali
