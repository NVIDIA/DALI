// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <utility>
#include <vector>
#include "dali/pipeline/operators/sequence/element_extract.h"

namespace dali {

namespace detail {

template <typename T>
void ElementExtractImpl(const TensorList<GPUBackend> &input,
                        TensorList<GPUBackend> &output,
                        int index,
                        hipStream_t hip_stream) {
    for (unsigned int i = 0; i < input.ntensor(); i++) {
        auto* output_data = output.mutable_tensor<T>(i);
        const auto* input_data = input.tensor<T>(i);
        const auto& tensor_shape = input.tensor_shape(i);
        const auto element_size = Product(tensor_shape) / tensor_shape[0];

        const auto input_offset = index * element_size;

        CUDA_CALL(hipMemcpyAsync(
            output_data,
            &input_data[input_offset],
            element_size * sizeof(T),
            hipMemcpyDeviceToDevice,
            hip_stream));
    }
}

std::vector<Dims> GetOutputShape(const TensorList<GPUBackend> &input,
                                 const std::vector<int>& element_map) {
    std::vector<Dims> output_shape;
    for (unsigned int i = 0; i < input.ntensor(); ++i) {
        auto shape = input.tensor_shape(i);
        CheckInputShape(shape, element_map);
        Dims element_shape(shape.begin() + 1, shape.end());
        output_shape.push_back(element_shape);
    }
    return output_shape;
}


}  // namespace detail

template <>
void ElementExtract<GPUBackend>::RunImpl(DeviceWorkspace *ws, int idx) {
    auto &input = ws->Input<GPUBackend>(idx);
    auto output_shape = detail::GetOutputShape(input, element_map_);
    auto element_layout = detail::GetElementLayout(input.GetLayout());
    int elements_per_sample = element_map_.size();
    int output_offset = idx * elements_per_sample;
    auto data_type = input.type().id();
    DALI_TYPE_SWITCH(data_type, Type,
        for (int k = 0; k < elements_per_sample; k++) {
            int element = element_map_[k];
            auto &output = ws->Output<GPUBackend>(output_offset + k);
            output.set_type(input.type());
            output.SetLayout(element_layout);
            output.Resize(output_shape);
            detail::ElementExtractImpl<Type>(
                input, output, element, ws->stream());
        }
    )
}

DALI_REGISTER_OPERATOR(ElementExtract, ElementExtract<GPUBackend>, GPU);

}  // namespace dali
