// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <utility>
#include <vector>
#include "dali/pipeline/operators/sequence/element_extract.h"

namespace dali {

namespace detail {

template <typename T>
void ElementExtractImpl(const TensorList<GPUBackend> &input,
                        TensorList<GPUBackend> &output,
                        const std::vector<int> &indexes,
                        hipStream_t hip_stream) {
    output.set_type(input.type());
    auto element_layout = GetElementLayout(input.GetLayout());
    output.SetLayout(element_layout);

    auto elements_per_sample = indexes.size();
    for (unsigned int i = 0; i < input.ntensor(); i++) {
        auto output_offset = elements_per_sample * i;
        auto* output_data = output.mutable_tensor<T>(output_offset);
        const auto* input_data = input.tensor<T>(i);
        const auto& tensor_shape = input.tensor_shape(i);
        const auto element_size = Product(tensor_shape) / tensor_shape[0];

        for (unsigned int k = 0; k < elements_per_sample; k++) {
            const auto input_offset = indexes[k] * element_size;

            CUDA_CALL(hipMemcpyAsync(
                output_data,
                &input_data[input_offset],
                element_size * sizeof(T),
                hipMemcpyDeviceToDevice,
                hip_stream));
        }
    }
}

std::vector<Dims> GetOutputShape(const TensorList<GPUBackend> &input,
                                 const std::vector<int>& element_map) {
    std::vector<Dims> output_shape;
    auto elements_per_sample = element_map.size();
    for (unsigned int i = 0; i < input.ntensor(); ++i) {
        auto shape = input.tensor_shape(i);
        CheckInputShape(shape, element_map);
        Dims element_shape(shape.begin() + 1, shape.end());
        for (std::size_t n = 0; n < elements_per_sample; n++) {
            output_shape.push_back(element_shape);
        }
    }
    return output_shape;
}


}  // namespace detail

template <>
void ElementExtract<GPUBackend>::RunImpl(DeviceWorkspace *ws, int idx) {
    auto &input = ws->Input<GPUBackend>(idx);
    auto &output = ws->Output<GPUBackend>(idx);

    auto output_shape = detail::GetOutputShape(input, element_map_);
    output.Resize(output_shape);

    auto data_type = input.type().id();
    DALI_TYPE_SWITCH(data_type, Type,
        detail::ElementExtractImpl<Type>(
            input, output, element_map_, ws->stream());
    )
}

DALI_REGISTER_OPERATOR(ElementExtract, ElementExtract<GPUBackend>, GPU);

}  // namespace dali
