#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "dali/pipeline/operators/detection/box_encoder.cuh"
#include <hip/hip_runtime.h>
#include <vector>
#include <utility>

namespace dali {
__host__ __device__ inline float4 ToCenterWidthHeight(const float4 &box) {
    return {
      0.5f * (box.x + box.z),
      0.5f * (box.y + box.w),
      box.z - box.x,
      box.w - box.y};
}

void BoxEncoder<GPUBackend>::PrepareAnchors(const vector<float> &anchors) {
  DALI_ENFORCE(
    (anchors.size() % BoundingBox::kSize) == 0,
    "Anchors size must be divisible by 4, actual value = " + std::to_string(anchors.size()));

  anchors_count_ = anchors.size() / BoundingBox::kSize;
  anchors_.Resize({anchors_count_, BoundingBox::kSize});
  anchors_as_center_wh_.Resize({anchors_count_, BoundingBox::kSize});

  auto anchors_data_cpu = reinterpret_cast<const float4 *>(anchors.data());

  vector<float4> anchors_as_center_wh(anchors_count_);
  for (unsigned int anchor = 0; anchor < anchors_count_; ++anchor)
    anchors_as_center_wh[anchor] = ToCenterWidthHeight(anchors_data_cpu[anchor]);

  auto anchors_data = anchors_.mutable_data<float>();
  auto anchors_as_center_wh_data = anchors_as_center_wh_.mutable_data<float>();
  MemCopy(anchors_data, anchors.data(), anchors_count_ * BoundingBox::kSize * sizeof(float));
  MemCopy(
    anchors_as_center_wh_data,
    anchors_as_center_wh.data(),
    anchors_count_ * BoundingBox::kSize * sizeof(float));
}

__device__ __forceinline__ float CalculateIou(const float4 &b1, const float4 &b2) {
  float l = max(b1.x, b2.x);
  float t = max(b1.y, b2.y);
  float r = min(b1.z, b2.z);
  float b = min(b1.w, b2.w);
  float first = max(r - l, 0.0f);
  float second = max(b - t, 0.0f);
  volatile float intersection = first * second;
  volatile float area1 = (b1.w - b1.y) * (b1.z - b1.x);
  volatile float area2 = (b2.w - b2.y) * (b2.z - b2.x);

  return intersection / (area1 + area2 - intersection);
}

__device__ inline void FindBestMatch(const int N, volatile float *vals, volatile int *idx) {
  for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (threadIdx.x < stride) {
      if (vals[threadIdx.x] <= vals[threadIdx.x + stride]) {
        if (vals[threadIdx.x] == vals[threadIdx.x + stride]) {
          idx[threadIdx.x] = max(idx[threadIdx.x], idx[threadIdx.x + stride]);
        } else {
          vals[threadIdx.x] = vals[threadIdx.x + stride];
          idx[threadIdx.x] = idx[threadIdx.x + stride];
        }
      }
    }
    __syncthreads();
  }
}

__device__ void WriteMatchesToOutput(
  int anchors_count, float criteria, int *labels_out, const int *labels_in,
  float4 *boxes_out, const float4 *boxes_in,
  volatile int *best_box_idx, volatile float *best_box_iou) {
  for (unsigned int anchor = threadIdx.x; anchor < anchors_count; anchor += blockDim.x) {
    if (best_box_iou[anchor] > criteria) {
      int box_idx = best_box_idx[anchor];
      labels_out[anchor] = labels_in[box_idx];
      float4 box = boxes_in[box_idx];

      boxes_out[anchor] = ToCenterWidthHeight(box);
    }
  }
}

__device__ void MatchBoxWithAnchors(
  const float4 &box, const int box_idx, unsigned int anchors_count, const float4 *anchors,
  volatile int *best_anchor_idx_tmp, volatile float *best_anchor_iou_tmp,
  volatile int *best_box_idx, volatile float *best_box_iou) {
  float best_anchor_iou = -1.0f;
  int best_anchor_idx = -1;

  for (unsigned int anchor = threadIdx.x; anchor < anchors_count; anchor += blockDim.x) {
    float new_val = CalculateIou(box, anchors[anchor]);

    if (new_val >= best_anchor_iou) {
        best_anchor_iou = new_val;
        best_anchor_idx = anchor;
    }

    if (new_val >= best_box_iou[anchor]) {
        best_box_iou[anchor] = new_val;
        best_box_idx[anchor] = box_idx;
    }
  }

  best_anchor_iou_tmp[threadIdx.x] = best_anchor_iou;
  best_anchor_idx_tmp[threadIdx.x] = best_anchor_idx;
}

template <int BLOCK_SIZE>
__global__ void Encode(
  const float4 *boxes_in, const int *labels_in, const int *offsets, const int anchors_count,
  const float4 *anchors, const float criteria, float4 *boxes_out,  int *labels_out,
  int *box_idx_buffer, float *box_iou_buffer) {
  const int sample = blockIdx.x;

  // Remark: This algorithm is very fragile to floating point arithmetic effects.
  // For now, excessive use of volatile in this code,
  // makes it conform to reference solution in terms of resulting encoding.

  __shared__ volatile int best_anchor_idx_tmp[BLOCK_SIZE];
  __shared__ volatile float best_anchor_iou_tmp[BLOCK_SIZE];

  volatile int *best_box_idx = box_idx_buffer + sample * anchors_count;
  volatile float *best_box_iou = box_iou_buffer + sample * anchors_count;

  int box_idx = 0;
  for (int box_global_idx = offsets[sample]; box_global_idx < offsets[sample+1]; ++box_global_idx) {
    MatchBoxWithAnchors(
      boxes_in[box_global_idx],
      box_idx,
      anchors_count,
      anchors,
      best_anchor_idx_tmp,
      best_anchor_iou_tmp,
      best_box_idx,
      best_box_iou);

    __syncthreads();

    FindBestMatch(blockDim.x, best_anchor_iou_tmp, best_anchor_idx_tmp);
    __syncthreads();

    if (threadIdx.x == 0) {
      int idx = best_anchor_idx_tmp[0];
      best_box_idx[idx] = box_idx;
      best_box_iou[idx] = 2.f;
    }
    __syncthreads();

    box_idx++;
  }
  __syncthreads();

  WriteMatchesToOutput(
    anchors_count,
    criteria,
    labels_out + sample * anchors_count,
    labels_in + offsets[sample],
    boxes_out + sample * anchors_count,
    boxes_in + offsets[sample],
    best_box_idx,
    best_box_iou);
}

std::pair<int *, float *> BoxEncoder<GPUBackend>::ClearBuffers(const hipStream_t &stream) {
  auto best_box_idx_data = best_box_idx_.mutable_data<int>();
  auto best_box_iou_data = best_box_iou_.mutable_data<float>();

  CUDA_CALL(hipMemsetAsync(best_box_idx_data, 0, batch_size_ * anchors_count_ * sizeof(int)));
  CUDA_CALL(hipMemsetAsync(best_box_iou_data, 0, batch_size_ * anchors_count_ * sizeof(float)));

  return {best_box_idx_data, best_box_iou_data};
}

void BoxEncoder<GPUBackend>::WriteAnchorsToOutput(
  float4 *boxes_out_data, int *labels_out_data,  const hipStream_t &stream) {
  CUDA_CALL(hipMemsetAsync(
    labels_out_data,
    0,
    batch_size_ * anchors_count_ * sizeof(int), stream));

  for (int sample = 0; sample < batch_size_; ++sample)
    MemCopy(
      boxes_out_data + sample * anchors_count_,
      anchors_as_center_wh_.data<float>(),
      anchors_count_ * BoundingBox::kSize * sizeof(float),
      stream);
}

std::pair<vector<Dims>, vector<Dims>> BoxEncoder<GPUBackend>::CalculateDims(
  const TensorList<GPUBackend> &boxes_input) {
  vector<Dims> boxes_output_dim;
  vector<Dims> labels_output_dim;
  for (const auto &sample_boxes_shape : boxes_input.shape()) {
    boxes_output_dim.push_back({anchors_count_, BoundingBox::kSize});
    labels_output_dim.push_back({anchors_count_});
  }

  return {boxes_output_dim, labels_output_dim};
}

int *BoxEncoder<GPUBackend>::CalculateOffsets(
  const TensorList<GPUBackend> &boxes_input, const hipStream_t &stream) {
  vector<int> offsets {0};
  for (const auto &sample_boxes_shape : boxes_input.shape())
    offsets.push_back(sample_boxes_shape[0] + offsets.back());

  auto offsets_data = boxes_offsets_.mutable_data<int>();
  MemCopy(offsets_data, offsets.data(), (batch_size_ + 1) * sizeof(int), stream);

  return offsets_data;
}

void BoxEncoder<GPUBackend>::RunImpl(Workspace<GPUBackend> *ws, const int idx) {
  const auto &boxes_input = ws->Input<GPUBackend>(0);
  const auto &labels_input = ws->Input<GPUBackend>(1);

  const auto anchors_data = reinterpret_cast<const float4 *>(anchors_.data<float>());
  const auto boxes_data = reinterpret_cast<const float4 *>(boxes_input.data<float>());
  const auto labels_data = labels_input.data<int>();

  const auto buffers = ClearBuffers(ws->stream());

  auto offsets_data = CalculateOffsets(boxes_input, ws->stream());
  auto dims = CalculateDims(boxes_input);

  auto &boxes_output = ws->Output<GPUBackend>(0);
  boxes_output.set_type(boxes_input.type());
  boxes_output.Resize(dims.first);
  auto boxes_out_data = reinterpret_cast<float4 *>(boxes_output.mutable_data<float>());

  auto &labels_output = ws->Output<GPUBackend>(1);
  labels_output.set_type(labels_input.type());
  labels_output.Resize(dims.second);
  auto labels_out_data = labels_output.mutable_data<int>();

  WriteAnchorsToOutput(boxes_out_data, labels_out_data, ws->stream());

  Encode<BlockSize><<<batch_size_, BlockSize, 0, ws->stream()>>>(
    boxes_data,
    labels_data,
    offsets_data,
    anchors_count_,
    anchors_data,
    criteria_,
    boxes_out_data,
    labels_out_data,
    buffers.first,
    buffers.second);
}

DALI_REGISTER_OPERATOR(BoxEncoder, BoxEncoder<GPUBackend>, GPU);

}  // namespace dali
