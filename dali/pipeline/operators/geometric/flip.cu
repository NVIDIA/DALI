// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "dali/pipeline/operators/geometric/flip.h"
#include <hip/hip_runtime_api.h>
#include <vector>
#include "dali/kernels/imgproc/flip_gpu.cuh"
#include "dali/pipeline/operators/geometric/flip_util.h"

namespace dali {

template <>
Flip<GPUBackend>::Flip(const OpSpec &spec) : Operator<GPUBackend>(spec) {}

void RunKernel(TensorList<GPUBackend> &output, const TensorList<GPUBackend> &input,
               const std::vector<int32> &horizontal,
               const std::vector<int32> &vertical, hipStream_t stream) {
  DALI_TYPE_SWITCH(
      input.type().id(), DType,
      auto in_shape = TransformShapes(input.shape(), input.GetLayout() == DALI_NHWC);
      kernels::InListGPU<DType, 4> in_view(input.data<DType>(), in_shape);
      kernels::KernelContext ctx;
      ctx.gpu.stream = stream;
      kernels::FlipGPU<DType> kernel;
      auto reqs = kernel.Setup(ctx, in_view);
      kernels::OutListGPU<DType, 4> out_view(output.mutable_data<DType>(),
          reqs.output_shapes[0].to_static<4>());
      std::vector<int32> flip_z(horizontal.size());
      kernel.Run(ctx, out_view, in_view, flip_z, vertical, horizontal);
  )
}

template <>
void Flip<GPUBackend>::RunImpl(Workspace<GPUBackend> *ws, const int idx) {
  const auto &input = ws->Input<GPUBackend>(idx);
  auto &output = ws->Output<GPUBackend>(idx);
  DALI_ENFORCE(input.GetLayout() == DALI_NHWC || input.GetLayout() == DALI_NCHW);
  output.SetLayout(input.GetLayout());
  output.set_type(input.type());
  output.ResizeLike(input);
  auto horizontal = GetHorizontal(ws);
  auto vertical = GetVertical(ws);
  RunKernel(output, input, horizontal, vertical, ws->stream());
}

DALI_REGISTER_OPERATOR(Flip, Flip<GPUBackend>, GPU);

}  // namespace dali
