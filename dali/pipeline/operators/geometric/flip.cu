#include "hip/hip_runtime.h"
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime_api.h>
#include "dali/pipeline/operators/geometric/flip.h"

namespace dali {

template <>
Flip<GPUBackend>::Flip(const OpSpec &spec) : Operator<GPUBackend>(spec), spec_(spec) {}

struct NoneIdxFlip {
  static __device__ size_t OutputIdx(size_t height, size_t width, size_t r, size_t c) {
    return r * width + c;
  }
};

struct HorizontalIdxFlip {
  static __device__ size_t OutputIdx(size_t height, size_t width, size_t r, size_t c) {
    return r * width + (width - c - 1);
  }
};

struct VerticalIdxFlip {
  static __device__ size_t OutputIdx(size_t height, size_t width, size_t r, size_t c) {
    return (height - r - 1) * width + c;
  }
};

struct HorizontalVerticalIdxFlip {
  static __device__ size_t OutputIdx(size_t height, size_t width, size_t r, size_t c) {
    return (height - r - 1) * width + (width - c - 1);
  }
};

template <typename IndexFlip, typename T>
__global__ void FlipKernel(T *__restrict__ output, const T *__restrict__ input, size_t height,
                           size_t width, size_t layers, size_t channels_per_layer) {
  size_t r = blockIdx.x * blockDim.x + threadIdx.x;
  size_t c = blockIdx.y * blockDim.y + threadIdx.y;
  size_t z = blockIdx.z * blockDim.z + threadIdx.z;
  if (r < height && c < width && z < layers * channels_per_layer) {
    size_t channel = z % channels_per_layer;
    size_t layer = z / channels_per_layer;
    size_t input_coord = r * width + c;
    size_t output_coord = IndexFlip::OutputIdx(height, width, r, c);
    size_t layer_origin = layer * height * width * channels_per_layer;
    output[layer_origin + output_coord * channels_per_layer + channel] =
        input[layer_origin + input_coord * channels_per_layer + channel];
  }
}

template <typename IndexFlip>
void RunKernel(TensorList<GPUBackend> &output, const TensorList<GPUBackend> &input,
               hipStream_t stream, size_t idx) {
  DALI_TYPE_SWITCH(
      input.type().id(), DType,
      const auto *input_ptr = input.tensor<DType>(idx);
      auto *output_ptr = output.mutable_tensor<DType>(idx);
      int64_t height, width, channels, layers;
      DALI_ENFORCE(input.tensor_shape(idx).size() == 3);
      if (input.GetLayout() == DALI_NHWC) {
        height = input.tensor_shape(idx)[0];
        width = input.tensor_shape(idx)[1];
        channels = input.tensor_shape(idx)[2];
        layers = 1;
      } else {
        height = input.tensor_shape(idx)[1];
        width = input.tensor_shape(idx)[2];
        channels = input.tensor_shape(idx)[0];
        layers = channels;
      }
      unsigned int block_x = height < 32 ? height : 32;
      unsigned int block_y = width < 32 ? width : 32;
      dim3 block(block_x, block_y, 1);
      dim3 grid((height + block_x - 1) / block_x, (width + block_y - 1) / block_y, channels);
      FlipKernel<IndexFlip>
        <<<grid, block, 0, stream>>>(output_ptr, input_ptr, height, width, layers, channels/layers);
  )
}

template <>
void Flip<GPUBackend>::RunImpl(Workspace<GPUBackend> *ws, const int idx) {
  const auto &input = ws->Input<GPUBackend>(idx);
  auto &output = ws->Output<GPUBackend>(idx);
  DALI_ENFORCE(input.GetLayout() == DALI_NHWC || input.GetLayout() == DALI_NCHW);
  output.SetLayout(input.GetLayout());
  output.set_type(input.type());
  output.ResizeLike(input);
  auto stream = ws->stream();
  for (size_t i = 0; i < input.ntensor(); ++i) {
    auto _horizontal = GetHorizontal(ws, i);
    auto _vertical = GetVertical(ws, i);
    if (_horizontal && _vertical) {
      RunKernel<HorizontalVerticalIdxFlip>(output, input, stream, i);
    } else if (_horizontal) {
      RunKernel<HorizontalIdxFlip>(output, input, stream, i);
    } else if (_vertical) {
      RunKernel<VerticalIdxFlip>(output, input, stream, i);
    } else {
      RunKernel<NoneIdxFlip>(output, input, stream, i);
    }
  }
}

DALI_REGISTER_OPERATOR(Flip, Flip<GPUBackend>, GPU);

}  // namespace dali
