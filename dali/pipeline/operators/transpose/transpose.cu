// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <memory>
#include <string>
#include <utility>
#include <vector>

#include "dali/pipeline/operators/transpose/transpose.h"
#include "dali/error_handling.h"
#include "dali/kernels/static_switch.h"

namespace dali {

#define cuttCheck(stmt) do {                                   \
  cuttResult err = stmt;                                       \
  if (err != CUTT_SUCCESS) {                                   \
    DALI_FAIL("Error while transposing " + std::string(#stmt)); \
  }                                                            \
} while (0)

using IntArr = std::unique_ptr<int[]>;

std::pair<IntArr, IntArr>
RowToColumnMajor(const int *dims, const int *perm, int rank) {
  IntArr new_dims(new int[rank]);
  IntArr new_perm(new int[rank]);

  for (int i = 0; i < rank; ++i) {
    new_dims[i] = dims[rank - 1 - i];
    new_perm[i] = rank - 1 - perm[rank - 1 - i];
  }
  return {std::move(new_dims), std::move(new_perm)};
}

namespace kernel {

template <typename T>
void cuTTKernel(const TensorList<GPUBackend>& input,
                TensorList<GPUBackend>& output,
                const std::vector<int>& permutation,
                hipStream_t stream) {
  int batch_size = static_cast<int>(input.ntensor());
  for (int i = 0; i < batch_size; ++i) {
    Dims tmp = input.tensor_shape(i);
    std::vector<int> input_shape(tmp.begin(), tmp.end());

    IntArr c_dims, c_perm;
    std::tie(c_dims, c_perm) = RowToColumnMajor(input_shape.data(),
                                                permutation.data(),
                                                input_shape.size());
    const void* in = input.raw_tensor(0);
    void* out = output.raw_mutable_tensor(0);
    cuttHandle plan;
    cuttCheck(cuttPlan(&plan, input_shape.size(), c_dims.get(), c_perm.get(), sizeof(T), stream));
    cuttCheck(cuttExecute(plan, in, out));
    CUDA_CALL(hipStreamSynchronize(stream));
    cuttCheck(cuttDestroy(plan));
  }
}

/* We insert an additional dim iff batch_size > 1 because cuTT require dim0 to be > 1 */
template <typename T>
void cuTTKernelBatched(const TensorList<GPUBackend>& input,
                       TensorList<GPUBackend>& output,
                       const std::vector<int>& permutation,
                       cuttHandle* plan,
                       hipStream_t stream) {
  int batch_size = static_cast<int>(input.ntensor());
  Dims tmp = input.tensor_shape(0);
  std::vector<int> input_shape(tmp.begin(), tmp.end());

  if (batch_size > 1) {
    input_shape.insert(input_shape.begin(), batch_size);
  }

  std::vector<int> batched_perm(permutation.begin(), permutation.end());
  if (batch_size > 1) {
    std::for_each(batched_perm.begin(), batched_perm.end(), [](int& i) { i++; });
    batched_perm.insert(batched_perm.begin(), 0);
  }

  IntArr c_dims, c_permutation;
  std::tie(c_dims, c_permutation) = RowToColumnMajor(input_shape.data(),
                                                     batched_perm.data(),
                                                     input_shape.size());

  if (*plan == 0) {
    cuttCheck(cuttPlan(plan,
                       batched_perm.size(),
                       c_dims.get(),
                       c_permutation.get(),
                       sizeof(T),
                       stream));
  }

  const void* in = input.raw_tensor(0);
  void* out = output.raw_mutable_tensor(0);
  cuttCheck(cuttExecute(*plan, in, out));
}
}  // namespace kernel

template <>
Transpose<GPUBackend>::~Transpose() {
  if (cutt_handle_ > 0) {
    cuttCheck(cuttDestroy(cutt_handle_));
  }
}

inline Dims GetPermutedDims(const Dims& dims, const std::vector<int>& permutation) {
  Dims permuted_dims;
  for (auto idx : permutation) {
    permuted_dims.push_back(dims[idx]);
  }
  return permuted_dims;
}

template<>
void Transpose<GPUBackend>::RunImpl(DeviceWorkspace* ws, int idx) {
  const auto& input = ws->Input<GPUBackend>(idx);
  auto& output = ws->Output<GPUBackend>(idx);

  TypeInfo itype = input.type();
  DALI_ENFORCE((itype.size() == 1 || itype.size() == 2 || itype.size() == 4 || itype.size() == 8),
      "cuTT transpose supports only [1-2-4-8] bytes types.");

  output.set_type(itype);
  // output.SetLayout(DALI_UNKNOWN);

  Dims input_shape = input.tensor_shape(0);
  DALI_ENFORCE(input_shape.size() == perm_.size(),
               "Transposed tensors rank should be equal to the permutation index list.");

  if (input.IsDenseTensor()) {
    if (cutt_handle_ != 0) {
      if (input_shape != previous_iter_shape_) {
        cuttCheck(cuttDestroy(cutt_handle_));
        cutt_handle_ = 0;
        previous_iter_shape_ = input_shape;
      }
    } else {
      previous_iter_shape_ = input_shape;
    }
    Dims permuted_dims = GetPermutedDims(input_shape, perm_);
    output.Resize(std::vector<Dims>(batch_size_, permuted_dims));
    if (itype.size() == 1) {
      kernel::cuTTKernelBatched<uint8_t>(input, output, perm_, &cutt_handle_, ws->stream());
    } else if (itype.size() == 2) {
      kernel::cuTTKernelBatched<uint16_t>(input, output, perm_, &cutt_handle_, ws->stream());
    } else if (itype.size() == 4) {
      kernel::cuTTKernelBatched<int32_t>(input, output, perm_, &cutt_handle_, ws->stream());
    } else {  // itype.size() == 8
      kernel::cuTTKernelBatched<int64_t>(input, output, perm_, &cutt_handle_, ws->stream());
    }
  } else {
    std::vector<Dims> tl_shape;
    for (int i = 0; i < batch_size_; ++i) {
      Dims in_shape = input.tensor_shape(i);
      tl_shape.emplace_back(GetPermutedDims(in_shape, perm_));
    }
    output.Resize(tl_shape);
    if (itype.size() == 1) {
      kernel::cuTTKernel<uint8_t>(input, output, perm_, ws->stream());
    } else if (itype.size() == 2) {
      kernel::cuTTKernel<uint16_t>(input, output, perm_, ws->stream());
    } else if (itype.size() == 4) {
      kernel::cuTTKernel<int32_t>(input, output, perm_, ws->stream());
    } else {  // itype.size() == 8
      kernel::cuTTKernel<int64_t>(input, output, perm_, ws->stream());
    }
  }
}

DALI_REGISTER_OPERATOR(Transpose, Transpose<GPUBackend>, GPU);

}  // namespace dali
