// Copyright (c) 2017-2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <utility>
#include <memory>
#include <list>

#include "dali/pipeline/operator/builtin/external_source.h"

namespace dali {

template <>
struct ExternalSource<GPUBackend>::RecycleFunctor {
  RecycleFunctor() = default;
  RecycleFunctor(const RecycleFunctor &) {
    assert(!"Should never happen");
  }
  RecycleFunctor(RecycleFunctor &&) = default;
  RecycleFunctor& operator=(const RecycleFunctor&) = default;
  RecycleFunctor& operator=(RecycleFunctor&&) = default;
  ~RecycleFunctor() = default;


  RecycleFunctor(ExternalSource<GPUBackend> *owner, std::list<uptr_cuda_event_type> event,
                 std::list<uptr_tl_type> ptr, std::list<uptr_cuda_event_type> internal_copy_to_gpu)
          : owner(owner), event(std::move(event)), copy_to_gpu(std::move(internal_copy_to_gpu)),
            ptr(std::move(ptr)) {}

  ExternalSource<GPUBackend> *owner;
  std::list<uptr_cuda_event_type> event, copy_to_gpu;
  std::list<uptr_tl_type> ptr;
  void operator()() {
    owner->RecycleBuffer(ptr, &event, &copy_to_gpu);
  }
};

template<>
void ExternalSource<GPUBackend>::RunImpl(DeviceWorkspace &ws) {
  std::list<uptr_tl_type> tensor_list_elm;
  std::list<uptr_tv_type> tensor_vector_elm;
  std::list<uptr_cuda_event_type> cuda_event, internal_copy_to_storage;
  ZeroCopyInfo copy_info;
  {
    std::unique_lock<std::mutex> busy_lock(busy_m_);
    copy_info = zero_copy_.front();
    zero_copy_.pop_front();
    if (copy_info.is_tensor_vector && copy_info.is_zero_copy) {
      tensor_vector_elm = tv_data_.PopFront();
    } else {
      tensor_list_elm = tl_data_.PopFront();
    }
    if (!copy_info.is_zero_copy) {
      internal_copy_to_storage = copy_to_storage_events_.PopFront();
      cuda_event = cuda_events_.GetEmpty();
    }
  }

  auto &output = ws.Output<GPUBackend>(0);
  if (!copy_info.is_zero_copy) {
    hipStream_t stream_used = ws.has_stream() ? ws.stream() : 0;
    CUDA_CALL(hipStreamWaitEvent(stream_used, *internal_copy_to_storage.front(), 0));
    output.Copy(*(tensor_list_elm.front()), stream_used);
    // record an event so Recycle can synchronize on it
    hipEventRecord(*cuda_event.front(), stream_used);
    sync_worker_.DoWork(RecycleFunctor{this, std::move(cuda_event), std::move(tensor_list_elm),
                                       std::move(internal_copy_to_storage)});
  } else {
    if (copy_info.is_tensor_vector) {
      // despite we share the input we need to do a copy internally to the continuous
      // piece of memory it should not hurt that much as it is D2D inside the pipeline
      hipStream_t stream_used = ws.has_stream() ? ws.stream() : 0;
      output.Copy(*(tensor_vector_elm.front()), stream_used);
      // empty tensor_list_elm, it is up to the user to keep memory alive so we can keep copying
      for (auto &t : *tensor_vector_elm.front()) {
        t->Reset();
      }
      // recycle right away as tensor_list_elm is only sharing data
      RecycleBuffer(tensor_vector_elm);
    } else {
      output.ShareData(tensor_list_elm.front().get());
      // empty tensor_list_elm
      tensor_list_elm.front()->Reset();
      // recycle right away as tensor_list_elm is only sharing data
      RecycleBuffer(tensor_list_elm);
    }
  }
}

DALI_REGISTER_OPERATOR(_ExternalSource, ExternalSource<GPUBackend>, GPU);
DALI_REGISTER_OPERATOR(ExternalSource, ExternalSource<GPUBackend>, GPU);

}  // namespace dali
