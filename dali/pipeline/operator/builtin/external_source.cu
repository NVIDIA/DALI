// Copyright (c) 2017-2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <utility>
#include <memory>
#include <list>

#include "dali/pipeline/operator/builtin/external_source.h"

namespace dali {

template <>
struct ExternalSource<GPUBackend>::RecycleFunctor {
  RecycleFunctor() = default;
  RecycleFunctor(const RecycleFunctor &) {
    assert(!"Should never happen");
  }
  RecycleFunctor(RecycleFunctor &&) = default;
  RecycleFunctor& operator=(const RecycleFunctor&) = default;
  RecycleFunctor& operator=(RecycleFunctor&&) = default;
  ~RecycleFunctor() = default;


  RecycleFunctor(ExternalSource<GPUBackend> *owner, std::list<uptr_cuda_event_type> event,
                 std::list<uptr_tl_type> ptr, std::list<uptr_cuda_event_type> internal_copy_to_gpu)
          : owner(owner), event(std::move(event)), copy_to_gpu(std::move(internal_copy_to_gpu)),
            ptr(std::move(ptr)) {}

  ExternalSource<GPUBackend> *owner;
  std::list<uptr_cuda_event_type> event, copy_to_gpu;
  std::list<uptr_tl_type> ptr;
  void operator()() {
    owner->RecycleBuffer(ptr, &event, &copy_to_gpu);
  }
};

template<>
void ExternalSource<GPUBackend>::RunImpl(DeviceWorkspace &ws) {
  std::list<uptr_tl_type> tensor_list_elm;
  std::list<uptr_cuda_event_type> cuda_event, internal_copy_to_storage;
  {
    std::unique_lock<std::mutex> busy_lock(busy_m_);
    tensor_list_elm = tl_data_.PopFront();
    internal_copy_to_storage = copy_to_storage_events_.PopFront();
    cuda_event = cuda_events_.GetEmpty();
  }

  auto &output = ws.Output<GPUBackend>(0);
  hipStream_t stream_used = ws.has_stream() ? ws.stream() : 0;
  CUDA_CALL(hipStreamWaitEvent(stream_used, *internal_copy_to_storage.front(), 0));
  output.Copy(*(tensor_list_elm.front()), stream_used);
  // record an event so Recycle can synchronize on it
  hipEventRecord(*cuda_event.front(), stream_used);
  sync_worker_.DoWork(RecycleFunctor{this, std::move(cuda_event), std::move(tensor_list_elm),
                                     std::move(internal_copy_to_storage)});
}

DALI_REGISTER_OPERATOR(_ExternalSource, ExternalSource<GPUBackend>, GPU);
DALI_REGISTER_OPERATOR(ExternalSource, ExternalSource<GPUBackend>, GPU);

}  // namespace dali
