// Copyright (c) 2017-2021, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <utility>
#include <memory>
#include <list>

#include "dali/pipeline/operator/builtin/external_source.h"

namespace dali {

template<>
void ExternalSource<GPUBackend>::RunImpl(DeviceWorkspace &ws) {
  std::list<uptr_tl_type> tensor_list_elm;
  std::list<uptr_cuda_event_type> internal_copy_to_storage;
  ExternalSourceState state_info;
  {
    std::unique_lock<std::mutex> busy_lock(busy_m_);
    tensor_list_elm = tl_data_.PopFront();
    state_info = state_.front();
    state_.pop_front();
    // even with no_copy we may have copied from TensorVector to TensorList and we
    // need to sync with that
    if (!state_info.no_copy || state_info.copied_shared_data) {
      internal_copy_to_storage = copy_to_storage_events_.PopFront();
    }
  }

  auto &output = ws.Output<GPUBackend>(0);
  hipStream_t stream_used = ws.has_stream() ? ws.stream() : 0;
  if (!state_info.no_copy || state_info.copied_shared_data) {
    CUDA_CALL(hipStreamWaitEvent(stream_used, *internal_copy_to_storage.front(), 0));
  }

  std::swap(output, *tensor_list_elm.front());

  if (!state_info.no_copy || state_info.copied_shared_data) {
    RecycleBuffer(tensor_list_elm, &internal_copy_to_storage);
  } else {
    RecycleBuffer(tensor_list_elm);
  }
}

DALI_REGISTER_OPERATOR(ExternalSource, ExternalSource<GPUBackend>, GPU);

}  // namespace dali
