#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "optical_flow_turing.h"

namespace dali {
namespace optical_flow {
namespace kernel {

constexpr size_t kFractionLength = 5;
constexpr size_t kDecimalLength = 10;
constexpr size_t kSignBit = 15;
constexpr size_t kBlockSize = 256;

__host__ __device__ int extract_bits(int number, int from, int howmany) {
  return (((1 << howmany) - 1) & (number >> (from)));
}


__host__ __device__ size_t count_digits(int number) {
  if (number < 0) number *= -1;
  return number > 0 ? static_cast<size_t>(log10(static_cast<double>(number))) + 1 : 1;
}


__host__ __device__ float decode_flow_component(int16_t value) {
  auto fra = extract_bits(value, 0, kFractionLength);
  auto dec = extract_bits(value, kFractionLength, kDecimalLength);
  return (dec + static_cast<float>(fra) / static_cast<float>(std::pow(10, count_digits(fra)))) *
         ((value & (1 << kSignBit)) ? -1.f : 1.f);
}


__global__ void DecodeFlowComponentKernel(const int16_t *input, float *output, size_t num_values) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  output[idx] = decode_flow_component(input[idx]);
}


void DecodeFlowComponents(const int16_t *input, float *output, size_t num_values) {
  size_t num_blocks = (num_values + kBlockSize - 1) / kBlockSize;
  DecodeFlowComponentKernel << < num_blocks, kBlockSize >> > (input, output, num_values);
  hipDeviceSynchronize();
}

}  // namespace kernel
}  // namespace optical_flow
}  // namespace dali

