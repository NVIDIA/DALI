#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "dali/aux/optical_flow/turing_of/optical_flow_turing.h"

namespace dali {
namespace optical_flow {
namespace kernel {

constexpr size_t kFractionLength = 5;
constexpr size_t kBlockSize = 256;


__host__ __device__ float decode_flow_component(int16_t value) {
  constexpr float precision = 1.0f / (1 << kFractionLength);
  return (value < 0 ? -precision : precision) * (value & 0x7fff);
}


__global__ void DecodeFlowComponentKernel(const int16_t *input, float *output) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  output[idx] = decode_flow_component(input[idx]);
}


void DecodeFlowComponents(const int16_t *input, float *output, size_t num_values) {
  size_t num_blocks = (num_values + kBlockSize - 1) / kBlockSize;
  DecodeFlowComponentKernel<<<num_blocks, kBlockSize>>>(input, output);
}

}  // namespace kernel
}  // namespace optical_flow
}  // namespace dali

