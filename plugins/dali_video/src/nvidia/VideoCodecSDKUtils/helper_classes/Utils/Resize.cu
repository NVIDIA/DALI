#include "hip/hip_runtime.h"
/*
 * This copyright notice applies to this file only
 *
 * SPDX-FileCopyrightText: Copyright (c) 2010-2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: MIT
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_runtime.h>
#include "NvCodecUtils.h"

template<typename YuvUnitx2>
static __global__ void Resize(hipTextureObject_t texY, hipTextureObject_t texUv,
        uint8_t *pDst, uint8_t *pDstUV, int nPitch, int nWidth, int nHeight,
        float fxScale, float fyScale)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x,
        iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= nWidth / 2 || iy >= nHeight / 2) {
        return;
    }

    int x = ix * 2, y = iy * 2;
    typedef decltype(YuvUnitx2::x) YuvUnit;
    const int MAX = (1 << (sizeof(YuvUnit) * 8)) - 1;
    *(YuvUnitx2 *)(pDst + y * nPitch + x * sizeof(YuvUnit)) = YuvUnitx2 {
        (YuvUnit)(tex2D<float>(texY, x / fxScale, y / fyScale) * MAX),
        (YuvUnit)(tex2D<float>(texY, (x + 1) / fxScale, y / fyScale) * MAX)
    };
    y++;
    *(YuvUnitx2 *)(pDst + y * nPitch + x * sizeof(YuvUnit)) = YuvUnitx2 {
        (YuvUnit)(tex2D<float>(texY, x / fxScale, y / fyScale) * MAX),
        (YuvUnit)(tex2D<float>(texY, (x + 1) / fxScale, y / fyScale) * MAX)
    };
    float2 uv = tex2D<float2>(texUv, ix / fxScale, (nHeight + iy) / fyScale + 0.5f);
    *(YuvUnitx2 *)(pDstUV + iy * nPitch + ix * 2 * sizeof(YuvUnit)) = YuvUnitx2{ (YuvUnit)(uv.x * MAX), (YuvUnit)(uv.y * MAX) };
}

template <typename YuvUnitx2>
static void Resize(unsigned char *dpDst, unsigned char* dpDstUV, int nDstPitch, int nDstWidth, int nDstHeight, unsigned char *dpSrc, int nSrcPitch, int nSrcWidth, int nSrcHeight) {
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = dpSrc;
    resDesc.res.pitch2D.desc = hipCreateChannelDesc<decltype(YuvUnitx2::x)>();
    resDesc.res.pitch2D.width = nSrcWidth;
    resDesc.res.pitch2D.height = nSrcHeight;
    resDesc.res.pitch2D.pitchInBytes = nSrcPitch;

    hipTextureDesc texDesc = {};
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeNormalizedFloat;

    hipTextureObject_t texY=0;
    ck(hipCreateTextureObject(&texY, &resDesc, &texDesc, NULL));

    resDesc.res.pitch2D.desc = hipCreateChannelDesc<YuvUnitx2>();
    resDesc.res.pitch2D.width = nSrcWidth / 2;
    resDesc.res.pitch2D.height = nSrcHeight * 3 / 2;

    hipTextureObject_t texUv=0;
    ck(hipCreateTextureObject(&texUv, &resDesc, &texDesc, NULL));

    Resize<YuvUnitx2> << <dim3((nDstWidth + 31) / 32, (nDstHeight + 31) / 32), dim3(16, 16) >> >(texY, texUv, dpDst, dpDstUV,
        nDstPitch, nDstWidth, nDstHeight, 1.0f * nDstWidth / nSrcWidth, 1.0f * nDstHeight / nSrcHeight);

    ck(hipDestroyTextureObject(texY));
    ck(hipDestroyTextureObject(texUv));
}

void ResizeNv12(unsigned char *dpDstNv12, int nDstPitch, int nDstWidth, int nDstHeight, unsigned char *dpSrcNv12, int nSrcPitch, int nSrcWidth, int nSrcHeight, unsigned char* dpDstNv12UV)
{
    unsigned char* dpDstUV = dpDstNv12UV ? dpDstNv12UV : dpDstNv12 + (nDstPitch*nDstHeight);
    return Resize<uchar2>(dpDstNv12, dpDstUV, nDstPitch, nDstWidth, nDstHeight, dpSrcNv12, nSrcPitch, nSrcWidth, nSrcHeight);
}


void ResizeP016(unsigned char *dpDstP016, int nDstPitch, int nDstWidth, int nDstHeight, unsigned char *dpSrcP016, int nSrcPitch, int nSrcWidth, int nSrcHeight, unsigned char* dpDstP016UV)
{
    unsigned char* dpDstUV = dpDstP016UV ? dpDstP016UV : dpDstP016 + (nDstPitch*nDstHeight);
    return Resize<ushort2>(dpDstP016, dpDstUV, nDstPitch, nDstWidth, nDstHeight, dpSrcP016, nSrcPitch, nSrcWidth, nSrcHeight);
}

static __global__ void Scale(hipTextureObject_t texSrc,
    uint8_t *pDst, int nPitch, int nWidth, int nHeight,
    float fxScale, float fyScale)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x,
        y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= nWidth || y >= nHeight)
    {
        return;
    }

    *(unsigned char*)(pDst + (y * nPitch) + x) = (unsigned char)(fminf((tex2D<float>(texSrc, x * fxScale, y * fyScale)) * 255.0f, 255.0f));
}

static __global__ void Scale_uv(hipTextureObject_t texSrc,
    uint8_t *pDst, int nPitch, int nWidth, int nHeight,
    float fxScale, float fyScale)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x,
        y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= nWidth || y >= nHeight)
    {
        return;
    }

    float2 uv = tex2D<float2>(texSrc, x * fxScale, y * fyScale);
    uchar2 uvOut = uchar2{ (unsigned char)(fminf(uv.x * 255.0f, 255.0f)), (unsigned char)(fminf(uv.y * 255.0f, 255.0f)) };

    *(uchar2*)(pDst + (y * nPitch) + 2 * x) = uvOut;
}

void ScaleKernelLaunch(unsigned char *dpDst, int nDstPitch, int nDstWidth, int nDstHeight, unsigned char *dpSrc, int nSrcPitch, int nSrcWidth, int nSrcHeight, bool bUVPlane = false) 
{
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = dpSrc;
    resDesc.res.pitch2D.desc = bUVPlane ? hipCreateChannelDesc<uchar2>() : hipCreateChannelDesc<unsigned char>();
    resDesc.res.pitch2D.width = nSrcWidth;
    resDesc.res.pitch2D.height = nSrcHeight;
    resDesc.res.pitch2D.pitchInBytes = nSrcPitch;

    hipTextureDesc texDesc = {};
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeNormalizedFloat;

    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;

    hipTextureObject_t texSrc = 0;
    ck(hipCreateTextureObject(&texSrc, &resDesc, &texDesc, NULL));

    dim3 blockSize(16, 16, 1);
    dim3 gridSize(((uint32_t)nDstWidth + blockSize.x - 1) / blockSize.x, ((uint32_t)nDstHeight + blockSize.y - 1) / blockSize.y, 1);

    if (bUVPlane)
    {
        Scale_uv << <gridSize, blockSize >> >(texSrc, dpDst,
            nDstPitch, nDstWidth, nDstHeight, 1.0f * nSrcWidth / nDstWidth, 1.0f * nSrcHeight / nDstHeight);
    }
    else
    {
        Scale << <gridSize, blockSize >> >(texSrc, dpDst,
            nDstPitch, nDstWidth, nDstHeight, 1.0f * nSrcWidth / nDstWidth, 1.0f * nSrcHeight / nDstHeight);
    }

    ck(hipGetLastError());
    ck(hipDestroyTextureObject(texSrc));
}

void ScaleYUV420(unsigned char *dpDstY,
                 unsigned char* dpDstU,
                unsigned char* dpDstV,
                int nDstPitch,
                int nDstChromaPitch,
                int nDstWidth,
                int nDstHeight,
                unsigned char *dpSrcY,
                unsigned char* dpSrcU,
                unsigned char* dpSrcV, 
                int nSrcPitch,
                int nSrcChromaPitch,
                int nSrcWidth,
                int nSrcHeight,
                bool bSemiplanar)
{
    int chromaWidthDst = (nDstWidth + 1) / 2;
    int chromaHeightDst = (nDstHeight + 1) / 2;

    int chromaWidthSrc = (nSrcWidth + 1) / 2;
    int chromaHeightSrc = (nSrcHeight + 1) / 2;

    ScaleKernelLaunch(dpDstY, nDstPitch, nDstWidth, nDstHeight, dpSrcY, nSrcPitch, nSrcWidth, nSrcHeight);

    if (bSemiplanar)
    {
        ScaleKernelLaunch(dpDstU, nDstChromaPitch, chromaWidthDst, chromaHeightDst, dpSrcU, nSrcChromaPitch, chromaWidthSrc, chromaHeightSrc, true);
    }
    else
    {
        ScaleKernelLaunch(dpDstU, nDstChromaPitch, chromaWidthDst, chromaHeightDst, dpSrcU, nSrcChromaPitch, chromaWidthSrc, chromaHeightSrc);
        ScaleKernelLaunch(dpDstV, nDstChromaPitch, chromaWidthDst, chromaHeightDst, dpSrcV, nSrcChromaPitch, chromaWidthSrc, chromaHeightSrc);
    }
}
