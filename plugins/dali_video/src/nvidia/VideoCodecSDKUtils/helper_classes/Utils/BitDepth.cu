/*
 * This copyright notice applies to this file only
 *
 * SPDX-FileCopyrightText: Copyright (c) 2010-2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: MIT
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

static __global__ void ConvertUInt8ToUInt16Kernel(uint8_t *dpUInt8, uint16_t *dpUInt16, int nSrcPitch, int nDestPitch, int nWidth, int nHeight)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x,
        y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= nWidth || y >= nHeight)
    {
        return;
    }
    int destStrideInPixels = nDestPitch / (sizeof(uint16_t));
    *(uchar2 *)&dpUInt16[y * destStrideInPixels + x] = uchar2{ 0, dpUInt8[y * nSrcPitch + x] };
}

static __global__ void ConvertUInt16ToUInt8Kernel(uint16_t *dpUInt16, uint8_t *dpUInt8, int nSrcPitch, int nDestPitch, int nWidth, int nHeight)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x,
        y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= nWidth || y >= nHeight)
    {
        return;
    }
    int srcStrideInPixels = nSrcPitch / (sizeof(uint16_t));
    dpUInt8[y * nDestPitch + x] = ((uchar2 *)&dpUInt16[y * srcStrideInPixels + x])->y;
}

void ConvertUInt8ToUInt16(uint8_t *dpUInt8, uint16_t *dpUInt16, int nSrcPitch, int nDestPitch, int nWidth, int nHeight)
{
    dim3 blockSize(16, 16, 1);
    dim3 gridSize(((uint32_t)nWidth + blockSize.x - 1) / blockSize.x, ((uint32_t)nHeight + blockSize.y - 1) / blockSize.y, 1);
    ConvertUInt8ToUInt16Kernel <<< gridSize, blockSize >>>(dpUInt8, dpUInt16, nSrcPitch, nDestPitch, nWidth, nHeight);
}

void ConvertUInt16ToUInt8(uint16_t *dpUInt16, uint8_t *dpUInt8, int nSrcPitch, int nDestPitch, int nWidth, int nHeight)
{
    dim3 blockSize(16, 16, 1);
    dim3 gridSize(((uint32_t)nWidth + blockSize.x - 1) / blockSize.x, ((uint32_t)nHeight + blockSize.y - 1) / blockSize.y, 1);
    ConvertUInt16ToUInt8Kernel <<<gridSize, blockSize >>>(dpUInt16, dpUInt8, nSrcPitch, nDestPitch, nWidth, nHeight);
}
